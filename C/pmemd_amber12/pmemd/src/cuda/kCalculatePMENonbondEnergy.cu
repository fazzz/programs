#include "hip/hip_runtime.h"
/***************************************************/
/*                                                 */
/*      AMBER NVIDIA CUDA CPU IMPLEMENTATION       */
/*                 PMEMD VERSION                   */
/*                     2010                        */
/*                      by                         */
/*             Scott Le Grand (NVIDIA)             */
/*               Duncan Poole (NVIDIA)             */
/*                Ross Walker (SDSC)               */
/*                                                 */
/***************************************************/

#include <hip/hip_runtime.h>
#include "gpu.h"
#include "ptxmacros.h"

//#define PME_VIRIAL
//#define PME_ENERGY

static __constant__ cudaSimulation cSim;

#ifndef use_DPDP
texture<float2, 1, hipReadModeElementType> xytexref;
texture<float, 1, hipReadModeElementType> ztexref;
texture<float, 1, hipReadModeElementType> qtexref;
texture<float2, 1, hipReadModeElementType> sigepstexref;
#endif

void SetkCalculatePMENonbondEnergySim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

void GetkCalculatePMENonBondEnergySim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}


#ifndef use_DPDP
static __forceinline__ __device__ float __internal_fmad(float a, float b, float c)
{
#if __CUDA_ARCH__ >= 200
  return __fmaf_rn (a, b, c);
#else /* __CUDA_ARCH__ >= 200 */
  return a * b + c;
#endif /* __CUDA_ARCH__ >= 200 */
}

// Faster ERFC approximation courtesy of Norbert Juffa. NVIDIA Corporation
static __forceinline__ __device__ PMEFloat fasterfc(PMEFloat a) 
{
  /* approximate log(erfc(a)) with rel. error < 7e-9 */
  PMEFloat t, x = a;
  t =                       (PMEFloat)-1.6488499458192755E-006;
  t = __internal_fmad(t, x, (PMEFloat)2.9524665006554534E-005);
  t = __internal_fmad(t, x, (PMEFloat)-2.3341951153749626E-004);
  t = __internal_fmad(t, x, (PMEFloat)1.0424943374047289E-003);
  t = __internal_fmad(t, x, (PMEFloat)-2.5501426008983853E-003);
  t = __internal_fmad(t, x, (PMEFloat)3.1979939710877236E-004);
  t = __internal_fmad(t, x, (PMEFloat)2.7605379075746249E-002);
  t = __internal_fmad(t, x, (PMEFloat)-1.4827402067461906E-001);
  t = __internal_fmad(t, x, (PMEFloat)-9.1844764013203406E-001);
  t = __internal_fmad(t, x, (PMEFloat)-1.6279070384382459E+000);
  t = t * x;
  return exp2f(t);
}
#endif




// Nonbond kernels

#define PME_ATOMS_PER_WARP (32)
#define PME_VIRIAL
__global__ void 
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_3X_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_2X_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_13_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#endif
kCalculatePMENonbondForcesVirial32_kernel()
#include "kNLCPNE.h"

#define PME_ENERGY

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_3X_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_2X_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_13_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#endif
kCalculatePMENonbondEnergyVirial32_kernel()
#include "kNLCPNE.h"

#undef PME_ENERGY

#define PME_IS_ORTHOGONAL
__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_3X_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_2X_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_13_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#endif
kCalculatePMEOrthogonalNonbondForcesVirial32_kernel()
#include "kNLCPNE.h"

#define PME_ENERGY

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_3X_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_2X_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_13_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#endif
kCalculatePMEOrthogonalNonbondEnergyVirial32_kernel()
#include "kNLCPNE.h"

#undef PME_ENERGY
#undef PME_IS_ORTHOGONAL
#undef PME_VIRIAL

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_3X_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_2X_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_13_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#endif
kCalculatePMENonbondForces32_kernel()
#include "kNLCPNE.h"

#define PME_ENERGY

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_3X_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_2X_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_13_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#endif
kCalculatePMENonbondEnergy32_kernel()
#include "kNLCPNE.h"

#undef PME_ENERGY

#define PME_IS_ORTHOGONAL
__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_3X_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_2X_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_13_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#endif
kCalculatePMEOrthogonalNonbondForces32_kernel()
#include "kNLCPNE.h"

#define PME_ENERGY

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_3X_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_2X_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_13_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#endif
kCalculatePMEOrthogonalNonbondEnergy32_kernel()
#include "kNLCPNE.h"

#undef PME_ENERGY
#undef PME_IS_ORTHOGONAL
#undef PME_ATOMS_PER_WARP



#define PME_ATOMS_PER_WARP (16)
#define PME_VIRIAL

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_3X_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_2X_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_13_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#endif
kCalculatePMENonbondForcesVirial16_kernel()
#include "kNLCPNE.h"

#define PME_ENERGY

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_3X_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_2X_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_13_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#endif
kCalculatePMENonbondEnergyVirial16_kernel()
#include "kNLCPNE.h"

#undef PME_ENERGY

#define PME_IS_ORTHOGONAL
__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_3X_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_2X_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_13_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#endif
kCalculatePMEOrthogonalNonbondForcesVirial16_kernel()
#include "kNLCPNE.h"

#define PME_ENERGY

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_3X_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_2X_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_13_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#endif
kCalculatePMEOrthogonalNonbondEnergyVirial16_kernel()
#include "kNLCPNE.h"

#undef PME_ENERGY
#undef PME_IS_ORTHOGONAL
#undef PME_VIRIAL

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_3X_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_2X_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_13_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#endif
kCalculatePMENonbondForces16_kernel()
#include "kNLCPNE.h"

#define PME_ENERGY

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_3X_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_2X_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_13_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#endif
kCalculatePMENonbondEnergy16_kernel()
#include "kNLCPNE.h"

#undef PME_ENERGY

#define PME_IS_ORTHOGONAL
__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_3X_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_2X_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_13_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#endif
kCalculatePMEOrthogonalNonbondForces16_kernel()
#include "kNLCPNE.h"

#define PME_ENERGY

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_3X_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_2X_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_PMENONBONDENERGY_THREADS_PER_BLOCK, SM_13_PMENONBONDENERGY_BLOCKS_MULTIPLIER)
#endif
kCalculatePMEOrthogonalNonbondEnergy16_kernel()
#include "kNLCPNE.h"

#undef PME_ENERGY
#undef PME_IS_ORTHOGONAL
#undef PME_ATOMS_PER_WARP





extern "C" void kCalculatePMENonbondForces(gpuContext gpu)
{   
    
#ifndef use_DPDP
    // Bind textures
    xytexref.normalized             = 0;
    xytexref.filterMode             = hipFilterModePoint;
    xytexref.addressMode[0]         = hipAddressModeClamp;
    xytexref.channelDesc.x          = 32;       
    xytexref.channelDesc.y          = 32;     
    xytexref.channelDesc.z          = 0;
    xytexref.channelDesc.w          = 0;
    hipBindTexture(NULL, xytexref, gpu->sim.pAtomXYSP, gpu->sim.stride * sizeof(float2));        
    ztexref.normalized              = 0;
    ztexref.filterMode              = hipFilterModePoint;
    ztexref.addressMode[0]          = hipAddressModeClamp;
    ztexref.channelDesc.x           = 32;       
    ztexref.channelDesc.y           = 0;     
    ztexref.channelDesc.z           = 0;
    ztexref.channelDesc.w           = 0;
    hipBindTexture(NULL, ztexref, gpu->sim.pAtomZSP, gpu->sim.stride * sizeof(float));   
    qtexref.normalized              = 0;
    qtexref.filterMode              = hipFilterModePoint;
    qtexref.addressMode[0]          = hipAddressModeClamp;
    qtexref.channelDesc.x           = 32;       
    qtexref.channelDesc.y           = 0;     
    qtexref.channelDesc.z           = 0;
    qtexref.channelDesc.w           = 0;
    hipBindTexture(NULL, qtexref, gpu->sim.pAtomChargeSP, gpu->sim.stride * sizeof(float));     
    sigepstexref.normalized         = 0;
    sigepstexref.filterMode         = hipFilterModePoint;
    sigepstexref.addressMode[0]     = hipAddressModeClamp;
    sigepstexref.channelDesc.x      = 32;       
    sigepstexref.channelDesc.y      = 32;     
    sigepstexref.channelDesc.z      = 0;
    sigepstexref.channelDesc.w      = 0;
    hipBindTexture(NULL, sigepstexref, gpu->sim.pImageSigEps, gpu->sim.stride * sizeof(float2)); 
#endif

    if (gpu->sim.NLAtomsPerWarp == 32)
    {
        if (gpu->sim.ntp > 0)
        {
             if (gpu->sim.is_orthog)
                kCalculatePMEOrthogonalNonbondForcesVirial32_kernel<<<gpu->PMENonbondBlocks, gpu->PMENonbondEnergyThreadsPerBlock>>>();   
            else    
                kCalculatePMENonbondForcesVirial32_kernel<<<gpu->PMENonbondBlocks, gpu->PMENonbondEnergyThreadsPerBlock>>>();      
        }
        else
        {
            if (gpu->sim.is_orthog)
                kCalculatePMEOrthogonalNonbondForces32_kernel<<<gpu->PMENonbondBlocks, gpu->PMENonbondEnergyThreadsPerBlock>>>();   
            else    
                kCalculatePMENonbondForces32_kernel<<<gpu->PMENonbondBlocks, gpu->PMENonbondEnergyThreadsPerBlock>>>();   
        }    
    }
    else
    {
        if (gpu->sim.ntp > 0)
        {
             if (gpu->sim.is_orthog)
                kCalculatePMEOrthogonalNonbondForcesVirial16_kernel<<<gpu->PMENonbondBlocks, gpu->PMENonbondEnergyThreadsPerBlock>>>();   
            else    
                kCalculatePMENonbondForcesVirial16_kernel<<<gpu->PMENonbondBlocks, gpu->PMENonbondEnergyThreadsPerBlock>>>();      
        }
        else
        {
            if (gpu->sim.is_orthog)
                kCalculatePMEOrthogonalNonbondForces16_kernel<<<gpu->PMENonbondBlocks, gpu->PMENonbondEnergyThreadsPerBlock>>>();   
            else    
                kCalculatePMENonbondForces16_kernel<<<gpu->PMENonbondBlocks, gpu->PMENonbondEnergyThreadsPerBlock>>>();   
        }    
    }
    LAUNCHERROR("kCalculatePMENonbondForces"); 

#ifndef use_DPDP
    // Unbind textures
    hipUnbindTexture(xytexref);
    hipUnbindTexture(ztexref);
    hipUnbindTexture(qtexref);
    hipUnbindTexture(sigepstexref);
#endif
}


extern "C" void kCalculatePMENonbondEnergy(gpuContext gpu)
{
#ifndef use_DPDP
    // Bind textures
    xytexref.normalized             = 0;
    xytexref.filterMode             = hipFilterModePoint;
    xytexref.addressMode[0]         = hipAddressModeClamp;
    xytexref.channelDesc.x          = 32;       
    xytexref.channelDesc.y          = 32;     
    xytexref.channelDesc.z          = 0;
    xytexref.channelDesc.w          = 0;
    hipBindTexture(NULL, xytexref, gpu->sim.pAtomXYSP, gpu->sim.stride * sizeof(float2));        
    ztexref.normalized              = 0;
    ztexref.filterMode              = hipFilterModePoint;
    ztexref.addressMode[0]          = hipAddressModeClamp;
    ztexref.channelDesc.x           = 32;       
    ztexref.channelDesc.y           = 0;     
    ztexref.channelDesc.z           = 0;
    ztexref.channelDesc.w           = 0;
    hipBindTexture(NULL, ztexref, gpu->sim.pAtomZSP, gpu->sim.stride * sizeof(float));   
    qtexref.normalized              = 0;
    qtexref.filterMode              = hipFilterModePoint;
    qtexref.addressMode[0]          = hipAddressModeClamp;
    qtexref.channelDesc.x           = 32;       
    qtexref.channelDesc.y           = 0;     
    qtexref.channelDesc.z           = 0;
    qtexref.channelDesc.w           = 0;
    hipBindTexture(NULL, qtexref, gpu->sim.pAtomChargeSP, gpu->sim.stride * sizeof(float));   
    sigepstexref.normalized         = 0;
    sigepstexref.filterMode         = hipFilterModePoint;
    sigepstexref.addressMode[0]     = hipAddressModeClamp;
    sigepstexref.channelDesc.x      = 32;       
    sigepstexref.channelDesc.y      = 32;     
    sigepstexref.channelDesc.z      = 0;
    sigepstexref.channelDesc.w      = 0;
    hipBindTexture(NULL, sigepstexref, gpu->sim.pImageSigEps, gpu->sim.stride * sizeof(float2)); 
#endif

    if (gpu->sim.NLAtomsPerWarp == 32)
    {
        if (gpu->sim.ntp > 0)
        {
            if (gpu->sim.is_orthog)
                kCalculatePMEOrthogonalNonbondEnergyVirial32_kernel<<<gpu->PMENonbondBlocks, gpu->PMENonbondEnergyThreadsPerBlock>>>();   
            else
                kCalculatePMENonbondEnergyVirial32_kernel<<<gpu->PMENonbondBlocks, gpu->PMENonbondEnergyThreadsPerBlock>>>();    
        }
        else
        {
            if (gpu->sim.is_orthog)
                kCalculatePMEOrthogonalNonbondEnergy32_kernel<<<gpu->PMENonbondBlocks, gpu->PMENonbondEnergyThreadsPerBlock>>>();   
            else
                kCalculatePMENonbondEnergy32_kernel<<<gpu->PMENonbondBlocks, gpu->PMENonbondEnergyThreadsPerBlock>>>(); 
        }    
    }
    else
    {
        if (gpu->sim.ntp > 0)
        {
            if (gpu->sim.is_orthog)
                kCalculatePMEOrthogonalNonbondEnergyVirial16_kernel<<<gpu->PMENonbondBlocks, gpu->PMENonbondEnergyThreadsPerBlock>>>();   
            else
                kCalculatePMENonbondEnergyVirial16_kernel<<<gpu->PMENonbondBlocks, gpu->PMENonbondEnergyThreadsPerBlock>>>();    
        }
        else
        {
            if (gpu->sim.is_orthog)
                kCalculatePMEOrthogonalNonbondEnergy16_kernel<<<gpu->PMENonbondBlocks, gpu->PMENonbondEnergyThreadsPerBlock>>>();   
            else
                kCalculatePMENonbondEnergy16_kernel<<<gpu->PMENonbondBlocks, gpu->PMENonbondEnergyThreadsPerBlock>>>(); 
        }
    }
    LAUNCHERROR("kCalculatePMENonbondEnergy");

#ifndef use_DPDP
    // Unbind textures
    hipUnbindTexture(xytexref);
    hipUnbindTexture(ztexref);
    hipUnbindTexture(qtexref);
    hipUnbindTexture(sigepstexref);
#endif
}


#define IPS_ATOMS_PER_WARP (32)
#define IPS_VIRIAL
__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_3X_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_2X_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_13_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#endif
kCalculateIPSNonbondForcesVirial32_kernel()
#include "kNLCINE.h"

#define IPS_ENERGY

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_3X_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_2X_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_13_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#endif
kCalculateIPSNonbondEnergyVirial32_kernel()
#include "kNLCINE.h"

#undef IPS_ENERGY

#define IPS_IS_ORTHOGONAL
__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_3X_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_2X_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_13_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#endif
kCalculateIPSOrthogonalNonbondForcesVirial32_kernel()
#include "kNLCINE.h"

#define IPS_ENERGY

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_3X_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_2X_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_13_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#endif
kCalculateIPSOrthogonalNonbondEnergyVirial32_kernel()
#include "kNLCINE.h"

#undef IPS_ENERGY
#undef IPS_IS_ORTHOGONAL
#undef IPS_VIRIAL

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_3X_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_2X_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_13_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#endif
kCalculateIPSNonbondForces32_kernel()
#include "kNLCINE.h"

#define IPS_ENERGY

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_3X_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_2X_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_13_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#endif
kCalculateIPSNonbondEnergy32_kernel()
#include "kNLCINE.h"

#undef IPS_ENERGY

#define IPS_IS_ORTHOGONAL
__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_3X_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_2X_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_13_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#endif
kCalculateIPSOrthogonalNonbondForces32_kernel()
#include "kNLCINE.h"

#define IPS_ENERGY

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_3X_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_2X_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_13_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#endif
kCalculateIPSOrthogonalNonbondEnergy32_kernel()
#include "kNLCINE.h"

#undef IPS_ENERGY
#undef IPS_IS_ORTHOGONAL
#undef IPS_ATOMS_PER_WARP



#define IPS_ATOMS_PER_WARP (16)
#define IPS_VIRIAL

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_3X_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_2X_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_13_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#endif
kCalculateIPSNonbondForcesVirial16_kernel()
#include "kNLCINE.h"

#define IPS_ENERGY

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_3X_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_2X_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_13_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#endif
kCalculateIPSNonbondEnergyVirial16_kernel()
#include "kNLCINE.h"

#undef IPS_ENERGY

#define IPS_IS_ORTHOGONAL
__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_3X_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_2X_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_13_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#endif
kCalculateIPSOrthogonalNonbondForcesVirial16_kernel()
#include "kNLCINE.h"

#define IPS_ENERGY

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_3X_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_2X_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_13_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#endif
kCalculateIPSOrthogonalNonbondEnergyVirial16_kernel()
#include "kNLCINE.h"

#undef IPS_ENERGY
#undef IPS_IS_ORTHOGONAL
#undef IPS_VIRIAL

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_3X_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_2X_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_13_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#endif
kCalculateIPSNonbondForces16_kernel()
#include "kNLCINE.h"

#define IPS_ENERGY

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_3X_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_2X_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_13_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#endif
kCalculateIPSNonbondEnergy16_kernel()
#include "kNLCINE.h"

#undef IPS_ENERGY

#define IPS_IS_ORTHOGONAL
__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_3X_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_2X_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_13_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#endif
kCalculateIPSOrthogonalNonbondForces16_kernel()
#include "kNLCINE.h"

#define IPS_ENERGY

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_3X_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_2X_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_IPSNONBONDENERGY_THREADS_PER_BLOCK, SM_13_IPSNONBONDENERGY_BLOCKS_MULTIPLIER)
#endif
kCalculateIPSOrthogonalNonbondEnergy16_kernel()
#include "kNLCINE.h"

#undef IPS_ENERGY
#undef IPS_IS_ORTHOGONAL
#undef IPS_ATOMS_PER_WARP

extern "C" void kCalculatePMENonbondEnergyInitKernels(gpuContext gpu)
{
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculatePMENonbondForcesVirial32_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculatePMENonbondEnergyVirial32_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculatePMEOrthogonalNonbondForcesVirial32_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculatePMEOrthogonalNonbondEnergyVirial32_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculatePMENonbondForces32_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculatePMENonbondEnergy32_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculatePMEOrthogonalNonbondForces32_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculatePMEOrthogonalNonbondEnergy32_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculatePMENonbondForcesVirial16_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculatePMENonbondEnergyVirial16_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculatePMEOrthogonalNonbondForcesVirial16_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculatePMEOrthogonalNonbondEnergyVirial16_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculatePMENonbondForces16_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculatePMENonbondEnergy16_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculatePMEOrthogonalNonbondForces16_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculatePMEOrthogonalNonbondEnergy16_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculateIPSNonbondForcesVirial32_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculateIPSNonbondEnergyVirial32_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculateIPSOrthogonalNonbondForcesVirial32_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculateIPSOrthogonalNonbondEnergyVirial32_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculateIPSNonbondForces32_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculateIPSNonbondEnergy32_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculateIPSOrthogonalNonbondForces32_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculateIPSOrthogonalNonbondEnergy32_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculateIPSNonbondForcesVirial16_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculateIPSNonbondEnergyVirial16_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculateIPSOrthogonalNonbondForcesVirial16_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculateIPSOrthogonalNonbondEnergyVirial16_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculateIPSNonbondForces16_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculateIPSNonbondEnergy16_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculateIPSOrthogonalNonbondForces16_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculateIPSOrthogonalNonbondEnergy16_kernel), hipSharedMemBankSizeEightByte);
}

extern "C" void kCalculateIPSNonbondForces(gpuContext gpu)
{  
#ifndef use_DPDP
    // Bind textures
    xytexref.normalized             = 0;
    xytexref.filterMode             = hipFilterModePoint;
    xytexref.addressMode[0]         = hipAddressModeClamp;
    xytexref.channelDesc.x          = 32;       
    xytexref.channelDesc.y          = 32;     
    xytexref.channelDesc.z          = 0;
    xytexref.channelDesc.w          = 0;
    hipBindTexture(NULL, xytexref, gpu->sim.pAtomXYSP, gpu->sim.stride * sizeof(float2));        
    ztexref.normalized              = 0;
    ztexref.filterMode              = hipFilterModePoint;
    ztexref.addressMode[0]          = hipAddressModeClamp;
    ztexref.channelDesc.x           = 32;       
    ztexref.channelDesc.y           = 0;     
    ztexref.channelDesc.z           = 0;
    ztexref.channelDesc.w           = 0;
    hipBindTexture(NULL, ztexref, gpu->sim.pAtomZSP, gpu->sim.stride * sizeof(float));   
    qtexref.normalized              = 0;
    qtexref.filterMode              = hipFilterModePoint;
    qtexref.addressMode[0]          = hipAddressModeClamp;
    qtexref.channelDesc.x           = 32;       
    qtexref.channelDesc.y           = 0;     
    qtexref.channelDesc.z           = 0;
    qtexref.channelDesc.w           = 0;
    hipBindTexture(NULL, qtexref, gpu->sim.pAtomChargeSP, gpu->sim.stride * sizeof(float));     
    sigepstexref.normalized         = 0;
    sigepstexref.filterMode         = hipFilterModePoint;
    sigepstexref.addressMode[0]     = hipAddressModeClamp;
    sigepstexref.channelDesc.x      = 32;       
    sigepstexref.channelDesc.y      = 32;     
    sigepstexref.channelDesc.z      = 0;
    sigepstexref.channelDesc.w      = 0;
    hipBindTexture(NULL, sigepstexref, gpu->sim.pImageSigEps, gpu->sim.stride * sizeof(float2)); 
#endif 

    if (gpu->sim.NLAtomsPerWarp == 32)
    {
        if (gpu->sim.ntp > 0)
        {
             if (gpu->sim.is_orthog)
                kCalculateIPSOrthogonalNonbondForcesVirial32_kernel<<<gpu->IPSNonbondBlocks, gpu->IPSNonbondEnergyThreadsPerBlock>>>();   
            else    
                kCalculateIPSNonbondForcesVirial32_kernel<<<gpu->IPSNonbondBlocks, gpu->IPSNonbondEnergyThreadsPerBlock>>>();      
        }
        else
        {
            if (gpu->sim.is_orthog)
                kCalculateIPSOrthogonalNonbondForces32_kernel<<<gpu->IPSNonbondBlocks, gpu->IPSNonbondEnergyThreadsPerBlock>>>();   
            else    
                kCalculateIPSNonbondForces32_kernel<<<gpu->IPSNonbondBlocks, gpu->IPSNonbondEnergyThreadsPerBlock>>>();   
        }    
    }
    else
    {
        if (gpu->sim.ntp > 0)
        {
             if (gpu->sim.is_orthog)
                kCalculateIPSOrthogonalNonbondForcesVirial16_kernel<<<gpu->IPSNonbondBlocks, gpu->IPSNonbondEnergyThreadsPerBlock>>>();   
            else    
                kCalculateIPSNonbondForcesVirial16_kernel<<<gpu->IPSNonbondBlocks, gpu->IPSNonbondEnergyThreadsPerBlock>>>();      
        }
        else
        {
            if (gpu->sim.is_orthog)
                kCalculateIPSOrthogonalNonbondForces16_kernel<<<gpu->IPSNonbondBlocks, gpu->IPSNonbondEnergyThreadsPerBlock>>>();   
            else    
                kCalculateIPSNonbondForces16_kernel<<<gpu->IPSNonbondBlocks, gpu->IPSNonbondEnergyThreadsPerBlock>>>();   
        }    
    }    
    LAUNCHERROR("kCalculateIPSNonbondForces");

#ifndef use_DPDP
    // Unbind textures
    hipUnbindTexture(xytexref);
    hipUnbindTexture(ztexref);
    hipUnbindTexture(qtexref);
    hipUnbindTexture(sigepstexref);
#endif  
}


extern "C" void kCalculateIPSNonbondEnergy(gpuContext gpu)
{
#ifndef use_DPDP
    // Bind textures
    xytexref.normalized             = 0;
    xytexref.filterMode             = hipFilterModePoint;
    xytexref.addressMode[0]         = hipAddressModeClamp;
    xytexref.channelDesc.x          = 32;       
    xytexref.channelDesc.y          = 32;     
    xytexref.channelDesc.z          = 0;
    xytexref.channelDesc.w          = 0;
    hipBindTexture(NULL, xytexref, gpu->sim.pAtomXYSP, gpu->sim.stride * sizeof(float2));        
    ztexref.normalized              = 0;
    ztexref.filterMode              = hipFilterModePoint;
    ztexref.addressMode[0]          = hipAddressModeClamp;
    ztexref.channelDesc.x           = 32;       
    ztexref.channelDesc.y           = 0;     
    ztexref.channelDesc.z           = 0;
    ztexref.channelDesc.w           = 0;
    hipBindTexture(NULL, ztexref, gpu->sim.pAtomZSP, gpu->sim.stride * sizeof(float));   
    qtexref.normalized              = 0;
    qtexref.filterMode              = hipFilterModePoint;
    qtexref.addressMode[0]          = hipAddressModeClamp;
    qtexref.channelDesc.x           = 32;       
    qtexref.channelDesc.y           = 0;     
    qtexref.channelDesc.z           = 0;
    qtexref.channelDesc.w           = 0;
    hipBindTexture(NULL, qtexref, gpu->sim.pAtomChargeSP, gpu->sim.stride * sizeof(float));     
    sigepstexref.normalized         = 0;
    sigepstexref.filterMode         = hipFilterModePoint;
    sigepstexref.addressMode[0]     = hipAddressModeClamp;
    sigepstexref.channelDesc.x      = 32;       
    sigepstexref.channelDesc.y      = 32;     
    sigepstexref.channelDesc.z      = 0;
    sigepstexref.channelDesc.w      = 0;
    hipBindTexture(NULL, sigepstexref, gpu->sim.pImageSigEps, gpu->sim.stride * sizeof(float2)); 
#endif

    if (gpu->sim.NLAtomsPerWarp == 32)
    {
        if (gpu->sim.ntp > 0)
        {
            if (gpu->sim.is_orthog)
                kCalculateIPSOrthogonalNonbondEnergyVirial32_kernel<<<gpu->IPSNonbondBlocks, gpu->IPSNonbondEnergyThreadsPerBlock>>>();   
            else
                kCalculateIPSNonbondEnergyVirial32_kernel<<<gpu->IPSNonbondBlocks, gpu->IPSNonbondEnergyThreadsPerBlock>>>();    
        }
        else
        {
            if (gpu->sim.is_orthog)
                kCalculateIPSOrthogonalNonbondEnergy32_kernel<<<gpu->IPSNonbondBlocks, gpu->IPSNonbondEnergyThreadsPerBlock>>>();   
            else
                kCalculateIPSNonbondEnergy32_kernel<<<gpu->IPSNonbondBlocks, gpu->IPSNonbondEnergyThreadsPerBlock>>>(); 
        }    
    }
    else
    {
        if (gpu->sim.ntp > 0)
        {
            if (gpu->sim.is_orthog)
                kCalculateIPSOrthogonalNonbondEnergyVirial16_kernel<<<gpu->IPSNonbondBlocks, gpu->IPSNonbondEnergyThreadsPerBlock>>>();   
            else
                kCalculateIPSNonbondEnergyVirial16_kernel<<<gpu->IPSNonbondBlocks, gpu->IPSNonbondEnergyThreadsPerBlock>>>();    
        }
        else
        {
            if (gpu->sim.is_orthog)
                kCalculateIPSOrthogonalNonbondEnergy16_kernel<<<gpu->IPSNonbondBlocks, gpu->IPSNonbondEnergyThreadsPerBlock>>>();   
            else
                kCalculateIPSNonbondEnergy16_kernel<<<gpu->IPSNonbondBlocks, gpu->IPSNonbondEnergyThreadsPerBlock>>>(); 
        }
    }
    LAUNCHERROR("kCalculateIPSNonbondEnergy");
  
#ifndef use_DPDP
    // Unbind textures
    hipUnbindTexture(xytexref);
    hipUnbindTexture(ztexref);
    hipUnbindTexture(qtexref);
    hipUnbindTexture(sigepstexref);
#endif
}






