/***************************************************/
/*                                                 */
/*      AMBER NVIDIA CUDA CPU IMPLEMENTATION       */
/*                 PMEMD VERSION                   */
/*                     2010                        */
/*                      by                         */
/*             Scott Le Grand (NVIDIA)             */
/*               Duncan Poole (NVIDIA)             */
/*                Ross Walker (SDSC)               */
/*                                                 */
/***************************************************/

#include <hip/hip_runtime.h>
#include "gpu.h"
#include <radix_sort/enactor.cuh>
#include <util/multi_buffer.cuh>

static b40c::radix_sort::Enactor* pSortEnactor = NULL;
static b40c::util::MultiBuffer<2, unsigned int, unsigned int>* pDeviceStorage = NULL;


extern "C" void kNLDeleteRadixSort(gpuContext gpu)
{
    if (pSortEnactor)
        delete pSortEnactor;
    pSortEnactor                                = NULL; 
    if (pDeviceStorage)
        delete pDeviceStorage;
    pDeviceStorage                              = NULL;
}

extern "C" void EnactSort(int bits, int sort_atoms)
{
	switch (bits)
	{
	    case 9:
	    case 10:
	        (*pSortEnactor).Sort<b40c::radix_sort::SMALL_PROBLEM, 10, 0>(*pDeviceStorage, sort_atoms);
	        break;
	    case 11:
	    case 12:	    
	        (*pSortEnactor).Sort<b40c::radix_sort::SMALL_PROBLEM, 12, 0>(*pDeviceStorage, sort_atoms);
	        break;
	    case 13:
	    case 14:
	    case 15:
	    case 16:
	        (*pSortEnactor).Sort<b40c::radix_sort::SMALL_PROBLEM, 16, 0>(*pDeviceStorage, sort_atoms);
	        break;
	    case 17:
	    case 18:
	    case 19:
	    case 20:
	        (*pSortEnactor).Sort<b40c::radix_sort::SMALL_PROBLEM, 20, 0>(*pDeviceStorage, sort_atoms);
	        break;
	    case 21:
	    case 22:
	    case 23:
	    case 24:
	        (*pSortEnactor).Sort<b40c::radix_sort::SMALL_PROBLEM, 24, 0>(*pDeviceStorage, sort_atoms);
	        break;	        	        
	    case 25:
	    case 26:
	    case 27:
	    case 28:
	        (*pSortEnactor).Sort<b40c::radix_sort::SMALL_PROBLEM, 28, 0>(*pDeviceStorage, sort_atoms);
	        break;
	    case 29:
	    case 30:
	        (*pSortEnactor).Sort<b40c::radix_sort::SMALL_PROBLEM, 30, 0>(*pDeviceStorage, sort_atoms);
	        break;
	    case 31:
	    case 32:
	        (*pSortEnactor).Sort<b40c::radix_sort::SMALL_PROBLEM, 32, 0>(*pDeviceStorage, sort_atoms);
	        break;	        
	}
}


extern "C" void kNLInitRadixSort(gpuContext gpu)
{


    // Delete old Radix sort
    hipDeviceSynchronize();
    kNLDeleteRadixSort(gpu);
    hipDeviceSynchronize();
   
    // Create new sort
    pDeviceStorage                              = new b40c::util::MultiBuffer<2, unsigned int, unsigned int>();
    pSortEnactor                                = new b40c::radix_sort::Enactor();
    hipDeviceSynchronize();
    pDeviceStorage->d_keys[pDeviceStorage->selector]                   = gpu->sim.pImageHash;
	pDeviceStorage->d_values[pDeviceStorage->selector]                 = gpu->sim.pImageIndex;
    pDeviceStorage->d_keys[pDeviceStorage->selector ^ 1]               = gpu->sim.pImageHash2;
	pDeviceStorage->d_values[pDeviceStorage->selector ^ 1]             = gpu->sim.pImageIndex2;
    hipDeviceSynchronize();
    EnactSort(gpu->neighborListBits, gpu->sim.atoms);
    hipDeviceSynchronize();
}




extern "C" void kNLRadixSort(gpuContext gpu)
{
    pDeviceStorage->d_keys[pDeviceStorage->selector]                   = gpu->sim.pImageHash;
	pDeviceStorage->d_values[pDeviceStorage->selector]                 = gpu->sim.pImageIndex;
    pDeviceStorage->d_keys[pDeviceStorage->selector ^ 1]               = gpu->sim.pImageHash2;
	pDeviceStorage->d_values[pDeviceStorage->selector ^ 1]             = gpu->sim.pImageIndex2;

    EnactSort(gpu->neighborListBits, gpu->sim.atoms);

    gpu->sim.pImageHash             = pDeviceStorage->d_keys[pDeviceStorage->selector];
	gpu->sim.pImageIndex            = pDeviceStorage->d_values[pDeviceStorage->selector];
    gpu->sim.pImageHash2            = pDeviceStorage->d_keys[pDeviceStorage->selector ^ 1];
	gpu->sim.pImageIndex2           = pDeviceStorage->d_values[pDeviceStorage->selector ^ 1];
}

