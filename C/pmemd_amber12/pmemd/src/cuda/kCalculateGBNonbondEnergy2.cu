#include "hip/hip_runtime.h"
/***************************************************/
/*                                                 */
/*      AMBER NVIDIA CUDA CPU IMPLEMENTATION       */
/*                 PMEMD VERSION                   */
/*                     2010                        */
/*                      by                         */
/*             Scott Le Grand (NVIDIA)             */
/*               Duncan Poole (NVIDIA)             */
/*                Ross Walker (SDSC)               */
/*                                                 */
/***************************************************/

#include <hip/hip_runtime.h>
#include "gpu.h"
#include "ptxmacros.h"
static __constant__ cudaSimulation cSim;
static __constant__ PMEFloat te                 = (PMEFloat)(4.0 / 3.0);
static __constant__ PMEFloat tf                 = (PMEFloat)(12.0 / 5.0);
static __constant__ PMEFloat tg                 = (PMEFloat)(24.0 / 7.0);
static __constant__ PMEFloat th                 = (PMEFloat)(40.0 / 9.0);
static __constant__ PMEFloat thh                = (PMEFloat)(60.0 / 11.0);

void SetkCalculateGBNonbondEnergy2Sim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

void GetkCalculateGBNonBondEnergy2Sim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

__global__ void 
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_GBNONBONDENERGY2_THREADS_PER_BLOCK, SM_3X_GBNONBONDENERGY2_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_GBNONBONDENERGY2_THREADS_PER_BLOCK, SM_2X_GBNONBONDENERGY2_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_GBNONBONDENERGY2_THREADS_PER_BLOCK, SM_13_GBNONBONDENERGY2_BLOCKS_MULTIPLIER)
#endif
kCalculateGBNonbondEnergy2_kernel()
#include "kCalculateGBNonbondEnergy2.h"

#define GB_IGB78
__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_GBNONBONDENERGY2_THREADS_PER_BLOCK, SM_3X_GBNONBONDENERGY2_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_GBNONBONDENERGY2_THREADS_PER_BLOCK, SM_2X_GBNONBONDENERGY2_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_GBNONBONDENERGY2_THREADS_PER_BLOCK, SM_13_GBNONBONDENERGY2_BLOCKS_MULTIPLIER)
#endif
kCalculateGBNonbondEnergy2IGB78_kernel()
#include "kCalculateGBNonbondEnergy2.h"
#undef IGB78

void kCalculateGBNonbondEnergy2InitKernels(gpuContext gpu)
{
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculateGBNonbondEnergy2_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculateGBNonbondEnergy2IGB78_kernel), hipSharedMemBankSizeEightByte);
    if (gpu->sm_version >= SM_3X)
    {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kCalculateGBNonbondEnergy2_kernel), hipFuncCachePreferEqual);
    }
}

void kCalculateGBNonbondEnergy2(gpuContext gpu)
{
    if ((gpu->sim.igb == 7) || (gpu->sim.igb == 8))
        kCalculateGBNonbondEnergy2IGB78_kernel<<<gpu->GBNonbondEnergy2Blocks, gpu->GBNonbondEnergy2IGB78ThreadsPerBlock>>>();   
    else
        kCalculateGBNonbondEnergy2_kernel<<<gpu->GBNonbondEnergy2Blocks, gpu->GBNonbondEnergy2ThreadsPerBlock>>>();
    LAUNCHERROR("kCalculateGBNonbondEnergy2");
}
