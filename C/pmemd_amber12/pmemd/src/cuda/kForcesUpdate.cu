#include "hip/hip_runtime.h"
/***************************************************/
/*                                                 */
/*      AMBER NVIDIA CUDA CPU IMPLEMENTATION       */
/*                 PMEMD VERSION                   */
/*                     2010                        */
/*                      by                         */
/*             Scott Le Grand (NVIDIA)             */
/*               Duncan Poole (NVIDIA)             */
/*                Ross Walker (SDSC)               */
/*                                                 */
/***************************************************/

#include <hip/hip_runtime.h>
#include "gpu.h"
#include "ptxmacros.h"

static __constant__ cudaSimulation cSim;

// kForces.cu
void SetkForcesUpdateSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

void GetkForcesUpdateSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

#ifdef MPI
__global__ void 
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_CLEARFORCES_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_CLEARFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_CLEARFORCES_THREADS_PER_BLOCK, 1)
#endif
kClearNBForces_kernel()
{
    unsigned int pos                                    = blockIdx.x * blockDim.x + threadIdx.x;   
#pragma unroll 16    
    while (pos < cSim.stride3)
    {
        cSim.pNBForce[pos]                              = (PMEDouble)0.0;
        pos                                            += blockDim.x * gridDim.x;
    }
}   

void kClearNBForces(gpuContext gpu)
{

    kClearNBForces_kernel<<<gpu->blocks, gpu->NLClearForcesThreadsPerBlock>>>(); 
    LAUNCHERROR("kClearNBForces");
}
#endif

__global__ void 
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_CLEARFORCES_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_CLEARFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_CLEARFORCES_THREADS_PER_BLOCK, 1)
#endif
kClearForces_kernel()
{
    unsigned int pos                                    = blockIdx.x * blockDim.x + threadIdx.x; 

    // Clear GB NB kernel counters
    if (pos < 3)
        cSim.pGBBRPosition[pos]                         = cSim.GBTotalWarps[pos];
  
    if (pos < ENERGYTERMS)
        cSim.pEnergyBuffer[pos]                         = 0;

#ifdef use_SPFP  
    unsigned int count                                  = cSim.stride3 * cSim.nonbondForceBuffers;
#pragma unroll 16 
    while (pos < count)
    {
        cSim.pForceAccumulator[pos]                     = (PMEAccumulator)0;
        pos                                            += blockDim.x * gridDim.x;
    }
#else
    while (pos < cSim.stride3)
    {
        unsigned int pos1                               = pos;
#pragma unroll 16        
        while (pos1 < cSim.stride3 * cSim.maxForceBuffers)
        {
            cSim.pForceBuffer[pos1]                     = (PMEDouble)0.0;
            pos1                                       += cSim.stride3;
        }
        pos                                            += blockDim.x * gridDim.x;
    }
#endif
}   

#ifdef use_SPFP
__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_CLEARFORCES_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_CLEARFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_CLEARFORCES_THREADS_PER_BLOCK, 1)
#endif
kNLClearForces_kernel()
{
    unsigned int pos                                    = blockIdx.x * blockDim.x + threadIdx.x; 
    unsigned int count                                  = cSim.stride3 * cSim.nonbondForceBuffers;  
    if (pos == 0)
        *(cSim.pNLPosition)                             = cSim.NLNonbondWarps;  

    if (pos < ENERGYTERMS)
        cSim.pEnergyBuffer[pos]                         = 0;

#pragma unroll 16   
    while (pos < count)
    {
        cSim.pForceAccumulator[pos]                     = (PMEAccumulator)0;
        pos                                            += blockDim.x * gridDim.x;
    }
}   


void kClearForces(gpuContext gpu)
{
    if (gpu->bNeighborList)
    {
        kNLClearForces_kernel<<<gpu->blocks, gpu->clearForcesThreadsPerBlock>>>();
    }
    else
    {
        kClearForces_kernel<<<gpu->blocks, gpu->clearForcesThreadsPerBlock>>>();  
    } 
    LAUNCHERROR("kClearForces");
}



__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_REDUCEFORCES_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_REDUCEFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_REDUCEFORCES_THREADS_PER_BLOCK, 1)
#endif
kReduceForces_kernel()
{
    unsigned int pos                                    = blockIdx.x * blockDim.x + threadIdx.x;  
#pragma unroll 16
    while (pos < cSim.stride3)
    {
        PMEAccumulator inForce                          = cSim.pForceAccumulator[pos];
        PMEDouble outForce                              = (PMEDouble)inForce * ONEOVERFORCESCALE;
        cSim.pForce[pos]                                = outForce;
        pos                                            += blockDim.x * gridDim.x;
    }
}

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_REDUCEFORCES_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_REDUCEFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_REDUCEFORCES_THREADS_PER_BLOCK, 1)
#endif
kNTPReduceForces_kernel()
{
    unsigned int pos                                    = blockIdx.x * blockDim.x + threadIdx.x;  
#pragma unroll 16
    while (pos < cSim.stride3)
    {
        PMEAccumulator inForce1                         = cSim.pBondedForceAccumulator[pos];
        PMEAccumulator inForce2                         = cSim.pNBForceAccumulator[pos];
        PMEDouble outForce                              = (PMEDouble)(inForce1 + inForce2) * ONEOVERFORCESCALE;
        cSim.pForce[pos]                                = outForce;
        PMEDouble NBForce                               = (PMEDouble)inForce2 * ONEOVERFORCESCALE;
        cSim.pNBForce[pos]                              = NBForce;
        pos                                            += blockDim.x * gridDim.x;
    }
}


void kReduceForces(gpuContext gpu)
{
    if (gpu->sim.ntp > 0)
    {
        kNTPReduceForces_kernel<<<gpu->blocks, gpu->reduceForcesThreadsPerBlock>>>();
    }
    else
    {
        kReduceForces_kernel<<<gpu->blocks, gpu->reduceForcesThreadsPerBlock>>>();  
    }
    LAUNCHERROR("kReduceForces");
}

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_REDUCEFORCES_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_REDUCEFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_REDUCEFORCES_THREADS_PER_BLOCK, 1)
#endif
kReduceNBForces_kernel()
{
    unsigned int pos                                    = blockIdx.x * blockDim.x + threadIdx.x;  
    while (pos < cSim.stride3)
    {
        PMEAccumulator inForce                          = cSim.pNBForceAccumulator[pos];
        PMEDouble NBForce                               = (PMEDouble)inForce * ONEOVERFORCESCALE;
        cSim.pNBForce[pos]                              = NBForce;
        pos                                            += blockDim.x * gridDim.x;
    }
}


void kReduceNBForces(gpuContext gpu)
{
    kReduceNBForces_kernel<<<gpu->blocks, gpu->reduceForcesThreadsPerBlock>>>();     
    LAUNCHERROR("kReduceNBForces");
}

#else  // use_SPFP


__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_NLCLEARFORCES_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_NLCLEARFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_NLCLEARFORCES_THREADS_PER_BLOCK, 1)
#endif
kNLClearForces_kernel()
#include "kNLCF.h"

#define CLEAR_NTP
__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_CLEARFORCES_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_CLEARFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_CLEARFORCES_THREADS_PER_BLOCK, 1)
#endif
kNLClearForcesNTP_kernel()
#include "kNLCF.h"

#define CLEAR_LARGE
__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_CLEARFORCES_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_CLEARFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_CLEARFORCES_THREADS_PER_BLOCK, 1)
#endif
kNLClearForcesNTPLarge_kernel()
#include "kNLCF.h"
#undef CLEAR_LARGE
#undef CLEAR_NTP


#if (__CUDA_ARCH__ >= 300) 
#define CLEARFUNC(FUNCNAME, YDIVISOR, XDIVISOR) __global__ void __launch_bounds__(SM_3X_CLEARFORCES_THREADS_PER_BLOCK, 1) kNLClear##FUNCNAME##_##YDIVISOR##_##XDIVISOR##_kernel() 
#elif (__CUDA_ARCH__ >= 200) 
#define CLEARFUNC(FUNCNAME, YDIVISOR, XDIVISOR) __global__ void __launch_bounds__(SM_2X_CLEARFORCES_THREADS_PER_BLOCK, 1) kNLClear##FUNCNAME##_##YDIVISOR##_##XDIVISOR##_kernel() 
#else
#define CLEARFUNC(FUNCNAME, YDIVISOR, XDIVISOR) __global__ void __launch_bounds__(SM_13_CLEARFORCES_THREADS_PER_BLOCK, 1) kNLClear##FUNCNAME##_##YDIVISOR##_##XDIVISOR##_kernel() 
#endif 

#define CLEAR_YDIVISOR 1
#define CLEAR_XDIVISOR 1
CLEARFUNC(Forces, 1, 1)
#include "kNLCF.h"

#define CLEAR_NTP
CLEARFUNC(ForcesNTP, 1, 1)
#include "kNLCF.h"

#define CLEAR_LARGE
CLEARFUNC(ForcesNTPLarge, 1, 1)
#include "kNLCF.h"
#undef CLEAR_LARGE
#undef CLEAR_NTP
#undef CLEAR_YDIVISOR
#undef CLEAR_XDIVISOR

#define CLEAR_YDIVISOR 2
#define CLEAR_XDIVISOR 1
CLEARFUNC(Forces, 2, 1)
#include "kNLCF.h"

#define CLEAR_NTP
CLEARFUNC(ForcesNTP, 2, 1)
#include "kNLCF.h"

#define CLEAR_LARGE
CLEARFUNC(ForcesNTPLarge, 2, 1)
#include "kNLCF.h"
#undef CLEAR_LARGE
#undef CLEAR_NTP
#undef CLEAR_YDIVISOR
#undef CLEAR_XDIVISOR

#define CLEAR_YDIVISOR 3
#define CLEAR_XDIVISOR 1
CLEARFUNC(Forces, 3, 1)
#include "kNLCF.h"

#define CLEAR_NTP
CLEARFUNC(ForcesNTP, 3, 1)
#include "kNLCF.h"

#define CLEAR_LARGE
CLEARFUNC(ForcesNTPLarge, 3, 1)
#include "kNLCF.h"
#undef CLEAR_LARGE
#undef CLEAR_NTP
#undef CLEAR_YDIVISOR
#undef CLEAR_XDIVISOR

#define CLEAR_YDIVISOR 3
#define CLEAR_XDIVISOR 2
CLEARFUNC(Forces, 3, 2)
#include "kNLCF.h"

#define CLEAR_NTP
CLEARFUNC(ForcesNTP, 3, 2)
#include "kNLCF.h"

#define CLEAR_LARGE
CLEARFUNC(ForcesNTPLarge, 3, 2)
#include "kNLCF.h"
#undef CLEAR_LARGE
#undef CLEAR_NTP
#undef CLEAR_YDIVISOR
#undef CLEAR_XDIVISOR

#define CLEAR_YDIVISOR 3
#define CLEAR_XDIVISOR 5
CLEARFUNC(Forces, 3, 5)
#include "kNLCF.h"

#define CLEAR_NTP
CLEARFUNC(ForcesNTP, 3, 5)
#include "kNLCF.h"

#define CLEAR_LARGE
CLEARFUNC(ForcesNTPLarge, 3, 5)
#include "kNLCF.h"
#undef CLEAR_LARGE
#undef CLEAR_NTP
#undef CLEAR_YDIVISOR
#undef CLEAR_XDIVISOR

#define CLEAR_YDIVISOR 4
#define CLEAR_XDIVISOR 5
CLEARFUNC(Forces, 4, 5)
#include "kNLCF.h"

#define CLEAR_NTP
CLEARFUNC(ForcesNTP, 4, 5)
#include "kNLCF.h"

#define CLEAR_LARGE
CLEARFUNC(ForcesNTPLarge, 4, 5)
#include "kNLCF.h"
#undef CLEAR_LARGE
#undef CLEAR_NTP
#undef CLEAR_YDIVISOR
#undef CLEAR_XDIVISOR

#define CLEAR_YDIVISOR 4
#define CLEAR_XDIVISOR 7
CLEARFUNC(Forces, 4, 7)
#include "kNLCF.h"

#define CLEAR_NTP
CLEARFUNC(ForcesNTP, 4, 7)
#include "kNLCF.h"

#define CLEAR_LARGE
CLEARFUNC(ForcesNTPLarge, 4, 7)
#include "kNLCF.h"
#undef CLEAR_LARGE
#undef CLEAR_NTP
#undef CLEAR_YDIVISOR
#undef CLEAR_XDIVISOR

#define CLEAR_YDIVISOR 4
#define CLEAR_XDIVISOR 14
CLEARFUNC(Forces, 4, 14)
#include "kNLCF.h"

#define CLEAR_NTP
CLEARFUNC(ForcesNTP, 4, 14)
#include "kNLCF.h"

#define CLEAR_LARGE
CLEARFUNC(ForcesNTPLarge, 4, 14)
#include "kNLCF.h"
#undef CLEAR_LARGE
#undef CLEAR_NTP
#undef CLEAR_YDIVISOR
#undef CLEAR_XDIVISOR

#define CLEAR_YDIVISOR 8
#define CLEAR_XDIVISOR 14
CLEARFUNC(Forces, 8, 14)
#include "kNLCF.h"

#define CLEAR_NTP
CLEARFUNC(ForcesNTP, 8, 14)
#include "kNLCF.h"

#define CLEAR_LARGE
CLEARFUNC(ForcesNTPLarge, 8, 14)
#include "kNLCF.h"
#undef CLEAR_LARGE
#undef CLEAR_NTP
#undef CLEAR_YDIVISOR
#undef CLEAR_XDIVISOR

typedef void (*KernelPointer)();
static KernelPointer spNLReduceForcesKernel = NULL;
static KernelPointer spNLClearForcesKernel = NULL;

void SetNLClearForcesKernel(gpuContext gpu)
{
    unsigned int clearType                          = gpu->sim.NLYDivisor * 100 + gpu->sim.NLXDivisor;
    if (gpu->sim.ntp > 0)
    {
        if (gpu->sim.NLCellBuffers >= gpu->sim.maxForceBuffers)
        {
            switch (clearType)
            {
                case 101:
                    spNLClearForcesKernel                   = &kNLClearForcesNTP_1_1_kernel;
                    break;
                        
                case 201:
                    spNLClearForcesKernel                   = &kNLClearForcesNTP_2_1_kernel;
                    break;
                        
                case 301:
                    spNLClearForcesKernel                   = &kNLClearForcesNTP_3_1_kernel;
                    break;
                        
                case 302:
                    spNLClearForcesKernel                   = &kNLClearForcesNTP_3_2_kernel;
                    break;
                case 305:
                    spNLClearForcesKernel                   = &kNLClearForcesNTP_3_5_kernel;
                    break;
                        
                case 405:
                    spNLClearForcesKernel                   = &kNLClearForcesNTP_4_5_kernel;
                    break;
                        
                case 407:
                    spNLClearForcesKernel                   = &kNLClearForcesNTP_4_7_kernel;
                    break;
                        
                case 414:
                    spNLClearForcesKernel                   = &kNLClearForcesNTP_4_14_kernel;
                    break;  
                                      
                case 814:
                    spNLClearForcesKernel                   = &kNLClearForcesNTP_8_14_kernel;
                    break;                                
                }                       
        }
        else
        {           
                switch (clearType)
                {
                case 101:
                    spNLClearForcesKernel                   = &kNLClearForcesNTPLarge_1_1_kernel;
                    break;
                        
                case 201:
                    spNLClearForcesKernel                   = &kNLClearForcesNTPLarge_2_1_kernel;
                    break;
                        
                case 301:
                    spNLClearForcesKernel                   = &kNLClearForcesNTPLarge_3_1_kernel;
                    break;
                        
                case 302:
                    spNLClearForcesKernel                   = &kNLClearForcesNTPLarge_3_2_kernel;
                        break;
                case 305:
                    spNLClearForcesKernel                   = &kNLClearForcesNTPLarge_3_5_kernel;
                    break;
                        
                case 405:
                    spNLClearForcesKernel                   = &kNLClearForcesNTPLarge_4_5_kernel;
                        break;
                        
                case 407:
                    spNLClearForcesKernel                   = &kNLClearForcesNTPLarge_4_7_kernel;
                    break;
                        
                case 414:
                    spNLClearForcesKernel                   = &kNLClearForcesNTPLarge_4_14_kernel;
                    break;  
                                     
                case 814:
                    spNLClearForcesKernel                   = &kNLClearForcesNTPLarge_8_14_kernel;
                    break;                                
            }                               
        }
    
    }
    else
    {
        switch (clearType)
        {
            case 101:
                spNLClearForcesKernel                       = &kNLClearForces_1_1_kernel;
                break;
                
            case 201:
                spNLClearForcesKernel                       = &kNLClearForces_2_1_kernel;
                break;
                
            case 301:
                spNLClearForcesKernel                       = &kNLClearForces_3_1_kernel;
                break;
                
            case 302:
                spNLClearForcesKernel                       = &kNLClearForces_3_2_kernel;
                break;
            case 305:
                spNLClearForcesKernel                       = &kNLClearForces_3_5_kernel;
                break;
                
            case 405:
                spNLClearForcesKernel                       = &kNLClearForces_4_5_kernel;
                break;
                
            case 407:
                spNLClearForcesKernel                       = &kNLClearForces_4_7_kernel;
                break;
                
            case 414:
                spNLClearForcesKernel                       = &kNLClearForces_4_14_kernel;
                break;  
                              
            case 814:
                spNLClearForcesKernel                       = &kNLClearForces_8_14_kernel;
                break;                                
        }
        
    }
    
}



void kClearForces(gpuContext gpu)
{
    if (gpu->bNeighborList)
    {
        spNLClearForcesKernel<<<gpu->blocks, gpu->NLClearForcesThreadsPerBlock>>>();
    }
    else
    {
        kClearForces_kernel<<<gpu->blocks, gpu->clearForcesThreadsPerBlock>>>();  
    }  
    LAUNCHERROR("kClearForces");
}

#if (__CUDA_ARCH__ >= 300) 
#define REDUCEFUNC(FUNCNAME, YDIVISOR, XDIVISOR) __global__ void __launch_bounds__(SM_3X_REDUCEFORCES_THREADS_PER_BLOCK, 1) kNLReduce##FUNCNAME##_##YDIVISOR##_##XDIVISOR##_kernel() 
#elif (__CUDA_ARCH__ >= 200) 
#define REDUCEFUNC(FUNCNAME, YDIVISOR, XDIVISOR) __global__ void __launch_bounds__(SM_2X_REDUCEFORCES_THREADS_PER_BLOCK, 1) kNLReduce##FUNCNAME##_##YDIVISOR##_##XDIVISOR##_kernel() 
#else
#define REDUCEFUNC(FUNCNAME, YDIVISOR, XDIVISOR) __global__ void __launch_bounds__(SM_13_REDUCEFORCES_THREADS_PER_BLOCK, 1) kNLReduce##FUNCNAME##_##YDIVISOR##_##XDIVISOR##_kernel() 
#endif 

#define REDUCE_YDIVISOR 1
#define REDUCE_XDIVISOR 1
REDUCEFUNC(Forces, 1, 1)
#include "kNLRF.h"

#define REDUCE_NTP
REDUCEFUNC(ForcesNTP, 1, 1)
#include "kNLRF.h"

#define REDUCE_LARGE
REDUCEFUNC(ForcesNTPLarge, 1, 1)
#include "kNLRF.h"

#ifdef MPI
#define MPI_REDUCE_NODE0
REDUCEFUNC(ForcesNTPLargeNode0, 1, 1)
#include "kNLRF.h"
#undef MPI_REDUCE_NODE0
#endif
#undef REDUCE_LARGE

#ifdef MPI
#define MPI_REDUCE_NODE0
REDUCEFUNC(ForcesNTPNode0, 1, 1)
#include "kNLRF.h"
#undef MPI_REDUCE_NODE0
#endif
#undef REDUCE_NTP
#undef REDUCE_YDIVISOR
#undef REDUCE_XDIVISOR




#define REDUCE_YDIVISOR 2
#define REDUCE_XDIVISOR 1
REDUCEFUNC(Forces, 2, 1)
#include "kNLRF.h"

#define REDUCE_NTP
REDUCEFUNC(ForcesNTP, 2, 1)
#include "kNLRF.h"

#define REDUCE_LARGE
REDUCEFUNC(ForcesNTPLarge, 2, 1)
#include "kNLRF.h"

#ifdef MPI
#define MPI_REDUCE_NODE0
REDUCEFUNC(ForcesNTPLargeNode0, 2, 1)
#include "kNLRF.h"
#undef MPI_REDUCE_NODE0
#endif
#undef REDUCE_LARGE

#ifdef MPI
#define MPI_REDUCE_NODE0
REDUCEFUNC(ForcesNTPNode0, 2, 1)
#include "kNLRF.h"
#undef MPI_REDUCE_NODE0
#endif
#undef REDUCE_NTP
#undef REDUCE_YDIVISOR
#undef REDUCE_XDIVISOR




#define REDUCE_YDIVISOR 3
#define REDUCE_XDIVISOR 1
REDUCEFUNC(Forces, 3, 1)
#include "kNLRF.h"

#define REDUCE_NTP
REDUCEFUNC(ForcesNTP, 3, 1)
#include "kNLRF.h"

#define REDUCE_LARGE
REDUCEFUNC(ForcesNTPLarge, 3, 1)
#include "kNLRF.h"

#ifdef MPI
#define MPI_REDUCE_NODE0
REDUCEFUNC(ForcesNTPLargeNode0, 3, 1)
#include "kNLRF.h"
#undef MPI_REDUCE_NODE0
#endif
#undef REDUCE_LARGE

#ifdef MPI
#define MPI_REDUCE_NODE0
REDUCEFUNC(ForcesNTPNode0, 3, 1)
#include "kNLRF.h"
#undef MPI_REDUCE_NODE0
#endif
#undef REDUCE_NTP
#undef REDUCE_YDIVISOR
#undef REDUCE_XDIVISOR



#define REDUCE_YDIVISOR 3
#define REDUCE_XDIVISOR 2
REDUCEFUNC(Forces, 3, 2)
#include "kNLRF.h"

#define REDUCE_NTP
REDUCEFUNC(ForcesNTP, 3, 2)
#include "kNLRF.h"

#define REDUCE_LARGE
REDUCEFUNC(ForcesNTPLarge, 3, 2)
#include "kNLRF.h"

#ifdef MPI
#define MPI_REDUCE_NODE0
REDUCEFUNC(ForcesNTPLargeNode0, 3, 2)
#include "kNLRF.h"
#undef MPI_REDUCE_NODE0
#endif
#undef REDUCE_LARGE

#ifdef MPI
#define MPI_REDUCE_NODE0
REDUCEFUNC(ForcesNTPNode0, 3, 2)
#include "kNLRF.h"
#undef MPI_REDUCE_NODE0
#endif
#undef REDUCE_NTP
#undef REDUCE_YDIVISOR
#undef REDUCE_XDIVISOR




#define REDUCE_YDIVISOR 3
#define REDUCE_XDIVISOR 5
REDUCEFUNC(Forces, 3, 5)
#include "kNLRF.h"

#define REDUCE_NTP
REDUCEFUNC(ForcesNTP, 3, 5)
#include "kNLRF.h"

#define REDUCE_LARGE
REDUCEFUNC(ForcesNTPLarge, 3, 5)
#include "kNLRF.h"

#ifdef MPI
#define MPI_REDUCE_NODE0
REDUCEFUNC(ForcesNTPLargeNode0, 3, 5)
#include "kNLRF.h"
#undef MPI_REDUCE_NODE0
#endif
#undef REDUCE_LARGE

#ifdef MPI
#define MPI_REDUCE_NODE0
REDUCEFUNC(ForcesNTPNode0, 3, 5)
#include "kNLRF.h"
#undef MPI_REDUCE_NODE0
#endif
#undef REDUCE_NTP
#undef REDUCE_YDIVISOR
#undef REDUCE_XDIVISOR


#define REDUCE_YDIVISOR 4
#define REDUCE_XDIVISOR 5
REDUCEFUNC(Forces, 4, 5)
#include "kNLRF.h"

#define REDUCE_NTP
REDUCEFUNC(ForcesNTP, 4, 5)
#include "kNLRF.h"

#define REDUCE_LARGE
REDUCEFUNC(ForcesNTPLarge, 4, 5)
#include "kNLRF.h"

#ifdef MPI
#define MPI_REDUCE_NODE0
REDUCEFUNC(ForcesNTPLargeNode0, 4, 5)
#include "kNLRF.h"
#undef MPI_REDUCE_NODE0
#endif
#undef REDUCE_LARGE

#ifdef MPI
#define MPI_REDUCE_NODE0
REDUCEFUNC(ForcesNTPNode0, 4, 5)
#include "kNLRF.h"
#undef MPI_REDUCE_NODE0
#endif
#undef REDUCE_NTP
#undef REDUCE_YDIVISOR
#undef REDUCE_XDIVISOR


#define REDUCE_YDIVISOR 4
#define REDUCE_XDIVISOR 7
REDUCEFUNC(Forces, 4, 7)
#include "kNLRF.h"

#define REDUCE_NTP
REDUCEFUNC(ForcesNTP, 4, 7)
#include "kNLRF.h"

#define REDUCE_LARGE
REDUCEFUNC(ForcesNTPLarge, 4, 7)
#include "kNLRF.h"

#ifdef MPI
#define MPI_REDUCE_NODE0
REDUCEFUNC(ForcesNTPLargeNode0, 4, 7)
#include "kNLRF.h"
#undef MPI_REDUCE_NODE0
#endif
#undef REDUCE_LARGE

#ifdef MPI
#define MPI_REDUCE_NODE0
REDUCEFUNC(ForcesNTPNode0, 4, 7)
#include "kNLRF.h"
#undef MPI_REDUCE_NODE0
#endif
#undef REDUCE_NTP
#undef REDUCE_YDIVISOR
#undef REDUCE_XDIVISOR


#define REDUCE_YDIVISOR 4
#define REDUCE_XDIVISOR 14
REDUCEFUNC(Forces, 4, 14)
#include "kNLRF.h"

#define REDUCE_NTP
REDUCEFUNC(ForcesNTP, 4, 14)
#include "kNLRF.h"

#define REDUCE_LARGE
REDUCEFUNC(ForcesNTPLarge, 4, 14)
#include "kNLRF.h"

#ifdef MPI
#define MPI_REDUCE_NODE0
REDUCEFUNC(ForcesNTPLargeNode0, 4, 14)
#include "kNLRF.h"
#undef MPI_REDUCE_NODE0
#endif
#undef REDUCE_LARGE

#ifdef MPI
#define MPI_REDUCE_NODE0
REDUCEFUNC(ForcesNTPNode0, 4, 14)
#include "kNLRF.h"
#undef MPI_REDUCE_NODE0
#endif
#undef REDUCE_NTP
#undef REDUCE_YDIVISOR
#undef REDUCE_XDIVISOR

#define REDUCE_YDIVISOR 8
#define REDUCE_XDIVISOR 14
REDUCEFUNC(Forces, 8, 14)
#include "kNLRF.h"

#define REDUCE_NTP
REDUCEFUNC(ForcesNTP, 8, 14)
#include "kNLRF.h"

#define REDUCE_LARGE
REDUCEFUNC(ForcesNTPLarge, 8, 14)
#include "kNLRF.h"

#ifdef MPI
#define MPI_REDUCE_NODE0
REDUCEFUNC(ForcesNTPLargeNode0, 8, 14)
#include "kNLRF.h"
#undef MPI_REDUCE_NODE0
#endif
#undef REDUCE_LARGE

#ifdef MPI
#define MPI_REDUCE_NODE0
REDUCEFUNC(ForcesNTPNode0, 8, 14)
#include "kNLRF.h"
#undef MPI_REDUCE_NODE0
#endif
#undef REDUCE_NTP
#undef REDUCE_YDIVISOR
#undef REDUCE_XDIVISOR

__global__ void kReduceForces_kernel()
{
    unsigned int pos                                    = blockIdx.x * blockDim.x + threadIdx.x;      
    while (pos < cSim.stride3)
    {
        unsigned int pos1                               = pos;
        PMEDouble force                                 = (PMEDouble)0.0;
        while (pos1 < cSim.stride3 * cSim.maxForceBuffers)
        {
            force                                      += cSim.pForceBuffer[pos1];
            pos1                                       += cSim.stride3;
        }
        cSim.pForce[pos]                                = force;
        pos                                            += blockDim.x * gridDim.x;
    }
}

void SetNLReduceForcesKernel(gpuContext gpu)
{
    unsigned int reductionType                          = gpu->sim.NLYDivisor * 100 + gpu->sim.NLXDivisor;
    if (gpu->sim.ntp > 0)
    {
        if (gpu->sim.NLCellBuffers >= gpu->sim.maxForceBuffers)
        {
#ifdef MPI
            if ((gpu->gpuID == 0) & !gpu->sim.bIPSActive)
            {
                switch (reductionType)
                {
                    case 101:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTPNode0_1_1_kernel;
                        break;
                        
                    case 201:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTPNode0_2_1_kernel;
                        break;
                        
                    case 301:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTPNode0_3_1_kernel;
                        break;
                        
                    case 302:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTPNode0_3_2_kernel;
                        break;
                    case 305:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTPNode0_3_5_kernel;
                        break;
                        
                    case 405:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTPNode0_4_5_kernel;
                        break;
                        
                    case 407:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTPNode0_4_7_kernel;
                        break;
                        
                    case 414:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTPNode0_4_14_kernel;
                        break;  
                                      
                    case 814:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTPNode0_8_14_kernel;
                        break;                                
                }
            }
            else
#endif            
            {
                switch (reductionType)
                {
                    case 101:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTP_1_1_kernel;
                        break;
                        
                    case 201:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTP_2_1_kernel;
                        break;
                        
                    case 301:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTP_3_1_kernel;
                        break;
                        
                    case 302:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTP_3_2_kernel;
                        break;
                    case 305:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTP_3_5_kernel;
                        break;
                        
                    case 405:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTP_4_5_kernel;
                        break;
                        
                    case 407:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTP_4_7_kernel;
                        break;
                        
                    case 414:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTP_4_14_kernel;
                        break;  
                                      
                    case 814:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTP_8_14_kernel;
                        break;                                
                }                       
            }
        }
        else
        {
#ifdef MPI
            if ((gpu->gpuID == 0) & !gpu->sim.bIPSActive)
            {
                switch (reductionType)
                {
                    case 101:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTPLargeNode0_1_1_kernel;
                        break;
                        
                    case 201:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTPLargeNode0_2_1_kernel;
                        break;
                        
                    case 301:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTPLargeNode0_3_1_kernel;
                        break;
                        
                    case 302:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTPLargeNode0_3_2_kernel;
                        break;
                    case 305:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTPLargeNode0_3_5_kernel;
                        break;
                        
                    case 405:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTPLargeNode0_4_5_kernel;
                        break;
                        
                    case 407:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTPLargeNode0_4_7_kernel;
                        break;
                        
                    case 414:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTPLargeNode0_4_14_kernel;
                        break;  
                                      
                    case 814:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTPLargeNode0_8_14_kernel;
                        break;                                
                }
            }
            else
#endif            
            {
                switch (reductionType)
                {
                    case 101:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTPLarge_1_1_kernel;
                        break;
                        
                    case 201:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTPLarge_2_1_kernel;
                        break;
                        
                    case 301:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTPLarge_3_1_kernel;
                        break;
                        
                    case 302:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTPLarge_3_2_kernel;
                        break;
                    case 305:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTPLarge_3_5_kernel;
                        break;
                        
                    case 405:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTPLarge_4_5_kernel;
                        break;
                        
                    case 407:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTPLarge_4_7_kernel;
                        break;
                        
                    case 414:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTPLarge_4_14_kernel;
                        break;  
                                      
                    case 814:
                        spNLReduceForcesKernel              = &kNLReduceForcesNTPLarge_8_14_kernel;
                        break;                                
                }                       
            }        
        
        }
    
    }
    else
    {
        switch (reductionType)
        {
            case 101:
                spNLReduceForcesKernel                      = &kNLReduceForces_1_1_kernel;
                break;
                
            case 201:
                spNLReduceForcesKernel                      = &kNLReduceForces_2_1_kernel;
                break;
                
            case 301:
                spNLReduceForcesKernel                      = &kNLReduceForces_3_1_kernel;
                break;
                
            case 302:
                spNLReduceForcesKernel                      = &kNLReduceForces_3_2_kernel;
                break;
            case 305:
                spNLReduceForcesKernel                      = &kNLReduceForces_3_5_kernel;
                break;
                
            case 405:
                spNLReduceForcesKernel                      = &kNLReduceForces_4_5_kernel;
                break;
                
            case 407:
                spNLReduceForcesKernel                      = &kNLReduceForces_4_7_kernel;
                break;
                
            case 414:
                spNLReduceForcesKernel                      = &kNLReduceForces_4_14_kernel;
                break;  
                              
            case 814:
                spNLReduceForcesKernel                      = &kNLReduceForces_8_14_kernel;
                break;                                
        }
        
    }
    
}


void kReduceForces(gpuContext gpu)
{
    

    if (gpu->bNeighborList)
    {
        spNLReduceForcesKernel<<<gpu->blocks, gpu->NLReduceForcesThreadsPerBlock>>>();
    }
    else
    {
        kReduceForces_kernel<<<gpu->blocks, gpu->reduceForcesThreadsPerBlock>>>();
    }
    LAUNCHERROR("kReduceForces"); 
}
#endif // use_SPFP

#ifdef MPI
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_REDUCEFORCES_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_REDUCEFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_REDUCEFORCES_THREADS_PER_BLOCK, 1)
#endif
__global__ void kTransposeForces_kernel()
{
#if (__CUDA_ARCH__ >= 300)
    __shared__ volatile PMEDouble sForce[3 * SM_3X_REDUCEFORCES_THREADS_PER_BLOCK];
#elif (__CUDA_ARCH__ >= 200)
    __shared__ volatile PMEDouble sForce[3 * SM_2X_REDUCEFORCES_THREADS_PER_BLOCK];
#else
    __shared__ volatile PMEDouble sForce[3 * SM_13_REDUCEFORCES_THREADS_PER_BLOCK];
#endif 
    int opos                                            = blockIdx.x * blockDim.x + threadIdx.x;
    while (opos < cSim.reducedAtoms)
    {
        int rpos                                        = opos + cSim.minReducedAtom;
        rpos                                            = (rpos < cSim.paddedNumberOfAtoms) ? rpos : rpos - cSim.paddedNumberOfAtoms;
        PMEDouble forceX                                = cSim.pForceX[rpos];
        PMEDouble forceY                                = cSim.pForceY[rpos];
        PMEDouble forceZ                                = cSim.pForceZ[rpos];
        volatile PMEDouble* psForce                     = &sForce[3 * threadIdx.x];
        *psForce++                                      = forceX;
        *psForce++                                      = forceY;
        *psForce                                        = forceZ;        
        int tgx                                         = threadIdx.x & GRIDBITSMASK;
        int tbx                                         = threadIdx.x - tgx;
        psForce                                         = &sForce[3 * tbx + tgx];     
        PMEDouble* pOutForce                            = &cSim.pOutForce[3 * (opos - tgx) + tgx];      
        *pOutForce                                      = *psForce;
        pOutForce                                      += GRID;
        psForce                                        += GRID;
        *pOutForce                                      = *psForce;
        pOutForce                                      += GRID;
        psForce                                        += GRID;
        *pOutForce                                      = *psForce;
        opos                                           += blockDim.x * gridDim.x;
    }
}

void kTransposeForces(gpuContext gpu)
{
    //printf("%06d %06d %06d %06d\n", gpu->gpuID, gpu->sim.minReducedAtom, gpu->sim.maxReducedAtom, gpu->sim.reducedAtoms);
    kTransposeForces_kernel<<<gpu->blocks, gpu->reduceForcesThreadsPerBlock>>>();
    LAUNCHERROR("kTransposeForces");  
}
#endif


// KUpdate.cu
static __constant__ PMEDouble boltz2                    = 0.00831441 * 0.5 / 4.184;

struct COM {
    PMEFloat xmin;
    PMEFloat ymin;
    PMEFloat zmin;
    PMEFloat xmax;
    PMEFloat ymax;
    PMEFloat zmax;
};

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_UPDATE_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_UPDATE_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_UPDATE_THREADS_PER_BLOCK, 1)
#endif
kRecenter_Molecule1_kernel()
{
#if (__CUDA_ARCH__ >= 300)
    __shared__ COM sA[SM_3X_UPDATE_THREADS_PER_BLOCK];
#elif (__CUDA_ARCH__ >= 200)
    __shared__ COM sA[SM_2X_UPDATE_THREADS_PER_BLOCK];
#else
    __shared__ COM sA[SM_13_UPDATE_THREADS_PER_BLOCK];
#endif 
    PMEFloat xmin                                       = (PMEFloat)999999999999.0;
    PMEFloat ymin                                       = (PMEFloat)999999999999.0;
    PMEFloat zmin                                       = (PMEFloat)999999999999.0;
    PMEFloat xmax                                       = (PMEFloat)-999999999999.0;
    PMEFloat ymax                                       = (PMEFloat)-999999999999.0;
    PMEFloat zmax                                       = (PMEFloat)-999999999999.0;
    unsigned int pos                                    = blockIdx.x * blockDim.x + threadIdx.x;
    
    
    // Perform individual sums
    while (pos < cSim.atoms)
    {
        PMEFloat2 xy                                    = cSim.pAtomXYSP[pos];
        PMEFloat z                                      = cSim.pAtomZSP[pos]; 
        xmax                                            = max(xy.x, xmax);
        xmin                                            = min(xy.x, xmin);
        ymax                                            = max(xy.y, ymax);
        ymin                                            = min(xy.y, ymin);
        zmax                                            = max(z,    zmax);
        zmin                                            = min(z,    zmin);
        pos                                            += blockDim.x * gridDim.x;
    }
    
    // Perform local reduction to thread 0
    sA[threadIdx.x].xmin                                = xmin;
    sA[threadIdx.x].ymin                                = ymin;
    sA[threadIdx.x].zmin                                = zmin;
    sA[threadIdx.x].xmax                                = xmax;
    sA[threadIdx.x].ymax                                = ymax;
    sA[threadIdx.x].zmax                                = zmax;
    __syncthreads();
    unsigned int m                                      = 1;
    while (m < blockDim.x)
    {
        int p                                           = threadIdx.x + m;
        PMEFloat xmin                                   = ((p < blockDim.x) ? sA[p].xmin : (PMEFloat)9999999999.0);
        PMEFloat ymin                                   = ((p < blockDim.x) ? sA[p].ymin : (PMEFloat)9999999999.0);
        PMEFloat zmin                                   = ((p < blockDim.x) ? sA[p].zmin : (PMEFloat)9999999999.0);
        PMEFloat xmax                                   = ((p < blockDim.x) ? sA[p].xmax : (PMEFloat)-9999999999.0);
        PMEFloat ymax                                   = ((p < blockDim.x) ? sA[p].ymax : (PMEFloat)-9999999999.0);
        PMEFloat zmax                                   = ((p < blockDim.x) ? sA[p].zmax : (PMEFloat)-9999999999.0);
        __syncthreads();
        sA[threadIdx.x].xmin                            = min(sA[threadIdx.x].xmin, xmin);
        sA[threadIdx.x].ymin                            = min(sA[threadIdx.x].ymin, ymin);
        sA[threadIdx.x].zmin                            = min(sA[threadIdx.x].zmin, zmin);
        sA[threadIdx.x].xmax                            = max(sA[threadIdx.x].xmax, xmax);
        sA[threadIdx.x].ymax                            = max(sA[threadIdx.x].ymax, ymax);
        sA[threadIdx.x].zmax                            = max(sA[threadIdx.x].zmax, zmax);
        __syncthreads();
        m                                              *= 2;
    }
    
    // Output sum if thread 0
    if (threadIdx.x == 0)
    {
        cSim.pXMin[blockIdx.x]                          = sA[threadIdx.x].xmin;
        cSim.pYMin[blockIdx.x]                          = sA[threadIdx.x].ymin;
        cSim.pZMin[blockIdx.x]                          = sA[threadIdx.x].zmin; 
        cSim.pXMax[blockIdx.x]                          = sA[threadIdx.x].xmax;
        cSim.pYMax[blockIdx.x]                          = sA[threadIdx.x].ymax;
        cSim.pZMax[blockIdx.x]                          = sA[threadIdx.x].zmax; 
    }
    
 
}

__global__ void 
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_UPDATE_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_UPDATE_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_UPDATE_THREADS_PER_BLOCK, 1)
#endif
kRecenter_Molecule2_kernel()
{
#if (__CUDA_ARCH__ >= 300)
    __shared__ COM sA[SM_3X_UPDATE_THREADS_PER_BLOCK];
#elif (__CUDA_ARCH__ >= 200)
    __shared__ COM sA[SM_2X_UPDATE_THREADS_PER_BLOCK];
#else
    __shared__ COM sA[SM_13_UPDATE_THREADS_PER_BLOCK];
#endif 
    // Read in local offsets
    unsigned int pos                                    = threadIdx.x;
    while (pos < gridDim.x)
    {
        sA[pos].xmin                                    = cSim.pXMin[pos]; 
        sA[pos].ymin                                    = cSim.pYMin[pos]; 
        sA[pos].zmin                                    = cSim.pZMin[pos]; 
        sA[pos].xmax                                    = cSim.pXMax[pos]; 
        sA[pos].ymax                                    = cSim.pYMax[pos]; 
        sA[pos].zmax                                    = cSim.pZMax[pos]; 
        pos                                            += blockDim.x;
    }
    __syncthreads();
    
    // Perform local reduction to thread 0
    unsigned int m                                      = 1;
    while (m < gridDim.x)
    {
        int p                                           = threadIdx.x + m;
        PMEFloat xmin                                   = ((p < gridDim.x) ? sA[p].xmin : (PMEFloat)9999999999.0);
        PMEFloat ymin                                   = ((p < gridDim.x) ? sA[p].ymin : (PMEFloat)9999999999.0);
        PMEFloat zmin                                   = ((p < gridDim.x) ? sA[p].zmin : (PMEFloat)9999999999.0);
        PMEFloat xmax                                   = ((p < gridDim.x) ? sA[p].xmax : (PMEFloat)-9999999999.0);
        PMEFloat ymax                                   = ((p < gridDim.x) ? sA[p].ymax : (PMEFloat)-9999999999.0);
        PMEFloat zmax                                   = ((p < gridDim.x) ? sA[p].zmax : (PMEFloat)-9999999999.0);
        __syncthreads();
        sA[threadIdx.x].xmin                            = min(sA[threadIdx.x].xmin, xmin);
        sA[threadIdx.x].ymin                            = min(sA[threadIdx.x].ymin, ymin);
        sA[threadIdx.x].zmin                            = min(sA[threadIdx.x].zmin, zmin);
        sA[threadIdx.x].xmax                            = max(sA[threadIdx.x].xmax, xmax);
        sA[threadIdx.x].ymax                            = max(sA[threadIdx.x].ymax, ymax);
        sA[threadIdx.x].zmax                            = max(sA[threadIdx.x].zmax, zmax);
        __syncthreads();
        m                                              *= 2;
    }
    PMEDouble xcenter                                   = (PMEFloat)-0.5 * (sA[0].xmin + sA[0].xmax);
    PMEDouble ycenter                                   = (PMEFloat)-0.5 * (sA[0].ymin + sA[0].ymax);
    PMEDouble zcenter                                   = (PMEFloat)-0.5 * (sA[0].zmin + sA[0].zmax);
      
    // Perform individual sums
    pos                                                 = blockIdx.x * blockDim.x + threadIdx.x;
    while (pos < cSim.atoms)
    {
        PMEDouble x                                     = cSim.pAtomX[pos];
        PMEDouble y                                     = cSim.pAtomY[pos];
        PMEDouble z                                     = cSim.pAtomZ[pos];
        x                                              += xcenter;
        y                                              += ycenter; 
        z                                              += zcenter; 
        
        PMEFloat2 xy                                    = {x, y};
        cSim.pAtomX[pos]                                = x;
        cSim.pAtomY[pos]                                = y;
        cSim.pAtomZ[pos]                                = z;
        cSim.pAtomXYSP[pos]                             = xy;
        cSim.pAtomZSP[pos]                              = z;
        pos                                            += blockDim.x * gridDim.x;
    }  
    
    // Fix restraints
    pos                                                 = blockIdx.x * blockDim.x + threadIdx.x;
    while (pos < cSim.constraints)
    {
        PMEDouble2 constraint1                          = cSim.pConstraint1[pos];
        PMEDouble2 constraint2                          = cSim.pConstraint2[pos];
        constraint1.y                                  += xcenter;
        constraint2.x                                  += ycenter;
        constraint2.y                                  += zcenter;
        cSim.pConstraint1[pos]                          = constraint1;
        cSim.pConstraint2[pos]                          = constraint2;
        pos                                            += blockDim.x * gridDim.x;
    }
    
}


extern "C" void kRecenter_Molecule(gpuContext gpu)
{
    kRecenter_Molecule1_kernel<<<gpu->blocks, gpu->updateThreadsPerBlock>>>();
    LAUNCHERROR("kRecenter_Molecule");
    kRecenter_Molecule2_kernel<<<gpu->blocks, gpu->updateThreadsPerBlock>>>();
    LAUNCHERROR("kRecenter_Molecule");
}

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_UPDATE_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_UPDATE_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_UPDATE_THREADS_PER_BLOCK, 1)
#endif
kUpdate_kernel(PMEDouble dt)
#include "kU.h"

#define LANGEVIN
__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_UPDATE_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_UPDATE_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_UPDATE_THREADS_PER_BLOCK, 1)
#endif
kLangevinUpdate_kernel(PMEDouble dt, PMEDouble temp0, PMEDouble gamma_ln)
#include "kU.h"
#undef LANGEVIN

#define PME
__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_UPDATE_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_UPDATE_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_UPDATE_THREADS_PER_BLOCK, 1)
#endif
kPMEUpdate_kernel(PMEDouble dt)
#include "kU.h"

#define LANGEVIN
__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_UPDATE_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_UPDATE_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_UPDATE_THREADS_PER_BLOCK, 1)
#endif
kPMELangevinUpdate_kernel(PMEDouble dt, PMEDouble temp0, PMEDouble gamma_ln)
#include "kU.h"
#undef LANGEVIN
#undef PME



void kUpdate(gpuContext gpu, PMEDouble dt, PMEDouble temp0, PMEDouble gamma_ln)
{
    // Choose Langevin update if necessary
    if (gpu->ntt == 3)
    {
        // Update random numbers if necessary
        if (gpu->randomCounter >= gpu->sim.randomSteps)
        {
#ifdef CPU_RANDOMS
            cpu_kRandom(gpu);
#else
            kRandom(gpu);
#endif
            gpu->randomCounter = 0;
        }
        
        if (gpu->bNeighborList)
            kPMELangevinUpdate_kernel<<<gpu->blocks, gpu->updateThreadsPerBlock>>>(dt, temp0, gamma_ln);
        else
            kLangevinUpdate_kernel<<<gpu->blocks, gpu->updateThreadsPerBlock>>>(dt, temp0, gamma_ln);
            
        gpu->randomCounter++; 
    }
    else
    {
        if (gpu->bNeighborList)
            kPMEUpdate_kernel<<<gpu->blocks, gpu->updateThreadsPerBlock>>>(dt);
        else
            kUpdate_kernel<<<gpu->blocks, gpu->updateThreadsPerBlock>>>(dt);
    }   
    LAUNCHERROR("kUpdate"); 
}


__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_UPDATE_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_UPDATE_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_UPDATE_THREADS_PER_BLOCK, 1)
#endif
kResetVelocities_kernel(PMEDouble temp, PMEDouble half_dtx)
{
    unsigned int pos                        = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int increment                  = gridDim.x * blockDim.x;
    unsigned int rpos                       = cSim.pRandomPos[blockIdx.x];
    PMEDouble boltz                         = 8.31441e-3 * temp / 4.184;
    while (pos < cSim.atoms)
    {  
        PMEDouble invMass                   = cSim.pAtomInvMass[pos];    
        PMEDouble forceX                    = cSim.pForceX[pos];
        PMEDouble forceY                    = cSim.pForceY[pos];
        PMEDouble forceZ                    = cSim.pForceZ[pos];
        PMEDouble velX, velY, velZ;
        
        // Zero velocities if it's really cold
        if (temp < 1.0e-6)
        {
            velX                            = (PMEDouble)0.0;
            velY                            = (PMEDouble)0.0;
            velZ                            = (PMEDouble)0.0;
        }
        else
        {
            PMEDouble gaussX                = cSim.pRandomX[rpos + pos];
            PMEDouble gaussY                = cSim.pRandomY[rpos + pos];
            PMEDouble gaussZ                = cSim.pRandomZ[rpos + pos]; 
            PMEDouble sd                    = sqrt(boltz * invMass);        
            velX                            = sd * gaussX;
            velY                            = sd * gaussY;
            velZ                            = sd * gaussZ;        
        }
             
        // Back velocities up a half-step
        PMEDouble wfac                      = invMass * half_dtx;
        velX                               -= forceX * wfac;
        velY                               -= forceY * wfac;
        velZ                               -= forceZ * wfac;
       
        // Write final velocities
        cSim.pVelX[pos]                     = velX;
        cSim.pVelY[pos]                     = velY;
        cSim.pVelZ[pos]                     = velZ;
        pos                                += increment;
    }
    
    // Update RNG position
    __syncthreads();
    if (threadIdx.x == 0)
        cSim.pRandomPos[blockIdx.x]         = rpos + cSim.paddedNumberOfAtoms;    
}

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_UPDATE_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_UPDATE_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_UPDATE_THREADS_PER_BLOCK, 1)
#endif
kNLResetVelocities_kernel(PMEDouble temp, PMEDouble half_dtx)
{
    unsigned int pos                        = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int increment                  = gridDim.x * blockDim.x;
    unsigned int rpos                       = cSim.pRandomPos[blockIdx.x];
    PMEDouble boltz                         = 8.31441e-3 * temp / 4.184;
    while (pos < cSim.atoms)
    {  
        PMEDouble invMass                   = cSim.pImageInvMass[pos];    
        PMEDouble forceX                    = cSim.pForceX[pos];
        PMEDouble forceY                    = cSim.pForceY[pos];
        PMEDouble forceZ                    = cSim.pForceZ[pos];
        PMEDouble velX, velY, velZ;
        
        // Zero velocities if it's really cold
        if (temp < 1.0e-6)
        {
            velX                            = (PMEDouble)0.0;
            velY                            = (PMEDouble)0.0;
            velZ                            = (PMEDouble)0.0;
        }
        else
        {
            PMEDouble gaussX                = cSim.pRandomX[rpos + pos];
            PMEDouble gaussY                = cSim.pRandomY[rpos + pos];
            PMEDouble gaussZ                = cSim.pRandomZ[rpos + pos]; 
            PMEDouble sd                    = sqrt(boltz * invMass);        
            velX                            = sd * gaussX;
            velY                            = sd * gaussY;
            velZ                            = sd * gaussZ;        
        }
        
        
        // Back velocities up a half-step
        PMEDouble wfac                      = invMass * half_dtx;
        velX                               -= forceX * wfac;
        velY                               -= forceY * wfac;
        velZ                               -= forceZ * wfac;
        
        // Write final velocities
        cSim.pImageVelX[pos]                = velX;
        cSim.pImageVelY[pos]                = velY;
        cSim.pImageVelZ[pos]                = velZ;
        pos                                += increment;
    }
    
    // Update RNG position
    __syncthreads();
    if (threadIdx.x == 0)
        cSim.pRandomPos[blockIdx.x]         = rpos + cSim.paddedNumberOfAtoms;    
}


void kResetVelocities(gpuContext gpu, double temp, double half_dtx)
{
  
    // Update random numbers if necessary
    if (gpu->randomCounter >= gpu->sim.randomSteps)
    {
#ifdef CPU_RANDOMS
        cpu_kRandom(gpu);
#else
        kRandom(gpu);
#endif
        gpu->randomCounter = 0;
    }
        
    if (gpu->bNeighborList)
        kNLResetVelocities_kernel<<<gpu->blocks, gpu->updateThreadsPerBlock>>>(temp, half_dtx);
    else
        kResetVelocities_kernel<<<gpu->blocks, gpu->updateThreadsPerBlock>>>(temp, half_dtx);
            
    gpu->randomCounter++; 
    LAUNCHERROR("kResetVelocities"); 
}

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_UPDATE_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_UPDATE_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_UPDATE_THREADS_PER_BLOCK, 1)
#endif
kRecalculateVelocities_kernel(PMEDouble dtx_inv)
{
    unsigned int pos                        = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int increment                  = gridDim.x * blockDim.x;

    while (pos < cSim.atoms)
    {  
        PMEDouble oldAtomX                  = cSim.pForceX[pos];
        PMEDouble atomX                     = cSim.pAtomX[pos];
        PMEDouble oldAtomY                  = cSim.pForceY[pos];
        PMEDouble atomY                     = cSim.pAtomY[pos];
        PMEDouble oldAtomZ                  = cSim.pForceZ[pos];
        PMEDouble atomZ                     = cSim.pAtomZ[pos];   
        PMEDouble velX                      = (atomX - oldAtomX) * dtx_inv;
        PMEDouble velY                      = (atomY - oldAtomY) * dtx_inv;
        PMEDouble velZ                      = (atomZ - oldAtomZ) * dtx_inv;
        cSim.pVelX[pos]                     = velX;
        cSim.pVelY[pos]                     = velY;
        cSim.pVelZ[pos]                     = velZ;
        pos                                += increment;
    }
}

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_UPDATE_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_UPDATE_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_UPDATE_THREADS_PER_BLOCK, 1)
#endif
kPMERecalculateVelocities_kernel(PMEDouble dtx_inv)
{
    unsigned int pos                                    = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int increment                              = gridDim.x * blockDim.x;

    while (pos < cSim.atoms)
    {  
        PMEDouble oldAtomX                  = cSim.pForceX[pos];
        PMEDouble atomX                     = cSim.pImageX[pos];
        PMEDouble oldAtomY                  = cSim.pForceY[pos];
        PMEDouble atomY                     = cSim.pImageY[pos];
        PMEDouble oldAtomZ                  = cSim.pForceZ[pos];
        PMEDouble atomZ                     = cSim.pImageZ[pos];   
        PMEDouble velX                      = (atomX - oldAtomX) * dtx_inv;
        PMEDouble velY                      = (atomY - oldAtomY) * dtx_inv;
        PMEDouble velZ                      = (atomZ - oldAtomZ) * dtx_inv;
        cSim.pImageVelX[pos]                = velX;
        cSim.pImageVelY[pos]                = velY;
        cSim.pImageVelZ[pos]                = velZ;
        pos                                += increment;
    }
}


void kRecalculateVelocities(gpuContext gpu, PMEDouble dtx_inv)
{
    if (gpu->bNeighborList)
    {
        kPMERecalculateVelocities_kernel<<<gpu->blocks, gpu->updateThreadsPerBlock>>>(dtx_inv);
    }
    else
    {
        kRecalculateVelocities_kernel<<<gpu->blocks, gpu->updateThreadsPerBlock>>>(dtx_inv);
    }
    LAUNCHERROR("kRecalculateVelocities");  
}

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_THREADS_PER_BLOCK, 1)
#endif
kCalculateKineticEnergy_kernel(PMEFloat c_ave)
{
extern __shared__ KineticEnergy sE[];

    PMEFloat eke                            = (PMEFloat)0.0;
    PMEFloat ekph                           = (PMEFloat)0.0;
    PMEFloat ekpbs                          = (PMEFloat)0.0;
    unsigned int pos                        = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Sum up kinetic energies
    while (pos < cSim.atoms)
    {
        PMEFloat mass                       = cSim.pAtomMass[pos];
        PMEFloat vx                         = cSim.pVelX[pos];
        PMEFloat lvx                        = cSim.pLVelX[pos];
        PMEFloat vy                         = cSim.pVelY[pos];
        PMEFloat lvy                        = cSim.pLVelY[pos];
        PMEFloat vz                         = cSim.pVelZ[pos];       
        PMEFloat lvz                        = cSim.pLVelZ[pos];
        PMEFloat svx                        = vx + lvx;
        PMEFloat svy                        = vy + lvy;
        PMEFloat svz                        = vz + lvz;
        eke                                += mass * (svx * svx + svy * svy + svz * svz);
        ekpbs                              += mass * (vx * lvx + vy * lvy + vz * lvz);
        ekph                               += mass * (vx * vx + vy * vy + vz * vz);
        pos                                += blockDim.x * gridDim.x;
    }       
    eke                                    *= (PMEFloat)0.125 * c_ave;
    ekph                                   *= (PMEFloat)0.5;
    ekpbs                                  *= (PMEFloat)0.5;
    sE[threadIdx.x].KE.EKE                  = eke;
    sE[threadIdx.x].KE.EKPH                 = ekph;
    sE[threadIdx.x].KE.EKPBS                = ekpbs;
        

    // Reduce per-thread kinetic energies
    __syncthreads();
    unsigned int m                          = 1;
    while (m < blockDim.x)
    {
        int p                               = threadIdx.x + m;
        eke                                 = ((p < blockDim.x) ? sE[p].KE.EKE : (PMEFloat)0.0f);
        ekph                                = ((p < blockDim.x) ? sE[p].KE.EKPH : (PMEFloat)0.0f);
        ekpbs                               = ((p < blockDim.x) ? sE[p].KE.EKPBS : (PMEFloat)0.0f);
        __syncthreads();
        sE[threadIdx.x].KE.EKE                += eke;
        sE[threadIdx.x].KE.EKPH               += ekph;
        sE[threadIdx.x].KE.EKPBS              += ekpbs;
        __syncthreads();
        m                                  *= 2;
    }

    // Save result
    if (threadIdx.x < 3)
    {
        cSim.pKineticEnergy[blockIdx.x].array[threadIdx.x]
                                            = sE[0].array[threadIdx.x];
    }
}

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_THREADS_PER_BLOCK, 1)
#endif
kPMECalculateKineticEnergy_kernel(PMEFloat c_ave)
{
extern __shared__ KineticEnergy sE[];

    PMEFloat eke                            = (PMEFloat)0.0;
    PMEFloat ekph                           = (PMEFloat)0.0;
    PMEFloat ekpbs                          = (PMEFloat)0.0;
    unsigned int pos                        = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Sum up kinetic energies
    while (pos < cSim.atoms)
    {
        PMEFloat mass                       = cSim.pImageMass[pos];
        PMEFloat vx                         = cSim.pImageVelX[pos];
        PMEFloat lvx                        = cSim.pImageLVelX[pos];
        PMEFloat vy                         = cSim.pImageVelY[pos];
        PMEFloat lvy                        = cSim.pImageLVelY[pos];
        PMEFloat vz                         = cSim.pImageVelZ[pos];       
        PMEFloat lvz                        = cSim.pImageLVelZ[pos];
        PMEFloat svx                        = vx + lvx;
        PMEFloat svy                        = vy + lvy;
        PMEFloat svz                        = vz + lvz;
        eke                                += mass * (svx * svx + svy * svy + svz * svz);
        ekpbs                              += mass * (vx * lvx + vy * lvy + vz * lvz);
        ekph                               += mass * (vx * vx + vy * vy + vz * vz);
        pos                                += blockDim.x * gridDim.x;
    }       
    eke                                    *= (PMEFloat)0.125 * c_ave;
    ekph                                   *= (PMEFloat)0.5;
    ekpbs                                  *= (PMEFloat)0.5;
    sE[threadIdx.x].KE.EKE                  = eke;
    sE[threadIdx.x].KE.EKPH                 = ekph;
    sE[threadIdx.x].KE.EKPBS                = ekpbs;   

    // Reduce per-thread kinetic energies
    __syncthreads();
    unsigned int m                          = 1;
    while (m < blockDim.x)
    {
        int p                               = threadIdx.x + m;
        eke                                 = ((p < blockDim.x) ? sE[p].KE.EKE : (PMEFloat)0.0);
        ekph                                = ((p < blockDim.x) ? sE[p].KE.EKPH : (PMEFloat)0.0);
        ekpbs                               = ((p < blockDim.x) ? sE[p].KE.EKPBS : (PMEFloat)0.0);
        __syncthreads();
        sE[threadIdx.x].KE.EKE             += eke;
        sE[threadIdx.x].KE.EKPH            += ekph;
        sE[threadIdx.x].KE.EKPBS           += ekpbs;
        __syncthreads();
        m                                  *= 2;
    }       
     
    // Save result
    if (threadIdx.x < 3)
    {
        cSim.pKineticEnergy[blockIdx.x].array[threadIdx.x]
                                            = sE[0].array[threadIdx.x];
    }
}

void kCalculateKineticEnergy(gpuContext gpu, PMEFloat c_ave)
{
    if (gpu->bNeighborList)
    {
        kPMECalculateKineticEnergy_kernel<<<gpu->blocks, gpu->threadsPerBlock, gpu->threadsPerBlock * sizeof(KineticEnergy)>>>(c_ave);
    }
    else
    {
        kCalculateKineticEnergy_kernel<<<gpu->blocks, gpu->threadsPerBlock, gpu->threadsPerBlock * sizeof(KineticEnergy)>>>(c_ave);
    }
    LAUNCHERROR("kCalculateKineticEnergy");
}

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_THREADS_PER_BLOCK, 1)
#endif
kScaleVelocities_kernel(PMEDouble scale)
{
    unsigned int pos                        = blockIdx.x * blockDim.x + threadIdx.x;
    
    while (pos < cSim.atoms)
    {
        double vx                           = cSim.pVelX[pos];
        double vy                           = cSim.pVelY[pos];
        double vz                           = cSim.pVelZ[pos];
        vx                                 *= scale;
        vy                                 *= scale;
        vz                                 *= scale;
        cSim.pVelX[pos]                     = vx;
        cSim.pVelY[pos]                     = vy;
        cSim.pVelZ[pos]                     = vz;
        pos                                += blockDim.x * gridDim.x;       
    }
}

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_THREADS_PER_BLOCK, 1)
#endif
kPMEScaleVelocities_kernel(PMEDouble scale)
{
    unsigned int pos                        = blockIdx.x * blockDim.x + threadIdx.x;
    
    while (pos < cSim.atoms)
    {
        double vx                           = cSim.pImageVelX[pos];
        double vy                           = cSim.pImageVelY[pos];
        double vz                           = cSim.pImageVelZ[pos];
        vx                                 *= scale;
        vy                                 *= scale;
        vz                                 *= scale;
        cSim.pImageVelX[pos]                = vx;
        cSim.pImageVelY[pos]                = vy;
        cSim.pImageVelZ[pos]                = vz;
        pos                                += blockDim.x * gridDim.x;       
    }
}

void kScaleVelocities(gpuContext gpu, PMEDouble scale)
{
    if (gpu->bNeighborList)
    {
        kPMEScaleVelocities_kernel<<<gpu->blocks, gpu->threadsPerBlock>>>(scale);
    }
    else
    {
        kScaleVelocities_kernel<<<gpu->blocks, gpu->threadsPerBlock>>>(scale);
    }
    LAUNCHERROR("kScaleVelocities");
}

#define KPMECALCULATECOM_KERNEL kPMECalculateCOM_kernel
#define KPMECALCULATESOLUTECOM_KERNEL kPMECalculateSoluteCOM_kernel
#define KPMECALCULATECOMKINETICENERGY_KERNEL kPMECalculateCOMKineticEnergy_kernel
#define KCALCULATEMOLECULARVIRIAL_KERNEL kCalculateMolecularVirial_kernel
#define KPRESSURESCALECOORDINATES_KERNEL kPressureScaleCoordinates_kernel
#include "kNTPKernels.h"
#undef KPMECALCULATECOM_KERNEL
#undef KPMECALCULATESOLUTECOM_KERNEL
#undef KPMECALCULATECOMKINETICENERGY_KERNEL
#undef KCALCULATEMOLECULARVIRIAL_KERNEL
#undef KPRESSURESCALECOORDINATES_KERNEL

#define NTP_LOTSOFMOLECULES
#define KPMECALCULATECOM_KERNEL kPMECalculateCOMLarge_kernel
#define KPMECALCULATESOLUTECOM_KERNEL kPMECalculateSoluteCOMLarge_kernel
#define KPMECALCULATECOMKINETICENERGY_KERNEL kPMECalculateCOMKineticEnergyLarge_kernel
#define KCALCULATEMOLECULARVIRIAL_KERNEL kCalculateMolecularVirialLarge_kernel
#define KPRESSURESCALECOORDINATES_KERNEL kPressureScaleCoordinatesLarge_kernel
#include "kNTPKernels.h"
#undef KPMECALCULATECOM_KERNEL
#undef KPMECALCULATESOLUTECOM_KERNEL
#undef KPMECALCULATECOMKINETICENERGY_KERNEL
#undef KCALCULATEMOLECULARVIRIAL_KERNEL
#undef KPRESSURESCALECOORDINATES_KERNEL
#undef NTP_LOTSOFMOLECULES

#include "kNTPCalls.h"
#include "kRandom.h"

#define EP_NEIGHBORLIST
__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_THREADS_PER_BLOCK, 1)
#endif
kNLOrientForces_kernel()
#include "kOrientForcesKernel.h"

#define EP_VIRIAL
__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_THREADS_PER_BLOCK, 1)
#endif
kNLOrientForcesVirial_kernel()
#include "kOrientForcesKernel.h"
#undef EP_VIRIAL
#undef EP_NEIGHBORLIST

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_THREADS_PER_BLOCK, 1)
#endif
kOrientForces_kernel()
#include "kOrientForcesKernel.h"

void kOrientForces(gpuContext gpu)
{
    if (gpu->bNeighborList)
    {
        if (gpu->sim.ntp > 0)
            kNLOrientForcesVirial_kernel<<<gpu->blocks, gpu->threadsPerBlock>>>();
        else
            kNLOrientForces_kernel<<<gpu->blocks, gpu->threadsPerBlock>>>();
    }
    else
        kOrientForces_kernel<<<gpu->blocks, gpu->threadsPerBlock>>>();
    LAUNCHERROR("kOrientForces");
}


#define EP_NEIGHBORLIST
__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_THREADS_PER_BLOCK, 1)
#endif
kNLLocalToGlobal_kernel()
#include "kLocalToGlobalKernel.h"
#undef EP_NEIGHBORLIST

__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_THREADS_PER_BLOCK, 1)
#endif
kLocalToGlobal_kernel()
#include "kLocalToGlobalKernel.h"

void kLocalToGlobal(gpuContext gpu)
{
    if (gpu->bNeighborList)
    {
        kNLLocalToGlobal_kernel<<<gpu->blocks, gpu->threadsPerBlock>>>();
    }
    else
        kLocalToGlobal_kernel<<<gpu->blocks, gpu->threadsPerBlock>>>();
    LAUNCHERROR("kLocalToGlobal");
}

