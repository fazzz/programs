#include "hip/hip_runtime.h"
/***************************************************/
/*                                                 */
/*      AMBER NVIDIA CUDA CPU IMPLEMENTATION       */
/*                 PMEMD VERSION                   */
/*                     2010                        */
/*                      by                         */
/*             Scott Le Grand (NVIDIA)             */
/*               Duncan Poole (NVIDIA)             */
/*                Ross Walker (SDSC)               */
/*                                                 */
/***************************************************/

#include <hip/hip_runtime.h>
#include "gpu.h"
#include "ptxmacros.h"
static __constant__ cudaSimulation cSim;
static __constant__ PMEFloat ta                 = (PMEFloat)(1.0 / 3.0);
static __constant__ PMEFloat tb                 = (PMEFloat)(2.0 / 5.0);
static __constant__ PMEFloat tc                 = (PMEFloat)(3.0 / 7.0);
static __constant__ PMEFloat td                 = (PMEFloat)(4.0 / 9.0);
static __constant__ PMEFloat tdd                = (PMEFloat)(5.0 / 11.0);

void SetkCalculateGBBornRadiiSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

void GetkCalculateGBBornRadiiSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

__global__ void 
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_GBBORNRADII_THREADS_PER_BLOCK, SM_3X_GBBORNRADII_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_GBBORNRADII_THREADS_PER_BLOCK, SM_2X_GBBORNRADII_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_GBBORNRADII_THREADS_PER_BLOCK, SM_13_GBBORNRADII_BLOCKS_MULTIPLIER)
#endif
kCalculateGBBornRadii_kernel()
#include "kCalculateGBBornRadii.h"

#define GB_IGB78
__global__ void 
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_GBBORNRADII_THREADS_PER_BLOCK, SM_3X_GBBORNRADII_BLOCKS_MULTIPLIER)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_GBBORNRADII_THREADS_PER_BLOCK, SM_2X_GBBORNRADII_BLOCKS_MULTIPLIER)
#else
__launch_bounds__(SM_13_GBBORNRADIIIGB78_THREADS_PER_BLOCK, SM_13_GBBORNRADII_BLOCKS_MULTIPLIER)
#endif
kCalculateGBBornRadiiIGB78_kernel()
#include "kCalculateGBBornRadii.h"
#undef GB_IGB78

void kCalculateGBBornRadiiInitKernels(gpuContext gpu)
{
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculateGBBornRadii_kernel), hipSharedMemBankSizeEightByte);
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculateGBBornRadiiIGB78_kernel), hipSharedMemBankSizeEightByte);
    if (gpu->sm_version >= SM_3X)
    {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kCalculateGBBornRadii_kernel), hipFuncCachePreferL1);
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kCalculateGBBornRadiiIGB78_kernel), hipFuncCachePreferL1);
    }
}

void kCalculateGBBornRadii(gpuContext gpu)
{
    if ((gpu->sim.igb == 7) || (gpu->sim.igb == 8))
        kCalculateGBBornRadiiIGB78_kernel<<<gpu->GBBornRadiiBlocks, gpu->GBBornRadiiIGB78ThreadsPerBlock>>>();
    else
        kCalculateGBBornRadii_kernel<<<gpu->GBBornRadiiBlocks, gpu->GBBornRadiiThreadsPerBlock>>>();
    LAUNCHERROR("kCalculateGBBornRadii");
}

__global__ void 
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_REDUCEBUFFER_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_REDUCEBUFFER_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_REDUCEBUFFER_THREADS_PER_BLOCK, 1)
#endif
kReduceGBBornRadii_kernel()
{
#ifndef MPI
    bool bIGB2578                                       = (cSim.igb == 2) || (cSim.igb == 5) || (cSim.igb == 7) || (cSim.igb == 8);
#endif
    unsigned int pos                                    = blockIdx.x * blockDim.x + threadIdx.x;   
    while (pos < cSim.atoms)
    {
        PMEDouble reff_i                                = (PMEDouble)0.0;
#ifndef MPI      
        PMEDouble rborn_i                               = cSim.pAtomRBorn[pos];
#endif

#ifdef use_SPFP
        reff_i                                          = (PMEDouble)cSim.pReffAccumulator[pos] * ONEOVERFORCESCALE;
#else
        unsigned int pos1                               = pos;
        while (pos1 < cSim.stride * cSim.nonbondForceBuffers)
        {
            reff_i                                     += cSim.pReffBuffer[pos1];
            pos1                                       += cSim.stride;
        }
#endif

#ifndef MPI        
        // Process final Born Radii
        PMEDouble ri                                    = rborn_i - cSim.offset;
        PMEDouble ri1i                                  = (PMEDouble)1.0 / ri;
       
        if (bIGB2578)
        {
            // apply the new Onufriev "gbalpha, gbbeta, gbgamma" correction:
            PMEDouble psi_i                             = -ri * reff_i;
            if (cSim.igb == 8)
                reff_i                                  = ri1i - tanh((cSim.pgb_alpha[pos] + cSim.pgb_gamma[pos] * psi_i * psi_i - cSim.pgb_beta[pos] * psi_i) * psi_i) / rborn_i;
            else
                reff_i                                  = ri1i - tanh((cSim.gb_alpha + cSim.gb_gamma * psi_i * psi_i - cSim.gb_beta * psi_i) * psi_i) / rborn_i;
            reff_i                                      = max(reff_i, (PMEDouble)1.0 / (PMEDouble)30.0);
            reff_i                                      = (PMEDouble)1.0 / reff_i;
            cSim.pPsi[pos]                              = psi_i;
        }
        else
        {
            // "standard" GB, including the "diagonal" term here:
            reff_i                                      = (PMEDouble)1.0 / (reff_i + ri1i); 
        }      

        cSim.pReffSP[pos]                               = reff_i;  
        cSim.pReff[pos]                                 = reff_i;      
#else
        cSim.pReffa[pos]                                = reff_i;
#endif
        pos                                            += blockDim.x * gridDim.x;
    }

}

void kReduceGBBornRadii(gpuContext gpu)
{
    kReduceGBBornRadii_kernel<<<gpu->blocks, gpu->reduceBufferThreadsPerBlock>>>();
    LAUNCHERROR("kReduceGBBornRadii");
}

#ifdef MPI
__global__ void
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_REDUCEBUFFER_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_REDUCEBUFFER_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_REDUCEBUFFER_THREADS_PER_BLOCK, 1)
#endif
kProcessGBBornRadii_kernel()
{
    unsigned int pos                                    = blockIdx.x * blockDim.x + threadIdx.x;
    bool bIGB2578                                       = (cSim.igb == 2) || (cSim.igb == 5) || (cSim.igb == 7) || (cSim.igb == 8);
    while (pos < cSim.atoms)
    {
        PMEDouble reff_i                                = cSim.pReffa[pos];
        PMEDouble rborn_i                               = cSim.pAtomRBorn[pos];
    
        // Process final Born Radii
        PMEDouble ri                                    = rborn_i - cSim.offset;
        PMEDouble ri1i                                  = (PMEDouble)1.0 / ri;
       
        if (bIGB2578)
        {
            // apply the new Onufriev "gbalpha, gbbeta, gbgamma" correction:
            PMEDouble psi_i                             = -ri * reff_i;
            if (cSim.igb == 8)
                reff_i                                  = ri1i - tanh((cSim.pgb_alpha[pos] + cSim.pgb_gamma[pos] * psi_i * psi_i - cSim.pgb_beta[pos] * psi_i) * psi_i) / rborn_i;
            else
                reff_i                                  = ri1i - tanh((cSim.gb_alpha + cSim.gb_gamma * psi_i * psi_i - cSim.gb_beta * psi_i) * psi_i) / rborn_i;
            reff_i                                      = max(reff_i, (PMEDouble)1.0 / (PMEDouble)30.0);
            reff_i                                      = (PMEDouble)1.0 / reff_i;
            cSim.pPsi[pos]                              = psi_i;
        }
        else
        {
            // "standard" GB, including the "diagonal" term here:
            reff_i                                      = (PMEDouble)1.0 / (reff_i + ri1i); 
        }      
        cSim.pReffSP[pos]                               = reff_i;  
        cSim.pReff[pos]                                 = reff_i;      
        pos                                            += blockDim.x * gridDim.x;
    }

}

void kProcessGBBornRadii(gpuContext gpu)
{
    kProcessGBBornRadii_kernel<<<gpu->blocks, gpu->reduceBufferThreadsPerBlock>>>();
    LAUNCHERROR("kProcessGBBornRadii");
}
#endif

__global__ void 
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_REDUCEBUFFER_THREADS_PER_BLOCK, 1)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_REDUCEBUFFER_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_REDUCEBUFFER_THREADS_PER_BLOCK, 1)
#endif
kClearGBBuffers_kernel()
{
    unsigned int pos                                    = blockIdx.x * blockDim.x + threadIdx.x;   
    while (pos < cSim.atoms)
    {

#ifdef use_SPFP
        cSim.pReffAccumulator[pos]                      = (PMEAccumulator)0;
        cSim.pSumdeijdaAccumulator[pos]                 = (PMEAccumulator)0;
#else
        unsigned int pos1                               = pos;
        while (pos1 < cSim.stride * cSim.nonbondForceBuffers)
        {
            cSim.pReffBuffer[pos1]                      = 0.0;
            cSim.pSumdeijdaBuffer[pos1]                 = 0.0;
            pos1                                       += cSim.stride;
        }
#endif

        pos                                            += blockDim.x * gridDim.x;
    }
}

void kClearGBBuffers(gpuContext gpu)
{
    kClearGBBuffers_kernel<<<gpu->blocks, gpu->reduceBufferThreadsPerBlock>>>();
    LAUNCHERROR("kClearGBBuffers");
}



