#include "hip/hip_runtime.h"
/***************************************************/
/*                                                 */
/*      AMBER NVIDIA CUDA CPU IMPLEMENTATION       */
/*                 PMEMD VERSION                   */
/*                     2010                        */
/*                      by                         */
/*             Scott Le Grand (NVIDIA)             */
/*               Duncan Poole (NVIDIA)             */
/*                Ross Walker (SDSC)               */
/*                                                 */
/***************************************************/

#include <hip/hip_runtime.h>
#include "gpu.h"

static __constant__ cudaSimulation cSim;
struct Atom 
{
    double invMassI;
    double xpl;
    double ypl;
    double zpl;
    double xil;
    double yil;
    double zil;
};

#if (__CUDA_ARCH__ >= 200)
#define INVMASSI invMassI
#define XPL xpl
#define YPL ypl
#define ZPL zpl
#define XIL xil
#define YIL yil
#define ZIL zil
#define XCOM xcom
#define YCOM ycom
#define ZCOM zcom
#define TRNS11 trns11
#define TRNS12 trns12
#define TRNS13 trns13
#define TRNS21 trns21
#define TRNS22 trns22
#define TRNS23 trns23
#define TRNS31 trns31
#else
#define INVMASSI psA->invMassI
#define XPL psA->xpl
#define YPL psA->ypl
#define ZPL psA->zpl
#define XIL psA->xil
#define YIL psA->yil
#define ZIL psA->zil
#define XCOM psA->xcom
#define YCOM psA->ycom
#define ZCOM psA->zcom
#define TRNS11 psA->trns11
#define TRNS12 psA->trns12
#define TRNS13 psA->trns13
#define TRNS21 psA->trns21
#define TRNS22 psA->trns22
#define TRNS23 psA->trns23
#define TRNS31 psA->trns31
#endif

// Texture reference for double-precision coordinates (disguised as int2 to work around HW limitations)
texture<int2, 1, hipReadModeElementType> texref;

void SetkShakeSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

void GetkShakeSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

__global__ void 
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_SHAKE_THREADS_PER_BLOCK, SM_3X_SHAKE_BLOCKS)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_SHAKE_THREADS_PER_BLOCK, SM_2X_SHAKE_BLOCKS)
#else
__launch_bounds__(SM_13_SHAKE_THREADS_PER_BLOCK, SM_13_SHAKE_BLOCKS)
#endif
kShake_kernel()
{
#if (__CUDA_ARCH__ >= 200)
    double invMassI;
    double xpl;
    double ypl;
    double zpl;
    double xil;
    double yil;
    double zil;
#else
    __shared__ Atom sA[SM_13_SHAKE_THREADS_PER_BLOCK];
    Atom* psA                                   = &sA[threadIdx.x];
#endif
    unsigned int pos                            = blockIdx.x * blockDim.x + threadIdx.x;   


    if (pos < cSim.shakeOffset)
    {
        if (pos < cSim.shakeConstraints)
        {   
            // Read SHAKE network data
            int4 shakeID                        = cSim.pShakeID[pos];
            double2 shakeParm                   = cSim.pShakeParm[pos];
            
            // Read SHAKE network components

#if defined(NODPTEXTURE)
            double xi                           = cSim.pForceX[shakeID.x];
            double yi                           = cSim.pForceY[shakeID.x];
            double zi                           = cSim.pForceZ[shakeID.x];
            double xij                          = cSim.pForceX[shakeID.y];
            double yij                          = cSim.pForceY[shakeID.y];
            double zij                          = cSim.pForceZ[shakeID.y];
#else        
            int2 ixi                            = tex1Dfetch(texref, shakeID.x);
            int2 iyi                            = tex1Dfetch(texref, shakeID.x + cSim.stride);
            int2 izi                            = tex1Dfetch(texref, shakeID.x + cSim.stride2);
            int2 ixij                           = tex1Dfetch(texref, shakeID.y);
            int2 iyij                           = tex1Dfetch(texref, shakeID.y + cSim.stride);
            int2 izij                           = tex1Dfetch(texref, shakeID.y + cSim.stride2);
            double xi                           = __hiloint2double(ixi.y, ixi.x);
            double yi                           = __hiloint2double(iyi.y, iyi.x);
            double zi                           = __hiloint2double(izi.y, izi.x);
            double xij                          = __hiloint2double(ixij.y, ixij.x);
            double yij                          = __hiloint2double(iyij.y, iyij.x);
            double zij                          = __hiloint2double(izij.y, izij.x); 
#endif                  
            double xpi                          = cSim.pAtomX[shakeID.x];
            double ypi                          = cSim.pAtomY[shakeID.x];
            double zpi                          = cSim.pAtomZ[shakeID.x];
            double xpj                          = cSim.pAtomX[shakeID.y];
            double ypj                          = cSim.pAtomY[shakeID.y];
            double zpj                          = cSim.pAtomZ[shakeID.y];               
            INVMASSI                            = shakeParm.x;
            double toler                        = shakeParm.y;
        
        
            // Optionally read 2nd hydrogen
            double xpk, ypk, zpk, xik, yik, zik;
            if (shakeID.z != -1)
            {
#if defined(NODPTEXTURE)
                xik                             = cSim.pForceX[shakeID.z];
                yik                             = cSim.pForceY[shakeID.z];
                zik                             = cSim.pForceZ[shakeID.z];   
#else
                int2 ixik                       = tex1Dfetch(texref, shakeID.z);
                int2 iyik                       = tex1Dfetch(texref, shakeID.z + cSim.stride);
                int2 izik                       = tex1Dfetch(texref, shakeID.z + cSim.stride2);
                xik                             = __hiloint2double(ixik.y, ixik.x);
                yik                             = __hiloint2double(iyik.y, iyik.x);
                zik                             = __hiloint2double(izik.y, izik.x);  
#endif 
                xpk                             = cSim.pAtomX[shakeID.z];
                ypk                             = cSim.pAtomY[shakeID.z];
                zpk                             = cSim.pAtomZ[shakeID.z];               
            }
            
            // Optionally read 3rd hydrogen into shared memory
            if (shakeID.w != -1)
            {
#if defined(NODPTEXTURE) 
                XIL                             = cSim.pForceX[shakeID.w];
                YIL                             = cSim.pForceY[shakeID.w];
                ZIL                             = cSim.pForceZ[shakeID.w];       
#else            
                int2 ixil                       = tex1Dfetch(texref, shakeID.w);
                int2 iyil                       = tex1Dfetch(texref, shakeID.w + cSim.stride);
                int2 izil                       = tex1Dfetch(texref, shakeID.w + cSim.stride2); 
                XIL                             = __hiloint2double(ixil.y, ixil.x);
                YIL                             = __hiloint2double(iyil.y, iyil.x);
                ZIL                             = __hiloint2double(izil.y, izil.x);   
#endif               
                XPL                             = cSim.pAtomX[shakeID.w];
                YPL                             = cSim.pAtomY[shakeID.w];
                ZPL                             = cSim.pAtomZ[shakeID.w];           
            }
            
            // Calculate unchanging quantities
            xij                                 = xi - xij;
            yij                                 = yi - yij;
            zij                                 = zi - zij;
            
            if (shakeID.z != -1)
            {
                xik                             = xi - xik;
                yik                             = yi - yik;
                zik                             = zi - zik;
            }        
             
            if (shakeID.w != -1)
            {
                XIL                             = xi - XIL;
                YIL                             = yi - YIL;
                ZIL                             = zi - ZIL;
            }      
       
            bool done                           = false;
            for (int i = 0; i < 3000; i++)
            {
                done                            = true;
                
                // Calculate nominal distance squared
                double xpxx                     = xpi - xpj;
                double ypxx                     = ypi - ypj;
                double zpxx                     = zpi - zpj;
                double rpxx2                    = xpxx * xpxx + ypxx * ypxx + zpxx * zpxx;
      
                // Apply correction
                double diff                     = toler - rpxx2;
                if (abs(diff) >= toler * cSim.tol)
                {
                    done                        = false;
                   
                    // Shake resetting of coordinate is done here
                    double rrpr                 = xij * xpxx + yij * ypxx + zij * zpxx;     
                    if (rrpr >= toler * 1.0e-06)
                    {
                    
                        double acor             = diff / (rrpr * 2.0 * (INVMASSI + cSim.invMassH));
                        double h                = xij * acor;
                        xpi                    += h * INVMASSI;
                        xpj                    -= h * cSim.invMassH;
                        h                       = yij * acor;
                        ypi                    += h * INVMASSI;
                        ypj                    -= h * cSim.invMassH;
                        h                       = zij * acor;
                        zpi                    += h * INVMASSI;
                        zpj                    -= h * cSim.invMassH;             
                    }
                }
      
                // Second bond if present
                if (shakeID.z != -1)
                {
                    xpxx                        = xpi - xpk;
                    ypxx                        = ypi - ypk;
                    zpxx                        = zpi - zpk;
                    rpxx2                       = xpxx * xpxx + ypxx * ypxx + zpxx * zpxx;
          
                    // Apply correction
                    diff                        = toler - rpxx2;
                    if (abs(diff) >= toler * cSim.tol)
                    {
                        done                    = false;
                   
                        // Shake resetting of coordinate is done here
                        double rrpr             = xik * xpxx + yik * ypxx + zik * zpxx;     
                        if (rrpr >= toler * 1.0e-06)
                        {
                    
                            double acor         = diff / (rrpr * 2.0 * (INVMASSI + cSim.invMassH));
                            double h            = xik * acor;
                            xpi                += h * INVMASSI;
                            xpk                -= h * cSim.invMassH;
                            h                   = yik * acor;
                            ypi                += h * INVMASSI;
                            ypk                -= h * cSim.invMassH;
                            h                   = zik * acor;
                            zpi                += h * INVMASSI;
                            zpk                -= h * cSim.invMassH;             
                        }
                    }
                }
            
                // Third bond if present
                if (shakeID.w != -1)
                {
                    xpxx                        = xpi - XPL;
                    ypxx                        = ypi - YPL;
                    zpxx                        = zpi - ZPL;
                    rpxx2                       = xpxx * xpxx + ypxx * ypxx + zpxx * zpxx;
          
                    // Apply correction
                    diff                        = toler - rpxx2;
                    if (abs(diff) >= toler * cSim.tol)
                    {
                        done                    = false;
                   
                        // Shake resetting of coordinate is done here
                        double rrpr             = XIL * xpxx + YIL * ypxx + ZIL * zpxx;     
                        if (rrpr >= toler * 1.0e-06)
                        {
                    
                            double acor         = diff / (rrpr * 2.0 * (INVMASSI + cSim.invMassH));
                            double h            = XIL * acor;
                            xpi                += h * INVMASSI;
                            XPL                -= h * cSim.invMassH;
                            h                   = YIL * acor;
                            ypi                += h * INVMASSI;
                            YPL                -= h * cSim.invMassH;
                            h                   = ZIL * acor;
                            zpi                += h * INVMASSI;
                            ZPL                -= h * cSim.invMassH;             
                        }
                    }
                }
                
                
                // Check for convergence
                if (done)
                    break;
            }
      
            // Write out results if converged, but there's no really good
            // way to indicate failure so we'll let the simulation heading
            // off to Neptune do that for us.  Wish there were a better way,
            // but until the CPU needs something from the GPU, those are the
            // the breaks.  I guess, technically, we could just set a flag to NOP
            // the simulation from here and then carry that result through upon
            // the next ntpr, ntwc, or ntwx update, but I leave that up to you 
            // guys to implement that (or not). 
            if (done)
            {
                cSim.pAtomX[shakeID.x]          = xpi;
                cSim.pAtomY[shakeID.x]          = ypi;
                cSim.pAtomZ[shakeID.x]          = zpi;
                PMEFloat2 xyi                   = {xpi, ypi};
                cSim.pAtomXYSP[shakeID.x]       = xyi;
                cSim.pAtomZSP[shakeID.x]        = zpi;
                
                cSim.pAtomX[shakeID.y]          = xpj;
                cSim.pAtomY[shakeID.y]          = ypj;
                cSim.pAtomZ[shakeID.y]          = zpj;
                PMEFloat2 xyj                   = {xpj, ypj};
                cSim.pAtomXYSP[shakeID.y]       = xyj;
                cSim.pAtomZSP[shakeID.y]        = zpj;

                if (shakeID.z != -1)
                {
                    cSim.pAtomX[shakeID.z]      = xpk;
                    cSim.pAtomY[shakeID.z]      = ypk;
                    cSim.pAtomZ[shakeID.z]      = zpk;
                    PMEFloat2 xyk               = {xpk, ypk};
                    cSim.pAtomXYSP[shakeID.z]   = xyk;
                    cSim.pAtomZSP[shakeID.z]    = zpk;
                }
            
                if (shakeID.w != -1)
                {
                    cSim.pAtomX[shakeID.w]      = XPL;
                    cSim.pAtomY[shakeID.w]      = YPL;
                    cSim.pAtomZ[shakeID.w]      = ZPL;
                    PMEFloat2 xyl               = {XPL, YPL};
                    cSim.pAtomXYSP[shakeID.w]   = xyl;
                    cSim.pAtomZSP[shakeID.w]    = ZPL;
                }
            
            }
        }
    } 
    else if (pos < cSim.fastShakeOffset)
    {    
        pos                                    -= cSim.shakeOffset;
        if (pos < cSim.fastShakeConstraints)
        {
            // Read atom data
            int4 shakeID                        = cSim.pFastShakeID[pos];
#if defined(NODPTEXTURE)
            double x1                           = cSim.pForceX[shakeID.x];
            double y1                           = cSim.pForceY[shakeID.x];
            double z1                           = cSim.pForceZ[shakeID.x]; 
            double x2                           = cSim.pForceX[shakeID.y];
            double y2                           = cSim.pForceY[shakeID.y];
            double z2                           = cSim.pForceZ[shakeID.y]; 
            double x3                           = cSim.pForceX[shakeID.z];
            double y3                           = cSim.pForceY[shakeID.z];
            double z3                           = cSim.pForceZ[shakeID.z]; 
#else           
            int2 ix1                            = tex1Dfetch(texref, shakeID.x);
            int2 iy1                            = tex1Dfetch(texref, shakeID.x + cSim.stride);
            int2 iz1                            = tex1Dfetch(texref, shakeID.x + cSim.stride2);
            int2 ix2                            = tex1Dfetch(texref, shakeID.y);
            int2 iy2                            = tex1Dfetch(texref, shakeID.y + cSim.stride);
            int2 iz2                            = tex1Dfetch(texref, shakeID.y + cSim.stride2);
            int2 ix3                            = tex1Dfetch(texref, shakeID.z);
            int2 iy3                            = tex1Dfetch(texref, shakeID.z + cSim.stride);
            int2 iz3                            = tex1Dfetch(texref, shakeID.z + cSim.stride2);
            double x1                           = __hiloint2double(ix1.y, ix1.x);
            double y1                           = __hiloint2double(iy1.y, iy1.x);
            double z1                           = __hiloint2double(iz1.y, iz1.x);
            double x2                           = __hiloint2double(ix2.y, ix2.x);
            double y2                           = __hiloint2double(iy2.y, iy2.x);
            double z2                           = __hiloint2double(iz2.y, iz2.x);
            double x3                           = __hiloint2double(ix3.y, ix3.x);
            double y3                           = __hiloint2double(iy3.y, iy3.x);
            double z3                           = __hiloint2double(iz3.y, iz3.x);  
#endif
            double xp1                          = cSim.pAtomX[shakeID.x];
            double yp1                          = cSim.pAtomY[shakeID.x];
            double zp1                          = cSim.pAtomZ[shakeID.x];
            double xp2                          = cSim.pAtomX[shakeID.y];
            double yp2                          = cSim.pAtomY[shakeID.y];
            double zp2                          = cSim.pAtomZ[shakeID.y];
            double xp3                          = cSim.pAtomX[shakeID.z];
            double yp3                          = cSim.pAtomY[shakeID.z];
            double zp3                          = cSim.pAtomZ[shakeID.z];
    
            // Step1  A1_prime:
            double xb0                          = x2 - x1;
            double yb0                          = y2 - y1;
            double zb0                          = z2 - z1;
            double xc0                          = x3 - x1;
            double yc0                          = y3 - y1;
            double zc0                          = z3 - z1;

            XPL                                 = xp1 * cSim.wo_div_wohh + (xp2 + xp3) * cSim.wh_div_wohh;
            YPL                                 = yp1 * cSim.wo_div_wohh + (yp2 + yp3) * cSim.wh_div_wohh;
            ZPL                                 = zp1 * cSim.wo_div_wohh + (zp2 + zp3) * cSim.wh_div_wohh;

            double xa1                          = xp1 - XPL;
            double ya1                          = yp1 - YPL;
            double za1                          = zp1 - ZPL;
            double xb1                          = xp2 - XPL;
            double yb1                          = yp2 - YPL;
            double zb1                          = zp2 - ZPL;
            double xc1                          = xp3 - XPL;
            double yc1                          = yp3 - YPL;
            double zc1                          = zp3 - ZPL;

            double xakszd                       = yb0 * zc0 - zb0 * yc0;
            double yakszd                       = zb0 * xc0 - xb0 * zc0;
            double zakszd                       = xb0 * yc0 - yb0 * xc0;
            double xaksxd                       = ya1 * zakszd - za1 * yakszd;
            double yaksxd                       = za1 * xakszd - xa1 * zakszd;
            double zaksxd                       = xa1 * yakszd - ya1 * xakszd;
            double xaksyd                       = yakszd * zaksxd - zakszd * yaksxd;
            double yaksyd                       = zakszd * xaksxd - xakszd * zaksxd;
            double zaksyd                       = xakszd * yaksxd - yakszd * xaksxd;

            double axlng_inv                    = rsqrt(xaksxd * xaksxd + yaksxd * yaksxd + zaksxd * zaksxd);
            double aylng_inv                    = rsqrt(xaksyd * xaksyd + yaksyd * yaksyd + zaksyd * zaksyd);
            double azlng_inv                    = rsqrt(xakszd * xakszd + yakszd * yakszd + zakszd * zakszd);

            double trns11                       = xaksxd * axlng_inv;
            double trns21                       = yaksxd * axlng_inv;
            double trns31                       = zaksxd * axlng_inv;
            double trns12                       = xaksyd * aylng_inv;
            double trns22                       = yaksyd * aylng_inv;
            double trns32                       = zaksyd * aylng_inv;
            double trns13                       = xakszd * azlng_inv;
            double trns23                       = yakszd * azlng_inv;
            double trns33                       = zakszd * azlng_inv;

            double xb0d                         = trns11 * xb0 + trns21 * yb0 + trns31 * zb0;
            double yb0d                         = trns12 * xb0 + trns22 * yb0 + trns32 * zb0;
            double xc0d                         = trns11 * xc0 + trns21 * yc0 + trns31 * zc0;
            double yc0d                         = trns12 * xc0 + trns22 * yc0 + trns32 * zc0;
            double za1d                         = trns13 * xa1 + trns23 * ya1 + trns33 * za1;
            double xb1d                         = trns11 * xb1 + trns21 * yb1 + trns31 * zb1;
            double yb1d                         = trns12 * xb1 + trns22 * yb1 + trns32 * zb1;
            double zb1d                         = trns13 * xb1 + trns23 * yb1 + trns33 * zb1;
            double xc1d                         = trns11 * xc1 + trns21 * yc1 + trns31 * zc1;
            double yc1d                         = trns12 * xc1 + trns22 * yc1 + trns32 * zc1;
            double zc1d                         = trns13 * xc1 + trns23 * yc1 + trns33 * zc1;

            // Step2  A2_prime:
            double sinphi                       = za1d * cSim.ra_inv;
            double cosphi                       = sqrt(1.0 - sinphi * sinphi);
            double sinpsi                       = (zb1d - zc1d) / (cSim.rc2 * cosphi);
            double cospsi                       = sqrt(1.0 - sinpsi * sinpsi);
 
            double ya2d                         =  cSim.ra * cosphi;
            double xb2d                         = -cSim.rc * cospsi;
            double yb2d                         = -cSim.rb * cosphi - cSim.rc * sinpsi * sinphi;
            double yc2d                         = -cSim.rb * cosphi + cSim.rc * sinpsi * sinphi;
            xb2d                                = -0.5 * sqrt(cSim.hhhh - (yb2d - yc2d) * (yb2d - yc2d) - (zb1d - zc1d) * (zb1d - zc1d));

            // Step3  al,be,ga:

            double alpa                         = (xb2d * (xb0d-xc0d) + yb0d * yb2d + yc0d * yc2d);
            double beta                         = (xb2d * (yc0d-yb0d) + xb0d * yb2d + xc0d * yc2d);
            double gama                         = xb0d * yb1d - xb1d * yb0d + xc0d * yc1d - xc1d * yc0d;

            double al2be2                       =  alpa * alpa + beta * beta;
            double sinthe                       = (alpa * gama - beta * sqrt(al2be2 - gama * gama)) / al2be2;

            // Step4  A3_prime:

            double costhe                       =  sqrt(1.0 - sinthe * sinthe);
            double xa3d                         = -ya2d * sinthe;
            double ya3d                         =  ya2d * costhe;
            double za3d                         =  za1d;
            double xb3d                         =  xb2d * costhe - yb2d * sinthe;
            double yb3d                         =  xb2d * sinthe + yb2d * costhe;
            double zb3d                         =  zb1d;
            double xc3d                         = -xb2d * costhe - yc2d * sinthe;
            double yc3d                         = -xb2d * sinthe + yc2d * costhe;
            double zc3d                         =  zc1d;

            // Step5  A3:
            cSim.pAtomX[shakeID.x]              = XPL + trns11 * xa3d + trns12 * ya3d + trns13 * za3d;
            cSim.pAtomY[shakeID.x]              = YPL + trns21 * xa3d + trns22 * ya3d + trns23 * za3d;
            cSim.pAtomZ[shakeID.x]              = ZPL + trns31 * xa3d + trns32 * ya3d + trns33 * za3d;
            cSim.pAtomX[shakeID.y]              = XPL + trns11 * xb3d + trns12 * yb3d + trns13 * zb3d;
            cSim.pAtomY[shakeID.y]              = YPL + trns21 * xb3d + trns22 * yb3d + trns23 * zb3d;
            cSim.pAtomZ[shakeID.y]              = ZPL + trns31 * xb3d + trns32 * yb3d + trns33 * zb3d;
            cSim.pAtomX[shakeID.z]              = XPL + trns11 * xc3d + trns12 * yc3d + trns13 * zc3d;
            cSim.pAtomY[shakeID.z]              = YPL + trns21 * xc3d + trns22 * yc3d + trns23 * zc3d;
            cSim.pAtomZ[shakeID.z]              = ZPL + trns31 * xc3d + trns32 * yc3d + trns33 * zc3d;                        
        }
    }
    else if ( pos < cSim.slowShakeOffset)
    {    
        pos                                    -= cSim.fastShakeOffset;

        if (pos < cSim.slowShakeConstraints)
        {
            int shakeID1;
		    int4 shakeID2;
            double toler;

            // Read SHAKE network data
            shakeID1                            = cSim.pSlowShakeID1[pos];
            shakeID2                            = cSim.pSlowShakeID2[pos];
            double2 shakeParm                   = cSim.pSlowShakeParm[pos];
        
            // Read SHAKE network components
#if defined(NODPTEXTURE)
            double xi                           = cSim.pForceX[shakeID1];
            double yi                           = cSim.pForceY[shakeID1];
            double zi                           = cSim.pForceZ[shakeID1];
            double xij                          = cSim.pForceX[shakeID2.x];
            double yij                          = cSim.pForceY[shakeID2.x];
            double zij                          = cSim.pForceZ[shakeID2.x];
            double xik                          = cSim.pForceX[shakeID2.y];
            double yik                          = cSim.pForceY[shakeID2.y];
            double zik                          = cSim.pForceZ[shakeID2.y];  
            XIL                                 = cSim.pForceX[shakeID2.z];
            YIL                                 = cSim.pForceY[shakeID2.z];
            ZIL                                 = cSim.pForceZ[shakeID2.z]; 
            double xim                          = cSim.pForceX[shakeID2.w];
            double yim                          = cSim.pForceY[shakeID2.w];
            double zim                          = cSim.pForceZ[shakeID2.w];       
#else        
            int2 ixi                            = tex1Dfetch(texref, shakeID1);
            int2 iyi                            = tex1Dfetch(texref, shakeID1 + cSim.stride);
            int2 izi                            = tex1Dfetch(texref, shakeID1 + cSim.stride2);
            int2 ixij                           = tex1Dfetch(texref, shakeID2.x);
            int2 iyij                           = tex1Dfetch(texref, shakeID2.x + cSim.stride);
            int2 izij                           = tex1Dfetch(texref, shakeID2.x + cSim.stride2);
            int2 ixik                           = tex1Dfetch(texref, shakeID2.y);
            int2 iyik                           = tex1Dfetch(texref, shakeID2.y + cSim.stride);
            int2 izik                           = tex1Dfetch(texref, shakeID2.y + cSim.stride2);
            int2 ixil                           = tex1Dfetch(texref, shakeID2.z);
            int2 iyil                           = tex1Dfetch(texref, shakeID2.z + cSim.stride);
            int2 izil                           = tex1Dfetch(texref, shakeID2.z + cSim.stride2); 
            int2 ixim                           = tex1Dfetch(texref, shakeID2.w);
            int2 iyim                           = tex1Dfetch(texref, shakeID2.w + cSim.stride);
            int2 izim                           = tex1Dfetch(texref, shakeID2.w + cSim.stride2); 
            double xi                           = __hiloint2double(ixi.y, ixi.x);
            double yi                           = __hiloint2double(iyi.y, iyi.x);
            double zi                           = __hiloint2double(izi.y, izi.x);
            double xij                          = __hiloint2double(ixij.y, ixij.x);
            double yij                          = __hiloint2double(iyij.y, iyij.x);
            double zij                          = __hiloint2double(izij.y, izij.x);
            double xik                          = __hiloint2double(ixik.y, ixik.x);
            double yik                          = __hiloint2double(iyik.y, iyik.x);
            double zik                          = __hiloint2double(izik.y, izik.x);   
            XIL                                 = __hiloint2double(ixil.y, ixil.x);
            YIL                                 = __hiloint2double(iyil.y, iyil.x);
            ZIL                                 = __hiloint2double(izil.y, izil.x);   
            double xim                          = __hiloint2double(ixim.y, ixim.x);
            double yim                          = __hiloint2double(iyim.y, iyim.x);
            double zim                          = __hiloint2double(izim.y, izim.x); 
#endif 
            double xpi                          = cSim.pAtomX[shakeID1];
            double ypi                          = cSim.pAtomY[shakeID1];
            double zpi                          = cSim.pAtomZ[shakeID1];
            double xpj                          = cSim.pAtomX[shakeID2.x];
            double ypj                          = cSim.pAtomY[shakeID2.x];
            double zpj                          = cSim.pAtomZ[shakeID2.x];
            double xpk                          = cSim.pAtomX[shakeID2.y];
            double ypk                          = cSim.pAtomY[shakeID2.y];
            double zpk                          = cSim.pAtomZ[shakeID2.y];
            XPL                                 = cSim.pAtomX[shakeID2.z];
            YPL                                 = cSim.pAtomY[shakeID2.z];
            ZPL                                 = cSim.pAtomZ[shakeID2.z];
            double xpm                          = cSim.pAtomX[shakeID2.w];
            double ypm                          = cSim.pAtomY[shakeID2.w];
            double zpm                          = cSim.pAtomZ[shakeID2.w];                    
            INVMASSI                            = shakeParm.x;
            toler                               = shakeParm.y;
            
            // Calculate unchanging quantities
            xij                                 = xi - xij;
            yij                                 = yi - yij;
            zij                                 = zi - zij;
            xik                                 = xi - xik;
            yik                                 = yi - yik;
            zik                                 = zi - zik; 
            XIL                                 = xi - XIL;
            YIL                                 = yi - YIL;
            ZIL                                 = zi - ZIL;
            xim                                 = xi - xim;
            yim                                 = yi - yim;
            zim                                 = zi - zim;               
           
            bool done                           = false;
            for (int i = 0; i < 3000; i++)
            {
                done = true;
                
                // Calculate nominal distance squared
                double xpxx                     = xpi - xpj;
                double ypxx                     = ypi - ypj;
                double zpxx                     = zpi - zpj;
                double rpxx2                    = xpxx * xpxx + ypxx * ypxx + zpxx * zpxx;
          
                // Apply correction to first hydrogen
                double diff                     = toler - rpxx2;
                if (abs(diff) >= toler * cSim.tol)
                {
                    done                        = false;
                   
                    // Shake resetting of coordinate is done here
                    double rrpr                 = xij * xpxx + yij * ypxx + zij * zpxx;     
                    if (rrpr >= toler * 1.0e-06)
                    {
                    
                        double acor             = diff / (rrpr * (double)2.0 * (INVMASSI + cSim.invMassH));
                        double h                = xij * acor;
                        xpi                    += h * INVMASSI;
                        xpj                    -= h * cSim.invMassH;
                        h                       = yij * acor;
                        ypi                    += h * INVMASSI;
                        ypj                    -= h * cSim.invMassH;
                        h                       = zij * acor;
                        zpi                    += h * INVMASSI;
                        zpj                    -= h * cSim.invMassH;             
                    }
                }
          
     
                xpxx                            = xpi - xpk;
                ypxx                            = ypi - ypk;
                zpxx                            = zpi - zpk;
                rpxx2                           = xpxx * xpxx + ypxx * ypxx + zpxx * zpxx;
          
                // Apply correction to second hydrogen
                diff                            = toler - rpxx2;
                if (abs(diff) >= toler * cSim.tol)
                {
                    done                        = false;
               
                    // Shake resetting of coordinate is done here
                    double rrpr                 = xik * xpxx + yik * ypxx + zik * zpxx;     
                    if (rrpr >= toler * 1.0e-06)
                    {
                    
                        double acor             = diff / (rrpr * 2.0 * (INVMASSI + cSim.invMassH));
                        double h                = xik * acor;
                        xpi                    += h * INVMASSI;
                        xpk                    -= h * cSim.invMassH;
                        h                       = yik * acor;
                        ypi                    += h * INVMASSI;
                        ypk                    -= h * cSim.invMassH;
                        h                       = zik * acor;
                        zpi                    += h * INVMASSI;
                        zpk                    -= h * cSim.invMassH;             
                    }
                }
                
 
                xpxx                            = xpi - XPL;
                ypxx                            = ypi - YPL;
                zpxx                            = zpi - ZPL;        
                rpxx2                           = xpxx * xpxx + ypxx * ypxx + zpxx * zpxx;
          
                // Apply correction to third hydrogen
                diff                            = toler - rpxx2;
                if (abs(diff) >= toler * cSim.tol)
                {
                    done                        = false;
                  
                    // Shake resetting of coordinate is done here
                    double rrpr                 = XIL * xpxx + YIL * ypxx + ZIL * zpxx;     
                    if (rrpr >= toler * 1.0e-06)
                    {             
                        double acor             = diff / (rrpr * (double)2.0 * (INVMASSI + cSim.invMassH));
                        double h                = XIL * acor;
                        xpi                    += h * INVMASSI;
                        XPL                    -= h * cSim.invMassH;
                        h                       = YIL * acor;
                        ypi                    += h * INVMASSI;
                        YPL                    -= h * cSim.invMassH;
                        h                       = ZIL * acor;
                        zpi                    += h * INVMASSI;
                        ZPL                    -= h * cSim.invMassH;             
                    }
                }

                xpxx                            = xpi - xpm;
                ypxx                            = ypi - ypm;
                zpxx                            = zpi - zpm;        
                rpxx2                           = xpxx * xpxx + ypxx * ypxx + zpxx * zpxx;

                // Apply correction to third hydrogen
                diff                            = toler - rpxx2;
                if (abs(diff) >= toler * cSim.tol)
                {
                    done                        = false;
                  
                    // Shake resetting of coordinate is done here
                    double rrpr                 = xim * xpxx + yim * ypxx + zim * zpxx;     
                    if (rrpr >= toler * 1.0e-06)
                    {             
                        double acor             = diff / (rrpr * (double)2.0 * (INVMASSI + cSim.invMassH));
                        double h                = xim * acor;
                        xpi                    += h * INVMASSI;
                        xpm                    -= h * cSim.invMassH;
                        h                       = yim * acor;
                        ypi                    += h * INVMASSI;
                        ypm                    -= h * cSim.invMassH;
                        h                       = zim * acor;
                        zpi                    += h * INVMASSI;
                        zpm                    -= h * cSim.invMassH;             
                    }
                }

                
                
                // Check for convergence
                if (done)
                    break;
            }
          
            // Write out results if converged, but there's no really good
            // way to indicate failure so we'll let the simulation heading
            // off to Neptune do that for us.  Wish there were a better way,
            // but until the CPU needs something from the GPU, those are the
            // the breaks.  I guess, technically, we could just set a flag to NOP
            // the simulation from here and then carry that result through upon
            // the next ntpr, ntwc, or ntwx update, but I leave that up to you 
            // guys to implement that (or not). 
            if (done)
            {
                cSim.pAtomX[shakeID1]           = xpi;
                cSim.pAtomY[shakeID1]           = ypi;
                cSim.pAtomZ[shakeID1]           = zpi;
                PMEFloat2 xyi                   = {xpi, ypi};
                cSim.pAtomXYSP[shakeID1]        = xyi;
                cSim.pAtomZSP[shakeID1]         = zpi;
                cSim.pAtomX[shakeID2.x]         = xpj;
                cSim.pAtomY[shakeID2.x]         = ypj;
                cSim.pAtomZ[shakeID2.x]         = zpj;
                PMEFloat2 xyj                   = {xpj, ypj};
                cSim.pAtomXYSP[shakeID2.x]      = xyj;
                cSim.pAtomZSP[shakeID2.x]       = zpj;
                cSim.pAtomX[shakeID2.y]         = xpk;
                cSim.pAtomY[shakeID2.y]         = ypk;
                cSim.pAtomZ[shakeID2.y]         = zpk;
                PMEFloat2 xyk                   = {xpk, ypk};
                cSim.pAtomXYSP[shakeID2.y]      = xyk;
                cSim.pAtomZSP[shakeID2.y]       = zpk;
                cSim.pAtomX[shakeID2.z]         = XPL;
                cSim.pAtomY[shakeID2.z]         = YPL;
                cSim.pAtomZ[shakeID2.z]         = ZPL;
                PMEFloat2 xyl                   = {XPL, YPL};
                cSim.pAtomXYSP[shakeID2.z]      = xyl;
                cSim.pAtomZSP[shakeID2.z]       = ZPL;
                cSim.pAtomX[shakeID2.w]         = xpm;
                cSim.pAtomY[shakeID2.w]         = ypm;
                cSim.pAtomZ[shakeID2.w]         = zpm;
                PMEFloat2 xym                   = {xpm, ypm};
                cSim.pAtomXYSP[shakeID2.w]      = xym;
                cSim.pAtomZSP[shakeID2.w]       = zpm;
            }                                      
        }
    }
}


#if (__CUDA_ARCH__ < 200)
struct PMEAtom 
{
    double invMassI;
    double xpl;
    double ypl;
    double zpl;
    double xil;
    double yil;
    double zil;
    double dummy1;
    double dummy2;
    double dummy3;
};


struct PMEFastAtom
{
    double xcom;
    double ycom;
    double zcom;
    double trns11;
    double trns12;
    double trns13;
    double trns21;
    double trns22;
    double trns23;
    double trns31;
};
#endif

__global__ void 
#if (__CUDA_ARCH__ >= 300)
__launch_bounds__(SM_3X_SHAKE_THREADS_PER_BLOCK, SM_3X_SHAKE_BLOCKS)
#elif (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_SHAKE_THREADS_PER_BLOCK, SM_2X_SHAKE_BLOCKS)
#else
__launch_bounds__(SM_13_SHAKE_THREADS_PER_BLOCK, SM_13_SHAKE_BLOCKS)
#endif
kPMEShake_kernel()
{
#if (__CUDA_ARCH__ < 200)
__shared__ PMEAtom sA[SM_13_SHAKE_THREADS_PER_BLOCK];
#endif

    unsigned int pos                            = blockIdx.x * blockDim.x + threadIdx.x;  
    if (pos < cSim.shakeOffset)
    { 
        if (pos < cSim.shakeConstraints)
        {
#if (__CUDA_ARCH__ >= 200)
            double invMassI;
            double xpl;
            double ypl;
            double zpl;
            double xil;
            double yil;
            double zil;
#else
            PMEAtom* psA                        = &sA[threadIdx.x];
#endif
            // Read SHAKE network data
            int4 shakeID                    	= cSim.pImageShakeID[pos];
            double2 shakeParm                   = cSim.pShakeParm[pos];
            
            // Read SHAKE network components
#if defined(NODPTEXTURE)
            double xi                           = cSim.pForceX[shakeID.x];
            double yi                           = cSim.pForceY[shakeID.x];
            double zi                           = cSim.pForceZ[shakeID.x];
            double xij                          = cSim.pForceX[shakeID.y];
            double yij                          = cSim.pForceY[shakeID.y];
            double zij                          = cSim.pForceZ[shakeID.y];
#else        
            int2 ixi                            = tex1Dfetch(texref, shakeID.x);
            int2 iyi                            = tex1Dfetch(texref, shakeID.x + cSim.stride);
            int2 izi                            = tex1Dfetch(texref, shakeID.x + cSim.stride2);
            int2 ixij                           = tex1Dfetch(texref, shakeID.y);
            int2 iyij                           = tex1Dfetch(texref, shakeID.y + cSim.stride);
            int2 izij                           = tex1Dfetch(texref, shakeID.y + cSim.stride2);
            double xi                           = __hiloint2double(ixi.y, ixi.x);
            double yi                           = __hiloint2double(iyi.y, iyi.x);
            double zi                           = __hiloint2double(izi.y, izi.x);
            double xij                          = __hiloint2double(ixij.y, ixij.x);
            double yij                          = __hiloint2double(iyij.y, iyij.x);
            double zij                          = __hiloint2double(izij.y, izij.x); 
#endif 
            double xpi                          = cSim.pImageX[shakeID.x];
            double ypi                          = cSim.pImageY[shakeID.x];
            double zpi                          = cSim.pImageZ[shakeID.x];
            double xpj                          = cSim.pImageX[shakeID.y];
            double ypj                          = cSim.pImageY[shakeID.y];
            double zpj                          = cSim.pImageZ[shakeID.y];                      
            INVMASSI                            = shakeParm.x;
            double toler                        = shakeParm.y;
            
            
            // Optionally read 2nd hydrogen
            double xpk, ypk, zpk, xik, yik, zik;
            if (shakeID.z != -1)
            {
#if defined(NODPTEXTURE)
                xik                             = cSim.pForceX[shakeID.z];
                yik                             = cSim.pForceY[shakeID.z];
                zik                             = cSim.pForceZ[shakeID.z];    
#else
                int2 ixik                       = tex1Dfetch(texref, shakeID.z);
                int2 iyik                       = tex1Dfetch(texref, shakeID.z + cSim.stride);
                int2 izik                       = tex1Dfetch(texref, shakeID.z + cSim.stride2);
                xik                             = __hiloint2double(ixik.y, ixik.x);
                yik                             = __hiloint2double(iyik.y, iyik.x);
                zik                             = __hiloint2double(izik.y, izik.x);  
#endif 
                xpk                             = cSim.pImageX[shakeID.z];
                ypk                             = cSim.pImageY[shakeID.z];
                zpk                             = cSim.pImageZ[shakeID.z];                
            }
        
            // Optionally read 3rd hydrogen into shared memory
            if (shakeID.w != -1)
            {
#if defined(NODPTEXTURE)
                XIL                             = cSim.pForceX[shakeID.w];
                YIL                             = cSim.pForceY[shakeID.w];
                ZIL                             = cSim.pForceZ[shakeID.w];      
#else            
                int2 ixil                       = tex1Dfetch(texref, shakeID.w);
                int2 iyil                       = tex1Dfetch(texref, shakeID.w + cSim.stride);
                int2 izil                       = tex1Dfetch(texref, shakeID.w + cSim.stride2); 
                XIL                             = __hiloint2double(ixil.y, ixil.x);
                YIL                             = __hiloint2double(iyil.y, iyil.x);
                ZIL                             = __hiloint2double(izil.y, izil.x);
#endif          
                XPL                             = cSim.pImageX[shakeID.w];
                YPL                             = cSim.pImageY[shakeID.w];
                ZPL                             = cSim.pImageZ[shakeID.w];          
            }
        
            // Calculate unchanging quantities
            xij                                 = xi - xij;
            yij                                 = yi - yij;
            zij                                 = zi - zij;
            
            if (shakeID.z != -1)
            {
                xik                             = xi - xik;
                yik                             = yi - yik;
                zik                             = zi - zik; 
            }        
             
            if (shakeID.w != -1)
            {
                XIL                             = xi - XIL;
                YIL                             = yi - YIL;
                ZIL                             = zi - ZIL;
            }      
       
            bool done                           = false;
            for (int i = 0; i < 3000; i++)
            {
                done                            = true;
                
                // Calculate nominal distance squared
                double xpxx                     = xpi - xpj;
                double ypxx                     = ypi - ypj;
                double zpxx                     = zpi - zpj;
                double rpxx2                    = xpxx * xpxx + ypxx * ypxx + zpxx * zpxx;
          
                // Apply correction
                double diff                     = toler - rpxx2;
                if (abs(diff) >= toler * cSim.tol)
                {
                    done                        = false;
                   
                    // Shake resetting of coordinate is done here
                    double rrpr                 = xij * xpxx + yij * ypxx + zij * zpxx;     
                    if (rrpr >= toler * 1.0e-06)
                    {
                    
                        double acor             = diff / (rrpr * (double)2.0 * (INVMASSI + cSim.invMassH));
                        double h                = xij * acor;
                        xpi                    += h * INVMASSI;
                        xpj                    -= h * cSim.invMassH;
                        h                       = yij * acor;
                        ypi                    += h * INVMASSI;
                        ypj                    -= h * cSim.invMassH;
                        h                       = zij * acor;
                        zpi                    += h * INVMASSI;
                        zpj                    -= h * cSim.invMassH;             
                    }
                }
      
                // Second bond if present
                if (shakeID.z != -1)
                {
                    xpxx                        = xpi - xpk;
                    ypxx                        = ypi - ypk;
                    zpxx                        = zpi - zpk;
                    rpxx2                       = xpxx * xpxx + ypxx * ypxx + zpxx * zpxx;
          
                    // Apply correction
                    diff                        = toler - rpxx2;
                    if (abs(diff) >= toler * cSim.tol)
                    {
                        done                    = false;
                   
                        // Shake resetting of coordinate is done here
                        double rrpr             = xik * xpxx + yik * ypxx + zik * zpxx;     
                        if (rrpr >= toler * 1.0e-06)
                        {
                    
                            double acor         = diff / (rrpr * 2.0 * (INVMASSI + cSim.invMassH));
                            double h            = xik * acor;
                            xpi                += h * INVMASSI;
                            xpk                -= h * cSim.invMassH;
                            h                   = yik * acor;
                            ypi                += h * INVMASSI;
                            ypk                -= h * cSim.invMassH;
                            h                   = zik * acor;
                            zpi                += h * INVMASSI;
                            zpk                -= h * cSim.invMassH;             
                        }
                    }
                }
            
                // Third bond if present
                if (shakeID.w != -1)
                {
                    xpxx                        = xpi - XPL;
                    ypxx                        = ypi - YPL;
                    zpxx                        = zpi - ZPL;        
                    rpxx2                       = xpxx * xpxx + ypxx * ypxx + zpxx * zpxx;
          
                    // Apply correction
                    diff                        = toler - rpxx2;
                    if (abs(diff) >= toler * cSim.tol)
                    {
                        done                    = false;
                   
                        // Shake resetting of coordinate is done here
                        double rrpr             = XIL * xpxx + YIL * ypxx + ZIL * zpxx;     
                        if (rrpr >= toler * 1.0e-06)
                        {
                    
                            double acor         = diff / (rrpr * (double)2.0 * (INVMASSI + cSim.invMassH));
                            double h            = XIL * acor;
                            xpi                += h * INVMASSI;
                            XPL                -= h * cSim.invMassH;
                            h                   = YIL * acor;
                            ypi                += h * INVMASSI;
                            YPL                -= h * cSim.invMassH;
                            h                   = ZIL * acor;
                            zpi                += h * INVMASSI;
                            ZPL                -= h * cSim.invMassH;             
                        }
                    }
                }
                
                
                // Check for convergence
                if (done)
                    break;
            }
      
            // Write out results if converged, but there's no really good
            // way to indicate failure so we'll let the simulation heading
            // off to Neptune do that for us.  Wish there were a better way,
            // but until the CPU needs something from the GPU, those are the
            // the breaks.  I guess, technically, we could just set a flag to NOP
            // the simulation from here and then carry that result through upon
            // the next ntpr, ntwc, or ntwx update, but I leave that up to you 
            // guys to implement that (or not). 
            if (done)
            {
                cSim.pImageX[shakeID.x]          = xpi;
                cSim.pImageY[shakeID.x]          = ypi;
                cSim.pImageZ[shakeID.x]          = zpi;
                cSim.pImageX[shakeID.y]          = xpj;
                cSim.pImageY[shakeID.y]          = ypj;
                cSim.pImageZ[shakeID.y]          = zpj;

                if (shakeID.z != -1)
                {
                    cSim.pImageX[shakeID.z]      = xpk;
                    cSim.pImageY[shakeID.z]      = ypk;
                    cSim.pImageZ[shakeID.z]      = zpk;
                }

                if (shakeID.w != -1)
                {
                    cSim.pImageX[shakeID.w]      = XPL;
                    cSim.pImageY[shakeID.w]      = YPL;
                    cSim.pImageZ[shakeID.w]      = ZPL;
                }
            }        
        }
    }
    else if (pos < cSim.fastShakeOffset)
    {    
        pos                                     -= cSim.shakeOffset;

        if (pos < cSim.fastShakeConstraints)
        {
#if (__CUDA_ARCH__ >= 200)
            double xcom;
            double ycom;
            double zcom;
            double trns11;
            double trns12;
            double trns13;
            double trns21;
            double trns22;
            double trns23;
            double trns31;
#else
            PMEFastAtom* psA                    = (PMEFastAtom*)&sA[threadIdx.x];
#endif            

            // Read atom data
            int4 shakeID                        = cSim.pImageFastShakeID[pos];
#if defined(NODPTEXTURE)
            double x1                           = cSim.pForceX[shakeID.x];
            double y1                           = cSim.pForceY[shakeID.x];
            double z1                           = cSim.pForceZ[shakeID.x]; 
            double x2                           = cSim.pForceX[shakeID.y];
            double y2                           = cSim.pForceY[shakeID.y];
            double z2                           = cSim.pForceZ[shakeID.y]; 
            double x3                           = cSim.pForceX[shakeID.z];
            double y3                           = cSim.pForceY[shakeID.z];
            double z3                           = cSim.pForceZ[shakeID.z]; 
#else           
            int2 ix1                            = tex1Dfetch(texref, shakeID.x);
            int2 iy1                            = tex1Dfetch(texref, shakeID.x + cSim.stride);
            int2 iz1                            = tex1Dfetch(texref, shakeID.x + cSim.stride2);
            int2 ix2                            = tex1Dfetch(texref, shakeID.y);
            int2 iy2                            = tex1Dfetch(texref, shakeID.y + cSim.stride);
            int2 iz2                            = tex1Dfetch(texref, shakeID.y + cSim.stride2);
            int2 ix3                            = tex1Dfetch(texref, shakeID.z);
            int2 iy3                            = tex1Dfetch(texref, shakeID.z + cSim.stride);
            int2 iz3                            = tex1Dfetch(texref, shakeID.z + cSim.stride2);  
            double x1                           = __hiloint2double(ix1.y, ix1.x);
            double y1                           = __hiloint2double(iy1.y, iy1.x);
            double z1                           = __hiloint2double(iz1.y, iz1.x);
            double x2                           = __hiloint2double(ix2.y, ix2.x);
            double y2                           = __hiloint2double(iy2.y, iy2.x);
            double z2                           = __hiloint2double(iz2.y, iz2.x);
            double x3                           = __hiloint2double(ix3.y, ix3.x);
            double y3                           = __hiloint2double(iy3.y, iy3.x);
            double z3                           = __hiloint2double(iz3.y, iz3.x);
#endif
            double xp1                          = cSim.pImageX[shakeID.x];
            double yp1                          = cSim.pImageY[shakeID.x];
            double zp1                          = cSim.pImageZ[shakeID.x];
            double xp2                          = cSim.pImageX[shakeID.y];
            double yp2                          = cSim.pImageY[shakeID.y];
            double zp2                          = cSim.pImageZ[shakeID.y];
            double xp3                          = cSim.pImageX[shakeID.z];
            double yp3                          = cSim.pImageY[shakeID.z];
            double zp3                          = cSim.pImageZ[shakeID.z];

            // Step1  A1_prime:
            double xb0                          = x2 - x1;
            double yb0                          = y2 - y1;
            double zb0                          = z2 - z1;
            double xc0                          = x3 - x1;
            double yc0                          = y3 - y1;
            double zc0                          = z3 - z1;
            XCOM                                = xp1 * cSim.wo_div_wohh + (xp2 + xp3) * cSim.wh_div_wohh;
            YCOM                                = yp1 * cSim.wo_div_wohh + (yp2 + yp3) * cSim.wh_div_wohh;
            ZCOM                                = zp1 * cSim.wo_div_wohh + (zp2 + zp3) * cSim.wh_div_wohh;

            double xa1                          = xp1 - XCOM;
            double ya1                          = yp1 - YCOM;
            double za1                          = zp1 - ZCOM;
            double xb1                          = xp2 - XCOM;
            double yb1                          = yp2 - YCOM;
            double zb1                          = zp2 - ZCOM;
            double xc1                          = xp3 - XCOM;
            double yc1                          = yp3 - YCOM;
            double zc1                          = zp3 - ZCOM;
            double xakszd                       = yb0 * zc0 - zb0 * yc0;
            double yakszd                       = zb0 * xc0 - xb0 * zc0;
            double zakszd                       = xb0 * yc0 - yb0 * xc0;
            double xaksxd                       = ya1 * zakszd - za1 * yakszd;
            double yaksxd                       = za1 * xakszd - xa1 * zakszd;
            double zaksxd                       = xa1 * yakszd - ya1 * xakszd;
            double xaksyd                       = yakszd * zaksxd - zakszd * yaksxd;
            double yaksyd                       = zakszd * xaksxd - xakszd * zaksxd;
            double zaksyd                       = xakszd * yaksxd - yakszd * xaksxd;

            double axlng_inv                    = rsqrt(xaksxd * xaksxd + yaksxd * yaksxd + zaksxd * zaksxd);
            double aylng_inv                    = rsqrt(xaksyd * xaksyd + yaksyd * yaksyd + zaksyd * zaksyd);
            double azlng_inv                    = rsqrt(xakszd * xakszd + yakszd * yakszd + zakszd * zakszd);

            TRNS11                              = xaksxd * axlng_inv;
            TRNS21                              = yaksxd * axlng_inv;
            TRNS31                              = zaksxd * axlng_inv;
            TRNS12                              = xaksyd * aylng_inv;
            TRNS22                              = yaksyd * aylng_inv;
			double trns32                       = zaksyd * aylng_inv;
            TRNS13                              = xakszd * azlng_inv;
            TRNS23                              = yakszd * azlng_inv;
			double trns33                       = zakszd * azlng_inv;

            double xb0d                         = TRNS11 * xb0 + TRNS21 * yb0 + TRNS31 * zb0;
            double yb0d                         = TRNS12 * xb0 + TRNS22 * yb0 + trns32 * zb0;
            double xc0d                         = TRNS11 * xc0 + TRNS21 * yc0 + TRNS31 * zc0;
            double yc0d                         = TRNS12 * xc0 + TRNS22 * yc0 + trns32 * zc0;
            double za1d                         = TRNS13 * xa1 + TRNS23 * ya1 + trns33 * za1;
            double xb1d                         = TRNS11 * xb1 + TRNS21 * yb1 + TRNS31 * zb1;
            double yb1d                         = TRNS12 * xb1 + TRNS22 * yb1 + trns32 * zb1;
            double zb1d                         = TRNS13 * xb1 + TRNS23 * yb1 + trns33 * zb1;
            double xc1d                         = TRNS11 * xc1 + TRNS21 * yc1 + TRNS31 * zc1;
            double yc1d                         = TRNS12 * xc1 + TRNS22 * yc1 + trns32 * zc1;
            double zc1d                         = TRNS13 * xc1 + TRNS23 * yc1 + trns33 * zc1;

            // Step2  A2_prime:
            double sinphi                       = za1d * cSim.ra_inv;
            double cosphi                       = sqrt(1.0 - sinphi * sinphi);
            double sinpsi                       = (zb1d - zc1d) / (cSim.rc2 * cosphi);
            double cospsi                       = sqrt(1.0 - sinpsi * sinpsi);
 
            double ya2d                         =  cSim.ra * cosphi;
            double xb2d                         = -cSim.rc * cospsi;
            double yb2d                         = -cSim.rb * cosphi - cSim.rc * sinpsi * sinphi;
            double yc2d                         = -cSim.rb * cosphi + cSim.rc * sinpsi * sinphi;
            xb2d                                = -0.5 * sqrt(cSim.hhhh - (yb2d-yc2d) * (yb2d - yc2d) - (zb1d - zc1d) * (zb1d - zc1d));

            // Step3  al,be,ga:
            double alpa                         = (xb2d * (xb0d-xc0d) + yb0d * yb2d + yc0d * yc2d);
            double beta                         = (xb2d * (yc0d-yb0d) + xb0d * yb2d + xc0d * yc2d);
            double gama                         = xb0d * yb1d - xb1d * yb0d + xc0d * yc1d - xc1d * yc0d;

            double al2be2                       =  alpa * alpa + beta * beta;
            double sinthe                       = (alpa * gama - beta * sqrt(al2be2 - gama * gama)) / al2be2;

            // Step4  A3_prime:
            double costhe                       =  sqrt(1.0 - sinthe * sinthe);
            double xa3d                         = -ya2d * sinthe;
            double ya3d                         =  ya2d * costhe;
            double za3d                         =  za1d;
            double xb3d                         =  xb2d * costhe - yb2d * sinthe;
            double yb3d                         =  xb2d * sinthe + yb2d * costhe;
            double zb3d                         =  zb1d;
            double xc3d                         = -xb2d * costhe - yc2d * sinthe;
            double yc3d                         = -xb2d * sinthe + yc2d * costhe;
            double zc3d                         =  zc1d;

            // Step5  A3:
            cSim.pImageX[shakeID.x]             = XCOM + TRNS11 * xa3d + TRNS12 * ya3d + TRNS13 * za3d;
            cSim.pImageY[shakeID.x]             = YCOM + TRNS21 * xa3d + TRNS22 * ya3d + TRNS23 * za3d;
            cSim.pImageZ[shakeID.x]             = ZCOM + TRNS31 * xa3d + trns32 * ya3d + trns33 * za3d;
            cSim.pImageX[shakeID.y]             = XCOM + TRNS11 * xb3d + TRNS12 * yb3d + TRNS13 * zb3d;
            cSim.pImageY[shakeID.y]             = YCOM + TRNS21 * xb3d + TRNS22 * yb3d + TRNS23 * zb3d;
            cSim.pImageZ[shakeID.y]             = ZCOM + TRNS31 * xb3d + trns32 * yb3d + trns33 * zb3d;
            cSim.pImageX[shakeID.z]             = XCOM + TRNS11 * xc3d + TRNS12 * yc3d + TRNS13 * zc3d;
            cSim.pImageY[shakeID.z]             = YCOM + TRNS21 * xc3d + TRNS22 * yc3d + TRNS23 * zc3d;
            cSim.pImageZ[shakeID.z]             = ZCOM + TRNS31 * xc3d + trns32 * yc3d + trns33 * zc3d;                                 
        }
    }
    else if (pos < cSim.slowShakeOffset)
    {    
        pos                                    -= cSim.fastShakeOffset;
        if (pos < cSim.slowShakeConstraints)
        {
            // Read SHAKE network data
#if (__CUDA_ARCH__ >= 200)
            double invMassI;
            double xpl;
            double ypl;
            double zpl;
            double xil;
            double yil;
            double zil;
#else
            PMEAtom* psA                        = &sA[threadIdx.x];
#endif
            int shakeID1                        = cSim.pImageSlowShakeID1[pos];
            int4 shakeID2                       = cSim.pImageSlowShakeID2[pos];
            double2 shakeParm                   = cSim.pSlowShakeParm[pos];
        
            // Read SHAKE network components
#if defined(NODPTEXTURE)
            double xi                           = cSim.pForceX[shakeID1];
            double yi                           = cSim.pForceY[shakeID1];
            double zi                           = cSim.pForceZ[shakeID1];
            double xij                          = cSim.pForceX[shakeID2.x];
            double yij                          = cSim.pForceY[shakeID2.x];
            double zij                          = cSim.pForceZ[shakeID2.x];
            double xik                          = cSim.pForceX[shakeID2.y];
            double yik                          = cSim.pForceY[shakeID2.y];
            double zik                          = cSim.pForceZ[shakeID2.y];  
            XIL                                 = cSim.pForceX[shakeID2.z];
            YIL                                 = cSim.pForceY[shakeID2.z];
            ZIL                                 = cSim.pForceZ[shakeID2.z]; 
            double xim                          = cSim.pForceX[shakeID2.w];
            double yim                          = cSim.pForceY[shakeID2.w];
            double zim                          = cSim.pForceZ[shakeID2.w];       
#else        
            int2 ixi                            = tex1Dfetch(texref, shakeID1);
            int2 iyi                            = tex1Dfetch(texref, shakeID1 + cSim.stride);
            int2 izi                            = tex1Dfetch(texref, shakeID1 + cSim.stride2);
            int2 ixij                           = tex1Dfetch(texref, shakeID2.x);
            int2 iyij                           = tex1Dfetch(texref, shakeID2.x + cSim.stride);
            int2 izij                           = tex1Dfetch(texref, shakeID2.x + cSim.stride2);
            int2 ixik                           = tex1Dfetch(texref, shakeID2.y);
            int2 iyik                           = tex1Dfetch(texref, shakeID2.y + cSim.stride);
            int2 izik                           = tex1Dfetch(texref, shakeID2.y + cSim.stride2);
            int2 ixil                           = tex1Dfetch(texref, shakeID2.z);
            int2 iyil                           = tex1Dfetch(texref, shakeID2.z + cSim.stride);
            int2 izil                           = tex1Dfetch(texref, shakeID2.z + cSim.stride2); 
            int2 ixim                           = tex1Dfetch(texref, shakeID2.w);
            int2 iyim                           = tex1Dfetch(texref, shakeID2.w + cSim.stride);
            int2 izim                           = tex1Dfetch(texref, shakeID2.w + cSim.stride2);
            double xi                           = __hiloint2double(ixi.y, ixi.x);
            double yi                           = __hiloint2double(iyi.y, iyi.x);
            double zi                           = __hiloint2double(izi.y, izi.x);
            double xij                          = __hiloint2double(ixij.y, ixij.x);
            double yij                          = __hiloint2double(iyij.y, iyij.x);
            double zij                          = __hiloint2double(izij.y, izij.x);
            double xik                          = __hiloint2double(ixik.y, ixik.x);
            double yik                          = __hiloint2double(iyik.y, iyik.x);
            double zik                          = __hiloint2double(izik.y, izik.x);   
            XIL                                 = __hiloint2double(ixil.y, ixil.x);
            YIL                                 = __hiloint2double(iyil.y, iyil.x);
            ZIL                                 = __hiloint2double(izil.y, izil.x);   
            double xim                          = __hiloint2double(ixim.y, ixim.x);
            double yim                          = __hiloint2double(iyim.y, iyim.x);
            double zim                          = __hiloint2double(izim.y, izim.x);   
#endif 
            double xpi                          = cSim.pImageX[shakeID1];
            double ypi                          = cSim.pImageY[shakeID1];
            double zpi                          = cSim.pImageZ[shakeID1];
            double xpj                          = cSim.pImageX[shakeID2.x];
            double ypj                          = cSim.pImageY[shakeID2.x];
            double zpj                          = cSim.pImageZ[shakeID2.x];
            double xpk                          = cSim.pImageX[shakeID2.y];
            double ypk                          = cSim.pImageY[shakeID2.y];
            double zpk                          = cSim.pImageZ[shakeID2.y];
            XPL                                 = cSim.pImageX[shakeID2.z];
            YPL                                 = cSim.pImageY[shakeID2.z];
            ZPL                                 = cSim.pImageZ[shakeID2.z];
            double xpm                          = cSim.pImageX[shakeID2.w];
            double ypm                          = cSim.pImageY[shakeID2.w];
            double zpm                          = cSim.pImageZ[shakeID2.w];                      
            INVMASSI                            = shakeParm.x;
            double toler                        = shakeParm.y;
            
            // Calculate unchanging quantities
            xij                                 = xi - xij;
            yij                                 = yi - yij;
            zij                                 = zi - zij;
            xik                                 = xi - xik;
            yik                                 = yi - yik;
            zik                                 = zi - zik; 
            XIL                                 = xi - XIL;
            YIL                                 = yi - YIL;
            ZIL                                 = zi - ZIL;
            xim                                 = xi - xim;
            yim                                 = yi - yim;
            zim                                 = zi - zim;               
           
            bool done                           = false;
            for (int i = 0; i < 3000; i++)
            {
                done                            = true;
                
                // Calculate nominal distance squared
                double xpxx                     = xpi - xpj;
                double ypxx                     = ypi - ypj;
                double zpxx                     = zpi - zpj;
                double rpxx2                    = xpxx * xpxx + ypxx * ypxx + zpxx * zpxx;
          
                // Apply correction to first hydrogen
                double diff                     = toler - rpxx2;
                if (abs(diff) >= toler * cSim.tol)
                {
                    done                        = false;
                   
                    // Shake resetting of coordinate is done here
                    double rrpr                 = xij * xpxx + yij * ypxx + zij * zpxx;     
                    if (rrpr >= toler * 1.0e-06)
                    {
                    
                        double acor             = diff / (rrpr * (double)2.0 * (INVMASSI + cSim.invMassH));
                        double h                = xij * acor;
                        xpi                    += h * INVMASSI;
                        xpj                    -= h * cSim.invMassH;
                        h                       = yij * acor;
                        ypi                    += h * INVMASSI;
                        ypj                    -= h * cSim.invMassH;
                        h                       = zij * acor;
                        zpi                    += h * INVMASSI;
                        zpj                    -= h * cSim.invMassH;             
                    }
                }
          
     
                xpxx                            = xpi - xpk;
                ypxx                            = ypi - ypk;
                zpxx                            = zpi - zpk;
                rpxx2                           = xpxx * xpxx + ypxx * ypxx + zpxx * zpxx;
          
                // Apply correction to second hydrogen
                diff                            = toler - rpxx2;
                if (abs(diff) >= toler * cSim.tol)
                {
                    done                        = false;
               
                    // Shake resetting of coordinate is done here
                    double rrpr                 = xik * xpxx + yik * ypxx + zik * zpxx;     
                    if (rrpr >= toler * 1.0e-06)
                    {
                    
                        double acor             = diff / (rrpr * 2.0 * (INVMASSI + cSim.invMassH));
                        double h                = xik * acor;
                        xpi                    += h * INVMASSI;
                        xpk                    -= h * cSim.invMassH;
                        h                       = yik * acor;
                        ypi                    += h * INVMASSI;
                        ypk                    -= h * cSim.invMassH;
                        h                       = zik * acor;
                        zpi                    += h * INVMASSI;
                        zpk                    -= h * cSim.invMassH;             
                    }
                }
                
 
                xpxx                            = xpi - XPL;
                ypxx                            = ypi - YPL;
                zpxx                            = zpi - ZPL;        
                rpxx2                           = xpxx * xpxx + ypxx * ypxx + zpxx * zpxx;
          
                // Apply correction to third hydrogen
                diff                            = toler - rpxx2;
                if (abs(diff) >= toler * cSim.tol)
                {
                    done                        = false;
                  
                    // Shake resetting of coordinate is done here
                    double rrpr                 = XIL * xpxx + YIL * ypxx + ZIL * zpxx;     
                    if (rrpr >= toler * 1.0e-06)
                    {             
                        double acor             = diff / (rrpr * (double)2.0 * (INVMASSI + cSim.invMassH));
                        double h                = XIL * acor;
                        xpi                    += h * INVMASSI;
                        XPL                    -= h * cSim.invMassH;
                        h                       = YIL * acor;
                        ypi                    += h * INVMASSI;
                        YPL                    -= h * cSim.invMassH;
                        h                       = ZIL * acor;
                        zpi                    += h * INVMASSI;
                        ZPL                    -= h * cSim.invMassH;             
                    }
                }

                xpxx                            = xpi - xpm;
                ypxx                            = ypi - ypm;
                zpxx                            = zpi - zpm;        
                rpxx2                           = xpxx * xpxx + ypxx * ypxx + zpxx * zpxx;

                // Apply correction to third hydrogen
                diff                            = toler - rpxx2;
                if (abs(diff) >= toler * cSim.tol)
                {
                    done                        = false;
                  
                    // Shake resetting of coordinate is done here
                    double rrpr                 = xim * xpxx + yim * ypxx + zim * zpxx;     
                    if (rrpr >= toler * 1.0e-06)
                    {             
                        double acor             = diff / (rrpr * (double)2.0 * (INVMASSI + cSim.invMassH));
                        double h                = xim * acor;
                        xpi                    += h * INVMASSI;
                        xpm                    -= h * cSim.invMassH;
                        h                       = yim * acor;
                        ypi                    += h * INVMASSI;
                        ypm                    -= h * cSim.invMassH;
                        h                       = zim * acor;
                        zpi                    += h * INVMASSI;
                        zpm                    -= h * cSim.invMassH;             
                    }
                }

                
                
                // Check for convergence
                if (done)
                    break;
            }
          
            // Write out results if converged, but there's no really good
            // way to indicate failure so we'll let the simulation heading
            // off to Neptune do that for us.  Wish there were a better way,
            // but until the CPU needs something from the GPU, those are the
            // the breaks.  I guess, technically, we could just set a flag to NOP
            // the simulation from here and then carry that result through upon
            // the next ntpr, ntwc, or ntwx update, but I leave that up to you 
            // guys to implement that (or not). 
            if (done)
            {
                cSim.pImageX[shakeID1]          = xpi;
                cSim.pImageY[shakeID1]          = ypi;
                cSim.pImageZ[shakeID1]          = zpi;
                cSim.pImageX[shakeID2.x]        = xpj;
                cSim.pImageY[shakeID2.x]        = ypj;
                cSim.pImageZ[shakeID2.x]        = zpj;
                cSim.pImageX[shakeID2.y]        = xpk;
                cSim.pImageY[shakeID2.y]        = ypk;
                cSim.pImageZ[shakeID2.y]        = zpk;
                cSim.pImageX[shakeID2.z]        = XPL;
                cSim.pImageY[shakeID2.z]        = YPL;
                cSim.pImageZ[shakeID2.z]        = ZPL;
                cSim.pImageX[shakeID2.w]        = xpm;
                cSim.pImageY[shakeID2.w]        = ypm;
                cSim.pImageZ[shakeID2.w]        = zpm;
            }                                    
        }
    }
}

void kShakeInitKernels(gpuContext gpu)
{
    if (gpu->sm_version >= SM_3X)
    {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kShake_kernel), hipFuncCachePreferL1);
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kPMEShake_kernel), hipFuncCachePreferL1);
    }
}

void kShake(gpuContext gpu)
{
    texref.normalized = 0;
    texref.filterMode = hipFilterModePoint;
    texref.addressMode[0] = hipAddressModeClamp;
    texref.channelDesc.x = 32;
    texref.channelDesc.y = 32;
    texref.channelDesc.z = 0;
    texref.channelDesc.w = 0;
    hipBindTexture(NULL, texref, (int2*)(gpu->sim.pForce), gpu->sim.stride3 * sizeof(int2));
    unsigned int totalConstraints = gpu->sim.slowShakeOffset;
    unsigned int totalBlocks = (totalConstraints + 63) / 64;
    if (gpu->bNeighborList)
    {
        kPMEShake_kernel<<<totalBlocks, 64>>>();  
    }
    else
    {    
        kShake_kernel<<<totalBlocks, 64>>>();  
    }
    LAUNCHERROR("kShake");
    hipUnbindTexture(texref);
}
