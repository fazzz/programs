#include "hip/hip_runtime.h"
/***************************************************/
/*                                                 */
/*      AMBER NVIDIA CUDA CPU IMPLEMENTATION       */
/*                 PMEMD VERSION                   */
/*                     2010                        */
/*                      by                         */
/*             Scott Le Grand (NVIDIA)             */
/*               Duncan Poole (NVIDIA)             */
/*                Ross Walker (SDSC)               */
/*                                                 */
/***************************************************/

#include <hip/hip_runtime.h>
#include "gpu.h"
static __constant__ cudaSimulation cSim;
static __constant__ PMEFloat te                 = (PMEFloat)(4.0 / 3.0);
static __constant__ PMEFloat tf                 = (PMEFloat)(12.0 / 5.0);
static __constant__ PMEFloat tg                 = (PMEFloat)(24.0 / 7.0);
static __constant__ PMEFloat th                 = (PMEFloat)(40.0 / 9.0);
static __constant__ PMEFloat thh                = (PMEFloat)(60.0 / 11.0);

void SetkCalculateGBNonbondEnergy2Sim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

void GetkCalculateGBNonBondEnergy2Sim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_GBNONBONDENERGY2_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_GBNONBONDENERGY2_THREADS_PER_BLOCK, 1)
#endif
kCalculateGBNonbondEnergy2_kernel()
#include "kCalculateGBNonbondEnergy2.h"

#define GB_IGB78
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_GBNONBONDENERGY2_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_GBNONBONDENERGY2IGB78_THREADS_PER_BLOCK, 1)
#endif
kCalculateGBNonbondEnergy2IGB78_kernel()
#include "kCalculateGBNonbondEnergy2.h"
#undef IGB78

void kCalculateGBNonbondEnergy2(gpuContext gpu)
{
    if ((gpu->sim.igb == 7) || (gpu->sim.igb == 8))
        kCalculateGBNonbondEnergy2IGB78_kernel<<<gpu->blocks, gpu->GBNonbondEnergy2IGB78ThreadsPerBlock>>>();   
    else
        kCalculateGBNonbondEnergy2_kernel<<<gpu->blocks, gpu->GBNonbondEnergy2ThreadsPerBlock>>>();   
    LAUNCHERROR("kCalculateGBNonbondEnergy2");
}
