/***************************************************/
/*                                                 */
/*      AMBER NVIDIA CUDA CPU IMPLEMENTATION       */
/*                 PMEMD VERSION                   */
/*                     2010                        */
/*                      by                         */
/*             Scott Le Grand (NVIDIA)             */
/*               Duncan Poole (NVIDIA)             */
/*                Ross Walker (SDSC)               */
/*                                                 */
/***************************************************/

#include <hip/hip_runtime.h>
#include "gpu.h"
#include <radixsort_single_grid.cu>
#include <radixsort_early_exit.cu>		

using namespace b40c;
static SingleGridRadixSortingEnactor<unsigned int, unsigned int>* pSortEnactor = NULL;
static MultiCtaRadixSortStorage<unsigned int, unsigned int>* pDeviceStorage = NULL;
extern "C" void kNLDeleteRadixSort(gpuContext gpu)
{
    if (pSortEnactor)
        delete pSortEnactor;
    pSortEnactor                                = NULL; 
    if (pDeviceStorage)
        delete pDeviceStorage;
    pDeviceStorage                              = NULL;
}

extern "C" void EnactSort(int bits)
{
	switch (bits)
	{
	    case 9:
	    case 10:
	    case 11:
	    case 12:	    
	        (*pSortEnactor).EnactSort<12>(*pDeviceStorage);
	        break;
	        
	    case 13:
	    case 14:
	    case 15:
	    case 16:
	        (*pSortEnactor).EnactSort<16>(*pDeviceStorage);
	        break;

	    case 17:
	    case 18:
	    case 19:
	    case 20:
	        (*pSortEnactor).EnactSort<20>(*pDeviceStorage);
	        break;

	    case 21:
	    case 22:
	    case 23:
	    case 24:
	        (*pSortEnactor).EnactSort<24>(*pDeviceStorage);
	        break;	        	        

	    case 25:
	    case 26:
	    case 27:
	    case 28:
	        (*pSortEnactor).EnactSort<28>(*pDeviceStorage);
	        break;

	    case 29:
	    case 30:
	    case 31:
	    case 32:
	        (*pSortEnactor).EnactSort<32>(*pDeviceStorage);
	        break;	        
	}
}


extern "C" void kNLInitRadixSort(gpuContext gpu)
{
    // Delete old Radix sort
    kNLDeleteRadixSort(gpu);
   
    // Create new sort
    pDeviceStorage                              = new MultiCtaRadixSortStorage<unsigned int, unsigned int>(gpu->sim.atoms);
    pSortEnactor                                = new SingleGridRadixSortingEnactor<unsigned int, unsigned int>;
    pDeviceStorage->d_keys[0]                   = gpu->sim.pImageHash;
	pDeviceStorage->d_values[0]                 = gpu->sim.pImageIndex;
    pDeviceStorage->d_keys[1]                   = gpu->sim.pImageHash2;
	pDeviceStorage->d_values[1]                 = gpu->sim.pImageIndex2;	
    EnactSort(gpu->neighborListBits);
}



extern "C" void kNLRadixSort(gpuContext gpu)
{
    pDeviceStorage->d_keys[0]                   = gpu->sim.pImageHash;
	pDeviceStorage->d_values[0]                 = gpu->sim.pImageIndex;
    pDeviceStorage->d_keys[1]                   = gpu->sim.pImageHash2;
	pDeviceStorage->d_values[1]                 = gpu->sim.pImageIndex2;	
    EnactSort(gpu->neighborListBits);
    gpu->sim.pImageHash                         = pDeviceStorage->d_keys[pDeviceStorage->selector];           
	gpu->sim.pImageIndex                        = pDeviceStorage->d_values[pDeviceStorage->selector];
    gpu->sim.pImageHash2                        = pDeviceStorage->d_keys[1 - pDeviceStorage->selector];           
	gpu->sim.pImageIndex2                       = pDeviceStorage->d_values[1 - pDeviceStorage->selector];
}

