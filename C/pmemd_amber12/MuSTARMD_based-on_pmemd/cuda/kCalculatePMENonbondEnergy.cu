#include "hip/hip_runtime.h"
/***************************************************/
/*                                                 */
/*      AMBER NVIDIA CUDA CPU IMPLEMENTATION       */
/*                 PMEMD VERSION                   */
/*                     2010                        */
/*                      by                         */
/*             Scott Le Grand (NVIDIA)             */
/*               Duncan Poole (NVIDIA)             */
/*                Ross Walker (SDSC)               */
/*                                                 */
/***************************************************/

#include <hip/hip_runtime.h>
#include "gpu.h"

//#define PME_VIRIAL
//#define PME_ENERGY


static __constant__ cudaSimulation cSim;

void SetkCalculatePMENonbondEnergySim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

void GetkCalculatePMENonBondEnergySim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}


#ifndef use_DPDP
static __forceinline__ __device__ float __internal_fmad(float a, float b, float c)
{
#if __CUDA_ARCH__ >= 200
  return __fmaf_rn (a, b, c);
#else /* __CUDA_ARCH__ >= 200 */
  return a * b + c;
#endif /* __CUDA_ARCH__ >= 200 */
}

// Faster ERFC approximation courtesy of Norbert Juffa. NVIDIA Corporation
static __forceinline__ __device__ PMEFloat fasterfc(PMEFloat a) 
{
  /* approximate log(erfc(a)) with rel. error < 7e-9 */
  PMEFloat t, x = a;
  t =                       (PMEFloat)-1.6488499458192755E-006;
  t = __internal_fmad(t, x, (PMEFloat)2.9524665006554534E-005);
  t = __internal_fmad(t, x, (PMEFloat)-2.3341951153749626E-004);
  t = __internal_fmad(t, x, (PMEFloat)1.0424943374047289E-003);
  t = __internal_fmad(t, x, (PMEFloat)-2.5501426008983853E-003);
  t = __internal_fmad(t, x, (PMEFloat)3.1979939710877236E-004);
  t = __internal_fmad(t, x, (PMEFloat)2.7605379075746249E-002);
  t = __internal_fmad(t, x, (PMEFloat)-1.4827402067461906E-001);
  t = __internal_fmad(t, x, (PMEFloat)-9.1844764013203406E-001);
  t = __internal_fmad(t, x, (PMEFloat)-1.6279070384382459E+000);
  t = t * x;
  return exp2f(t);
}
#endif




// Nonbond kernels

#define PME_ATOMS_PER_WARP (32)
#define PME_VIRIAL
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_PMENONBONDENERGY_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_PMENONBONDENERGY_THREADS_PER_BLOCK, 1)
#endif
kCalculatePMENonbondForcesVirial32_kernel()
#include "kNLCPNE.h"

#define PME_ENERGY

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_PMENONBONDENERGY_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_PMENONBONDENERGY_THREADS_PER_BLOCK, 1)
#endif
kCalculatePMENonbondEnergyVirial32_kernel()
#include "kNLCPNE.h"

#undef PME_ENERGY

#define PME_IS_ORTHOGONAL
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_PMENONBONDENERGY_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_PMENONBONDENERGY_THREADS_PER_BLOCK, 1)
#endif
kCalculatePMEOrthogonalNonbondForcesVirial32_kernel()
#include "kNLCPNE.h"

#define PME_ENERGY

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_PMENONBONDENERGY_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_PMENONBONDENERGY_THREADS_PER_BLOCK, 1)
#endif
kCalculatePMEOrthogonalNonbondEnergyVirial32_kernel()
#include "kNLCPNE.h"

#undef PME_ENERGY
#undef PME_IS_ORTHOGONAL
#undef PME_VIRIAL

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_PMENONBONDENERGY_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_PMENONBONDENERGY_THREADS_PER_BLOCK, 1)
#endif
kCalculatePMENonbondForces32_kernel()
#include "kNLCPNE.h"

#define PME_ENERGY

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_PMENONBONDENERGY_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_PMENONBONDENERGY_THREADS_PER_BLOCK, 1)
#endif
kCalculatePMENonbondEnergy32_kernel()
#include "kNLCPNE.h"

#undef PME_ENERGY

#define PME_IS_ORTHOGONAL
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_PMENONBONDENERGY_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_PMENONBONDENERGY_THREADS_PER_BLOCK, 1)
#endif
kCalculatePMEOrthogonalNonbondForces32_kernel()
#include "kNLCPNE.h"

#define PME_ENERGY

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_PMENONBONDENERGY_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_PMENONBONDENERGY_THREADS_PER_BLOCK, 1)
#endif
kCalculatePMEOrthogonalNonbondEnergy32_kernel()
#include "kNLCPNE.h"

#undef PME_ENERGY
#undef PME_IS_ORTHOGONAL
#undef PME_ATOMS_PER_WARP



#define PME_ATOMS_PER_WARP (16)
#define PME_VIRIAL

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_PMENONBONDENERGY_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_PMENONBONDENERGY_THREADS_PER_BLOCK, 1)
#endif
kCalculatePMENonbondForcesVirial16_kernel()
#include "kNLCPNE.h"

#define PME_ENERGY

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_PMENONBONDENERGY_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_PMENONBONDENERGY_THREADS_PER_BLOCK, 1)
#endif
kCalculatePMENonbondEnergyVirial16_kernel()
#include "kNLCPNE.h"

#undef PME_ENERGY

#define PME_IS_ORTHOGONAL
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_PMENONBONDENERGY_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_PMENONBONDENERGY_THREADS_PER_BLOCK, 1)
#endif
kCalculatePMEOrthogonalNonbondForcesVirial16_kernel()
#include "kNLCPNE.h"

#define PME_ENERGY

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_PMENONBONDENERGY_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_PMENONBONDENERGY_THREADS_PER_BLOCK, 1)
#endif
kCalculatePMEOrthogonalNonbondEnergyVirial16_kernel()
#include "kNLCPNE.h"

#undef PME_ENERGY
#undef PME_IS_ORTHOGONAL
#undef PME_VIRIAL

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_PMENONBONDENERGY_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_PMENONBONDENERGY_THREADS_PER_BLOCK, 1)
#endif
kCalculatePMENonbondForces16_kernel()
#include "kNLCPNE.h"

#define PME_ENERGY

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_PMENONBONDENERGY_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_PMENONBONDENERGY_THREADS_PER_BLOCK, 1)
#endif
kCalculatePMENonbondEnergy16_kernel()
#include "kNLCPNE.h"

#undef PME_ENERGY

#define PME_IS_ORTHOGONAL
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_PMENONBONDENERGY_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_PMENONBONDENERGY_THREADS_PER_BLOCK, 1)
#endif
kCalculatePMEOrthogonalNonbondForces16_kernel()
#include "kNLCPNE.h"

#define PME_ENERGY

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_PMENONBONDENERGY_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_PMENONBONDENERGY_THREADS_PER_BLOCK, 1)
#endif
kCalculatePMEOrthogonalNonbondEnergy16_kernel()
#include "kNLCPNE.h"

#undef PME_ENERGY
#undef PME_IS_ORTHOGONAL
#undef PME_ATOMS_PER_WARP





extern "C" void kCalculatePMENonbondForces(gpuContext gpu)
{   
    if (gpu->sim.NLAtomsPerWarp == 32)
    {
        if (gpu->sim.ntp > 0)
        {
             if (gpu->sim.is_orthog)
                kCalculatePMEOrthogonalNonbondForcesVirial32_kernel<<<gpu->blocks, gpu->PMENonbondEnergyThreadsPerBlock>>>();   
            else    
                kCalculatePMENonbondForcesVirial32_kernel<<<gpu->blocks, gpu->PMENonbondEnergyThreadsPerBlock>>>();      
        }
        else
        {
            if (gpu->sim.is_orthog)
                kCalculatePMEOrthogonalNonbondForces32_kernel<<<gpu->blocks, gpu->PMENonbondEnergyThreadsPerBlock>>>();   
            else    
                kCalculatePMENonbondForces32_kernel<<<gpu->blocks, gpu->PMENonbondEnergyThreadsPerBlock>>>();   
        }    
    }
    else
    {
        if (gpu->sim.ntp > 0)
        {
             if (gpu->sim.is_orthog)
                kCalculatePMEOrthogonalNonbondForcesVirial16_kernel<<<gpu->blocks, gpu->PMENonbondEnergyThreadsPerBlock>>>();   
            else    
                kCalculatePMENonbondForcesVirial16_kernel<<<gpu->blocks, gpu->PMENonbondEnergyThreadsPerBlock>>>();      
        }
        else
        {
            if (gpu->sim.is_orthog)
                kCalculatePMEOrthogonalNonbondForces16_kernel<<<gpu->blocks, gpu->PMENonbondEnergyThreadsPerBlock>>>();   
            else    
                kCalculatePMENonbondForces16_kernel<<<gpu->blocks, gpu->PMENonbondEnergyThreadsPerBlock>>>();   
        }    
    }


    
    
    
#ifdef MPI 
    LAUNCHERROR_NONBLOCKING("kCalculatePMENonbondForces");
#else
    LAUNCHERROR("kCalculatePMENonbondForces");
#endif  
}


extern "C" void kCalculatePMENonbondEnergy(gpuContext gpu)
{
    if (gpu->sim.NLAtomsPerWarp == 32)
    {
        if (gpu->sim.ntp > 0)
        {
            if (gpu->sim.is_orthog)
                kCalculatePMEOrthogonalNonbondEnergyVirial32_kernel<<<gpu->blocks, gpu->PMENonbondEnergyThreadsPerBlock>>>();   
            else
                kCalculatePMENonbondEnergyVirial32_kernel<<<gpu->blocks, gpu->PMENonbondEnergyThreadsPerBlock>>>();    
        }
        else
        {
            if (gpu->sim.is_orthog)
                kCalculatePMEOrthogonalNonbondEnergy32_kernel<<<gpu->blocks, gpu->PMENonbondEnergyThreadsPerBlock>>>();   
            else
                kCalculatePMENonbondEnergy32_kernel<<<gpu->blocks, gpu->PMENonbondEnergyThreadsPerBlock>>>(); 
        }    
    }
    else
    {
        if (gpu->sim.ntp > 0)
        {
            if (gpu->sim.is_orthog)
                kCalculatePMEOrthogonalNonbondEnergyVirial16_kernel<<<gpu->blocks, gpu->PMENonbondEnergyThreadsPerBlock>>>();   
            else
                kCalculatePMENonbondEnergyVirial16_kernel<<<gpu->blocks, gpu->PMENonbondEnergyThreadsPerBlock>>>();    
        }
        else
        {
            if (gpu->sim.is_orthog)
                kCalculatePMEOrthogonalNonbondEnergy16_kernel<<<gpu->blocks, gpu->PMENonbondEnergyThreadsPerBlock>>>();   
            else
                kCalculatePMENonbondEnergy16_kernel<<<gpu->blocks, gpu->PMENonbondEnergyThreadsPerBlock>>>(); 
        }
    }
#ifdef MPI 
    LAUNCHERROR_NONBLOCKING("kCalculatePMENonbondEnergy");
#else
    LAUNCHERROR("kCalculatePMENonbondEnergy");
#endif  
}


#define IPS_ATOMS_PER_WARP (32)
#define IPS_VIRIAL
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_IPSNONBONDENERGY_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_IPSNONBONDENERGY_THREADS_PER_BLOCK, 1)
#endif
kCalculateIPSNonbondForcesVirial32_kernel()
#include "kNLCINE.h"

#define IPS_ENERGY

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_IPSNONBONDENERGY_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_IPSNONBONDENERGY_THREADS_PER_BLOCK, 1)
#endif
kCalculateIPSNonbondEnergyVirial32_kernel()
#include "kNLCINE.h"

#undef IPS_ENERGY

#define IPS_IS_ORTHOGONAL
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_IPSNONBONDENERGY_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_IPSNONBONDENERGY_THREADS_PER_BLOCK, 1)
#endif
kCalculateIPSOrthogonalNonbondForcesVirial32_kernel()
#include "kNLCINE.h"

#define IPS_ENERGY

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_IPSNONBONDENERGY_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_IPSNONBONDENERGY_THREADS_PER_BLOCK, 1)
#endif
kCalculateIPSOrthogonalNonbondEnergyVirial32_kernel()
#include "kNLCINE.h"

#undef IPS_ENERGY
#undef IPS_IS_ORTHOGONAL
#undef IPS_VIRIAL

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_IPSNONBONDENERGY_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_IPSNONBONDENERGY_THREADS_PER_BLOCK, 1)
#endif
kCalculateIPSNonbondForces32_kernel()
#include "kNLCINE.h"

#define IPS_ENERGY

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_IPSNONBONDENERGY_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_IPSNONBONDENERGY_THREADS_PER_BLOCK, 1)
#endif
kCalculateIPSNonbondEnergy32_kernel()
#include "kNLCINE.h"

#undef IPS_ENERGY

#define IPS_IS_ORTHOGONAL
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_IPSNONBONDENERGY_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_IPSNONBONDENERGY_THREADS_PER_BLOCK, 1)
#endif
kCalculateIPSOrthogonalNonbondForces32_kernel()
#include "kNLCINE.h"

#define IPS_ENERGY

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_IPSNONBONDENERGY_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_IPSNONBONDENERGY_THREADS_PER_BLOCK, 1)
#endif
kCalculateIPSOrthogonalNonbondEnergy32_kernel()
#include "kNLCINE.h"

#undef IPS_ENERGY
#undef IPS_IS_ORTHOGONAL
#undef IPS_ATOMS_PER_WARP



#define IPS_ATOMS_PER_WARP (16)
#define IPS_VIRIAL

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_IPSNONBONDENERGY_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_IPSNONBONDENERGY_THREADS_PER_BLOCK, 1)
#endif
kCalculateIPSNonbondForcesVirial16_kernel()
#include "kNLCINE.h"

#define IPS_ENERGY

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_IPSNONBONDENERGY_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_IPSNONBONDENERGY_THREADS_PER_BLOCK, 1)
#endif
kCalculateIPSNonbondEnergyVirial16_kernel()
#include "kNLCINE.h"

#undef IPS_ENERGY

#define IPS_IS_ORTHOGONAL
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_IPSNONBONDENERGY_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_IPSNONBONDENERGY_THREADS_PER_BLOCK, 1)
#endif
kCalculateIPSOrthogonalNonbondForcesVirial16_kernel()
#include "kNLCINE.h"

#define IPS_ENERGY

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_IPSNONBONDENERGY_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_IPSNONBONDENERGY_THREADS_PER_BLOCK, 1)
#endif
kCalculateIPSOrthogonalNonbondEnergyVirial16_kernel()
#include "kNLCINE.h"

#undef IPS_ENERGY
#undef IPS_IS_ORTHOGONAL
#undef IPS_VIRIAL

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_IPSNONBONDENERGY_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_IPSNONBONDENERGY_THREADS_PER_BLOCK, 1)
#endif
kCalculateIPSNonbondForces16_kernel()
#include "kNLCINE.h"

#define IPS_ENERGY

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_IPSNONBONDENERGY_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_IPSNONBONDENERGY_THREADS_PER_BLOCK, 1)
#endif
kCalculateIPSNonbondEnergy16_kernel()
#include "kNLCINE.h"

#undef IPS_ENERGY

#define IPS_IS_ORTHOGONAL
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_IPSNONBONDENERGY_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_IPSNONBONDENERGY_THREADS_PER_BLOCK, 1)
#endif
kCalculateIPSOrthogonalNonbondForces16_kernel()
#include "kNLCINE.h"

#define IPS_ENERGY

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_IPSNONBONDENERGY_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_IPSNONBONDENERGY_THREADS_PER_BLOCK, 1)
#endif
kCalculateIPSOrthogonalNonbondEnergy16_kernel()
#include "kNLCINE.h"

#undef IPS_ENERGY
#undef IPS_IS_ORTHOGONAL
#undef IPS_ATOMS_PER_WARP





extern "C" void kCalculateIPSNonbondForces(gpuContext gpu)
{   
    if (gpu->sim.NLAtomsPerWarp == 32)
    {
        if (gpu->sim.ntp > 0)
        {
             if (gpu->sim.is_orthog)
                kCalculateIPSOrthogonalNonbondForcesVirial32_kernel<<<gpu->blocks, gpu->IPSNonbondEnergyThreadsPerBlock>>>();   
            else    
                kCalculateIPSNonbondForcesVirial32_kernel<<<gpu->blocks, gpu->IPSNonbondEnergyThreadsPerBlock>>>();      
        }
        else
        {
            if (gpu->sim.is_orthog)
                kCalculateIPSOrthogonalNonbondForces32_kernel<<<gpu->blocks, gpu->IPSNonbondEnergyThreadsPerBlock>>>();   
            else    
                kCalculateIPSNonbondForces32_kernel<<<gpu->blocks, gpu->IPSNonbondEnergyThreadsPerBlock>>>();   
        }    
    }
    else
    {
        if (gpu->sim.ntp > 0)
        {
             if (gpu->sim.is_orthog)
                kCalculateIPSOrthogonalNonbondForcesVirial16_kernel<<<gpu->blocks, gpu->IPSNonbondEnergyThreadsPerBlock>>>();   
            else    
                kCalculateIPSNonbondForcesVirial16_kernel<<<gpu->blocks, gpu->IPSNonbondEnergyThreadsPerBlock>>>();      
        }
        else
        {
            if (gpu->sim.is_orthog)
                kCalculateIPSOrthogonalNonbondForces16_kernel<<<gpu->blocks, gpu->IPSNonbondEnergyThreadsPerBlock>>>();   
            else    
                kCalculateIPSNonbondForces16_kernel<<<gpu->blocks, gpu->IPSNonbondEnergyThreadsPerBlock>>>();   
        }    
    }


    
    
    
#ifdef MPI 
    LAUNCHERROR_NONBLOCKING("kCalculateIPSNonbondForces");
#else
    LAUNCHERROR("kCalculateIPSNonbondForces");
#endif  
}


extern "C" void kCalculateIPSNonbondEnergy(gpuContext gpu)
{
    if (gpu->sim.NLAtomsPerWarp == 32)
    {
        if (gpu->sim.ntp > 0)
        {
            if (gpu->sim.is_orthog)
                kCalculateIPSOrthogonalNonbondEnergyVirial32_kernel<<<gpu->blocks, gpu->IPSNonbondEnergyThreadsPerBlock>>>();   
            else
                kCalculateIPSNonbondEnergyVirial32_kernel<<<gpu->blocks, gpu->IPSNonbondEnergyThreadsPerBlock>>>();    
        }
        else
        {
            if (gpu->sim.is_orthog)
                kCalculateIPSOrthogonalNonbondEnergy32_kernel<<<gpu->blocks, gpu->IPSNonbondEnergyThreadsPerBlock>>>();   
            else
                kCalculateIPSNonbondEnergy32_kernel<<<gpu->blocks, gpu->IPSNonbondEnergyThreadsPerBlock>>>(); 
        }    
    }
    else
    {
        if (gpu->sim.ntp > 0)
        {
            if (gpu->sim.is_orthog)
                kCalculateIPSOrthogonalNonbondEnergyVirial16_kernel<<<gpu->blocks, gpu->IPSNonbondEnergyThreadsPerBlock>>>();   
            else
                kCalculateIPSNonbondEnergyVirial16_kernel<<<gpu->blocks, gpu->IPSNonbondEnergyThreadsPerBlock>>>();    
        }
        else
        {
            if (gpu->sim.is_orthog)
                kCalculateIPSOrthogonalNonbondEnergy16_kernel<<<gpu->blocks, gpu->IPSNonbondEnergyThreadsPerBlock>>>();   
            else
                kCalculateIPSNonbondEnergy16_kernel<<<gpu->blocks, gpu->IPSNonbondEnergyThreadsPerBlock>>>(); 
        }
    }
#ifdef MPI 
    LAUNCHERROR_NONBLOCKING("kCalculateIPSNonbondEnergy");
#else
    LAUNCHERROR("kCalculateIPSNonbondEnergy");
#endif  
}






