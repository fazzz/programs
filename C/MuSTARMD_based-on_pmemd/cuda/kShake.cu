#include "hip/hip_runtime.h"
/***************************************************/
/*                                                 */
/*      AMBER NVIDIA CUDA CPU IMPLEMENTATION       */
/*                 PMEMD VERSION                   */
/*                     2010                        */
/*                      by                         */
/*             Scott Le Grand (NVIDIA)             */
/*               Duncan Poole (NVIDIA)             */
/*                Ross Walker (SDSC)               */
/*                                                 */
/***************************************************/

#include <hip/hip_runtime.h>
#include "gpu.h"

static __constant__ cudaSimulation cSim;
struct Atom 
{
    double invMassI;
    double xpl;
    double ypl;
    double zpl;
    double xil;
    double yil;
    double zil;
};

// Texture reference for double-precision coordinates (disguised as int2 to work around HW limitations)
#ifndef use_SPSP
texture<int2, 1, hipReadModeElementType> texref;
#else
texture<float, 1, hipReadModeElementType> texref;
#endif

void SetkShakeSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

void GetkShakeSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_SHAKE_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_SHAKE_THREADS_PER_BLOCK, 1)
#endif
kShake_kernel()
{
#if (__CUDA_ARCH__ >= 200)
    __shared__ Atom sA[SM_2X_SHAKE_THREADS_PER_BLOCK];
#else
    __shared__ Atom sA[SM_13_SHAKE_THREADS_PER_BLOCK];
#endif
    Atom* psA                                   = &sA[threadIdx.x];
    unsigned int pos                            = blockIdx.x * blockDim.x + threadIdx.x;   
    while (pos < cSim.shakeConstraints)
    {
    
        // Read SHAKE network data
        int4 shakeID                            = cSim.pShakeID[pos];
        double2 shakeParm                       = cSim.pShakeParm[pos];
        
        // Read SHAKE network components
#ifdef use_SPSP        
        double xi                               = tex1Dfetch(texref, shakeID.x);
        double yi                               = tex1Dfetch(texref, shakeID.x + cSim.stride);
        double zi                               = tex1Dfetch(texref, shakeID.x + cSim.stride2);
        double xij                              = tex1Dfetch(texref, shakeID.y);
        double yij                              = tex1Dfetch(texref, shakeID.y + cSim.stride);
        double zij                              = tex1Dfetch(texref, shakeID.y + cSim.stride2);
#elif defined(NODPTEXTURE) && (__CUDA_ARCH__ < 200)
        double xi                               = cSim.pForceX[shakeID.x];
        double yi                               = cSim.pForceY[shakeID.x];
        double zi                               = cSim.pForceZ[shakeID.x];
        double xij                              = cSim.pForceX[shakeID.y];
        double yij                              = cSim.pForceY[shakeID.y];
        double zij                              = cSim.pForceZ[shakeID.y];
#else        
        int2 ixi                                = tex1Dfetch(texref, shakeID.x);
        int2 iyi                                = tex1Dfetch(texref, shakeID.x + cSim.stride);
        int2 izi                                = tex1Dfetch(texref, shakeID.x + cSim.stride2);
        int2 ixij                               = tex1Dfetch(texref, shakeID.y);
        int2 iyij                               = tex1Dfetch(texref, shakeID.y + cSim.stride);
        int2 izij                               = tex1Dfetch(texref, shakeID.y + cSim.stride2);
#endif                  
        double xpi                              = cSim.pAtomX[shakeID.x];
        double ypi                              = cSim.pAtomY[shakeID.x];
        double zpi                              = cSim.pAtomZ[shakeID.x];
        double xpj                              = cSim.pAtomX[shakeID.y];
        double ypj                              = cSim.pAtomY[shakeID.y];
        double zpj                              = cSim.pAtomZ[shakeID.y];
#if !defined(use_SPSP) && (!defined(NODPTEXTURE) || (__CUDA_ARCH__ >= 200))
        double xi                               = __hiloint2double(ixi.y, ixi.x);
        double yi                               = __hiloint2double(iyi.y, iyi.x);
        double zi                               = __hiloint2double(izi.y, izi.x);
        double xij                              = __hiloint2double(ixij.y, ixij.x);
        double yij                              = __hiloint2double(iyij.y, iyij.x);
        double zij                              = __hiloint2double(izij.y, izij.x); 
#endif               
        psA->invMassI                           = shakeParm.x;
        double toler                            = shakeParm.y;
        
        
        // Optionally read 2nd hydrogen
        double xpk, ypk, zpk, xik, yik, zik;
        if (shakeID.z != -1)
        {
#ifdef use_SPSP
            xik                                 = tex1Dfetch(texref, shakeID.z);
            yik                                 = tex1Dfetch(texref, shakeID.z + cSim.stride);
            zik                                 = tex1Dfetch(texref, shakeID.z + cSim.stride2); 
#elif defined(NODPTEXTURE) && (__CUDA_ARCH__ < 200)
            xik                                 = cSim.pForceX[shakeID.z];
            yik                                 = cSim.pForceY[shakeID.z];
            zik                                 = cSim.pForceZ[shakeID.z];    
#else
            int2 ixik                           = tex1Dfetch(texref, shakeID.z);
            int2 iyik                           = tex1Dfetch(texref, shakeID.z + cSim.stride);
            int2 izik                           = tex1Dfetch(texref, shakeID.z + cSim.stride2);
#endif 
            xpk                                 = cSim.pAtomX[shakeID.z];
            ypk                                 = cSim.pAtomY[shakeID.z];
            zpk                                 = cSim.pAtomZ[shakeID.z];
#if !defined(use_SPSP) && (!defined(NODPTEXTURE) || (__CUDA_ARCH__ >= 200))
            xik                                 = __hiloint2double(ixik.y, ixik.x);
            yik                                 = __hiloint2double(iyik.y, iyik.x);
            zik                                 = __hiloint2double(izik.y, izik.x);  
#endif                  
        }
        
        // Optionally read 3rd hydrogen into shared memory
        if (shakeID.w != -1)
        {
#ifdef use_SPSP
            psA->xil                            = tex1Dfetch(texref, shakeID.w);
            psA->yil                            = tex1Dfetch(texref, shakeID.w + cSim.stride);
            psA->zil                            = tex1Dfetch(texref, shakeID.w + cSim.stride2); 
#elif defined(NODPTEXTURE) && (__CUDA_ARCH__ < 200)     
            psA->xil                            = cSim.pForceX[shakeID.w];
            psA->yil                            = cSim.pForceY[shakeID.w];
            psA->zil                            = cSim.pForceZ[shakeID.w];       
#else            
            int2 ixil                           = tex1Dfetch(texref, shakeID.w);
            int2 iyil                           = tex1Dfetch(texref, shakeID.w + cSim.stride);
            int2 izil                           = tex1Dfetch(texref, shakeID.w + cSim.stride2); 
#endif               
            psA->xpl                            = cSim.pAtomX[shakeID.w];
            psA->ypl                            = cSim.pAtomY[shakeID.w];
            psA->zpl                            = cSim.pAtomZ[shakeID.w]; 
#if !defined(use_SPSP) && (!defined(NODPTEXTURE) || (__CUDA_ARCH__ >= 200))
            psA->xil                            = __hiloint2double(ixil.y, ixil.x);
            psA->yil                            = __hiloint2double(iyil.y, iyil.x);
            psA->zil                            = __hiloint2double(izil.y, izil.x);   
#endif          
        }
        
        // Calculate unchanging quantities
        xij                                     = xi - xij;
        yij                                     = yi - yij;
        zij                                     = zi - zij;
        
        if (shakeID.z != -1)
        {
            xik                                 = xi - xik;
            yik                                 = yi - yik;
            zik                                 = zi - zik;
        }        
         
        if (shakeID.w != -1)
        {
            psA->xil                            = xi - psA->xil;
            psA->yil                            = yi - psA->yil;
            psA->zil                            = zi - psA->zil;
        }      
       
        bool done                               = false;
        for (int i = 0; i < 3000; i++)
        {
            done = true;
            
            // Calculate nominal distance squared
            double xpxx                         = xpi - xpj;
            double ypxx                         = ypi - ypj;
            double zpxx                         = zpi - zpj;
            double rpxx2                        = xpxx * xpxx + ypxx * ypxx + zpxx * zpxx;
      
            // Apply correction
            double diff                         = toler - rpxx2;
            if (abs(diff) >= toler * cSim.tol)
            {
                done                            = false;
               
                // Shake resetting of coordinate is done here
                double rrpr                     = xij * xpxx + yij * ypxx + zij * zpxx;     
                if (rrpr >= toler * 1.0e-06)
                {
                
                    double acor                 = diff / (rrpr * 2.0 * (psA->invMassI + cSim.invMassH));
                    double h                    = xij * acor;
                    xpi                        += h * psA->invMassI;
                    xpj                        -= h * cSim.invMassH;
                    h                           = yij * acor;
                    ypi                        += h * psA->invMassI;
                    ypj                        -= h * cSim.invMassH;
                    h                           = zij * acor;
                    zpi                        += h * psA->invMassI;
                    zpj                        -= h * cSim.invMassH;             
                }
            }
      
            // Second bond if present
            if (shakeID.z != -1)
            {
                xpxx                            = xpi - xpk;
                ypxx                            = ypi - ypk;
                zpxx                            = zpi - zpk;
                rpxx2                           = xpxx * xpxx + ypxx * ypxx + zpxx * zpxx;
      
                // Apply correction
                diff                            = toler - rpxx2;
                if (abs(diff) >= toler * cSim.tol)
                {
                    done                        = false;
               
                    // Shake resetting of coordinate is done here
                    double rrpr              = xik * xpxx + yik * ypxx + zik * zpxx;     
                    if (rrpr >= toler * 1.0e-06)
                    {
                
                        double acor          = diff / (rrpr * 2.0 * (psA->invMassI + cSim.invMassH));
                        double h             = xik * acor;
                        xpi                    += h * psA->invMassI;
                        xpk                    -= h * cSim.invMassH;
                        h                       = yik * acor;
                        ypi                    += h * psA->invMassI;
                        ypk                    -= h * cSim.invMassH;
                        h                       = zik * acor;
                        zpi                    += h * psA->invMassI;
                        zpk                    -= h * cSim.invMassH;             
                    }
                }
            }
            
            // Third bond if present
            if (shakeID.w != -1)
            {
                xpxx                            = xpi - psA->xpl;
                ypxx                            = ypi - psA->ypl;
                zpxx                            = zpi - psA->zpl;
                rpxx2                           = xpxx * xpxx + ypxx * ypxx + zpxx * zpxx;
      
                // Apply correction
                diff                            = toler - rpxx2;
                if (abs(diff) >= toler * cSim.tol)
                {
                    done                        = false;
               
                    // Shake resetting of coordinate is done here
                    double rrpr              = psA->xil * xpxx + psA->yil * ypxx + psA->zil * zpxx;     
                    if (rrpr >= toler * 1.0e-06)
                    {
                
                        double acor             = diff / (rrpr * 2.0 * (psA->invMassI + cSim.invMassH));
                        double h                = psA->xil * acor;
                        xpi                    += h * psA->invMassI;
                        psA->xpl               -= h * cSim.invMassH;
                        h                       = psA->yil * acor;
                        ypi                    += h * psA->invMassI;
                        psA->ypl               -= h * cSim.invMassH;
                        h                       = psA->zil * acor;
                        zpi                    += h * psA->invMassI;
                        psA->zpl               -= h * cSim.invMassH;             
                    }
                }
            }
            
            
            // Check for convergence
            if (done)
                break;
        }
      
        // Write out results if converged, but there's no really good
        // way to indicate failure so we'll let the simulation heading
        // off to Neptune do that for us.  Wish there were a better way,
        // but until the CPU needs something from the GPU, those are the
        // the breaks.  I guess, technically, we could just set a flag to NOP
        // the simulation from here and then carry that result through upon
        // the next ntpr, ntwc, or ntwx update, but I leave that up to you 
        // guys to implement that (or not). 
        if (done)
        {
            cSim.pAtomX[shakeID.x]              = xpi;
            cSim.pAtomY[shakeID.x]              = ypi;
            cSim.pAtomZ[shakeID.x]              = zpi;
            PMEFloat2 xyi                       = {xpi, ypi};
            cSim.pAtomXYSP[shakeID.x]           = xyi;
            cSim.pAtomZSP[shakeID.x]            = zpi;
            
            cSim.pAtomX[shakeID.y]              = xpj;
            cSim.pAtomY[shakeID.y]              = ypj;
            cSim.pAtomZ[shakeID.y]              = zpj;
            PMEFloat2 xyj                       = {xpj, ypj};
            cSim.pAtomXYSP[shakeID.y]           = xyj;
            cSim.pAtomZSP[shakeID.y]            = zpj;

            if (shakeID.z != -1)
            {
                cSim.pAtomX[shakeID.z]          = xpk;
                cSim.pAtomY[shakeID.z]          = ypk;
                cSim.pAtomZ[shakeID.z]          = zpk;
                PMEFloat2 xyk                   = {xpk, ypk};
                cSim.pAtomXYSP[shakeID.z]       = xyk;
                cSim.pAtomZSP[shakeID.z]        = zpk;
            }
        
            if (shakeID.w != -1)
            {
                cSim.pAtomX[shakeID.w]          = psA->xpl;
                cSim.pAtomY[shakeID.w]          = psA->ypl;
                cSim.pAtomZ[shakeID.w]          = psA->zpl;
                PMEFloat2 xyl                   = {psA->xpl, psA->ypl};
                cSim.pAtomXYSP[shakeID.w]       = xyl;
                cSim.pAtomZSP[shakeID.w]        = psA->zpl;
            }
        
        }
        
        pos                                    += gridDim.x * blockDim.x;
    }

    if (cSim.fastShakeConstraints > 0)
    {    
        while (pos < cSim.shakeOffset)
        {
            pos                                += gridDim.x * blockDim.x;
        }
        pos                                    -= cSim.shakeOffset;

        while (pos < cSim.fastShakeConstraints)
        {
            
            // Read atom data
            int4 shakeID                        = cSim.pFastShakeID[pos];
#ifdef use_SPSP
            double x1                           = tex1Dfetch(texref, shakeID.x);
            double y1                           = tex1Dfetch(texref, shakeID.x + cSim.stride);
            double z1                           = tex1Dfetch(texref, shakeID.x + cSim.stride2);
            double x2                           = tex1Dfetch(texref, shakeID.y);
            double y2                           = tex1Dfetch(texref, shakeID.y + cSim.stride);
            double z2                           = tex1Dfetch(texref, shakeID.y + cSim.stride2);
            double x3                           = tex1Dfetch(texref, shakeID.z);
            double y3                           = tex1Dfetch(texref, shakeID.z + cSim.stride);
            double z3                           = tex1Dfetch(texref, shakeID.z + cSim.stride2);  
#elif defined(NODPTEXTURE) && (__CUDA_ARCH__ < 200)
            double x1                           = cSim.pForceX[shakeID.x];
            double y1                           = cSim.pForceY[shakeID.x];
            double z1                           = cSim.pForceZ[shakeID.x]; 
            double x2                           = cSim.pForceX[shakeID.y];
            double y2                           = cSim.pForceY[shakeID.y];
            double z2                           = cSim.pForceZ[shakeID.y]; 
            double x3                           = cSim.pForceX[shakeID.z];
            double y3                           = cSim.pForceY[shakeID.z];
            double z3                           = cSim.pForceZ[shakeID.z]; 
#else           
            int2 ix1                            = tex1Dfetch(texref, shakeID.x);
            int2 iy1                            = tex1Dfetch(texref, shakeID.x + cSim.stride);
            int2 iz1                            = tex1Dfetch(texref, shakeID.x + cSim.stride2);
            int2 ix2                            = tex1Dfetch(texref, shakeID.y);
            int2 iy2                            = tex1Dfetch(texref, shakeID.y + cSim.stride);
            int2 iz2                            = tex1Dfetch(texref, shakeID.y + cSim.stride2);
            int2 ix3                            = tex1Dfetch(texref, shakeID.z);
            int2 iy3                            = tex1Dfetch(texref, shakeID.z + cSim.stride);
            int2 iz3                            = tex1Dfetch(texref, shakeID.z + cSim.stride2);  
#endif
            double xp1                          = cSim.pAtomX[shakeID.x];
            double yp1                          = cSim.pAtomY[shakeID.x];
            double zp1                          = cSim.pAtomZ[shakeID.x];
            double xp2                          = cSim.pAtomX[shakeID.y];
            double yp2                          = cSim.pAtomY[shakeID.y];
            double zp2                          = cSim.pAtomZ[shakeID.y];
            double xp3                          = cSim.pAtomX[shakeID.z];
            double yp3                          = cSim.pAtomY[shakeID.z];
            double zp3                          = cSim.pAtomZ[shakeID.z];
#if !defined(use_SPSP) && (!defined(NODPTEXTURE) || (__CUDA_ARCH__ >= 200))
            double x1                           = __hiloint2double(ix1.y, ix1.x);
            double y1                           = __hiloint2double(iy1.y, iy1.x);
            double z1                           = __hiloint2double(iz1.y, iz1.x);
            double x2                           = __hiloint2double(ix2.y, ix2.x);
            double y2                           = __hiloint2double(iy2.y, iy2.x);
            double z2                           = __hiloint2double(iz2.y, iz2.x);
            double x3                           = __hiloint2double(ix3.y, ix3.x);
            double y3                           = __hiloint2double(iy3.y, iy3.x);
            double z3                           = __hiloint2double(iz3.y, iz3.x);
#endif
    
            // Step1  A1_prime:
            double xb0                          = x2 - x1;
            double yb0                          = y2 - y1;
            double zb0                          = z2 - z1;
            double xc0                          = x3 - x1;
            double yc0                          = y3 - y1;
            double zc0                          = z3 - z1;

            psA->xpl                            = xp1 * cSim.wo_div_wohh + (xp2 + xp3) * cSim.wh_div_wohh;
            psA->ypl                            = yp1 * cSim.wo_div_wohh + (yp2 + yp3) * cSim.wh_div_wohh;
            psA->zpl                            = zp1 * cSim.wo_div_wohh + (zp2 + zp3) * cSim.wh_div_wohh;

            double xa1                          = xp1 - psA->xpl;
            double ya1                          = yp1 - psA->ypl;
            double za1                          = zp1 - psA->zpl;
            double xb1                          = xp2 - psA->xpl;
            double yb1                          = yp2 - psA->ypl;
            double zb1                          = zp2 - psA->zpl;
            double xc1                          = xp3 - psA->xpl;
            double yc1                          = yp3 - psA->ypl;
            double zc1                          = zp3 - psA->zpl;

            double xakszd                       = yb0 * zc0 - zb0 * yc0;
            double yakszd                       = zb0 * xc0 - xb0 * zc0;
            double zakszd                       = xb0 * yc0 - yb0 * xc0;
            double xaksxd                       = ya1 * zakszd - za1 * yakszd;
            double yaksxd                       = za1 * xakszd - xa1 * zakszd;
            double zaksxd                       = xa1 * yakszd - ya1 * xakszd;
            double xaksyd                       = yakszd * zaksxd - zakszd * yaksxd;
            double yaksyd                       = zakszd * xaksxd - xakszd * zaksxd;
            double zaksyd                       = xakszd * yaksxd - yakszd * xaksxd;

            double axlng_inv                    = rsqrt(xaksxd * xaksxd + yaksxd * yaksxd + zaksxd * zaksxd);
            double aylng_inv                    = rsqrt(xaksyd * xaksyd + yaksyd * yaksyd + zaksyd * zaksyd);
            double azlng_inv                    = rsqrt(xakszd * xakszd + yakszd * yakszd + zakszd * zakszd);

            double trns11                       = xaksxd * axlng_inv;
            double trns21                       = yaksxd * axlng_inv;
            double trns31                       = zaksxd * axlng_inv;
            double trns12                       = xaksyd * aylng_inv;
            double trns22                       = yaksyd * aylng_inv;
            double trns32                       = zaksyd * aylng_inv;
            double trns13                       = xakszd * azlng_inv;
            double trns23                       = yakszd * azlng_inv;
            double trns33                       = zakszd * azlng_inv;

            double xb0d                         = trns11 * xb0 + trns21 * yb0 + trns31 * zb0;
            double yb0d                         = trns12 * xb0 + trns22 * yb0 + trns32 * zb0;
            double xc0d                         = trns11 * xc0 + trns21 * yc0 + trns31 * zc0;
            double yc0d                         = trns12 * xc0 + trns22 * yc0 + trns32 * zc0;
            double za1d                         = trns13 * xa1 + trns23 * ya1 + trns33 * za1;
            double xb1d                         = trns11 * xb1 + trns21 * yb1 + trns31 * zb1;
            double yb1d                         = trns12 * xb1 + trns22 * yb1 + trns32 * zb1;
            double zb1d                         = trns13 * xb1 + trns23 * yb1 + trns33 * zb1;
            double xc1d                         = trns11 * xc1 + trns21 * yc1 + trns31 * zc1;
            double yc1d                         = trns12 * xc1 + trns22 * yc1 + trns32 * zc1;
            double zc1d                         = trns13 * xc1 + trns23 * yc1 + trns33 * zc1;

            // Step2  A2_prime:
            double sinphi                       = za1d * cSim.ra_inv;
            double cosphi                       = sqrt(1.0 - sinphi * sinphi);
            double sinpsi                       = (zb1d - zc1d) / (cSim.rc2 * cosphi);
            double cospsi                       = sqrt(1.0 - sinpsi * sinpsi);
 
            double ya2d                         =  cSim.ra * cosphi;
            double xb2d                         = -cSim.rc * cospsi;
            double yb2d                         = -cSim.rb * cosphi - cSim.rc * sinpsi * sinphi;
            double yc2d                         = -cSim.rb * cosphi + cSim.rc * sinpsi * sinphi;
            xb2d                                = -0.5 * sqrt(cSim.hhhh - (yb2d - yc2d) * (yb2d - yc2d) - (zb1d - zc1d) * (zb1d - zc1d));

            // Step3  al,be,ga:

            double alpa                         = (xb2d * (xb0d-xc0d) + yb0d * yb2d + yc0d * yc2d);
            double beta                         = (xb2d * (yc0d-yb0d) + xb0d * yb2d + xc0d * yc2d);
            double gama                         = xb0d * yb1d - xb1d * yb0d + xc0d * yc1d - xc1d * yc0d;

            double al2be2                       =  alpa * alpa + beta * beta;
            double sinthe                       = (alpa * gama - beta * sqrt(al2be2 - gama * gama)) / al2be2;

            // Step4  A3_prime:

            double costhe                       =  sqrt(1.0 - sinthe * sinthe);
            double xa3d                         = -ya2d * sinthe;
            double ya3d                         =  ya2d * costhe;
            double za3d                         =  za1d;
            double xb3d                         =  xb2d * costhe - yb2d * sinthe;
            double yb3d                         =  xb2d * sinthe + yb2d * costhe;
            double zb3d                         =  zb1d;
            double xc3d                         = -xb2d * costhe - yc2d * sinthe;
            double yc3d                         = -xb2d * sinthe + yc2d * costhe;
            double zc3d                         =  zc1d;

            // Step5  A3:
            cSim.pAtomX[shakeID.x]              = psA->xpl + trns11 * xa3d + trns12 * ya3d + trns13 * za3d;
            cSim.pAtomY[shakeID.x]              = psA->ypl + trns21 * xa3d + trns22 * ya3d + trns23 * za3d;
            cSim.pAtomZ[shakeID.x]              = psA->zpl + trns31 * xa3d + trns32 * ya3d + trns33 * za3d;
            cSim.pAtomX[shakeID.y]              = psA->xpl + trns11 * xb3d + trns12 * yb3d + trns13 * zb3d;
            cSim.pAtomY[shakeID.y]              = psA->ypl + trns21 * xb3d + trns22 * yb3d + trns23 * zb3d;
            cSim.pAtomZ[shakeID.y]              = psA->zpl + trns31 * xb3d + trns32 * yb3d + trns33 * zb3d;
            cSim.pAtomX[shakeID.z]              = psA->xpl + trns11 * xc3d + trns12 * yc3d + trns13 * zc3d;
            cSim.pAtomY[shakeID.z]              = psA->ypl + trns21 * xc3d + trns22 * yc3d + trns23 * zc3d;
            cSim.pAtomZ[shakeID.z]              = psA->zpl + trns31 * xc3d + trns32 * yc3d + trns33 * zc3d;

            pos                                += gridDim.x * blockDim.x;                                     
        }
    }

    if (cSim.slowShakeConstraints > 0)
    {    
        while (pos < cSim.fastShakeOffset)
        {
            pos                                += gridDim.x * blockDim.x;
        }
        pos                                    -= cSim.fastShakeOffset;

        while (pos < cSim.slowShakeConstraints)
        {
            int shakeID1;
		    int4 shakeID2;
            double toler;

            // Read SHAKE network data
            Atom* psA                           = &sA[threadIdx.x];
            shakeID1                            = cSim.pSlowShakeID1[pos];
            shakeID2                            = cSim.pSlowShakeID2[pos];
            double2 shakeParm                   = cSim.pSlowShakeParm[pos];
        
            // Read SHAKE network components
#ifdef use_SPSP        
            double xi                           = tex1Dfetch(texref, shakeID1);
            double yi                           = tex1Dfetch(texref, shakeID1 + cSim.stride);
            double zi                           = tex1Dfetch(texref, shakeID1 + cSim.stride2);
            double xij                          = tex1Dfetch(texref, shakeID2.x);
            double yij                          = tex1Dfetch(texref, shakeID2.x + cSim.stride);
            double zij                          = tex1Dfetch(texref, shakeID2.x + cSim.stride2);
            double xik                          = tex1Dfetch(texref, shakeID2.y);
            double yik                          = tex1Dfetch(texref, shakeID2.y + cSim.stride);
            double zik                          = tex1Dfetch(texref, shakeID2.y + cSim.stride2); 
            psA->xil                            = tex1Dfetch(texref, shakeID2.z);
            psA->yil                            = tex1Dfetch(texref, shakeID2.z + cSim.stride);
            psA->zil                            = tex1Dfetch(texref, shakeID2.z + cSim.stride2); 
            double xim                          = tex1Dfetch(texref, shakeID2.w);
            double yim                          = tex1Dfetch(texref, shakeID2.w + cSim.stride);
            double zim                          = tex1Dfetch(texref, shakeID2.w + cSim.stride2); 
#elif defined(NODPTEXTURE) && (__CUDA_ARCH__ < 200)
            double xi                           = cSim.pForceX[shakeID1];
            double yi                           = cSim.pForceY[shakeID1];
            double zi                           = cSim.pForceZ[shakeID1];
            double xij                          = cSim.pForceX[shakeID2.x];
            double yij                          = cSim.pForceY[shakeID2.x];
            double zij                          = cSim.pForceZ[shakeID2.x];
            double xik                          = cSim.pForceX[shakeID2.y];
            double yik                          = cSim.pForceY[shakeID2.y];
            double zik                          = cSim.pForceZ[shakeID2.y];  
            psA->xil                            = cSim.pForceX[shakeID2.z];
            psA->yil                            = cSim.pForceY[shakeID2.z];
            psA->zil                            = cSim.pForceZ[shakeID2.z]; 
            double xim                          = cSim.pForceX[shakeID2.w];
            double yim                          = cSim.pForceY[shakeID2.w];
            double zim                          = cSim.pForceZ[shakeID2.w];       
#else        
            int2 ixi                            = tex1Dfetch(texref, shakeID1);
            int2 iyi                            = tex1Dfetch(texref, shakeID1 + cSim.stride);
            int2 izi                            = tex1Dfetch(texref, shakeID1 + cSim.stride2);
            int2 ixij                           = tex1Dfetch(texref, shakeID2.x);
            int2 iyij                           = tex1Dfetch(texref, shakeID2.x + cSim.stride);
            int2 izij                           = tex1Dfetch(texref, shakeID2.x + cSim.stride2);
            int2 ixik                           = tex1Dfetch(texref, shakeID2.y);
            int2 iyik                           = tex1Dfetch(texref, shakeID2.y + cSim.stride);
            int2 izik                           = tex1Dfetch(texref, shakeID2.y + cSim.stride2);
            int2 ixil                           = tex1Dfetch(texref, shakeID2.z);
            int2 iyil                           = tex1Dfetch(texref, shakeID2.z + cSim.stride);
            int2 izil                           = tex1Dfetch(texref, shakeID2.z + cSim.stride2); 
            int2 ixim                           = tex1Dfetch(texref, shakeID2.w);
            int2 iyim                           = tex1Dfetch(texref, shakeID2.w + cSim.stride);
            int2 izim                           = tex1Dfetch(texref, shakeID2.w + cSim.stride2); 
#endif 
            double xpi                          = cSim.pAtomX[shakeID1];
            double ypi                          = cSim.pAtomY[shakeID1];
            double zpi                          = cSim.pAtomZ[shakeID1];
            double xpj                          = cSim.pAtomX[shakeID2.x];
            double ypj                          = cSim.pAtomY[shakeID2.x];
            double zpj                          = cSim.pAtomZ[shakeID2.x];
            double xpk                          = cSim.pAtomX[shakeID2.y];
            double ypk                          = cSim.pAtomY[shakeID2.y];
            double zpk                          = cSim.pAtomZ[shakeID2.y];
            psA->xpl                            = cSim.pAtomX[shakeID2.z];
            psA->ypl                            = cSim.pAtomY[shakeID2.z];
            psA->zpl                            = cSim.pAtomZ[shakeID2.z];
            double xpm                          = cSim.pAtomX[shakeID2.w];
            double ypm                          = cSim.pAtomY[shakeID2.w];
            double zpm                          = cSim.pAtomZ[shakeID2.w];
#if !defined(use_SPSP) && (!defined(NODPTEXTURE) || (__CUDA_ARCH__ >= 200))
            double xi                           = __hiloint2double(ixi.y, ixi.x);
            double yi                           = __hiloint2double(iyi.y, iyi.x);
            double zi                           = __hiloint2double(izi.y, izi.x);
            double xij                          = __hiloint2double(ixij.y, ixij.x);
            double yij                          = __hiloint2double(iyij.y, iyij.x);
            double zij                          = __hiloint2double(izij.y, izij.x);
            double xik                          = __hiloint2double(ixik.y, ixik.x);
            double yik                          = __hiloint2double(iyik.y, iyik.x);
            double zik                          = __hiloint2double(izik.y, izik.x);   
            psA->xil                            = __hiloint2double(ixil.y, ixil.x);
            psA->yil                            = __hiloint2double(iyil.y, iyil.x);
            psA->zil                            = __hiloint2double(izil.y, izil.x);   
            double xim                          = __hiloint2double(ixim.y, ixim.x);
            double yim                          = __hiloint2double(iyim.y, iyim.x);
            double zim                          = __hiloint2double(izim.y, izim.x);   
#endif                       
            psA->invMassI                       = shakeParm.x;
            toler                               = shakeParm.y;
            
            // Calculate unchanging quantities
            xij                                 = xi - xij;
            yij                                 = yi - yij;
            zij                                 = zi - zij;
            xik                                 = xi - xik;
            yik                                 = yi - yik;
            zik                                 = zi - zik; 
            psA->xil                            = xi - psA->xil;
            psA->yil                            = yi - psA->yil;
            psA->zil                            = zi - psA->zil;
            xim                                 = xi - xim;
            yim                                 = yi - yim;
            zim                                 = zi - zim;               
           
            bool done                           = false;
            for (int i = 0; i < 3000; i++)
            {
                done = true;
                
                // Calculate nominal distance squared
                double xpxx                     = xpi - xpj;
                double ypxx                     = ypi - ypj;
                double zpxx                     = zpi - zpj;
                double rpxx2                    = xpxx * xpxx + ypxx * ypxx + zpxx * zpxx;
          
                // Apply correction to first hydrogen
                double diff                     = toler - rpxx2;
                if (abs(diff) >= toler * cSim.tol)
                {
                    done                        = false;
                   
                    // Shake resetting of coordinate is done here
                    double rrpr                 = xij * xpxx + yij * ypxx + zij * zpxx;     
                    if (rrpr >= toler * 1.0e-06)
                    {
                    
                        double acor             = diff / (rrpr * (double)2.0 * (psA->invMassI + cSim.invMassH));
                        double h                = xij * acor;
                        xpi                    += h * psA->invMassI;
                        xpj                    -= h * cSim.invMassH;
                        h                       = yij * acor;
                        ypi                    += h * psA->invMassI;
                        ypj                    -= h * cSim.invMassH;
                        h                       = zij * acor;
                        zpi                    += h * psA->invMassI;
                        zpj                    -= h * cSim.invMassH;             
                    }
                }
          
     
                xpxx                            = xpi - xpk;
                ypxx                            = ypi - ypk;
                zpxx                            = zpi - zpk;
                rpxx2                           = xpxx * xpxx + ypxx * ypxx + zpxx * zpxx;
          
                // Apply correction to second hydrogen
                diff                            = toler - rpxx2;
                if (abs(diff) >= toler * cSim.tol)
                {
                    done                        = false;
               
                    // Shake resetting of coordinate is done here
                    double rrpr                 = xik * xpxx + yik * ypxx + zik * zpxx;     
                    if (rrpr >= toler * 1.0e-06)
                    {
                    
                        double acor             = diff / (rrpr * 2.0 * (psA->invMassI + cSim.invMassH));
                        double h                = xik * acor;
                        xpi                    += h * psA->invMassI;
                        xpk                    -= h * cSim.invMassH;
                        h                       = yik * acor;
                        ypi                    += h * psA->invMassI;
                        ypk                    -= h * cSim.invMassH;
                        h                       = zik * acor;
                        zpi                    += h * psA->invMassI;
                        zpk                    -= h * cSim.invMassH;             
                    }
                }
                
 
                xpxx                            = xpi - psA->xpl;
                ypxx                            = ypi - psA->ypl;
                zpxx                            = zpi - psA->zpl;        
                rpxx2                           = xpxx * xpxx + ypxx * ypxx + zpxx * zpxx;
          
                // Apply correction to third hydrogen
                diff                            = toler - rpxx2;
                if (abs(diff) >= toler * cSim.tol)
                {
                    done                        = false;
                  
                    // Shake resetting of coordinate is done here
                    double rrpr                 = psA->xil * xpxx + psA->yil * ypxx + psA->zil * zpxx;     
                    if (rrpr >= toler * 1.0e-06)
                    {             
                        double acor             = diff / (rrpr * (double)2.0 * (psA->invMassI + cSim.invMassH));
                        double h                = psA->xil * acor;
                        xpi                    += h * psA->invMassI;
                        psA->xpl               -= h * cSim.invMassH;
                        h                       = psA->yil * acor;
                        ypi                    += h * psA->invMassI;
                        psA->ypl               -= h * cSim.invMassH;
                        h                       = psA->zil * acor;
                        zpi                    += h * psA->invMassI;
                        psA->zpl               -= h * cSim.invMassH;             
                    }
                }

                xpxx                            = xpi - xpm;
                ypxx                            = ypi - ypm;
                zpxx                            = zpi - zpm;        
                rpxx2                           = xpxx * xpxx + ypxx * ypxx + zpxx * zpxx;

                // Apply correction to third hydrogen
                diff                            = toler - rpxx2;
                if (abs(diff) >= toler * cSim.tol)
                {
                    done                        = false;
                  
                    // Shake resetting of coordinate is done here
                    double rrpr                 = xim * xpxx + yim * ypxx + zim * zpxx;     
                    if (rrpr >= toler * 1.0e-06)
                    {             
                        double acor             = diff / (rrpr * (double)2.0 * (psA->invMassI + cSim.invMassH));
                        double h                = xim * acor;
                        xpi                    += h * psA->invMassI;
                        xpm                    -= h * cSim.invMassH;
                        h                       = yim * acor;
                        ypi                    += h * psA->invMassI;
                        ypm                    -= h * cSim.invMassH;
                        h                       = zim * acor;
                        zpi                    += h * psA->invMassI;
                        zpm                    -= h * cSim.invMassH;             
                    }
                }

                
                
                // Check for convergence
                if (done)
                    break;
            }
          
            // Write out results if converged, but there's no really good
            // way to indicate failure so we'll let the simulation heading
            // off to Neptune do that for us.  Wish there were a better way,
            // but until the CPU needs something from the GPU, those are the
            // the breaks.  I guess, technically, we could just set a flag to NOP
            // the simulation from here and then carry that result through upon
            // the next ntpr, ntwc, or ntwx update, but I leave that up to you 
            // guys to implement that (or not). 
            if (done)
            {
                cSim.pAtomX[shakeID1]           = xpi;
                cSim.pAtomY[shakeID1]           = ypi;
                cSim.pAtomZ[shakeID1]           = zpi;
                PMEFloat2 xyi                   = {xpi, ypi};
                cSim.pAtomXYSP[shakeID1]        = xyi;
                cSim.pAtomZSP[shakeID1]         = zpi;
                cSim.pAtomX[shakeID2.x]         = xpj;
                cSim.pAtomY[shakeID2.x]         = ypj;
                cSim.pAtomZ[shakeID2.x]         = zpj;
                PMEFloat2 xyj                   = {xpj, ypj};
                cSim.pAtomXYSP[shakeID2.x]      = xyj;
                cSim.pAtomZSP[shakeID2.x]       = zpj;
                cSim.pAtomX[shakeID2.y]         = xpk;
                cSim.pAtomY[shakeID2.y]         = ypk;
                cSim.pAtomZ[shakeID2.y]         = zpk;
                PMEFloat2 xyk                   = {xpk, ypk};
                cSim.pAtomXYSP[shakeID2.y]      = xyk;
                cSim.pAtomZSP[shakeID2.y]       = zpk;
                cSim.pAtomX[shakeID2.z]         = psA->xpl;
                cSim.pAtomY[shakeID2.z]         = psA->ypl;
                cSim.pAtomZ[shakeID2.z]         = psA->zpl;
                PMEFloat2 xyl                   = {psA->xpl, psA->ypl};
                cSim.pAtomXYSP[shakeID2.z]      = xyl;
                cSim.pAtomZSP[shakeID2.z]       = psA->zpl;
                cSim.pAtomX[shakeID2.w]         = xpm;
                cSim.pAtomY[shakeID2.w]         = ypm;
                cSim.pAtomZ[shakeID2.w]         = zpm;
                PMEFloat2 xym                   = {xpm, ypm};
                cSim.pAtomXYSP[shakeID2.w]      = xym;
                cSim.pAtomZSP[shakeID2.w]       = zpm;
            }        

    
            pos                                += gridDim.x * blockDim.x;                                 
        }
    }


}

#if (__CUDA_ARCH__ >= 200)
struct PMEAtom 
{
    double invMassI;
    double xpl;
    double ypl;
    double zpl;
    double xil;
    double yil;
    double zil;
    double toler;
    int4 shakeID;
    double dummy1;
    double dummy2;
};


struct PMEFastAtom
{
    double xcom;
    double ycom;
    double zcom;
    double trns11;
    double trns12;
    double trns13;
    double trns21;
    double trns22;
    double trns23;
    double trns31;
    double trns32;
    double trns33;
};
#else
struct PMEAtom 
{
    double invMassI;
    double xpl;
    double ypl;
    double zpl;
    double xil;
    double yil;
    double zil;
    double dummy1;
    double dummy2;
    double dummy3;
};


struct PMEFastAtom
{
    double xcom;
    double ycom;
    double zcom;
    double trns11;
    double trns12;
    double trns13;
    double trns21;
    double trns22;
    double trns23;
    double trns31;
};
#endif

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_SHAKE_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_SHAKE_THREADS_PER_BLOCK, 1)
#endif
kPMEShake_kernel()
{
#if (__CUDA_ARCH__ >= 200)
__shared__ PMEAtom sA[SM_2X_SHAKE_THREADS_PER_BLOCK];
#else
__shared__ PMEAtom sA[SM_13_SHAKE_THREADS_PER_BLOCK];
#endif

    unsigned int pos                            = blockIdx.x * blockDim.x + threadIdx.x;   
    while (pos < cSim.shakeConstraints)
    {
        PMEAtom* psA                            = &sA[threadIdx.x];
#if (__CUDA_ARCH__ >= 200)
#define TOLER psA->toler
#define SHAKEID psA->shakeID
#define SHAKEIDX psA->shakeID.x
#define SHAKEIDY psA->shakeID.y
#define SHAKEIDZ psA->shakeID.z
#define SHAKEIDW psA->shakeID.w
#else
		int4 shakeID;
        double toler;
#define TOLER toler
#define SHAKEID shakeID
#define SHAKEIDX shakeID.x
#define SHAKEIDY shakeID.y
#define SHAKEIDZ shakeID.z
#define SHAKEIDW shakeID.w
#endif
        // Read SHAKE network data
        SHAKEID                            		= cSim.pImageShakeID[pos];
        double2 shakeParm                       = cSim.pShakeParm[pos];
        
        // Read SHAKE network components
#ifdef use_SPSP        
        double xi                               = tex1Dfetch(texref, SHAKEIDX);
        double yi                               = tex1Dfetch(texref, SHAKEIDX + cSim.stride);
        double zi                               = tex1Dfetch(texref, SHAKEIDX + cSim.stride2);
        double xij                              = tex1Dfetch(texref, SHAKEIDY);
        double yij                              = tex1Dfetch(texref, SHAKEIDY + cSim.stride);
        double zij                              = tex1Dfetch(texref, SHAKEIDY + cSim.stride2);
#elif defined(NODPTEXTURE) && (__CUDA_ARCH__ < 200)
        double xi                               = cSim.pForceX[SHAKEIDX];
        double yi                               = cSim.pForceY[SHAKEIDX];
        double zi                               = cSim.pForceZ[SHAKEIDX];
        double xij                              = cSim.pForceX[SHAKEIDY];
        double yij                              = cSim.pForceY[SHAKEIDY];
        double zij                              = cSim.pForceZ[SHAKEIDY];
#else        
        int2 ixi                                = tex1Dfetch(texref, SHAKEIDX);
        int2 iyi                                = tex1Dfetch(texref, SHAKEIDX + cSim.stride);
        int2 izi                                = tex1Dfetch(texref, SHAKEIDX + cSim.stride2);
        int2 ixij                               = tex1Dfetch(texref, SHAKEIDY);
        int2 iyij                               = tex1Dfetch(texref, SHAKEIDY + cSim.stride);
        int2 izij                               = tex1Dfetch(texref, SHAKEIDY + cSim.stride2);
#endif 
        double xpi                              = cSim.pImageX[SHAKEIDX];
        double ypi                              = cSim.pImageY[SHAKEIDX];
        double zpi                              = cSim.pImageZ[SHAKEIDX];
        double xpj                              = cSim.pImageX[SHAKEIDY];
        double ypj                              = cSim.pImageY[SHAKEIDY];
        double zpj                              = cSim.pImageZ[SHAKEIDY];
#if !defined(use_SPSP) && (!defined(NODPTEXTURE) || (__CUDA_ARCH__ >= 200))
        double xi                               = __hiloint2double(ixi.y, ixi.x);
        double yi                               = __hiloint2double(iyi.y, iyi.x);
        double zi                               = __hiloint2double(izi.y, izi.x);
        double xij                              = __hiloint2double(ixij.y, ixij.x);
        double yij                              = __hiloint2double(iyij.y, iyij.x);
        double zij                              = __hiloint2double(izij.y, izij.x); 
#endif                       
        psA->invMassI                           = shakeParm.x;
        TOLER                                   = shakeParm.y;
        
        
        // Optionally read 2nd hydrogen
        double xpk, ypk, zpk, xik, yik, zik;
        if (SHAKEIDZ != -1)
        {
#ifdef use_SPSP
            xik                                 = tex1Dfetch(texref, SHAKEIDZ);
            yik                                 = tex1Dfetch(texref, SHAKEIDZ + cSim.stride);
            zik                                 = tex1Dfetch(texref, SHAKEIDZ + cSim.stride2); 
#elif defined(NODPTEXTURE) && (__CUDA_ARCH__ < 200)
            xik                                 = cSim.pForceX[SHAKEIDZ];
            yik                                 = cSim.pForceY[SHAKEIDZ];
            zik                                 = cSim.pForceZ[SHAKEIDZ];    
#else
            int2 ixik                           = tex1Dfetch(texref, SHAKEIDZ);
            int2 iyik                           = tex1Dfetch(texref, SHAKEIDZ + cSim.stride);
            int2 izik                           = tex1Dfetch(texref, SHAKEIDZ + cSim.stride2);
#endif 
            xpk                                 = cSim.pImageX[SHAKEIDZ];
            ypk                                 = cSim.pImageY[SHAKEIDZ];
            zpk                                 = cSim.pImageZ[SHAKEIDZ];
#if !defined(use_SPSP) && (!defined(NODPTEXTURE) || (__CUDA_ARCH__ >= 200))
            xik                                 = __hiloint2double(ixik.y, ixik.x);
            yik                                 = __hiloint2double(iyik.y, iyik.x);
            zik                                 = __hiloint2double(izik.y, izik.x);  
#endif                  
        }
        
        // Optionally read 3rd hydrogen into shared memory
        if (SHAKEIDW != -1)
        {
#ifdef use_SPSP
            psA->xil                            = tex1Dfetch(texref, SHAKEIDW);
            psA->yil                            = tex1Dfetch(texref, SHAKEIDW + cSim.stride);
            psA->zil                            = tex1Dfetch(texref, SHAKEIDW + cSim.stride2); 
#elif defined(NODPTEXTURE) && (__CUDA_ARCH__ < 200)
            psA->xil                            = cSim.pForceX[SHAKEIDW];
            psA->yil                            = cSim.pForceY[SHAKEIDW];
            psA->zil                            = cSim.pForceZ[SHAKEIDW];      
#else            
            int2 ixil                           = tex1Dfetch(texref, SHAKEIDW);
            int2 iyil                           = tex1Dfetch(texref, SHAKEIDW + cSim.stride);
            int2 izil                           = tex1Dfetch(texref, SHAKEIDW + cSim.stride2); 
#endif          
            psA->xpl                            = cSim.pImageX[SHAKEIDW];
            psA->ypl                            = cSim.pImageY[SHAKEIDW];
            psA->zpl                            = cSim.pImageZ[SHAKEIDW];
#if !defined(use_SPSP) && (!defined(NODPTEXTURE) || (__CUDA_ARCH__ >= 200))
            psA->xil                            = __hiloint2double(ixil.y, ixil.x);
            psA->yil                            = __hiloint2double(iyil.y, iyil.x);
            psA->zil                            = __hiloint2double(izil.y, izil.x);   
#endif          
        }
        
        // Calculate unchanging quantities
        xij                                     = xi - xij;
        yij                                     = yi - yij;
        zij                                     = zi - zij;
        
        if (SHAKEIDZ != -1)
        {
            xik                                 = xi - xik;
            yik                                 = yi - yik;
            zik                                 = zi - zik; 
        }        
         
        if (SHAKEIDW != -1)
        {
            psA->xil                            = xi - psA->xil;
            psA->yil                            = yi - psA->yil;
            psA->zil                            = zi - psA->zil;
        }      
       
        bool done                               = false;
        for (int i = 0; i < 3000; i++)
        {
            done = true;
            
            // Calculate nominal distance squared
            double xpxx                         = xpi - xpj;
            double ypxx                         = ypi - ypj;
            double zpxx                         = zpi - zpj;
            double rpxx2                        = xpxx * xpxx + ypxx * ypxx + zpxx * zpxx;
      
            // Apply correction
            double diff                         = TOLER - rpxx2;
            if (abs(diff) >= TOLER * cSim.tol)
            {
                done                            = false;
               
                // Shake resetting of coordinate is done here
                double rrpr                     = xij * xpxx + yij * ypxx + zij * zpxx;     
                if (rrpr >= TOLER * 1.0e-06)
                {
                
                    double acor                 = diff / (rrpr * (double)2.0 * (psA->invMassI + cSim.invMassH));
                    double h                    = xij * acor;
                    xpi                        += h * psA->invMassI;
                    xpj                        -= h * cSim.invMassH;
                    h                           = yij * acor;
                    ypi                        += h * psA->invMassI;
                    ypj                        -= h * cSim.invMassH;
                    h                           = zij * acor;
                    zpi                        += h * psA->invMassI;
                    zpj                        -= h * cSim.invMassH;             
                }
            }
      
            // Second bond if present
            if (SHAKEIDZ != -1)
            {
                xpxx                            = xpi - xpk;
                ypxx                            = ypi - ypk;
                zpxx                            = zpi - zpk;
                rpxx2                           = xpxx * xpxx + ypxx * ypxx + zpxx * zpxx;
      
                // Apply correction
                diff                            = TOLER - rpxx2;
                if (abs(diff) >= TOLER * cSim.tol)
                {
                    done                        = false;
               
                    // Shake resetting of coordinate is done here
                    double rrpr                 = xik * xpxx + yik * ypxx + zik * zpxx;     
                    if (rrpr >= TOLER * 1.0e-06)
                    {
                
                        double acor             = diff / (rrpr * 2.0 * (psA->invMassI + cSim.invMassH));
                        double h                = xik * acor;
                        xpi                    += h * psA->invMassI;
                        xpk                    -= h * cSim.invMassH;
                        h                       = yik * acor;
                        ypi                    += h * psA->invMassI;
                        ypk                    -= h * cSim.invMassH;
                        h                       = zik * acor;
                        zpi                    += h * psA->invMassI;
                        zpk                    -= h * cSim.invMassH;             
                    }
                }
            }
            
            // Third bond if present
            if (SHAKEIDW != -1)
            {
                xpxx                            = xpi - psA->xpl;
                ypxx                            = ypi - psA->ypl;
                zpxx                            = zpi - psA->zpl;        
                rpxx2                           = xpxx * xpxx + ypxx * ypxx + zpxx * zpxx;
      
                // Apply correction
                diff                            = TOLER - rpxx2;
                if (abs(diff) >= TOLER * cSim.tol)
                {
                    done                        = false;
               
                    // Shake resetting of coordinate is done here
                    double rrpr                 = psA->xil * xpxx + psA->yil * ypxx + psA->zil * zpxx;     
                    if (rrpr >= TOLER * 1.0e-06)
                    {
                
                        double acor             = diff / (rrpr * (double)2.0 * (psA->invMassI + cSim.invMassH));
                        double h                = psA->xil * acor;
                        xpi                    += h * psA->invMassI;
                        psA->xpl               -= h * cSim.invMassH;
                        h                       = psA->yil * acor;
                        ypi                    += h * psA->invMassI;
                        psA->ypl               -= h * cSim.invMassH;
                        h                       = psA->zil * acor;
                        zpi                    += h * psA->invMassI;
                        psA->zpl               -= h * cSim.invMassH;             
                    }
                }
            }
            
            
            // Check for convergence
            if (done)
                break;
        }
      
        // Write out results if converged, but there's no really good
        // way to indicate failure so we'll let the simulation heading
        // off to Neptune do that for us.  Wish there were a better way,
        // but until the CPU needs something from the GPU, those are the
        // the breaks.  I guess, technically, we could just set a flag to NOP
        // the simulation from here and then carry that result through upon
        // the next ntpr, ntwc, or ntwx update, but I leave that up to you 
        // guys to implement that (or not). 
        if (done)
        {
            cSim.pImageX[SHAKEIDX]              = xpi;
            cSim.pImageY[SHAKEIDX]              = ypi;
            cSim.pImageZ[SHAKEIDX]              = zpi;
            cSim.pImageX[SHAKEIDY]              = xpj;
            cSim.pImageY[SHAKEIDY]              = ypj;
            cSim.pImageZ[SHAKEIDY]              = zpj;

            if (SHAKEIDZ != -1)
            {
                cSim.pImageX[SHAKEIDZ]          = xpk;
                cSim.pImageY[SHAKEIDZ]          = ypk;
                cSim.pImageZ[SHAKEIDZ]          = zpk;
            }

            if (SHAKEIDW != -1)
            {
                cSim.pImageX[SHAKEIDW]          = psA->xpl;
                cSim.pImageY[SHAKEIDW]          = psA->ypl;
                cSim.pImageZ[SHAKEIDW]          = psA->zpl;
            }
        }        
        pos                                    += gridDim.x * blockDim.x;
    }

#undef TOLER
#undef SHAKEID
#undef SHAKEIDX
#undef SHAKEIDY
#undef SHAKEIDZ
#undef SHAKEIDW
#if (__CUDA_ARCH__ >= 200)
#define TRNS32 psA->trns32
#define TRNS33 psA->trns33
#else
#define TRNS32 trns32
#define TRNS33 trns33
#endif
    if (cSim.fastShakeConstraints > 0)
    {    
        while (pos < cSim.shakeOffset)
        {
            pos                                += gridDim.x * blockDim.x;
        }
        pos                                    -= cSim.shakeOffset;

        while (pos < cSim.fastShakeConstraints)
        {
            PMEFastAtom* psA                    = (PMEFastAtom*)&sA[threadIdx.x];
            
            // Read atom data
            int4 shakeID                        = cSim.pImageFastShakeID[pos];
#ifdef use_SPSP
            double x1                           = tex1Dfetch(texref, shakeID.x);
            double y1                           = tex1Dfetch(texref, shakeID.x + cSim.stride);
            double z1                           = tex1Dfetch(texref, shakeID.x + cSim.stride2);
            double x2                           = tex1Dfetch(texref, shakeID.y);
            double y2                           = tex1Dfetch(texref, shakeID.y + cSim.stride);
            double z2                           = tex1Dfetch(texref, shakeID.y + cSim.stride2);
            double x3                           = tex1Dfetch(texref, shakeID.z);
            double y3                           = tex1Dfetch(texref, shakeID.z + cSim.stride);
            double z3                           = tex1Dfetch(texref, shakeID.z + cSim.stride2);  
#elif defined(NODPTEXTURE) && (__CUDA_ARCH__ < 200)
            double x1                           = cSim.pForceX[shakeID.x];
            double y1                           = cSim.pForceY[shakeID.x];
            double z1                           = cSim.pForceZ[shakeID.x]; 
            double x2                           = cSim.pForceX[shakeID.y];
            double y2                           = cSim.pForceY[shakeID.y];
            double z2                           = cSim.pForceZ[shakeID.y]; 
            double x3                           = cSim.pForceX[shakeID.z];
            double y3                           = cSim.pForceY[shakeID.z];
            double z3                           = cSim.pForceZ[shakeID.z]; 
#else           
            int2 ix1                            = tex1Dfetch(texref, shakeID.x);
            int2 iy1                            = tex1Dfetch(texref, shakeID.x + cSim.stride);
            int2 iz1                            = tex1Dfetch(texref, shakeID.x + cSim.stride2);
            int2 ix2                            = tex1Dfetch(texref, shakeID.y);
            int2 iy2                            = tex1Dfetch(texref, shakeID.y + cSim.stride);
            int2 iz2                            = tex1Dfetch(texref, shakeID.y + cSim.stride2);
            int2 ix3                            = tex1Dfetch(texref, shakeID.z);
            int2 iy3                            = tex1Dfetch(texref, shakeID.z + cSim.stride);
            int2 iz3                            = tex1Dfetch(texref, shakeID.z + cSim.stride2);  
#endif
            double xp1                          = cSim.pImageX[shakeID.x];
            double yp1                          = cSim.pImageY[shakeID.x];
            double zp1                          = cSim.pImageZ[shakeID.x];
            double xp2                          = cSim.pImageX[shakeID.y];
            double yp2                          = cSim.pImageY[shakeID.y];
            double zp2                          = cSim.pImageZ[shakeID.y];
            double xp3                          = cSim.pImageX[shakeID.z];
            double yp3                          = cSim.pImageY[shakeID.z];
            double zp3                          = cSim.pImageZ[shakeID.z];
#if !defined(use_SPSP) && (!defined(NODPTEXTURE) || (__CUDA_ARCH__ >= 200))
            double x1                           = __hiloint2double(ix1.y, ix1.x);
            double y1                           = __hiloint2double(iy1.y, iy1.x);
            double z1                           = __hiloint2double(iz1.y, iz1.x);
            double x2                           = __hiloint2double(ix2.y, ix2.x);
            double y2                           = __hiloint2double(iy2.y, iy2.x);
            double z2                           = __hiloint2double(iz2.y, iz2.x);
            double x3                           = __hiloint2double(ix3.y, ix3.x);
            double y3                           = __hiloint2double(iy3.y, iy3.x);
            double z3                           = __hiloint2double(iz3.y, iz3.x);
#endif

            // Step1  A1_prime:
            double xb0                          = x2 - x1;
            double yb0                          = y2 - y1;
            double zb0                          = z2 - z1;
            double xc0                          = x3 - x1;
            double yc0                          = y3 - y1;
            double zc0                          = z3 - z1;
            psA->xcom                           = xp1 * cSim.wo_div_wohh + (xp2 + xp3) * cSim.wh_div_wohh;
            psA->ycom                           = yp1 * cSim.wo_div_wohh + (yp2 + yp3) * cSim.wh_div_wohh;
            psA->zcom                           = zp1 * cSim.wo_div_wohh + (zp2 + zp3) * cSim.wh_div_wohh;

            double xa1                          = xp1 - psA->xcom;
            double ya1                          = yp1 - psA->ycom;
            double za1                          = zp1 - psA->zcom;
            double xb1                          = xp2 - psA->xcom;
            double yb1                          = yp2 - psA->ycom;
            double zb1                          = zp2 - psA->zcom;
            double xc1                          = xp3 - psA->xcom;
            double yc1                          = yp3 - psA->ycom;
            double zc1                          = zp3 - psA->zcom;
            double xakszd                       = yb0 * zc0 - zb0 * yc0;
            double yakszd                       = zb0 * xc0 - xb0 * zc0;
            double zakszd                       = xb0 * yc0 - yb0 * xc0;
            double xaksxd                       = ya1 * zakszd - za1 * yakszd;
            double yaksxd                       = za1 * xakszd - xa1 * zakszd;
            double zaksxd                       = xa1 * yakszd - ya1 * xakszd;
            double xaksyd                       = yakszd * zaksxd - zakszd * yaksxd;
            double yaksyd                       = zakszd * xaksxd - xakszd * zaksxd;
            double zaksyd                       = xakszd * yaksxd - yakszd * xaksxd;

            double axlng_inv                    = rsqrt(xaksxd * xaksxd + yaksxd * yaksxd + zaksxd * zaksxd);
            double aylng_inv                    = rsqrt(xaksyd * xaksyd + yaksyd * yaksyd + zaksyd * zaksyd);
            double azlng_inv                    = rsqrt(xakszd * xakszd + yakszd * yakszd + zakszd * zakszd);

            psA->trns11                         = xaksxd * axlng_inv;
            psA->trns21                         = yaksxd * axlng_inv;
            psA->trns31                         = zaksxd * axlng_inv;
            psA->trns12                         = xaksyd * aylng_inv;
            psA->trns22                         = yaksyd * aylng_inv;
#if (__CUDA_ARCH__ < 200)
			double trns32;
#endif
            TRNS32                              = zaksyd * aylng_inv;
            psA->trns13                         = xakszd * azlng_inv;
            psA->trns23                         = yakszd * azlng_inv;
#if (__CUDA_ARCH__ < 200)
			double trns33;
#endif
            TRNS33                              = zakszd * azlng_inv;

            double xb0d                         = psA->trns11 * xb0 + psA->trns21 * yb0 + psA->trns31 * zb0;
            double yb0d                         = psA->trns12 * xb0 + psA->trns22 * yb0 + TRNS32      * zb0;
            double xc0d                         = psA->trns11 * xc0 + psA->trns21 * yc0 + psA->trns31 * zc0;
            double yc0d                         = psA->trns12 * xc0 + psA->trns22 * yc0 + TRNS32      * zc0;
            double za1d                         = psA->trns13 * xa1 + psA->trns23 * ya1 + TRNS33      * za1;
            double xb1d                         = psA->trns11 * xb1 + psA->trns21 * yb1 + psA->trns31 * zb1;
            double yb1d                         = psA->trns12 * xb1 + psA->trns22 * yb1 + TRNS32      * zb1;
            double zb1d                         = psA->trns13 * xb1 + psA->trns23 * yb1 + TRNS33      * zb1;
            double xc1d                         = psA->trns11 * xc1 + psA->trns21 * yc1 + psA->trns31 * zc1;
            double yc1d                         = psA->trns12 * xc1 + psA->trns22 * yc1 + TRNS32      * zc1;
            double zc1d                         = psA->trns13 * xc1 + psA->trns23 * yc1 + TRNS33      * zc1;

            // Step2  A2_prime:
            double sinphi                       = za1d * cSim.ra_inv;
            double cosphi                       = sqrt(1.0 - sinphi * sinphi);
            double sinpsi                       = (zb1d - zc1d) / (cSim.rc2 * cosphi);
            double cospsi                       = sqrt(1.0 - sinpsi * sinpsi);
 
            double ya2d                         =  cSim.ra * cosphi;
            double xb2d                         = -cSim.rc * cospsi;
            double yb2d                         = -cSim.rb * cosphi - cSim.rc * sinpsi * sinphi;
            double yc2d                         = -cSim.rb * cosphi + cSim.rc * sinpsi * sinphi;
            xb2d                                = -0.5 * sqrt(cSim.hhhh - (yb2d-yc2d) * (yb2d - yc2d) - (zb1d - zc1d) * (zb1d - zc1d));

            // Step3  al,be,ga:
            double alpa                         = (xb2d * (xb0d-xc0d) + yb0d * yb2d + yc0d * yc2d);
            double beta                         = (xb2d * (yc0d-yb0d) + xb0d * yb2d + xc0d * yc2d);
            double gama                         = xb0d * yb1d - xb1d * yb0d + xc0d * yc1d - xc1d * yc0d;

            double al2be2                       =  alpa * alpa + beta * beta;
            double sinthe                       = (alpa * gama - beta * sqrt(al2be2 - gama * gama)) / al2be2;

            // Step4  A3_prime:
            double costhe                       =  sqrt(1.0 - sinthe * sinthe);
            double xa3d                         = -ya2d * sinthe;
            double ya3d                         =  ya2d * costhe;
            double za3d                         =  za1d;
            double xb3d                         =  xb2d * costhe - yb2d * sinthe;
            double yb3d                         =  xb2d * sinthe + yb2d * costhe;
            double zb3d                         =  zb1d;
            double xc3d                         = -xb2d * costhe - yc2d * sinthe;
            double yc3d                         = -xb2d * sinthe + yc2d * costhe;
            double zc3d                         =  zc1d;

            // Step5  A3:
            cSim.pImageX[shakeID.x]             = psA->xcom + psA->trns11 * xa3d + psA->trns12 * ya3d + psA->trns13 * za3d;
            cSim.pImageY[shakeID.x]             = psA->ycom + psA->trns21 * xa3d + psA->trns22 * ya3d + psA->trns23 * za3d;
            cSim.pImageZ[shakeID.x]             = psA->zcom + psA->trns31 * xa3d + TRNS32      * ya3d + TRNS33      * za3d;
            cSim.pImageX[shakeID.y]             = psA->xcom + psA->trns11 * xb3d + psA->trns12 * yb3d + psA->trns13 * zb3d;
            cSim.pImageY[shakeID.y]             = psA->ycom + psA->trns21 * xb3d + psA->trns22 * yb3d + psA->trns23 * zb3d;
            cSim.pImageZ[shakeID.y]             = psA->zcom + psA->trns31 * xb3d + TRNS32      * yb3d + TRNS33      * zb3d;
            cSim.pImageX[shakeID.z]             = psA->xcom + psA->trns11 * xc3d + psA->trns12 * yc3d + psA->trns13 * zc3d;
            cSim.pImageY[shakeID.z]             = psA->ycom + psA->trns21 * xc3d + psA->trns22 * yc3d + psA->trns23 * zc3d;
            cSim.pImageZ[shakeID.z]             = psA->zcom + psA->trns31 * xc3d + TRNS32      * yc3d + TRNS33      * zc3d;
            pos                                += gridDim.x * blockDim.x;                                     
        }
    }
#undef TRNS32
#undef TRNS33

    if (cSim.slowShakeConstraints > 0)
    {    
        while (pos < cSim.fastShakeOffset)
        {
            pos                                += gridDim.x * blockDim.x;
        }
        pos                                    -= cSim.fastShakeOffset;

        while (pos < cSim.slowShakeConstraints)
        {

            int shakeID1;
#if (__CUDA_ARCH__ >= 200)
#define TOLER psA->toler
#define SHAKEID2 psA->shakeID
#define SHAKEID2X psA->shakeID.x
#define SHAKEID2Y psA->shakeID.y
#define SHAKEID2Z psA->shakeID.z
#define SHAKEID2W psA->shakeID.w
#else
		    int4 shakeID2;
            double toler;
#define TOLER toler
#define SHAKEID2 shakeID2
#define SHAKEID2X shakeID2.x
#define SHAKEID2Y shakeID2.y
#define SHAKEID2Z shakeID2.z
#define SHAKEID2W shakeID2.w
#endif
            // Read SHAKE network data
            PMEAtom* psA                        = &sA[threadIdx.x];
            shakeID1                            = cSim.pImageSlowShakeID1[pos];
            SHAKEID2                            = cSim.pImageSlowShakeID2[pos];
            double2 shakeParm                   = cSim.pSlowShakeParm[pos];
        
            // Read SHAKE network components
#ifdef use_SPSP        
            double xi                           = tex1Dfetch(texref, shakeID1);
            double yi                           = tex1Dfetch(texref, shakeID1 + cSim.stride);
            double zi                           = tex1Dfetch(texref, shakeID1 + cSim.stride2);
            double xij                          = tex1Dfetch(texref, SHAKEID2X);
            double yij                          = tex1Dfetch(texref, SHAKEID2X + cSim.stride);
            double zij                          = tex1Dfetch(texref, SHAKEID2X + cSim.stride2);
            double xik                          = tex1Dfetch(texref, SHAKEID2Y);
            double yik                          = tex1Dfetch(texref, SHAKEID2Y + cSim.stride);
            double zik                          = tex1Dfetch(texref, SHAKEID2Y + cSim.stride2); 
            psA->xil                            = tex1Dfetch(texref, SHAKEID2Z);
            psA->yil                            = tex1Dfetch(texref, SHAKEID2Z + cSim.stride);
            psA->zil                            = tex1Dfetch(texref, SHAKEID2Z + cSim.stride2); 
            double xim                          = tex1Dfetch(texref, SHAKEID2W);
            double yim                          = tex1Dfetch(texref, SHAKEID2W + cSim.stride);
            double zim                          = tex1Dfetch(texref, SHAKEID2W + cSim.stride2); 
#elif defined(NODPTEXTURE) && (__CUDA_ARCH__ < 200)
            double xi                           = cSim.pForceX[shakeID1];
            double yi                           = cSim.pForceY[shakeID1];
            double zi                           = cSim.pForceZ[shakeID1];
            double xij                          = cSim.pForceX[SHAKEID2X];
            double yij                          = cSim.pForceY[SHAKEID2X];
            double zij                          = cSim.pForceZ[SHAKEID2X];
            double xik                          = cSim.pForceX[SHAKEID2Y];
            double yik                          = cSim.pForceY[SHAKEID2Y];
            double zik                          = cSim.pForceZ[SHAKEID2Y];  
            psA->xil                            = cSim.pForceX[SHAKEID2Z];
            psA->yil                            = cSim.pForceY[SHAKEID2Z];
            psA->zil                            = cSim.pForceZ[SHAKEID2Z]; 
            double xim                          = cSim.pForceX[SHAKEID2W];
            double yim                          = cSim.pForceY[SHAKEID2W];
            double zim                          = cSim.pForceZ[SHAKEID2W];       
#else        
            int2 ixi                            = tex1Dfetch(texref, shakeID1);
            int2 iyi                            = tex1Dfetch(texref, shakeID1 + cSim.stride);
            int2 izi                            = tex1Dfetch(texref, shakeID1 + cSim.stride2);
            int2 ixij                           = tex1Dfetch(texref, SHAKEID2X);
            int2 iyij                           = tex1Dfetch(texref, SHAKEID2X + cSim.stride);
            int2 izij                           = tex1Dfetch(texref, SHAKEID2X + cSim.stride2);
            int2 ixik                           = tex1Dfetch(texref, SHAKEID2Y);
            int2 iyik                           = tex1Dfetch(texref, SHAKEID2Y + cSim.stride);
            int2 izik                           = tex1Dfetch(texref, SHAKEID2Y + cSim.stride2);
            int2 ixil                           = tex1Dfetch(texref, SHAKEID2Z);
            int2 iyil                           = tex1Dfetch(texref, SHAKEID2Z + cSim.stride);
            int2 izil                           = tex1Dfetch(texref, SHAKEID2Z + cSim.stride2); 
            int2 ixim                           = tex1Dfetch(texref, SHAKEID2W);
            int2 iyim                           = tex1Dfetch(texref, SHAKEID2W + cSim.stride);
            int2 izim                           = tex1Dfetch(texref, SHAKEID2W + cSim.stride2); 
#endif 
            double xpi                          = cSim.pImageX[shakeID1];
            double ypi                          = cSim.pImageY[shakeID1];
            double zpi                          = cSim.pImageZ[shakeID1];
            double xpj                          = cSim.pImageX[SHAKEID2X];
            double ypj                          = cSim.pImageY[SHAKEID2X];
            double zpj                          = cSim.pImageZ[SHAKEID2X];
            double xpk                          = cSim.pImageX[SHAKEID2Y];
            double ypk                          = cSim.pImageY[SHAKEID2Y];
            double zpk                          = cSim.pImageZ[SHAKEID2Y];
            psA->xpl                            = cSim.pImageX[SHAKEID2Z];
            psA->ypl                            = cSim.pImageY[SHAKEID2Z];
            psA->zpl                            = cSim.pImageZ[SHAKEID2Z];
            double xpm                          = cSim.pImageX[SHAKEID2W];
            double ypm                          = cSim.pImageY[SHAKEID2W];
            double zpm                          = cSim.pImageZ[SHAKEID2W];
#if !defined(use_SPSP) && (!defined(NODPTEXTURE) || (__CUDA_ARCH__ >= 200))
            double xi                           = __hiloint2double(ixi.y, ixi.x);
            double yi                           = __hiloint2double(iyi.y, iyi.x);
            double zi                           = __hiloint2double(izi.y, izi.x);
            double xij                          = __hiloint2double(ixij.y, ixij.x);
            double yij                          = __hiloint2double(iyij.y, iyij.x);
            double zij                          = __hiloint2double(izij.y, izij.x);
            double xik                          = __hiloint2double(ixik.y, ixik.x);
            double yik                          = __hiloint2double(iyik.y, iyik.x);
            double zik                          = __hiloint2double(izik.y, izik.x);   
            psA->xil                            = __hiloint2double(ixil.y, ixil.x);
            psA->yil                            = __hiloint2double(iyil.y, iyil.x);
            psA->zil                            = __hiloint2double(izil.y, izil.x);   
            double xim                          = __hiloint2double(ixim.y, ixim.x);
            double yim                          = __hiloint2double(iyim.y, iyim.x);
            double zim                          = __hiloint2double(izim.y, izim.x);   
#endif                       
            psA->invMassI                       = shakeParm.x;
            TOLER                               = shakeParm.y;
            
            // Calculate unchanging quantities
            xij                                 = xi - xij;
            yij                                 = yi - yij;
            zij                                 = zi - zij;
            xik                                 = xi - xik;
            yik                                 = yi - yik;
            zik                                 = zi - zik; 
            psA->xil                            = xi - psA->xil;
            psA->yil                            = yi - psA->yil;
            psA->zil                            = zi - psA->zil;
            xim                                 = xi - xim;
            yim                                 = yi - yim;
            zim                                 = zi - zim;               
           
            bool done                           = false;
            for (int i = 0; i < 3000; i++)
            {
                done = true;
                
                // Calculate nominal distance squared
                double xpxx                     = xpi - xpj;
                double ypxx                     = ypi - ypj;
                double zpxx                     = zpi - zpj;
                double rpxx2                    = xpxx * xpxx + ypxx * ypxx + zpxx * zpxx;
          
                // Apply correction to first hydrogen
                double diff                     = TOLER - rpxx2;
                if (abs(diff) >= TOLER * cSim.tol)
                {
                    done                        = false;
                   
                    // Shake resetting of coordinate is done here
                    double rrpr                 = xij * xpxx + yij * ypxx + zij * zpxx;     
                    if (rrpr >= TOLER * 1.0e-06)
                    {
                    
                        double acor             = diff / (rrpr * (double)2.0 * (psA->invMassI + cSim.invMassH));
                        double h                = xij * acor;
                        xpi                    += h * psA->invMassI;
                        xpj                    -= h * cSim.invMassH;
                        h                       = yij * acor;
                        ypi                    += h * psA->invMassI;
                        ypj                    -= h * cSim.invMassH;
                        h                       = zij * acor;
                        zpi                    += h * psA->invMassI;
                        zpj                    -= h * cSim.invMassH;             
                    }
                }
          
     
                xpxx                            = xpi - xpk;
                ypxx                            = ypi - ypk;
                zpxx                            = zpi - zpk;
                rpxx2                           = xpxx * xpxx + ypxx * ypxx + zpxx * zpxx;
          
                // Apply correction to second hydrogen
                diff                            = TOLER - rpxx2;
                if (abs(diff) >= TOLER * cSim.tol)
                {
                    done                        = false;
               
                    // Shake resetting of coordinate is done here
                    double rrpr                 = xik * xpxx + yik * ypxx + zik * zpxx;     
                    if (rrpr >= TOLER * 1.0e-06)
                    {
                    
                        double acor             = diff / (rrpr * 2.0 * (psA->invMassI + cSim.invMassH));
                        double h                = xik * acor;
                        xpi                    += h * psA->invMassI;
                        xpk                    -= h * cSim.invMassH;
                        h                       = yik * acor;
                        ypi                    += h * psA->invMassI;
                        ypk                    -= h * cSim.invMassH;
                        h                       = zik * acor;
                        zpi                    += h * psA->invMassI;
                        zpk                    -= h * cSim.invMassH;             
                    }
                }
                
 
                xpxx                            = xpi - psA->xpl;
                ypxx                            = ypi - psA->ypl;
                zpxx                            = zpi - psA->zpl;        
                rpxx2                           = xpxx * xpxx + ypxx * ypxx + zpxx * zpxx;
          
                // Apply correction to third hydrogen
                diff                            = TOLER - rpxx2;
                if (abs(diff) >= TOLER * cSim.tol)
                {
                    done                        = false;
                  
                    // Shake resetting of coordinate is done here
                    double rrpr                 = psA->xil * xpxx + psA->yil * ypxx + psA->zil * zpxx;     
                    if (rrpr >= TOLER * 1.0e-06)
                    {             
                        double acor             = diff / (rrpr * (double)2.0 * (psA->invMassI + cSim.invMassH));
                        double h                = psA->xil * acor;
                        xpi                    += h * psA->invMassI;
                        psA->xpl               -= h * cSim.invMassH;
                        h                       = psA->yil * acor;
                        ypi                    += h * psA->invMassI;
                        psA->ypl               -= h * cSim.invMassH;
                        h                       = psA->zil * acor;
                        zpi                    += h * psA->invMassI;
                        psA->zpl               -= h * cSim.invMassH;             
                    }
                }

                xpxx                            = xpi - xpm;
                ypxx                            = ypi - ypm;
                zpxx                            = zpi - zpm;        
                rpxx2                           = xpxx * xpxx + ypxx * ypxx + zpxx * zpxx;

                // Apply correction to third hydrogen
                diff                            = TOLER - rpxx2;
                if (abs(diff) >= TOLER * cSim.tol)
                {
                    done                        = false;
                  
                    // Shake resetting of coordinate is done here
                    double rrpr                 = xim * xpxx + yim * ypxx + zim * zpxx;     
                    if (rrpr >= TOLER * 1.0e-06)
                    {             
                        double acor             = diff / (rrpr * (double)2.0 * (psA->invMassI + cSim.invMassH));
                        double h                = xim * acor;
                        xpi                    += h * psA->invMassI;
                        xpm                    -= h * cSim.invMassH;
                        h                       = yim * acor;
                        ypi                    += h * psA->invMassI;
                        ypm                    -= h * cSim.invMassH;
                        h                       = zim * acor;
                        zpi                    += h * psA->invMassI;
                        zpm                    -= h * cSim.invMassH;             
                    }
                }

                
                
                // Check for convergence
                if (done)
                    break;
            }
          
            // Write out results if converged, but there's no really good
            // way to indicate failure so we'll let the simulation heading
            // off to Neptune do that for us.  Wish there were a better way,
            // but until the CPU needs something from the GPU, those are the
            // the breaks.  I guess, technically, we could just set a flag to NOP
            // the simulation from here and then carry that result through upon
            // the next ntpr, ntwc, or ntwx update, but I leave that up to you 
            // guys to implement that (or not). 
            if (done)
            {
                cSim.pImageX[shakeID1]          = xpi;
                cSim.pImageY[shakeID1]          = ypi;
                cSim.pImageZ[shakeID1]          = zpi;
                cSim.pImageX[SHAKEID2X]         = xpj;
                cSim.pImageY[SHAKEID2X]         = ypj;
                cSim.pImageZ[SHAKEID2X]         = zpj;
                cSim.pImageX[SHAKEID2Y]         = xpk;
                cSim.pImageY[SHAKEID2Y]         = ypk;
                cSim.pImageZ[SHAKEID2Y]         = zpk;
                cSim.pImageX[SHAKEID2Z]         = psA->xpl;
                cSim.pImageY[SHAKEID2Z]         = psA->ypl;
                cSim.pImageZ[SHAKEID2Z]         = psA->zpl;
                cSim.pImageX[SHAKEID2W]         = xpm;
                cSim.pImageY[SHAKEID2W]         = ypm;
                cSim.pImageZ[SHAKEID2W]         = zpm;
            }        

    
            pos                                += gridDim.x * blockDim.x;  
#undef TOLER
#undef SHAKEID2
#undef SHAKEID2X
#undef SHAKEID2Y
#undef SHAKEID2Z
#undef SHAKEID2W                                  
        }
    }

}

void kShake(gpuContext gpu)
{
    texref.normalized = 0;
    texref.filterMode = hipFilterModePoint;
    texref.addressMode[0] = hipAddressModeClamp;
    texref.channelDesc.x = 32;
#ifndef use_SPSP    
    texref.channelDesc.y = 32;
#else    
    texref.channelDesc.y = 0;
#endif
    texref.channelDesc.z = 0;
    texref.channelDesc.w = 0;
#ifndef use_SPSP
    hipBindTexture(NULL, texref, (int2*)(gpu->sim.pForce), gpu->sim.stride3 * sizeof(int2));
#else
    hipBindTexture(NULL, texref, gpu->sim.pForce, gpu->sim.stride3 * sizeof(float));
#endif
    if (gpu->bNeighborList)
    {
        kPMEShake_kernel<<<gpu->blocks, gpu->shakeThreadsPerBlock>>>();  
    }
    else
    {    
        kShake_kernel<<<gpu->blocks, gpu->shakeThreadsPerBlock>>>();  
    }
    LAUNCHERROR("kShake");
    hipUnbindTexture(texref);
}
