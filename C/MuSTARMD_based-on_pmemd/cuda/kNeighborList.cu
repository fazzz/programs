#include "hip/hip_runtime.h"
/***************************************************/
/*                                                 */
/*      AMBER NVIDIA CUDA CPU IMPLEMENTATION       */
/*                 PMEMD VERSION                   */
/*                     2010                        */
/*                      by                         */
/*             Scott Le Grand (NVIDIA)             */
/*               Duncan Poole (NVIDIA)             */
/*                Ross Walker (SDSC)               */
/*                                                 */
/***************************************************/

#include <hip/hip_runtime.h>
#include "gpu.h"
static __constant__ cudaSimulation cSim;

struct Atom {
    PMEFloat xmin;
    PMEFloat xmax;
    PMEFloat ymin;
    PMEFloat ymax;
    PMEFloat zmin;
    PMEFloat zmax;
};

void SetkNeighborListSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

void GetkNeighborListSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

struct stuff
{
    double fx;
    double fy;
    double fz;
    unsigned int ix;
    unsigned int iy;
    unsigned int iz;
    unsigned int ox;
    unsigned int oy;
    unsigned int oz;
    unsigned int cellHash;
    unsigned int hash;
};

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_THREADS_PER_BLOCK, 1)
#endif
kNLGenerateSpatialHash_kernel()
{
    __shared__ unsigned int sCellHash[CELLHASHCELLS];
    __shared__ PMEFloat sRecipf[9];
    unsigned int pos                            = blockIdx.x * blockDim.x + threadIdx.x; 
    unsigned int increment                      = gridDim.x * blockDim.x;
    
    // Clear atom list/exclusion mask space counter
    if (pos == 0)
    {
        *(cSim.pNLTotalOffset)                  = 0;
        *(cSim.pNLPosition)                     = cSim.NLBuildWarps;
    }
    
    // Read cell hash
    if (threadIdx.x < CELLHASHCELLS)
    {
        sCellHash[threadIdx.x]                  = cSim.pNLCellHash[threadIdx.x];
    }

    if (cSim.ntp > 0)
    {
        if (threadIdx.x < 9)
        {
            sRecipf[threadIdx.x]                = cSim.pNTPData->recipf[threadIdx.x];
        }    
         __syncthreads();
    
        while (pos < cSim.atoms)
        {
            PMEFloat x                          = cSim.pImageX[pos];
            PMEFloat y                          = cSim.pImageY[pos];
            PMEFloat z                          = cSim.pImageZ[pos];
            
            // Orthogonal/nonorthogonal handled in the same code (3 single precision multiplies and adds? Who cares and why?)
            PMEFloat fx                         = sRecipf[0] * x + sRecipf[3] * y + sRecipf[6] * z;
            PMEFloat fy                         =                  sRecipf[4] * y + sRecipf[7] * z;
            PMEFloat fz                         =                                   sRecipf[8] * z;

            // Account for minimum image convention  
            fx                                  = fx - round(fx) + (PMEFloat)0.5;
            fy                                  = fy - round(fy) + (PMEFloat)0.5;
            fz                                  = fz - round(fz) + (PMEFloat)0.5;
            fx                                  = (fx < (PMEFloat)1.0 ? fx : (PMEFloat)0.0);
            fy                                  = (fy < (PMEFloat)1.0 ? fy : (PMEFloat)0.0);
            fz                                  = (fz < (PMEFloat)1.0 ? fz : (PMEFloat)0.0);
                
            // Generate box coordinates
            cSim.pImageIndex[pos]               = pos;
            unsigned int ix                     = fx * cSim.xcells;
            unsigned int iy                     = fy * cSim.ycells;
            unsigned int iz                     = fz * cSim.zcells;
            cSim.pImageCellID[pos]              = ix + (iy << CELLIDYSHIFT) + (iz << CELLIDZSHIFT);
            unsigned int ox                     = min(CELLHASHX - 1, (unsigned int)((PMEFloat)CELLHASHX * ((fx - ix * cSim.oneOverXcellsf) * cSim.xcells))); 
            unsigned int oy                     = min(CELLHASHY - 1, (unsigned int)((PMEFloat)CELLHASHY * ((fy - iy * cSim.oneOverYcellsf) * cSim.ycells))) * CELLHASHX; 
            unsigned int oz                     = min(CELLHASHZ - 1, (unsigned int)((PMEFloat)CELLHASHZ * ((fz - iz * cSim.oneOverZcellsf) * cSim.zcells))) * CELLHASHXY; 
            unsigned int cellHash               = sCellHash[ox + oy + oz];
            unsigned int hash                   = (((iz * cSim.ycells + iy) * cSim.xcells + ix) << CELLHASHBITS) | cellHash;
            cSim.pImageHash[pos]                = hash;
            pos                                += increment;
        }   
    }
    else
    {
        __syncthreads();
        while (pos < cSim.atoms)
        {
            PMEFloat x                          = cSim.pImageX[pos];
            PMEFloat y                          = cSim.pImageY[pos];
            PMEFloat z                          = cSim.pImageZ[pos];
            
            // Orthogonal/nonorthogonal handled in the same code (3 single precision multiplies and adds? Who cares and why?)
            PMEFloat fx                         = cSim.recipf[0][0] * x + cSim.recipf[1][0] * y + cSim.recipf[2][0] * z;
            PMEFloat fy                         =                         cSim.recipf[1][1] * y + cSim.recipf[2][1] * z;
            PMEFloat fz                         =                                                 cSim.recipf[2][2] * z;

            // Account for minimum image convention  
            fx                                  = fx - round(fx) + (PMEFloat)0.5;
            fy                                  = fy - round(fy) + (PMEFloat)0.5;
            fz                                  = fz - round(fz) + (PMEFloat)0.5;
            fx                                  = (fx < (PMEFloat)1.0 ? fx : (PMEFloat)0.0);
            fy                                  = (fy < (PMEFloat)1.0 ? fy : (PMEFloat)0.0);
            fz                                  = (fz < (PMEFloat)1.0 ? fz : (PMEFloat)0.0);
                
            // Generate box coordinates
            cSim.pImageIndex[pos]               = pos;
            unsigned int ix                     = fx * cSim.xcells;
            unsigned int iy                     = fy * cSim.ycells;
            unsigned int iz                     = fz * cSim.zcells;
            cSim.pImageCellID[pos]              = ix + (iy << CELLIDYSHIFT) + (iz << CELLIDZSHIFT);
            unsigned int ox                     = min(CELLHASHX - 1, (unsigned int)((PMEFloat)CELLHASHX * ((fx - ix * cSim.oneOverXcellsf) * cSim.xcells))); 
            unsigned int oy                     = min(CELLHASHY - 1, (unsigned int)((PMEFloat)CELLHASHY * ((fy - iy * cSim.oneOverYcellsf) * cSim.ycells))) * CELLHASHX; 
            unsigned int oz                     = min(CELLHASHZ - 1, (unsigned int)((PMEFloat)CELLHASHZ * ((fz - iz * cSim.oneOverZcellsf) * cSim.zcells))) * CELLHASHXY; 
            unsigned int cellHash               = sCellHash[ox + oy + oz];
            unsigned int hash                   = (((iz * cSim.ycells + iy) * cSim.xcells + ix) << CELLHASHBITS) | cellHash;
            cSim.pImageHash[pos]                = hash;
            pos                                += increment;
        }   
    }
    
    
    
}

extern "C" void kNLGenerateSpatialHash(gpuContext gpu)
{
    kNLGenerateSpatialHash_kernel<<<gpu->blocks, gpu->threadsPerBlock>>>();   
    LAUNCHERROR("kNLGenerateSpatialHash");
    
#if 0
    gpu->pbFraction->Download();
    gpu->pbImageIndex->Download();
    for (int i = 0; i < gpu->sim.atoms; i++)
    {
        PMEFloat fx = gpu->pbFraction->_pSysData[i];
        PMEFloat fy = gpu->pbFraction->_pSysData[i + gpu->sim.stride];
        PMEFloat fz = gpu->pbFraction->_pSysData[i + gpu->sim.stride2];
        unsigned int ix                         = fx * gpu->sim.xcells;
        unsigned int iy                         = fy * gpu->sim.ycells;
        unsigned int iz                         = fz * gpu->sim.zcells;
        unsigned int ox                         = (unsigned int)((PMEFloat)CELLHASHX * ((fx - ix * gpu->sim.oneOverXcells) * gpu->sim.xcells)); 
        unsigned int oy                         = (unsigned int)((PMEFloat)CELLHASHY * ((fy - iy * gpu->sim.oneOverYcells) * gpu->sim.ycells)) * CELLHASHX; 
        unsigned int oz                         = (unsigned int)((PMEFloat)CELLHASHZ * ((fz - iz * gpu->sim.oneOverZcells) * gpu->sim.zcells)) * CELLHASHXY; 
    
    
        printf("%6d 0x%08x %10.7f %10.7f %10.7f |  %6d %6d %6d | %6d %6d %6d\n", i, gpu->pbImageIndex->_pSysData[i + gpu->sim.stride3], fx, fy, fz, ix, iy, iz, ox, oy, oz); 
    }
    exit(-1);
#endif
    
    
}



__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_THREADS_PER_BLOCK, 1)
#endif
kNLRemapImage_kernel(unsigned int* pImageIndex)
{
    unsigned int pos                            = blockIdx.x * blockDim.x + threadIdx.x; 
    unsigned int increment                      = gridDim.x * blockDim.x;
    unsigned int index                          = 0;
    unsigned int newindex;
    
    if (pos < cSim.atoms)
        index                                   = pImageIndex[pos];

    while (pos < cSim.atoms)
    {
        unsigned int newpos                     = pos + increment;
        if (newpos < cSim.atoms)
            newindex                            = pImageIndex[newpos];
      
        // Read new data
        unsigned int atom                       = cSim.pImageAtom[index];
        PMEDouble x                             = cSim.pImageX[index];
        PMEDouble y                             = cSim.pImageY[index];
        PMEDouble z                             = cSim.pImageZ[index];
        PMEDouble vx                            = cSim.pImageVelX[index];
        PMEDouble vy                            = cSim.pImageVelY[index];
        PMEDouble vz                            = cSim.pImageVelZ[index];
        PMEDouble lvx                           = cSim.pImageLVelX[index];
        PMEDouble lvy                           = cSim.pImageLVelY[index];
        PMEDouble lvz                           = cSim.pImageLVelZ[index];
        PMEDouble q                             = cSim.pImageCharge[index];
        PMEDouble m                             = cSim.pImageMass[index];
        PMEFloat2 sigeps                        = cSim.pImageSigEps[index];
        unsigned int outputBuffers              = cSim.pImageOutputBuffers[index];
        unsigned int cellID                     = cSim.pImageCellID[index];
        cSim.pImageX2[pos]                      = x;
        cSim.pImageY2[pos]                      = y;
        cSim.pImageZ2[pos]                      = z;
        cSim.pImageAtom2[pos]                   = atom;
        cSim.pImageAtomLookup[atom]             = pos;
        PMEFloat2 xy                            = {x, y};
        cSim.pAtomXYSaveSP[pos]                 = xy;
        cSim.pAtomZSaveSP[pos]                  = z;
        cSim.pImageVelX2[pos]                   = vx;
        cSim.pImageVelY2[pos]                   = vy;
        cSim.pImageVelZ2[pos]                   = vz;
        cSim.pImageLVelX2[pos]                  = lvx;
        cSim.pImageLVelY2[pos]                  = lvy;
        cSim.pImageLVelZ2[pos]                  = lvz;
        cSim.pImageCharge2[pos]                 = q;
        cSim.pAtomChargeSP[pos]                 = q;
        cSim.pImageMass2[pos]                   = m;
        cSim.pImageInvMass2[pos]                = (m != (PMEDouble)0.0 ? (PMEDouble)1.0 / m : (PMEDouble)0.0);
        cSim.pImageSigEps2[pos]                 = sigeps;
        cSim.pImageOutputBuffers2[pos]          = outputBuffers;
        cSim.pImageCellID2[pos]                 = cellID;
        
        // Advance to next atom
        index                                   = newindex;    
        pos                                     = newpos;    
    }
}


extern "C" void kNLRemapImage(gpuContext gpu)
{
    kNLRemapImage_kernel<<<gpu->blocks, gpu->threadsPerBlock>>>(gpu->sim.pImageIndex);
    LAUNCHERROR("kNLRemapImage");

    unsigned int *pUint;
    PMEDouble* pPMEDouble;
    PMEFloat2* pPMEFloat2;
    
    // Remap constant memory pointers
    pUint                           = gpu->sim.pImageAtom;
    gpu->sim.pImageAtom             = gpu->sim.pImageAtom2;
    gpu->sim.pImageAtom2            = pUint;
    pPMEDouble                      = gpu->sim.pImageX;
    gpu->sim.pImageX                = gpu->sim.pImageX2;
    gpu->sim.pImageX2               = pPMEDouble;
    pPMEDouble                      = gpu->sim.pImageY;
    gpu->sim.pImageY                = gpu->sim.pImageY2;
    gpu->sim.pImageY2               = pPMEDouble;
    pPMEDouble                      = gpu->sim.pImageZ;
    gpu->sim.pImageZ                = gpu->sim.pImageZ2;
    gpu->sim.pImageZ2               = pPMEDouble; 
    pPMEDouble                      = gpu->sim.pImageVelX;
    gpu->sim.pImageVelX             = gpu->sim.pImageVelX2;
    gpu->sim.pImageVelX2            = pPMEDouble;
    pPMEDouble                      = gpu->sim.pImageVelY;
    gpu->sim.pImageVelY             = gpu->sim.pImageVelY2;
    gpu->sim.pImageVelY2            = pPMEDouble;
    pPMEDouble                      = gpu->sim.pImageVelZ;
    gpu->sim.pImageVelZ             = gpu->sim.pImageVelZ2;
    gpu->sim.pImageVelZ2            = pPMEDouble;
    pPMEDouble                      = gpu->sim.pImageLVelX;
    gpu->sim.pImageLVelX            = gpu->sim.pImageLVelX2;
    gpu->sim.pImageLVelX2           = pPMEDouble;
    pPMEDouble                      = gpu->sim.pImageLVelY;
    gpu->sim.pImageLVelY            = gpu->sim.pImageLVelY2;
    gpu->sim.pImageLVelY2           = pPMEDouble;
    pPMEDouble                      = gpu->sim.pImageLVelZ;
    gpu->sim.pImageLVelZ            = gpu->sim.pImageLVelZ2;
    gpu->sim.pImageLVelZ2           = pPMEDouble;    
    pPMEDouble                      = gpu->sim.pImageCharge;
    gpu->sim.pImageCharge           = gpu->sim.pImageCharge2;
    gpu->sim.pImageCharge2          = pPMEDouble;
    pPMEDouble                      = gpu->sim.pImageMass;
    gpu->sim.pImageMass             = gpu->sim.pImageMass2;
    gpu->sim.pImageMass2            = pPMEDouble;
    pPMEDouble                      = gpu->sim.pImageInvMass;
    gpu->sim.pImageInvMass          = gpu->sim.pImageInvMass2;
    gpu->sim.pImageInvMass2         = pPMEDouble;
    pPMEFloat2                      = gpu->sim.pImageSigEps;
    gpu->sim.pImageSigEps           = gpu->sim.pImageSigEps2;
    gpu->sim.pImageSigEps2          = pPMEFloat2;
    pUint                           = gpu->sim.pImageOutputBuffers;
    gpu->sim.pImageOutputBuffers    = gpu->sim.pImageOutputBuffers2;
    gpu->sim.pImageOutputBuffers2   = pUint;
    pUint                           = gpu->sim.pImageCellID;
    gpu->sim.pImageCellID           = gpu->sim.pImageCellID2;
    gpu->sim.pImageCellID2          = pUint;
}

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_THREADS_PER_BLOCK, 1)
#endif
kNLRemapLocalInteractions_kernel()
{
    unsigned int pos                = blockIdx.x * blockDim.x + threadIdx.x;    
    
    // Remap bond forces
    while (pos < cSim.bondOffset)
    {
        if (pos < cSim.bonds)
        {
            int4 atom                       = cSim.pBondID[pos];
            atom.x                          = cSim.pImageAtomLookup[atom.x];
            atom.y                          = cSim.pImageAtomLookup[atom.y];
            atom.z                         += atom.x;
            atom.w                         += atom.y;         
            cSim.pImageBondID[pos]          = atom;
        }
        pos                                += blockDim.x * gridDim.x;
    }

    
    // Remap bond angle forces
    while (pos < cSim.bondAngleOffset)
    {
        pos                                -= cSim.bondOffset;
        if (pos < cSim.bondAngles)
        {
            int4 atom1                      = cSim.pBondAngleID1[pos];
            int2 atom2                      = cSim.pBondAngleID2[pos];
            atom1.x                         = cSim.pImageAtomLookup[atom1.x];
            atom1.y                         = cSim.pImageAtomLookup[atom1.y];
            atom1.z                         = cSim.pImageAtomLookup[atom1.z];
            atom1.w                        += atom1.x;
            atom2.x                        += atom1.y;
            atom2.y                        += atom1.z;
            cSim.pImageBondAngleID1[pos]    = atom1;
            cSim.pImageBondAngleID2[pos]    = atom2;
        }
        pos                                += cSim.bondOffset + blockDim.x * gridDim.x;
    }
    
    // Remap dihedral forces
    while (pos < cSim.dihedralOffset)
    {
        pos                                -= cSim.bondAngleOffset;
        if (pos < cSim.dihedrals)
        {
            int4 atom1                      = cSim.pDihedralID1[pos];
            int4 atom2                      = cSim.pDihedralID2[pos];
            atom1.x                         = cSim.pImageAtomLookup[atom1.x];
            atom1.y                         = cSim.pImageAtomLookup[atom1.y];
            atom1.z                         = cSim.pImageAtomLookup[atom1.z];
            atom1.w                         = cSim.pImageAtomLookup[atom1.w];
            atom2.x                        += atom1.x;
            atom2.y                        += atom1.y;
            atom2.z                        += atom1.z;
            atom2.w                        += atom1.w;
            cSim.pImageDihedralID1[pos]     = atom1;
            cSim.pImageDihedralID2[pos]     = atom2;
        }
        pos                                += cSim.bondAngleOffset + blockDim.x * gridDim.x;
    }
   
   
    // Remap 1-4 forces
    while (pos < cSim.nb14Offset)
    {
        pos                                -= cSim.dihedralOffset;
        if (pos < cSim.nb14s)
        {
            int4 atom                       = cSim.pNb14ID[pos];
            atom.x                          = cSim.pImageAtomLookup[atom.x];
            atom.y                          = cSim.pImageAtomLookup[atom.y];
            atom.z                         += atom.x;
            atom.w                         += atom.y;
            cSim.pImageNb14ID[pos]          = atom;
        }
        pos                                += cSim.dihedralOffset + blockDim.x * gridDim.x;
    }    
    
    // Remap Constraint forces
    while (pos < cSim.constraintOffset)
    {
        pos                                -= cSim.nb14Offset;
        if (pos < cSim.constraints)
        {
            int2 atom                       = cSim.pConstraintID[pos];
            atom.x                          = cSim.pImageAtomLookup[atom.x];
            atom.y                         += atom.x;
            cSim.pImageConstraintID[pos]    = atom;
        }
        pos                                += cSim.nb14Offset + blockDim.x * gridDim.x;
    }
    pos                                    -= cSim.constraintOffset;
    
    // Remap Charmm interactions
    while (pos < cSim.UBAngleOffset)
    {
        if (pos < cSim.UBAngles)
        {
            int4 atom                       = cSim.pUBAngleID[pos];
            atom.x                          = cSim.pImageAtomLookup[atom.x];
            atom.y                          = cSim.pImageAtomLookup[atom.y];
            atom.z                         += atom.x;
            atom.w                         += atom.y;         
            cSim.pImageUBAngleID[pos]       = atom;
        }
        pos                                += blockDim.x * gridDim.x;    
    }
    
    while (pos < cSim.impDihedralOffset)
    {
        pos                                -= cSim.UBAngleOffset;
        if (pos < cSim.impDihedrals)
        {
            int4 atom1                      = cSim.pImpDihedralID1[pos];
            int4 atom2                      = cSim.pImpDihedralID2[pos];
            atom1.x                         = cSim.pImageAtomLookup[atom1.x];
            atom1.y                         = cSim.pImageAtomLookup[atom1.y];
            atom1.z                         = cSim.pImageAtomLookup[atom1.z];
            atom1.w                         = cSim.pImageAtomLookup[atom1.w];
            atom2.x                        += atom1.x;
            atom2.y                        += atom1.y;
            atom2.z                        += atom1.z;
            atom2.w                        += atom1.w;
            cSim.pImageImpDihedralID1[pos]  = atom1;
            cSim.pImageImpDihedralID2[pos]  = atom2;
        }
        pos                                += cSim.UBAngleOffset + blockDim.x * gridDim.x;    
    }
    
    while (pos < cSim.cmapOffset)
    {
        pos                                -= cSim.impDihedralOffset;
        if (pos < cSim.cmaps)
        {       
            int4 atom1                      = cSim.pCmapID1[pos];
            int4 atom2                      = cSim.pCmapID2[pos];
            int2 atom3                      = cSim.pCmapID3[pos];
            atom1.x                         = cSim.pImageAtomLookup[atom1.x];
            atom1.y                         = cSim.pImageAtomLookup[atom1.y];
            atom1.z                         = cSim.pImageAtomLookup[atom1.z];
            atom1.w                         = cSim.pImageAtomLookup[atom1.w];
            atom2.x                         = cSim.pImageAtomLookup[atom2.x];
            atom2.y                        += atom1.x;
            atom2.z                        += atom1.y;
            atom2.w                        += atom1.z;
            atom3.x                        += atom1.w;
            atom3.y                        += atom2.x;
            cSim.pImageCmapID1[pos]         = atom1;
            cSim.pImageCmapID2[pos]         = atom2;
            cSim.pImageCmapID3[pos]         = atom3;
        }
        pos                                += cSim.impDihedralOffset + blockDim.x * gridDim.x;    
    }    
    pos                                    -= cSim.cmapOffset;

    // Remap Shake constraints
    while (pos < cSim.shakeOffset)
    {
        if (pos < cSim.shakeConstraints)
        {
            int4 atom                       = cSim.pShakeID[pos];
            atom.x                          = cSim.pImageAtomLookup[atom.x];
            atom.y                          = cSim.pImageAtomLookup[atom.y];
            if (atom.z != -1)
                atom.z                      = cSim.pImageAtomLookup[atom.z];
            if (atom.w != -1)
                atom.w                      = cSim.pImageAtomLookup[atom.w];
            cSim.pImageShakeID[pos]         = atom;
        }
        pos                                += blockDim.x * gridDim.x;
    }
    pos                                    -= cSim.shakeOffset;

    while (pos < cSim.fastShakeOffset)
    {
        if (pos < cSim.fastShakeConstraints)
        {
            int4 atom                       = cSim.pFastShakeID[pos];
            atom.x                          = cSim.pImageAtomLookup[atom.x];
            atom.y                          = cSim.pImageAtomLookup[atom.y];
            atom.z                          = cSim.pImageAtomLookup[atom.z];
            cSim.pImageFastShakeID[pos]     = atom;
        }
        pos                                += blockDim.x * gridDim.x;
    }
    pos                                    -= cSim.fastShakeOffset;

    while (pos < cSim.slowShakeOffset)
    {
        if (pos < cSim.slowShakeConstraints)
        {
            int atom1                       = cSim.pSlowShakeID1[pos];
            int4 atom2                      = cSim.pSlowShakeID2[pos];
            atom1                           = cSim.pImageAtomLookup[atom1];
            atom2.x                         = cSim.pImageAtomLookup[atom2.x];
            atom2.y                         = cSim.pImageAtomLookup[atom2.y];
            atom2.z                         = cSim.pImageAtomLookup[atom2.z];
            atom2.w                         = cSim.pImageAtomLookup[atom2.w];
            cSim.pImageSlowShakeID1[pos]    = atom1;
            cSim.pImageSlowShakeID2[pos]    = atom2;
        }
        pos                                += blockDim.x * gridDim.x;
    }
    pos                                    -= cSim.slowShakeOffset;
    
    while (pos < cSim.soluteAtoms)          // Solute atoms already padded to warp width
    {
        int atom                            = cSim.pSoluteAtomID[pos];
        if (atom != -1)
            atom                            = cSim.pImageAtomLookup[atom];
        cSim.pImageSoluteAtomID[pos]        = atom;         
        pos                                += blockDim.x * gridDim.x;
    }
    pos                                    -= cSim.soluteAtoms;

    while (pos < cSim.solventMoleculeStride)
    {
        if (pos < cSim.solventMolecules)
        {
            int4 atom                           = cSim.pSolventAtomID[pos];
            atom.x                              = cSim.pImageAtomLookup[atom.x];
            if (atom.y != -1)
                atom.y                          = cSim.pImageAtomLookup[atom.y];
            if (atom.z != -1)
                atom.z                          = cSim.pImageAtomLookup[atom.z];
            if (atom.w != -1)
                atom.w                          = cSim.pImageAtomLookup[atom.w];
            cSim.pImageSolventAtomID[pos]       = atom;
        }
        pos                                    += blockDim.x * gridDim.x;
    }
    pos                                        -= cSim.solventMoleculeStride;

    while (pos < cSim.EP11Offset)
    {  
        if (pos < cSim.EP11s)
        {
            int4 frame                          = cSim.pExtraPoint11Frame[pos];
            int index                           = cSim.pExtraPoint11Index[pos];   
            frame.x                             = cSim.pImageAtomLookup[frame.x];
            frame.y                             = cSim.pImageAtomLookup[frame.y];
            frame.z                             = cSim.pImageAtomLookup[frame.z];
            frame.w                             = cSim.pImageAtomLookup[frame.w];
            index                               = cSim.pImageAtomLookup[index];
            cSim.pImageExtraPoint11Frame[pos]   = frame;
            cSim.pImageExtraPoint11Index[pos]   = index;
        }
        pos                                    += blockDim.x * gridDim.x;
    }
    while (pos < cSim.EP12Offset)
    {
        pos                                    -= cSim.EP11Offset;
        if (pos < cSim.EP12s)
        {
            int4 frame                          = cSim.pExtraPoint12Frame[pos];
            int index                           = cSim.pExtraPoint12Index[pos];   
            frame.x                             = cSim.pImageAtomLookup[frame.x];
            frame.y                             = cSim.pImageAtomLookup[frame.y];
            frame.z                             = cSim.pImageAtomLookup[frame.z];
            frame.w                             = cSim.pImageAtomLookup[frame.w];
            index                               = cSim.pImageAtomLookup[index];
            cSim.pImageExtraPoint12Frame[pos]   = frame;
            cSim.pImageExtraPoint12Index[pos]   = index;
        }
        pos                                    += cSim.EP11Offset + blockDim.x * gridDim.x;
    }
    while (pos < cSim.EP21Offset)
    {
        pos                                    -= cSim.EP12Offset;        
        if (pos < cSim.EP21s)
        {
            int4 frame                          = cSim.pExtraPoint21Frame[pos];
            int2 index                          = cSim.pExtraPoint21Index[pos];   
            frame.x                             = cSim.pImageAtomLookup[frame.x];
            frame.y                             = cSim.pImageAtomLookup[frame.y];
            frame.z                             = cSim.pImageAtomLookup[frame.z];
            frame.w                             = cSim.pImageAtomLookup[frame.w];
            index.x                             = cSim.pImageAtomLookup[index.x];
            index.y                             = cSim.pImageAtomLookup[index.y];
            cSim.pImageExtraPoint21Frame[pos]   = frame;
            cSim.pImageExtraPoint21Index[pos]   = index;
        }
        pos                                    += cSim.EP12Offset + blockDim.x * gridDim.x;
    }   
    while (pos < cSim.EP22Offset)
    {
        pos                                    -= cSim.EP21Offset;
        if (pos < cSim.EP22s)
        {
            int4 frame                          = cSim.pExtraPoint22Frame[pos];
            int2 index                          = cSim.pExtraPoint22Index[pos];   
            frame.x                             = cSim.pImageAtomLookup[frame.x];
            frame.y                             = cSim.pImageAtomLookup[frame.y];
            frame.z                             = cSim.pImageAtomLookup[frame.z];
            frame.w                             = cSim.pImageAtomLookup[frame.w];
            index.x                             = cSim.pImageAtomLookup[index.x];
            index.y                             = cSim.pImageAtomLookup[index.y];
            cSim.pImageExtraPoint22Frame[pos]   = frame;
            cSim.pImageExtraPoint22Index[pos]   = index;
        }
        pos                                    += cSim.EP21Offset + blockDim.x * gridDim.x;
    }
}

extern "C" void kNLRemapLocalInteractions(gpuContext gpu)
{
    kNLRemapLocalInteractions_kernel<<<gpu->blocks, gpu->threadsPerBlock>>>();
    LAUNCHERROR("kNLRemapLocalInteractions");   
}

// Clear all cell boundaries in case some are empty
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_THREADS_PER_BLOCK, 1)
#endif
kNLClearCellBoundaries_kernel()
{
    unsigned int pos                        = blockIdx.x * blockDim.x + threadIdx.x;
    uint2 nulldata                          = {0, 0};
    while (pos < cSim.cells)
    {
        cSim.pNLNonbondCellStartEnd[pos]    = nulldata;
        pos                                += blockDim.x * gridDim.x;
    }
}

extern "C" void kNLClearCellBoundaries(gpuContext gpu)
{
    kNLClearCellBoundaries_kernel<<<gpu->blocks, gpu->threadsPerBlock>>>();
    LAUNCHERROR("kNLClearCellBoundaries");   
}


__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_THREADS_PER_BLOCK, 1)
#endif
kNLCalculateCellBoundaries_kernel(unsigned int* pImageHash)
{
#if (__CUDA_ARCH__ >= 200)
const int cSpan = 12000;
#else
const int cSpan = 4000;
#endif
__shared__ unsigned int sHash[cSpan];
    int pos                                                 = ((cSim.atoms + 1) * blockIdx.x) / gridDim.x - 1;
    int end                                                 = ((cSim.atoms + 1) * (blockIdx.x + 1)) / gridDim.x;
    while (pos < end)
    {
        // Read span to check for transitions
        int pos1                                            = pos + threadIdx.x;
        int spos                                            = threadIdx.x;
        int span                                            = min(end, pos + cSpan);
        int lSpan                                           = min(end, pos + cSpan) - pos;
        while (pos1 < span)
        {
            // Read hash data or 0s on either end to force transitions
            if ((pos1 >= 0) && (pos1 < cSim.atoms)) 
                sHash[spos]                                 = pImageHash[pos1];
            else
                sHash[spos]                                 = 0;
            pos1                                           += blockDim.x;
            spos                                           += blockDim.x;
        }
        __syncthreads();
        spos                                                = threadIdx.x + 1;
        while (spos < lSpan)
        {
            int oldHash                                     = sHash[spos - 1] >> CELLHASHBITS;
            int newHash                                     = sHash[spos] >> CELLHASHBITS;
            if (oldHash != newHash)
            {
                if (pos + spos != cSim.atoms)
                    cSim.pNLNonbondCellStartEnd[newHash].x  = pos + spos;
                if (pos + spos != 0)
                    cSim.pNLNonbondCellStartEnd[oldHash].y  = pos + spos;   
            }
            spos                                           += blockDim.x;
        }      
        __syncthreads();  
        pos                                                 = min(end, pos + cSpan - 1);        
    }
}

extern "C" void kNLCalculateCellBoundaries(gpuContext gpu)
{
    kNLCalculateCellBoundaries_kernel<<<gpu->blocks, gpu->threadsPerBlock>>>(gpu->sim.pImageHash);   
    LAUNCHERROR("kNLCalculateCellBoundaries");
}



__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_THREADS_PER_BLOCK, 1)
#endif
kNLClearExclusionMasks_kernel()
{

}

extern "C" void kNLClearExclusionMasks(gpuContext gpu)
{
    kNLClearExclusionMasks_kernel<<<gpu->blocks, gpu->threadsPerBlock>>>();   
    LAUNCHERROR("kNLClearExclusionMasks");
}

struct GEData {
    unsigned int workUnit;
    unsigned int xCellStart;
    unsigned int xCellEnd;
    unsigned int yCellStart;
    unsigned int yCellEnd;
    unsigned int exclusionMap;
    unsigned int imageAtom;
    unsigned int atom;
};

#if 0
__global__ void
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_NLCALCULATE_OFFSETS_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_NLCALCULATE_OFFSETS_THREADS_PER_BLOCK, 1)
#endif
kNLCalculateOffsets_kernel()
{
#if (__CUDA_ARCH__ >= 200)
__shared__ volatile SNLRecord sNLRecord[SM_2X_NLCALCULATE_OFFSETS_THREADS_PER_BLOCK];
#else
__shared__ volatile SNLRecord sNLRecord[SM_13_NLCALCULATE_OFFSETS_THREADS_PER_BLOCK];
#endif
    unsigned int pos                            = blockIdx.x * blockDim.x;
    unsigned int warps                          = blockDim.x >> 4;

    while (pos < cSim.NLSize)
    {
        // Read NLRecord entries
        unsigned int rpos                       = threadIdx.x >> 4;
        unsigned int tpos                       = threadIdx.x - (rpos << 4);
        unsigned int reads                      = min(blockDim.x, cSim.NLSize - pos);
        
        while (rpos < reads)
        {
            sNLRecord[rpos].array[tpos]         = cSim.pNLRecord[pos + rpos].array[tpos];
            rpos                               += warps;
        }
        __syncthreads();
        
        // Read y cell data
        if (pos + threadIdx.x < cSim.NLSize)
        {
            uint2 homeCell                      = cSim.pNLNonbondCellStartEnd[sNLRecord[threadIdx.x].NL.homeCell >> NLCELLSHIFT];
            int startOffset                     = sNLRecord[threadIdx.x].NL.homeCell & NLBUFFERMASK;
            int ysize                           = max(0, (int)(((homeCell.y - homeCell.x + (GRID - 1)) >> GRIDBITS)) - startOffset) / cSim.NLYDivisor;
            
            // Calculate maximum required space
            int xsize                           = 0;
            for (int i = 0; i < sNLRecord[i].NL.neighborCells; i++)
            {
                uint2 cell                      = cSim.pNLNonbondCellStartEnd[sNLRecord[threadIdx.x].NL.neighborCell[i] >> NLCELLSHIFT];
                xsize                          += cell.y - cell.x;
            }
            xsize                               = ((xsize + GRID - 1) >> GRIDBITS) << GRIDBITS;
            cSim.pNLOffset[pos + threadIdx.x]   =  xsize * ysize;
        }
    
        pos                                    += blockDim.x * gridDim.x;
    }
}

extern "C" void kNLCalculateOffsets(gpuContext gpu)
{
    kNLCalculateOffsets_kernel<<<gpu->blocks, gpu->threadsPerBlock>>>();   
    LAUNCHERROR("kNLCalculateOffsets");
    
    hipDeviceSynchronize();
    gpu->pbNLOffset->Download();
    for (int i = 0; i < gpu->sim.NLSize; i++)
        printf("%4d %6d\n", i, gpu->pbNLOffset->_pSysData[i]);
#if 0        
    gpu->pbNLNonbondCellStartEnd->Download();  
    for (int i = 0; i < gpu->sim.cells; i++)
        printf("%4d %6d\n", i, gpu->pbNLNonbondCellStartEnd->_pSysData[i].y - gpu->pbNLNonbondCellStartEnd->_pSysData[i].x);
#endif
        
    exit(-1);
    
}
#endif

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_UPDATE_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_UPDATE_THREADS_PER_BLOCK, 1)
#endif
kNLCalculateCellCoordinates_kernel()
#include "kCCC.h"

#define PME_ORTHOGONAL
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_UPDATE_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_UPDATE_THREADS_PER_BLOCK, 1)
#endif
kNLCalculateCellCoordinatesOrthogonal_kernel()
#include "kCCC.h"
#undef PME_ORTHOGONAL

#define PME_NTP
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_UPDATE_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_UPDATE_THREADS_PER_BLOCK, 1)
#endif
kNLCalculateCellCoordinatesNTP_kernel()
#include "kCCC.h"

#define PME_ORTHOGONAL
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_UPDATE_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_UPDATE_THREADS_PER_BLOCK, 1)
#endif
kNLCalculateCellCoordinatesOrthogonalNTP_kernel()
#include "kCCC.h"
#undef PME_ORTHOGONAL
#undef PME_NTP

extern "C" void kNLCalculateCellCoordinates(gpuContext gpu)
{
    if (gpu->sim.ntp > 0)
    {
        if (gpu->sim.is_orthog)    
            kNLCalculateCellCoordinatesOrthogonalNTP_kernel<<<gpu->blocks, gpu->updateThreadsPerBlock>>>();  
        else
            kNLCalculateCellCoordinatesNTP_kernel<<<gpu->blocks, gpu->updateThreadsPerBlock>>>();         
    }
    else
    {
        if (gpu->sim.is_orthog)    
            kNLCalculateCellCoordinatesOrthogonal_kernel<<<gpu->blocks, gpu->updateThreadsPerBlock>>>();  
        else
            kNLCalculateCellCoordinates_kernel<<<gpu->blocks, gpu->updateThreadsPerBlock>>>(); 
    } 
    hipDeviceSynchronize();

#if 0
    gpu->pbAtomXYSP->Download();
    gpu->pbAtomZSP->Download();
    for (int i = 0; i < gpu->sim.atoms; i++)
    {
        printf("%6d: %16.8f %16.8f %16.8f\n", i, gpu->pbAtomXYSP->_pSysData[i].x, gpu->pbAtomXYSP->_pSysData[i].y, gpu->pbAtomZSP->_pSysData[i]);
    }
#endif    
    
    LAUNCHERROR_NONBLOCKING("kNLCalculateCellCoordinates");
}


#define PME_ATOMS_PER_WARP (32)
#define PME_IS_ORTHOGONAL
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_NLBUILD_NEIGHBORLIST32_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_NLBUILD_NEIGHBORLIST32_THREADS_PER_BLOCK, 1)
#endif
kNLBuildNeighborListOrthogonal32_kernel()
#include "kBNL.h"

#define PME_VIRIAL
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_NLBUILD_NEIGHBORLIST32_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_NLBUILD_NEIGHBORLIST32_THREADS_PER_BLOCK, 1)
#endif
kNLBuildNeighborListOrthogonalNTP32_kernel()
#include "kBNL.h"
#undef PME_VIRIAL
#undef PME_IS_ORTHOGONAL

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_NLBUILD_NEIGHBORLIST32_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_NLBUILD_NEIGHBORLIST32_THREADS_PER_BLOCK, 1)
#endif
kNLBuildNeighborList32_kernel()
#include "kBNL.h"

#define PME_VIRIAL
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_NLBUILD_NEIGHBORLIST32_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_NLBUILD_NEIGHBORLIST32_THREADS_PER_BLOCK, 1)
#endif
kNLBuildNeighborListNTP32_kernel()
#include "kBNL.h"
#undef PME_VIRIAL
#undef PME_ATOMS_PER_WARP


#define PME_ATOMS_PER_WARP (16)
#define PME_IS_ORTHOGONAL
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_NLBUILD_NEIGHBORLIST16_THREADS_PER_BLOCK + 64, 1)
#else
__launch_bounds__(SM_13_NLBUILD_NEIGHBORLIST16_THREADS_PER_BLOCK, 1)
#endif
kNLBuildNeighborListOrthogonal16_kernel()
#include "kBNL.h"

#define PME_VIRIAL
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_NLBUILD_NEIGHBORLIST16_THREADS_PER_BLOCK + 64, 1)
#else
__launch_bounds__(SM_13_NLBUILD_NEIGHBORLIST16_THREADS_PER_BLOCK, 1)
#endif
kNLBuildNeighborListOrthogonalNTP16_kernel()
#include "kBNL.h"
#undef PME_VIRIAL
#undef PME_IS_ORTHOGONAL

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_NLBUILD_NEIGHBORLIST16_THREADS_PER_BLOCK + 64, 1)
#else
__launch_bounds__(SM_13_NLBUILD_NEIGHBORLIST16_THREADS_PER_BLOCK, 1)
#endif
kNLBuildNeighborList16_kernel()
#include "kBNL.h"

#define PME_VIRIAL
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_NLBUILD_NEIGHBORLIST16_THREADS_PER_BLOCK + 64, 1)
#else
__launch_bounds__(SM_13_NLBUILD_NEIGHBORLIST16_THREADS_PER_BLOCK, 1)
#endif
kNLBuildNeighborListNTP16_kernel()
#include "kBNL.h"
#undef PME_VIRIAL
#undef PME_ATOMS_PER_WARP

extern "C" void kNLBuildNeighborList(gpuContext gpu)
{
//    cudaThreadSetLimit(hipLimitPrintfFifoSize, 200000000);   
//    printf("%d %d %d\n", gpu->sim.NLAtomsPerWarp, gpu->sim.ntp, gpu->sim.is_orthog);
//    printf("%06d %06d\n", gpu->NLBuildNeighborList16ThreadsPerBlock, sizeof(uint) * gpu->sim.NLMaxExclusionsPerWarp * gpu->NLBuildNeighborList16ThreadsPerBlock / GRID);
//    printf("%06d %06d\n", gpu->NLBuildNeighborList32ThreadsPerBlock, sizeof(uint) * gpu->sim.NLMaxExclusionsPerWarp * gpu->NLBuildNeighborList32ThreadsPerBlock / GRID);
    if (gpu->sim.NLAtomsPerWarp == 32)
    {
        if (gpu->sim.ntp > 0)
        {
            if (gpu->sim.is_orthog)    
                kNLBuildNeighborListOrthogonalNTP32_kernel<<<gpu->blocks, gpu->NLBuildNeighborList32ThreadsPerBlock, sizeof(uint) * gpu->sim.NLMaxExclusionsPerWarp * gpu->NLBuildNeighborList32ThreadsPerBlock / GRID>>>();  
            else
                kNLBuildNeighborListNTP32_kernel<<<gpu->blocks, gpu->NLBuildNeighborList32ThreadsPerBlock, sizeof(uint) * gpu->sim.NLMaxExclusionsPerWarp * gpu->NLBuildNeighborList32ThreadsPerBlock / GRID>>>();         
        }
        else
        {
            if (gpu->sim.is_orthog)    
                kNLBuildNeighborListOrthogonal32_kernel<<<gpu->blocks, gpu->NLBuildNeighborList32ThreadsPerBlock, sizeof(uint) * gpu->sim.NLMaxExclusionsPerWarp * gpu->NLBuildNeighborList32ThreadsPerBlock / GRID>>>();  
            else
                kNLBuildNeighborList32_kernel<<<gpu->blocks, gpu->NLBuildNeighborList32ThreadsPerBlock, sizeof(uint) * gpu->sim.NLMaxExclusionsPerWarp * gpu->NLBuildNeighborList32ThreadsPerBlock / GRID>>>(); 
        }     
    }
    else
    {
        if (gpu->sim.ntp > 0)
        {
            if (gpu->sim.is_orthog)    
                kNLBuildNeighborListOrthogonalNTP16_kernel<<<gpu->blocks, gpu->NLBuildNeighborList16ThreadsPerBlock, sizeof(uint) * gpu->sim.NLMaxExclusionsPerWarp * gpu->NLBuildNeighborList16ThreadsPerBlock / GRID>>>();  
            else
                kNLBuildNeighborListNTP16_kernel<<<gpu->blocks, gpu->NLBuildNeighborList16ThreadsPerBlock, sizeof(uint) * gpu->sim.NLMaxExclusionsPerWarp * gpu->NLBuildNeighborList16ThreadsPerBlock / GRID>>>();         
        }
        else
        {
            if (gpu->sim.is_orthog)    
                kNLBuildNeighborListOrthogonal16_kernel<<<gpu->blocks, gpu->NLBuildNeighborList16ThreadsPerBlock, sizeof(uint) * gpu->sim.NLMaxExclusionsPerWarp * gpu->NLBuildNeighborList16ThreadsPerBlock / GRID>>>();  
            else
                kNLBuildNeighborList16_kernel<<<gpu->blocks, gpu->NLBuildNeighborList16ThreadsPerBlock, sizeof(uint) * gpu->sim.NLMaxExclusionsPerWarp * gpu->NLBuildNeighborList16ThreadsPerBlock / GRID>>>(); 
        }     
    }

    LAUNCHERROR("kNLBuildNeighborList");    

#if 0
    hipDeviceSynchronize();  
    gpu->pbNLTotalOffset->Download();
    fprintf(stdout, "Total: %d, maximum %d\n", gpu->pbNLTotalOffset->_pSysData[0], gpu->sim.NLMaxTotalOffset);
#endif
    
#if 0   
    hipDeviceSynchronize();
    gpu->pbNLOffset->Download();  
    for (int i = 0; i < gpu->sim.NLSize; i++)
        printf("%4d %12u\n", i, gpu->pbNLOffset->_pSysData[i]);
#endif

#if 0        
    gpu->pbNLNonbondCellStartEnd->Download();  
    for (int i = 0; i < gpu->sim.cells; i++)
        printf("%4d %6d\n", i, gpu->pbNLNonbondCellStartEnd->_pSysData[i].y - gpu->pbNLNonbondCellStartEnd->_pSysData[i].x);
#endif   
}

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_UPDATE_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_UPDATE_THREADS_PER_BLOCK, 1)
#endif
kNLSkinTest_kernel()
{
    __shared__ volatile bool sbFail;
    __shared__ PMEFloat sOne_half_nonbond_skin_squared;
    unsigned int pos                        = blockIdx.x * blockDim.x + threadIdx.x;
    sbFail                                  = false; 
    if ((cSim.ntp > 0) && (threadIdx.x == 0))
        sOne_half_nonbond_skin_squared      = cSim.pNTPData->one_half_nonbond_skin_squared;
    __syncthreads();
    
    if (cSim.ntp > 0)
    {
        while (pos < cSim.atoms)
        {
            PMEFloat x                      = cSim.pImageX[pos];
            PMEFloat y                      = cSim.pImageY[pos];
            PMEFloat2 oldxy                 = cSim.pAtomXYSaveSP[pos];
            PMEFloat z                      = cSim.pImageZ[pos];
            PMEFloat oldz                   = cSim.pAtomZSaveSP[pos];
            PMEFloat dx                     = x - oldxy.x;
            PMEFloat dy                     = y - oldxy.y;
            PMEFloat dz                     = z - oldz;
            
            PMEFloat r2                     = dx * dx + dy * dy + dz * dz;
            if (r2 >= sOne_half_nonbond_skin_squared)
                sbFail                      = true;                  

            pos                            += blockDim.x * gridDim.x;    
        }    
    }
    else
    {
        while (pos < cSim.atoms)
        {
            PMEFloat x                      = cSim.pImageX[pos];
            PMEFloat y                      = cSim.pImageY[pos];
            PMEFloat2 oldxy                 = cSim.pAtomXYSaveSP[pos];
            PMEFloat z                      = cSim.pImageZ[pos];
            PMEFloat oldz                   = cSim.pAtomZSaveSP[pos];
            
            PMEFloat dx                     = x - oldxy.x;
            PMEFloat dy                     = y - oldxy.y;
            PMEFloat dz                     = z - oldz;
            
            PMEFloat r2                     = dx * dx + dy * dy + dz * dz;
            if (r2 >= cSim.one_half_nonbond_skin_squared)
                sbFail                      = true;                  

            pos                            += blockDim.x * gridDim.x;    
        }
    }
    
    __syncthreads();
    if ((threadIdx.x == 0) && sbFail)
        *cSim.pNLbSkinTestFail              = true;
}

extern "C" void kNLSkinTest(gpuContext gpu)
{
    static int counter = 0;
 
    *(gpu->pbNLbSkinTestFail->_pSysData) = false;    
    kNLSkinTest_kernel<<<gpu->blocks, gpu->updateThreadsPerBlock>>>();   
    LAUNCHERROR_NONBLOCKING("kNLSkinTest");
    hipDeviceSynchronize();
    
    if (*(gpu->pbNLbSkinTestFail->_pSysData))
    {
     //   printf("%d %f\n", i, gpu->sim.one_half_nonbond_skin_squared);
        gpu->bNeedNewNeighborList = true;
     //   fprintf(stdout, "%06d yes build\n", counter);   
    }
    // else    
    //    fprintf(stdout, "%06d no build\n", counter);

    counter++;
}

