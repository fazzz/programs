#include "hip/hip_runtime.h"
/***************************************************/
/*                                                 */
/*      AMBER NVIDIA CUDA CPU IMPLEMENTATION       */
/*                 PMEMD VERSION                   */
/*                     2011                        */
/*                      by                         */
/*                Romelia Salomon (SDSC)           */
/*                                                 */
/***************************************************/

#include <hip/hip_runtime.h>
#include "gpu.h"

static __constant__ cudaSimulation cSim;
static __constant__ PMEDouble tenm3             = (PMEDouble)(1.0e-03);
static __constant__ PMEDouble tm24              = (PMEDouble)(1.0e-18);
static __constant__ PMEDouble one               = (PMEDouble)(1.0);
static __constant__ PMEDouble zero              = (PMEDouble)(0.0);

// Texture reference for PMEDouble-precision coordinates (disguised as int2 to work around HW limitations)
#ifndef use_SPSP
texture<int2, 1, hipReadModeElementType> texref;
#else
texture<float, 1, hipReadModeElementType> texref;
#endif

void SetkCalculateAMDWeightsSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

void GetkCalculateAMDWeightssSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}


__device__ void faster_sincos2(double a, double *sptr, double *cptr) 
#include "kFastCosSin.h"


__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_UPDATE_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_UPDATE_THREADS_PER_BLOCK, 1)
#endif
kAMDCalcWeightAndScaleForces_kernel(PMEDouble pot_ene_tot, PMEDouble dih_ene_tot, PMEDouble fwgt)
{

  //calculate AMD weight, seting dihedral boost (tboost) to zero for now

    unsigned int pos                        = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int increment                              = gridDim.x * blockDim.x;    
    while (pos < cSim.atoms)
    {
        PMEDouble forceX                            = cSim.pForceX[pos];
        PMEDouble forceY                            = cSim.pForceY[pos];
        PMEDouble forceZ                            = cSim.pForceZ[pos];

        forceX                                 *= fwgt;
        forceY                                 *= fwgt;
        forceZ                                 *= fwgt;

        cSim.pForceX[pos]                = forceX;
        cSim.pForceY[pos]                = forceY;
        cSim.pForceZ[pos]                = forceZ;
        pos                                += increment;       
    }
}

void kAMDCalcWeightAndScaleForces(gpuContext gpu, PMEDouble pot_ene_tot, PMEDouble dih_ene_tot, PMEDouble fwgt)
{

      kAMDCalcWeightAndScaleForces_kernel<<<gpu->blocks, gpu->updateThreadsPerBlock>>>(pot_ene_tot,dih_ene_tot,fwgt);

    LAUNCHERROR("kAMDScaleForces");

}

#define LOCAL_ENERGY
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_LOCALFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_LOCALFORCES_THREADS_PER_BLOCK, 1)
#endif
kCalculateAmdDihedralEnergy_kernel()
#include "kCLFdih.h"
#undef LOCAL_ENERGY

#define LOCAL_NEIGHBORLIST
#define LOCAL_ENERGY
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_LOCALFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_LOCALFORCES_THREADS_PER_BLOCK, 1)
#endif
kCalculatePMEAmdDihedralEnergy_kernel()
#include "kCLFdih.h"
#undef LOCAL_ENERGY
#undef LOCAL_NEIGHBORLIST

// Consumer Fermi kernels
#define NODPTEXTURE
#define LOCAL_ENERGY
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_LOCALFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_LOCALFORCES_THREADS_PER_BLOCK, 1)
#endif
kCalculateAmdDihedralEnergyFermi_kernel()
#include "kCLFdih.h"
#undef LOCAL_ENERGY

#define LOCAL_NEIGHBORLIST
#define LOCAL_ENERGY
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_LOCALFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_LOCALFORCES_THREADS_PER_BLOCK, 1)
#endif
kCalculatePMEAmdDihedralEnergyFermi_kernel()
#include "kCLFdih.h"
#undef LOCAL_ENERGY
#undef LOCAL_NEIGHBORLIST
#undef NODPTEXTURE


extern "C" void kCalculateAmdDihedralEnergy(gpuContext gpu)
{

//Calculate dihedral energy
    if (gpu->bLocalInteractions)
    {
#ifndef use_SPSP    
        if (!gpu->bECCSupport && (gpu->sm_version == SM_2X))
        {
            if (gpu->bNeighborList)
            {
                    kCalculatePMEAmdDihedralEnergyFermi_kernel<<<gpu->blocks, gpu->localForcesThreadsPerBlock>>>();
            }
            else
                kCalculateAmdDihedralEnergyFermi_kernel<<<gpu->blocks, gpu->localForcesThreadsPerBlock>>>();
#ifdef MPI 
            LAUNCHERROR_NONBLOCKING("kCalculateAmdDihedralEnergyFermi");
#else
            LAUNCHERROR("kCalculateAmdDihedralEnergyFermi");
#endif          
        }
        else
#endif
        {
            texref.normalized       = 0;
            texref.filterMode       = hipFilterModePoint;
            texref.addressMode[0]   = hipAddressModeClamp;
            texref.channelDesc.x    = 32;
#ifndef use_SPSP        
            texref.channelDesc.y    = 32;
#else
            texref.channelDesc.y    = 0;
#endif        
            texref.channelDesc.z    = 0;
            texref.channelDesc.w    = 0;
        
#ifndef use_SPSP       
            int2* pX;
            if (gpu->bNeighborList)
                pX                  = (int2*)gpu->sim.pImageX;
            else
                pX                  = (int2*)gpu->sim.pAtomX;
            hipBindTexture(NULL, texref, pX, gpu->sim.stride3 * sizeof(int2));
#else
            float* pX;
            if (gpu->bNeighborList)
                pX                  = (float*)gpu->sim.pImageX;
            else
                pX                  = (float*)gpu->sim.pAtomX;
            hipBindTexture(NULL, texref, pX, gpu->sim.stride3 * sizeof(float));
#endif        
            if (gpu->bNeighborList)
            {
                kCalculatePMEAmdDihedralEnergy_kernel<<<gpu->blocks, gpu->localForcesThreadsPerBlock>>>();
            }
            else
                kCalculateAmdDihedralEnergy_kernel<<<gpu->blocks, gpu->localForcesThreadsPerBlock>>>();
#ifdef MPI 
            LAUNCHERROR_NONBLOCKING("kCalculateAmdDihedralEnergy");
#else
            LAUNCHERROR("kCalculateAmdDihedralEnergy");
#endif  
            hipUnbindTexture(texref);
        }
    }

}

