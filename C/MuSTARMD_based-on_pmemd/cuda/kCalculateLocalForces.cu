#include "hip/hip_runtime.h"
/***************************************************/
/*                                                 */
/*      AMBER NVIDIA CUDA CPU IMPLEMENTATION       */
/*                 PMEMD VERSION                   */
/*                     2010                        */
/*                      by                         */
/*             Scott Le Grand (NVIDIA)             */
/*               Duncan Poole (NVIDIA)             */
/*                Ross Walker (SDSC)               */
/*                                                 */
/***************************************************/

#include <hip/hip_runtime.h>
#include "gpu.h"

//extern __shared__ Vectors sV[];
static __constant__ cudaSimulation cSim;
static __constant__ PMEDouble pt999             = (PMEDouble)(0.9990);
static __constant__ PMEDouble tm06              = (PMEDouble)(1.0e-06);
static __constant__ PMEDouble tenm3             = (PMEDouble)(1.0e-03);
static __constant__ PMEDouble tm24              = (PMEDouble)(1.0e-18);
static __constant__ PMEDouble one               = (PMEDouble)(1.0);
static __constant__ PMEDouble zero              = (PMEDouble)(0.0);
static __constant__ PMEFloat rad_to_deg_coeff   = (PMEDouble)180.0 / ((PMEDouble)CMAPSTEPSIZE * (PMEDouble)PI_VAL);

// Texture reference for PMEDouble-precision coordinates (disguised as int2 to work around HW limitations)
#ifndef use_SPSP
texture<int2, 1, hipReadModeElementType> texref;
#else
texture<float, 1, hipReadModeElementType> texref;
#endif

void SetkCalculateLocalForcesSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

void GetkCalculateLocalForcesSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

/*
 * This special version of sincos is designed for |a| < 6*PI. On a GTX 285
 * it is about 25% faster than sincos from the CUDA math library. Also uses
 * 8 fewer registers than the CUDA math library's sincos. Maximum observed
 * error is 2 ulps across range stated above. Infinities and negative zero
 * are not handled according to C99 specifications. NaNs are handled fine.
 */
__device__ void faster_sincos(double a, double *sptr, double *cptr) {
  double t, u, s, c, j, a2;
  int i;

  i = __double2int_rn (a * 6.3661977236758138e-1);
  j = (double)i;
  a = __fma_rn (-j, 1.57079632679489660e+000, a); /* PIO2_HI */
  a = __fma_rn (-j, 6.12323399573676600e-017, a); /* PIO2_LO */
  a2 = a * a;
  u =                  -1.136788825395985E-011;   
  u = __fma_rn (u, a2,  2.087588480545065E-009);
  u = __fma_rn (u, a2, -2.755731555403950E-007);
  u = __fma_rn (u, a2,  2.480158729365970E-005);
  u = __fma_rn (u, a2, -1.388888888888074E-003);
  u = __fma_rn (u, a2,  4.166666666666664E-002);
  u = __fma_rn (u, a2, -5.000000000000000E-001);
  u = __fma_rn (u, a2,  1.000000000000000E+000);
  t =                   1.5896230157221844E-010;
  t = __fma_rn (t, a2, -2.5050747762850355E-008);
  t = __fma_rn (t, a2,  2.7557313621385676E-006);
  t = __fma_rn (t, a2, -1.9841269829589539E-004);
  t = __fma_rn (t, a2,  8.3333333333221182E-003);
  t = __fma_rn (t, a2, -1.6666666666666630E-001);
  t = t * a2;
  t = __fma_rn(t, a, a);
  if (i & 1) {
    s = u;
    c = t;
  } else {
    s = t;
    c = u;
  }
  if (i & 2) {
    s = -s;
  }
  i++;
  if (i & 2) {
    c = -c;
  }
  *sptr = s;
  *cptr = c;
}

struct Energy {
    PMEDouble bond;
    PMEDouble angle;
    PMEDouble dihedral;
    PMEDouble el14;
    PMEDouble nb14;
    PMEDouble restraint;
};

struct Virial {
    PMEDouble vir_11;
    PMEDouble vir_22;
    PMEDouble vir_33;
};

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_LOCALFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_LOCALFORCES_THREADS_PER_BLOCK, 1)
#endif
kCalculateLocalForces_kernel()
#include "kCLF.h"

#define LOCAL_ENERGY
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_LOCALFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_LOCALFORCES_THREADS_PER_BLOCK, 1)
#endif
kCalculateLocalEnergy_kernel()
#include "kCLF.h"
#undef LOCAL_ENERGY

#define LOCAL_NEIGHBORLIST
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_LOCALFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_LOCALFORCES_THREADS_PER_BLOCK, 1)
#endif
kCalculatePMELocalForces_kernel()
#include "kCLF.h"

#define LOCAL_ENERGY
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_LOCALFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_LOCALFORCES_THREADS_PER_BLOCK, 1)
#endif
kCalculatePMELocalEnergy_kernel()
#include "kCLF.h"
#undef LOCAL_ENERGY

#define LOCAL_VIRIAL
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_LOCALFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_LOCALFORCES_THREADS_PER_BLOCK, 1)
#endif
kCalculatePMELocalForcesVirial_kernel()
#include "kCLF.h"

#define LOCAL_ENERGY
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_LOCALFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_LOCALFORCES_THREADS_PER_BLOCK, 1)
#endif
kCalculatePMELocalEnergyVirial_kernel()
#include "kCLF.h"
#undef LOCAL_ENERGY
#undef LOCAL_VIRIAL
#undef LOCAL_NEIGHBORLIST

// Consumer Fermi kernels
#define NODPTEXTURE
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_LOCALFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_LOCALFORCES_THREADS_PER_BLOCK, 1)
#endif
kCalculateLocalForcesFermi_kernel()
#include "kCLF.h"

#define LOCAL_ENERGY
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_LOCALFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_LOCALFORCES_THREADS_PER_BLOCK, 1)
#endif
kCalculateLocalEnergyFermi_kernel()
#include "kCLF.h"
#undef LOCAL_ENERGY

#define LOCAL_NEIGHBORLIST
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_LOCALFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_LOCALFORCES_THREADS_PER_BLOCK, 1)
#endif
kCalculatePMELocalForcesFermi_kernel()
#include "kCLF.h"

#define LOCAL_ENERGY
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_LOCALFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_LOCALFORCES_THREADS_PER_BLOCK, 1)
#endif
kCalculatePMELocalEnergyFermi_kernel()
#include "kCLF.h"
#undef LOCAL_ENERGY

#define LOCAL_VIRIAL
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_LOCALFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_LOCALFORCES_THREADS_PER_BLOCK, 1)
#endif
kCalculatePMELocalForcesVirialFermi_kernel()
#include "kCLF.h"

#define LOCAL_ENERGY
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_LOCALFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_LOCALFORCES_THREADS_PER_BLOCK, 1)
#endif
kCalculatePMELocalEnergyVirialFermi_kernel()
#include "kCLF.h"
#undef LOCAL_ENERGY
#undef LOCAL_VIRIAL
#undef LOCAL_NEIGHBORLIST
#undef NODPTEXTURE


extern "C" void kCalculateLocalForces(gpuContext gpu)
{
    if (gpu->bLocalInteractions)
    {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kCalculateLocalForces_kernel), hipFuncCachePreferL1);
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kCalculatePMELocalForces_kernel), hipFuncCachePreferL1);
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kCalculatePMELocalForcesVirial_kernel), hipFuncCachePreferL1);

#ifndef use_SPSP        
        // Pathetic kludge to workaround GTX4xx/GTX5xx HW bug
        if (!gpu->bECCSupport && (gpu->sm_version == SM_2X))
        {
            if (gpu->bNeighborList)
            {
                if (gpu->sim.ntp > 0)
                    kCalculatePMELocalForcesVirialFermi_kernel<<<gpu->blocks, gpu->localForcesThreadsPerBlock>>>();
                else    
                    kCalculatePMELocalForcesFermi_kernel<<<gpu->blocks, gpu->localForcesThreadsPerBlock>>>();
            }
            else
                kCalculateLocalForcesFermi_kernel<<<gpu->blocks, gpu->localForcesThreadsPerBlock>>>();
#ifdef MPI 
            LAUNCHERROR_NONBLOCKING("kCalculateLocalForcesFermi");
#else
            LAUNCHERROR("kCalculateLocalForcesFermi");
#endif          
        }
        else
#endif
        {      
            texref.normalized       = 0;
            texref.filterMode       = hipFilterModePoint;
            texref.addressMode[0]   = hipAddressModeClamp;
            texref.channelDesc.x    = 32;
#ifndef use_SPSP        
            texref.channelDesc.y    = 32;
#else
            texref.channelDesc.y    = 0;
#endif        
            texref.channelDesc.z    = 0;
            texref.channelDesc.w    = 0;
#ifndef use_SPSP       
            int2* pX;
            if (gpu->bNeighborList)
                pX                  = (int2*)gpu->sim.pImageX;
            else
                pX                  = (int2*)gpu->sim.pAtomX;
            hipBindTexture(NULL, texref, pX, gpu->sim.stride3 * sizeof(int2));
#else
            float* pX;
            if (gpu->bNeighborList)
                pX                  = (float*)gpu->sim.pImageX;
            else
                pX                  = (float*)gpu->sim.pAtomX;
            hipBindTexture(NULL, texref, pX, gpu->sim.stride3 * sizeof(float));
#endif            
            if (gpu->bNeighborList)
            {
                if (gpu->sim.ntp > 0)
                    kCalculatePMELocalForcesVirial_kernel<<<gpu->blocks, gpu->localForcesThreadsPerBlock>>>();
                else    
                    kCalculatePMELocalForces_kernel<<<gpu->blocks, gpu->localForcesThreadsPerBlock>>>();
            }
            else
                kCalculateLocalForces_kernel<<<gpu->blocks, gpu->localForcesThreadsPerBlock>>>();
#ifdef MPI 
            LAUNCHERROR_NONBLOCKING("kCalculateLocalForces");
#else
            LAUNCHERROR("kCalculateLocalForces");
#endif  
            hipUnbindTexture(texref);
        }
    }
}

extern "C" void kCalculateLocalEnergy(gpuContext gpu)
{
    if (gpu->bLocalInteractions)
    {
#ifndef use_SPSP    
        if (!gpu->bECCSupport && (gpu->sm_version == SM_2X))
        {
            if (gpu->bNeighborList)
            {
                if (gpu->sim.ntp > 0)
                    kCalculatePMELocalEnergyVirialFermi_kernel<<<gpu->blocks, gpu->localForcesThreadsPerBlock>>>();
                else    
                    kCalculatePMELocalEnergyFermi_kernel<<<gpu->blocks, gpu->localForcesThreadsPerBlock>>>();
            }
            else
                kCalculateLocalEnergyFermi_kernel<<<gpu->blocks, gpu->localForcesThreadsPerBlock>>>();
#ifdef MPI 
            LAUNCHERROR_NONBLOCKING("kCalculateLocalEnergyFermi");
#else
            LAUNCHERROR("kCalculateLocalEnergyFermi");
#endif          
        }
        else
#endif
        {
            texref.normalized       = 0;
            texref.filterMode       = hipFilterModePoint;
            texref.addressMode[0]   = hipAddressModeClamp;
            texref.channelDesc.x    = 32;
#ifndef use_SPSP        
            texref.channelDesc.y    = 32;
#else
            texref.channelDesc.y    = 0;
#endif        
            texref.channelDesc.z    = 0;
            texref.channelDesc.w    = 0;
        
#ifndef use_SPSP       
            int2* pX;
            if (gpu->bNeighborList)
                pX                  = (int2*)gpu->sim.pImageX;
            else
                pX                  = (int2*)gpu->sim.pAtomX;
            hipBindTexture(NULL, texref, pX, gpu->sim.stride3 * sizeof(int2));
#else
            float* pX;
            if (gpu->bNeighborList)
                pX                  = (float*)gpu->sim.pImageX;
            else
                pX                  = (float*)gpu->sim.pAtomX;
            hipBindTexture(NULL, texref, pX, gpu->sim.stride3 * sizeof(float));
#endif        
            if (gpu->bNeighborList)
            {
                if (gpu->sim.ntp > 0)
                    kCalculatePMELocalEnergyVirial_kernel<<<gpu->blocks, gpu->localForcesThreadsPerBlock>>>();
                else    
                    kCalculatePMELocalEnergy_kernel<<<gpu->blocks, gpu->localForcesThreadsPerBlock>>>();
            }
            else
                kCalculateLocalEnergy_kernel<<<gpu->blocks, gpu->localForcesThreadsPerBlock>>>();
#ifdef MPI 
            LAUNCHERROR_NONBLOCKING("kCalculateLocalEnergy");
#else
            LAUNCHERROR("kCalculateLocalEnergy");
#endif 
            hipUnbindTexture(texref);
        }
    }
}



#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_CHARMMFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_CHARMMFORCES_THREADS_PER_BLOCK, 1)
#endif
__global__ void kCalculateCHARMMForces_kernel()
#include "kCCF.h"

#define CHARMM_ENERGY
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_CHARMMFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_CHARMMFORCES_THREADS_PER_BLOCK, 1)
#endif
__global__ void kCalculateCHARMMEnergy_kernel()
#include "kCCF.h"
#undef CHARMM_ENERGY

#define CHARMM_NEIGHBORLIST
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_CHARMMFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_CHARMMFORCES_THREADS_PER_BLOCK, 1)
#endif
__global__ void kCalculatePMECHARMMForces_kernel()
#include "kCCF.h"

#define CHARMM_ENERGY
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_CHARMMFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_CHARMMFORCES_THREADS_PER_BLOCK, 1)
#endif
__global__ void kCalculatePMECHARMMEnergy_kernel()
#include "kCCF.h"
#undef CHARMM_ENERGY

#define CHARMM_VIRIAL
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_CHARMMFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_CHARMMFORCES_THREADS_PER_BLOCK, 1)
#endif
__global__ void kCalculatePMECHARMMForcesVirial_kernel()
#include "kCCF.h"

#define CHARMM_ENERGY
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_CHARMMFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_CHARMMFORCES_THREADS_PER_BLOCK, 1)
#endif
__global__ void kCalculatePMECHARMMEnergyVirial_kernel()
#include "kCCF.h"
#undef CHARMM_ENERGY
#undef CHARMM_VIRIAL
#undef CHARMM_NEIGHBORLIST


#define NODPTEXTURE
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_CHARMMFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_CHARMMFORCES_THREADS_PER_BLOCK, 1)
#endif
__global__ void kCalculateCHARMMForcesFermi_kernel()
#include "kCCF.h"

#define CHARMM_ENERGY
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_CHARMMFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_CHARMMFORCES_THREADS_PER_BLOCK, 1)
#endif
__global__ void kCalculateCHARMMEnergyFermi_kernel()
#include "kCCF.h"
#undef CHARMM_ENERGY

#define CHARMM_NEIGHBORLIST
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_CHARMMFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_CHARMMFORCES_THREADS_PER_BLOCK, 1)
#endif
__global__ void kCalculatePMECHARMMForcesFermi_kernel()
#include "kCCF.h"

#define CHARMM_ENERGY
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_CHARMMFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_CHARMMFORCES_THREADS_PER_BLOCK, 1)
#endif
__global__ void kCalculatePMECHARMMEnergyFermi_kernel()
#include "kCCF.h"
#undef CHARMM_ENERGY

#define CHARMM_VIRIAL
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_CHARMMFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_CHARMMFORCES_THREADS_PER_BLOCK, 1)
#endif
__global__ void kCalculatePMECHARMMForcesVirialFermi_kernel()
#include "kCCF.h"

#define CHARMM_ENERGY
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_CHARMMFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_CHARMMFORCES_THREADS_PER_BLOCK, 1)
#endif
__global__ void kCalculatePMECHARMMEnergyVirialFermi_kernel()
#include "kCCF.h"
#undef CHARMM_ENERGY
#undef CHARMM_NEIGHBORLIST
#undef NODPTEXTURE



extern "C" void kCalculateCHARMMForces(gpuContext gpu)
{
    if (gpu->bCharmmInteractions)
    {
#ifndef use_SPSP
        if (!gpu->bECCSupport && (gpu->sm_version == SM_2X))
        {
            if (gpu->bNeighborList)
            {
                if (gpu->sim.ntp > 0)
                    kCalculatePMECHARMMForcesVirialFermi_kernel<<<gpu->blocks, gpu->CHARMMForcesThreadsPerBlock>>>();
                else    
                    kCalculatePMECHARMMForcesFermi_kernel<<<gpu->blocks, gpu->CHARMMForcesThreadsPerBlock>>>();
            }
            else
                kCalculateCHARMMForcesFermi_kernel<<<gpu->blocks, gpu->CHARMMForcesThreadsPerBlock>>>();
#ifdef MPI 
            LAUNCHERROR_NONBLOCKING("kCalculateCHARMMForcesFermi");
#else
            LAUNCHERROR("kCalculateCHARMMForcesFermi");
#endif         
        }
        else
#endif
        {
            texref.normalized       = 0;
            texref.filterMode       = hipFilterModePoint;
            texref.addressMode[0]   = hipAddressModeClamp;
            texref.channelDesc.x    = 32;
#ifndef use_SPSP        
            texref.channelDesc.y    = 32;
#else
            texref.channelDesc.y    = 0;
#endif        
            texref.channelDesc.z    = 0;
            texref.channelDesc.w    = 0;
#ifndef use_SPSP       
            int2* pX;
            if (gpu->bNeighborList)
                pX                  = (int2*)gpu->sim.pImageX;
            else
                pX                  = (int2*)gpu->sim.pAtomX;
            hipBindTexture(NULL, texref, pX, gpu->sim.stride3 * sizeof(int2));
#else
            float* pX;
            if (gpu->bNeighborList)
                pX                  = (float*)gpu->sim.pImageX;
            else
                pX                  = (float*)gpu->sim.pAtomX;
            hipBindTexture(NULL, texref, pX, gpu->sim.stride3 * sizeof(float));
#endif            
            if (gpu->bNeighborList)
            {
                if (gpu->sim.ntp > 0)
                    kCalculatePMECHARMMForcesVirial_kernel<<<gpu->blocks, gpu->CHARMMForcesThreadsPerBlock>>>();
                else
                    kCalculatePMECHARMMForces_kernel<<<gpu->blocks, gpu->CHARMMForcesThreadsPerBlock>>>();
            }
            else
                kCalculateCHARMMForces_kernel<<<gpu->blocks, gpu->CHARMMForcesThreadsPerBlock>>>();
#ifdef MPI 
            LAUNCHERROR_NONBLOCKING("kCalculateCHARMMForces");
#else
            LAUNCHERROR("kCalculateCHARMMForces");
#endif 
            hipUnbindTexture(texref);
        }
    }
}

extern "C" void kCalculateCHARMMEnergy(gpuContext gpu)
{    
    if (gpu->bCharmmInteractions)
    {
#ifndef use_SPSP   
        if (!gpu->bECCSupport && (gpu->sm_version == SM_2X))
        {
            if (gpu->bNeighborList)
            {
                if (gpu->sim.ntp > 0)
                    kCalculatePMECHARMMEnergyVirialFermi_kernel<<<gpu->blocks, gpu->CHARMMForcesThreadsPerBlock>>>();
                else
                    kCalculatePMECHARMMEnergyFermi_kernel<<<gpu->blocks, gpu->CHARMMForcesThreadsPerBlock>>>();
            }
            else
                kCalculateCHARMMEnergyFermi_kernel<<<gpu->blocks, gpu->CHARMMForcesThreadsPerBlock>>>();
#ifdef MPI 
            LAUNCHERROR_NONBLOCKING("kCalculateCHARMMEnergyFermi");
#else
            LAUNCHERROR("kCalculateCHARMMEnergyFermi");
#endif         
        }
        else
#endif
        {    
            texref.normalized       = 0;
            texref.filterMode       = hipFilterModePoint;
            texref.addressMode[0]   = hipAddressModeClamp;
            texref.channelDesc.x    = 32;
#ifndef use_SPSP        
            texref.channelDesc.y    = 32;
#else
            texref.channelDesc.y    = 0;
#endif        
            texref.channelDesc.z    = 0;
            texref.channelDesc.w    = 0;
        
#ifndef use_SPSP       
            int2* pX;
            if (gpu->bNeighborList)
                pX                  = (int2*)gpu->sim.pImageX;
            else
                pX                  = (int2*)gpu->sim.pAtomX;
            hipBindTexture(NULL, texref, pX, gpu->sim.stride3 * sizeof(int2));
#else
            float* pX;
            if (gpu->bNeighborList)
                pX                  = (float*)gpu->sim.pImageX;
            else
                pX                  = (float*)gpu->sim.pAtomX;
            hipBindTexture(NULL, texref, pX, gpu->sim.stride3 * sizeof(float));
#endif        
            if (gpu->bNeighborList)
            {
                if (gpu->sim.ntp > 0)
                    kCalculatePMECHARMMEnergyVirial_kernel<<<gpu->blocks, gpu->CHARMMForcesThreadsPerBlock>>>();
                else
                    kCalculatePMECHARMMEnergy_kernel<<<gpu->blocks, gpu->CHARMMForcesThreadsPerBlock>>>();
            }
            else
                kCalculateCHARMMEnergy_kernel<<<gpu->blocks, gpu->CHARMMForcesThreadsPerBlock>>>();
#ifdef MPI 
            LAUNCHERROR_NONBLOCKING("kCalculateCHARMMEnergy");
#else
            LAUNCHERROR("kCalculateCHARMMEnergy");
#endif 
            hipUnbindTexture(texref);
        }
    }
}

