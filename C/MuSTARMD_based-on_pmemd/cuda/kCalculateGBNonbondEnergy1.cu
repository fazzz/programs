#include "hip/hip_runtime.h"
/***************************************************/
/*                                                 */
/*      AMBER NVIDIA CUDA CPU IMPLEMENTATION       */
/*                 PMEMD VERSION                   */
/*                     2010                        */
/*                      by                         */
/*             Scott Le Grand (NVIDIA)             */
/*               Duncan Poole (NVIDIA)             */
/*                Ross Walker (SDSC)               */
/*                                                 */
/***************************************************/

#include <hip/hip_runtime.h>
#include "gpu.h"
static __constant__ cudaSimulation cSim;

void SetkCalculateGBNonbondEnergy1Sim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

void GetkCalculateGBNonBondEnergy1Sim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_GBNONBONDENERGY1_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_GBNONBONDENERGY1_THREADS_PER_BLOCK, 1)
#endif
kCalculateGBNonbondForces1_kernel()
#include "kCalculateGBNonbondEnergy1.h"

extern "C" void kCalculateGBNonbondForces1(gpuContext gpu)
{
    // Refresh texture if necessary
    
    kCalculateGBNonbondForces1_kernel<<<gpu->blocks, gpu->GBNonbondEnergy1ThreadsPerBlock>>>();   
    LAUNCHERROR("kCalculateGBNonbondForces1");
}

#define GB_ENERGY
__global__ 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_GBNONBONDENERGY1_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_GBNONBONDENERGY1_THREADS_PER_BLOCK, 1)
#endif
void kCalculateGBNonbondEnergy1_kernel()
#include "kCalculateGBNonbondEnergy1.h"
#undef GB_ENERGY


extern "C" void kCalculateGBNonbondEnergy1(gpuContext gpu)
{
    // Refresh texture if necessary
  
    kCalculateGBNonbondEnergy1_kernel<<<gpu->blocks, gpu->GBNonbondEnergy1ThreadsPerBlock>>>();   
    LAUNCHERROR("kCalculateGBNonbondEnergy1");
}

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_REDUCEBUFFER_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_REDUCEBUFFER_THREADS_PER_BLOCK, 1)
#endif
kReduceGBTemp7_kernel()
{
#ifndef MPI
    bool bIGB2578                                       = (cSim.igb == 2) || (cSim.igb == 5) || (cSim.igb == 7) || (cSim.igb == 8);    
#endif
    unsigned int pos                                    = blockIdx.x * blockDim.x + threadIdx.x;   
    while (pos < cSim.atoms)
    {
        unsigned int pos1                               = pos;
#ifndef MPI        
        PMEFloat reff_i                                 = cSim.pReffSP[pos];
        PMEFloat psi_i                                  = cSim.pPsi[pos];
        PMEFloat rborn_i                                = cSim.pAtomRBorn[pos];
        PMEFloat qi                                     = cSim.pAtomChargeSP[pos];
#endif        
        PMEDouble sumdeijda_i                           = (PMEDouble)0.0;
        while (pos1 < cSim.stride * cSim.nonbondForceBuffers)
        {
            sumdeijda_i                                += cSim.pSumdeijdaBuffer[pos1];
            pos1                                       += cSim.stride;
        }       
#ifndef MPI        
        // Process Temp7 component       
        PMEFloat expmkf                                = exp(-cSim.gb_kappa * reff_i) * cSim.extdiel_inv;
        PMEFloat dl                                    = cSim.intdiel_inv - expmkf;
        PMEFloat qi2h                                  = (PMEFloat)0.50 * qi * qi;
        PMEFloat qid2h                                 = qi2h * dl;
        sumdeijda_i                                    = -sumdeijda_i + qid2h - cSim.gb_kappa * qi2h * expmkf * reff_i;
        if (cSim.alpb == 0)
        {
            // egb                                        -= qid2h / reff_i;
           
        }
        else
        {
            // egb                                        -= qid2h * (1.0 / reff_i + cSim.one_arad_beta);
            sumdeijda_i                                *= ((PMEFloat)1.0 + cSim.one_arad_beta * reff_i);
        }
         
        if (bIGB2578)
        {
            
            // new onufriev: we have to later scale values by a
            //               alpha,beta,gamma -dependent factor:           
            PMEFloat thi, thi2;
            if (cSim.igb == 8)
            {
                PMEFloat alpha                          = cSim.pgb_alpha[pos];
                PMEFloat gamma                          = cSim.pgb_gamma[pos];
                PMEFloat beta                           = cSim.pgb_beta[pos];
                thi                                     = tanh((alpha + gamma * psi_i * psi_i - beta * psi_i) * psi_i);
                thi2                                    = (alpha + (PMEFloat)3.0 * gamma * psi_i * psi_i - (PMEFloat)2.0 * beta * psi_i) * ((PMEFloat)1.0 - thi * thi) * (rborn_i - cSim.offset) / rborn_i;
            }
            else
            {
                thi                                     = tanh((cSim.gb_alpha + cSim.gb_gamma * psi_i * psi_i - cSim.gb_beta * psi_i) * psi_i);
                thi2                                    = (cSim.gb_alpha + (PMEFloat)3.0 * cSim.gb_gamma * psi_i * psi_i - (PMEFloat)2.0 * cSim.gb_beta * psi_i) * ((PMEFloat)1.0 - thi * thi) * (rborn_i - cSim.offset) / rborn_i;
            }
            sumdeijda_i                                *= thi2;
        }
        cSim.pTemp7[pos]                                = sumdeijda_i;
#else
        cSim.pTemp7a[pos]                               = sumdeijda_i;
#endif
        pos                                            += blockDim.x * gridDim.x;
    }
}

extern "C" void kReduceGBTemp7(gpuContext gpu)
{
    kReduceGBTemp7_kernel<<<gpu->blocks, gpu->reduceBufferThreadsPerBlock>>>();
    LAUNCHERROR("kReduceGBTemp7");
}

#ifdef MPI
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_REDUCEBUFFER_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_REDUCEBUFFER_THREADS_PER_BLOCK, 1)
#endif
kProcessGBTemp7_kernel()
{
    bool bIGB2578                                       = (cSim.igb == 2) || (cSim.igb == 5) || (cSim.igb == 7) || (cSim.igb == 8);    
    unsigned int pos                                    = blockIdx.x * blockDim.x + threadIdx.x;   
    while (pos < cSim.atoms)
    {
        PMEFloat reff_i                                 = cSim.pReffSP[pos];
        PMEFloat psi_i                                  = cSim.pPsi[pos];
        PMEFloat rborn_i                                = cSim.pAtomRBorn[pos];
        PMEFloat qi                                     = cSim.pAtomChargeSP[pos];
        PMEDouble sumdeijda_i                           = cSim.pTemp7a[pos];

        // Process Temp7 component       
        PMEFloat expmkf                                = exp(-cSim.gb_kappa * reff_i) * cSim.extdiel_inv;
        PMEFloat dl                                    = cSim.intdiel_inv - expmkf;
        PMEFloat qi2h                                  = (PMEFloat)0.50 * qi * qi;
        PMEFloat qid2h                                 = qi2h * dl;
        sumdeijda_i                                    = -sumdeijda_i + qid2h - cSim.gb_kappa * qi2h * expmkf * reff_i;
        if (cSim.alpb == 0)
        {
            // egb                                        -= qid2h / reff_i;
           
        }
        else
        {
            // egb                                        -= qid2h * (1.0 / reff_i + cSim.one_arad_beta);
            sumdeijda_i                                *= ((PMEFloat)1.0 + cSim.one_arad_beta * reff_i);
        }
         
        if (bIGB2578)
        {
            
            // new onufriev: we have to later scale values by a
            //               alpha,beta,gamma -dependent factor:
            PMEFloat thi, thi2;
            if (cSim.igb == 8)
            {
                PMEFloat alpha                          = cSim.pgb_alpha[pos];
                PMEFloat gamma                          = cSim.pgb_gamma[pos];
                PMEFloat beta                           = cSim.pgb_beta[pos];
                thi                                     = tanh((alpha + gamma * psi_i * psi_i - beta * psi_i) * psi_i);
                thi2                                    = (alpha + (PMEFloat)3.0 * gamma * psi_i * psi_i - (PMEFloat)2.0 * beta * psi_i) * ((PMEFloat)1.0 - thi * thi) * (rborn_i - cSim.offset) / rborn_i;
            }
            else
            {
                thi                                     = tanh((cSim.gb_alpha + cSim.gb_gamma * psi_i * psi_i - cSim.gb_beta * psi_i) * psi_i);
                thi2                                    = (cSim.gb_alpha + (PMEFloat)3.0 * cSim.gb_gamma * psi_i * psi_i - (PMEFloat)2.0 * cSim.gb_beta * psi_i) * ((PMEFloat)1.0 - thi * thi) * (rborn_i - cSim.offset) / rborn_i;
            }
            sumdeijda_i                                *= thi2;
        }
      
        cSim.pTemp7[pos]                                = sumdeijda_i;
        pos                                            += blockDim.x * gridDim.x;
    }
}

extern "C" void kProcessGBTemp7(gpuContext gpu)
{
    kProcessGBTemp7_kernel<<<gpu->blocks, gpu->reduceBufferThreadsPerBlock>>>();
    LAUNCHERROR("kProcessGBTemp7");
}
#endif


__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_REDUCEBUFFER_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_REDUCEBUFFER_THREADS_PER_BLOCK, 1)
#endif
kReduceGBTemp7Energy_kernel()
{
#ifndef MPI
    volatile __shared__ PMEDouble sE[1024];
    bool bIGB2578                                       = (cSim.igb == 2) || (cSim.igb == 5) || (cSim.igb == 7) || (cSim.igb == 8);    
#endif
    unsigned int pos                                    = blockIdx.x * blockDim.x + threadIdx.x; 
#ifndef MPI
    PMEDouble egb                                       = (PMEDouble)0.0;  
#endif

    while (pos < cSim.atoms)
    {
        unsigned int pos1                               = pos;
#ifndef MPI
        PMEFloat reff_i                                 = cSim.pReffSP[pos];
        PMEFloat psi_i                                  = cSim.pPsi[pos];
        PMEFloat rborn_i                                = cSim.pAtomRBorn[pos];
        PMEFloat qi                                     = cSim.pAtomChargeSP[pos];
#endif        
        PMEDouble sumdeijda_i                           = (PMEDouble)0.0;
        while (pos1 < cSim.stride * cSim.nonbondForceBuffers)
        {
            sumdeijda_i                                += cSim.pSumdeijdaBuffer[pos1];
            pos1                                       += cSim.stride;
        }       
#ifndef MPI        
        // Process Temp7 component       
        PMEFloat expmkf                                = exp(-cSim.gb_kappa * reff_i) * cSim.extdiel_inv;
        PMEFloat dl                                    = cSim.intdiel_inv - expmkf;
        PMEFloat qi2h                                  = (PMEFloat)0.50 * qi * qi;
        PMEFloat qid2h                                 = qi2h * dl;
        sumdeijda_i                                    = -sumdeijda_i + qid2h - cSim.gb_kappa * qi2h * expmkf * reff_i;
        if (cSim.alpb == 0)
        {
           egb                                        -= qid2h / reff_i;
        }
        else
        {
            egb                                        -= qid2h * ((PMEFloat)1.0 / reff_i + cSim.one_arad_beta);
            sumdeijda_i                                *= ((PMEFloat)1.0 + cSim.one_arad_beta * reff_i);
        }
         
        if (bIGB2578)
        {
            
            // new onufriev: we have to later scale values by a
            //               alpha,beta,gamma-dependent factor:
            PMEFloat thi, thi2;
            if (cSim.igb == 8)
            {
                PMEFloat alpha                          = cSim.pgb_alpha[pos];
                PMEFloat gamma                          = cSim.pgb_gamma[pos];
                PMEFloat beta                           = cSim.pgb_beta[pos];
                thi                                     = tanh((alpha + gamma * psi_i * psi_i - beta * psi_i) * psi_i);
                thi2                                    = (alpha + (PMEFloat)3.0 * gamma * psi_i * psi_i - (PMEFloat)2.0 * beta * psi_i) * ((PMEFloat)1.0 - thi * thi) * (rborn_i - cSim.offset) / rborn_i;
            }
            else
            {
                thi                                     = tanh((cSim.gb_alpha + cSim.gb_gamma * psi_i * psi_i - cSim.gb_beta * psi_i) * psi_i);
                thi2                                    = (cSim.gb_alpha + (PMEFloat)3.0 * cSim.gb_gamma * psi_i * psi_i - (PMEFloat)2.0 * cSim.gb_beta * psi_i) * ((PMEFloat)1.0 - thi * thi) * (rborn_i - cSim.offset) / rborn_i;
            }
            sumdeijda_i                                *= thi2;
        }
        cSim.pTemp7[pos]                                = sumdeijda_i;
#else     
        cSim.pTemp7a[pos]                               = sumdeijda_i;
#endif
        pos                                            += blockDim.x * gridDim.x;
    }
    
#ifndef MPI    
    // Reduce Generalized Born energy
    sE[threadIdx.x]                                     = egb;
    __syncthreads();
    unsigned int m                                      = 1;
    while (m < blockDim.x)
    {
        int p                                           = threadIdx.x + m;
        PMEDouble d                                     = ((p < blockDim.x) ? sE[p] : (PMEDouble)0.0);
        __syncthreads();
        sE[threadIdx.x]                                += d;
        __syncthreads();
        m                                              *= 2;
    }
    egb                                                 = sE[threadIdx.x];
    unsigned long long int val                          = (unsigned long long int)(fabs(egb) * ENERGYSCALE + (PMEDouble)0.5);
    if (egb < (PMEDouble)0.0)
        val                                             = 0ull - val;
    if (threadIdx.x == 0)
        atomicAdd(cSim.pEGB, val);  
#endif
}


extern "C" void kReduceGBTemp7Energy(gpuContext gpu)
{
    kReduceGBTemp7Energy_kernel<<<gpu->blocks, gpu->reduceBufferThreadsPerBlock>>>();
    LAUNCHERROR("kReduceGBTemp7Energy");
}

#ifdef MPI
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_REDUCEBUFFER_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_REDUCEBUFFER_THREADS_PER_BLOCK, 1)
#endif
kProcessGBTemp7Energy_kernel()
{
    volatile __shared__ PMEDouble sE[1024];
    bool bIGB2578                                       = (cSim.igb == 2) || (cSim.igb == 5) || (cSim.igb == 7) || (cSim.igb == 8);    
    unsigned int pos                                    = blockIdx.x * blockDim.x + threadIdx.x; 
    PMEDouble egb                                       = (PMEDouble)0.0;  
    while (pos < cSim.atoms)
    {
        PMEFloat reff_i                                 = cSim.pReffSP[pos];
        PMEFloat psi_i                                  = cSim.pPsi[pos];
        PMEFloat rborn_i                                = cSim.pAtomRBorn[pos];
        PMEFloat qi                                     = cSim.pAtomChargeSP[pos];        
        PMEDouble sumdeijda_i                           = cSim.pTemp7a[pos];
      
        // Process Temp7 component       
        PMEFloat expmkf                                = exp(-cSim.gb_kappa * reff_i) * cSim.extdiel_inv;
        PMEFloat dl                                    = cSim.intdiel_inv - expmkf;
        PMEFloat qi2h                                  = (PMEFloat)0.50 * qi * qi;
        PMEFloat qid2h                                 = qi2h * dl;
        sumdeijda_i                                    = -sumdeijda_i + qid2h - cSim.gb_kappa * qi2h * expmkf * reff_i;
        if (cSim.alpb == 0)
        {
           egb                                        -= qid2h / reff_i;
           
        }
        else
        {
            egb                                        -= qid2h * ((PMEFloat)1.0 / reff_i + cSim.one_arad_beta);
            sumdeijda_i                                *= ((PMEFloat)1.0 + cSim.one_arad_beta * reff_i);
        }
         
        if (bIGB2578)
        {
            
            // new onufriev: we have to later scale values by a
            //               alpha,beta,gamma -dependent factor:          
            PMEFloat thi, thi2;
            if (cSim.igb == 8)
            {
                PMEFloat alpha                          = cSim.pgb_alpha[pos];
                PMEFloat gamma                          = cSim.pgb_gamma[pos];
                PMEFloat beta                           = cSim.pgb_beta[pos];
                thi                                     = tanh((alpha + gamma * psi_i * psi_i - beta * psi_i) * psi_i);
                thi2                                    = (alpha + (PMEFloat)3.0 * gamma * psi_i * psi_i - (PMEFloat)2.0 * beta * psi_i) * ((PMEFloat)1.0 - thi * thi) * (rborn_i - cSim.offset) / rborn_i;
            }
            else
            {
                thi                                     = tanh((cSim.gb_alpha + cSim.gb_gamma * psi_i * psi_i - cSim.gb_beta * psi_i) * psi_i);
                thi2                                    = (cSim.gb_alpha + (PMEFloat)3.0 * cSim.gb_gamma * psi_i * psi_i - (PMEFloat)2.0 * cSim.gb_beta * psi_i) * ((PMEFloat)1.0 - thi * thi) * (rborn_i - cSim.offset) / rborn_i;
            }

            sumdeijda_i                                *= thi2;
        }      
        cSim.pTemp7[pos]                                = sumdeijda_i;
        pos                                            += blockDim.x * gridDim.x;
    }
   
    // Reduce Generalized Born energy
    sE[threadIdx.x]                                     = egb;
    __syncthreads();
    unsigned int m                                      = 1;
    while (m < blockDim.x)
    {
        int p                                           = threadIdx.x + m;
        PMEDouble d                                     = ((p < blockDim.x) ? sE[p] : (PMEDouble)0.0);
        __syncthreads();
        sE[threadIdx.x]                                += d;
        __syncthreads();
        m                                              *= 2;
    }
    egb                                                 = sE[threadIdx.x];
    unsigned long long int val                          = (unsigned long long int)(fabs(egb) * ENERGYSCALE + (PMEDouble)0.5);
    if (egb < (PMEDouble)0.0)
        val                                             = 0ull - val;
    if (threadIdx.x == 0)
        atomicAdd(cSim.pEGB, val);  
}


extern "C" void kProcessGBTemp7Energy(gpuContext gpu)
{
    kProcessGBTemp7Energy_kernel<<<gpu->blocks, gpu->reduceBufferThreadsPerBlock>>>();
    LAUNCHERROR("kProcessGBTemp7Energy");
}
#endif

