#include "hip/hip_runtime.h"
/***************************************************/
/*                                                 */
/*      AMBER NVIDIA CUDA CPU IMPLEMENTATION       */
/*                 PMEMD VERSION                   */
/*                     2010                        */
/*                      by                         */
/*             Scott Le Grand (NVIDIA)             */
/*               Duncan Poole (NVIDIA)             */
/*                Ross Walker (SDSC)               */
/*                                                 */
/***************************************************/

#include <hip/hip_runtime.h>
#include "gpu.h"
static __constant__ cudaSimulation cSim;

void SetkPMEInterpolationSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(cudaSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

void GetkPMEInterpolationSim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(cudaSimulation));     
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_UPDATE_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_UPDATE_THREADS_PER_BLOCK, 1)
#endif
kPMEGetGridWeights_kernel()
#include "kPGGW.h"

#define PME_ORTHOGONAL
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_UPDATE_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_UPDATE_THREADS_PER_BLOCK, 1)
#endif
kPMEGetGridWeightsOrthogonal_kernel()
#include "kPGGW.h"
#undef PME_ORTHOGONAL

#define PME_NTP
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_UPDATE_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_UPDATE_THREADS_PER_BLOCK, 1)
#endif
kPMEGetGridWeightsNTP_kernel()
#include "kPGGW.h"

#define PME_ORTHOGONAL
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_UPDATE_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_UPDATE_THREADS_PER_BLOCK, 1)
#endif
kPMEGetGridWeightsOrthogonalNTP_kernel()
#include "kPGGW.h"
#undef PME_ORTHOGONAL
#undef PME_NTP

#define PME_SMALLBOX
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_UPDATE_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_UPDATE_THREADS_PER_BLOCK, 1)
#endif
kPMEGetGridWeightsSmall_kernel()
#include "kPGGW.h"

#define PME_ORTHOGONAL
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_UPDATE_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_UPDATE_THREADS_PER_BLOCK, 1)
#endif
kPMEGetGridWeightsSmallOrthogonal_kernel()
#include "kPGGW.h"
#undef PME_ORTHOGONAL

#define PME_NTP
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_UPDATE_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_UPDATE_THREADS_PER_BLOCK, 1)
#endif
kPMEGetGridWeightsSmallNTP_kernel()
#include "kPGGW.h"

#define PME_ORTHOGONAL
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_UPDATE_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_UPDATE_THREADS_PER_BLOCK, 1)
#endif
kPMEGetGridWeightsSmallOrthogonalNTP_kernel()
#include "kPGGW.h"
#undef PME_ORTHOGONAL
#undef PME_NTP
#undef PME_SMALLBOX


extern "C" void kPMEGetGridWeights(gpuContext gpu)
{

    if (gpu->bSmallBox)
    {
        if (gpu->sim.ntp > 0)
        {
            if (gpu->sim.is_orthog)    
                kPMEGetGridWeightsSmallOrthogonalNTP_kernel<<<gpu->blocks, gpu->updateThreadsPerBlock>>>();  
            else
                kPMEGetGridWeightsSmallNTP_kernel<<<gpu->blocks, gpu->updateThreadsPerBlock>>>();         
        }
        else
        {
            if (gpu->sim.is_orthog)    
                kPMEGetGridWeightsSmallOrthogonal_kernel<<<gpu->blocks, gpu->updateThreadsPerBlock>>>();  
            else
                kPMEGetGridWeightsSmall_kernel<<<gpu->blocks, gpu->updateThreadsPerBlock>>>(); 
        }     
    }
    else 
    {
        if (gpu->sim.ntp > 0)
        {
            if (gpu->sim.is_orthog)    
                kPMEGetGridWeightsOrthogonalNTP_kernel<<<gpu->blocks, gpu->updateThreadsPerBlock>>>();  
            else
                kPMEGetGridWeightsNTP_kernel<<<gpu->blocks, gpu->updateThreadsPerBlock>>>();         
        }
        else
        {
            if (gpu->sim.is_orthog)    
                kPMEGetGridWeightsOrthogonal_kernel<<<gpu->blocks, gpu->updateThreadsPerBlock>>>();  
            else
                kPMEGetGridWeights_kernel<<<gpu->blocks, gpu->updateThreadsPerBlock>>>(); 
        } 
    }
#ifdef MPI 
    LAUNCHERROR_NONBLOCKING("kPMEGetGridWeights");
#else
    LAUNCHERROR("kPMEGetGridWeights");
#endif  

#if 0
    hipDeviceSynchronize();
    gpu->pbAtomXYSP->Download();
    gpu->pbAtomZSP->Download();
    for (int i = 0; i < gpu->sim.atoms; i++)
    {
        printf("%06d %20.10f %20.10f %20.10f\n", i, gpu->pbAtomXYSP->_pSysData[i].x, gpu->pbAtomXYSP->_pSysData[i].y, gpu->pbAtomZSP->_pSysData[i]);
    }
   // exit(-1);
#endif
}

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_UPDATE_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_UPDATE_THREADS_PER_BLOCK, 1)
#endif
kPMEClearChargeGridBuffer27_kernel()
{
extern __shared__ int sOddBufferOverlapFlag[];
    int* psOddXBufferOverlapFlag                = &sOddBufferOverlapFlag[0];
    int* psOddYBufferOverlapFlag                = &sOddBufferOverlapFlag[cSim.nfft1];
    int* psOddZBufferOverlapFlag                = &sOddBufferOverlapFlag[cSim.nfft1 + cSim.nfft2];


    // Read axis dependent buffer flags
    unsigned int pos                            = threadIdx.x;
    unsigned int end                            = cSim.nfft1 + cSim.nfft2 + cSim.nfft3;
    while (pos < end)
    {
        sOddBufferOverlapFlag[pos]              = cSim.pNLOddBufferOverlapFlag[pos];
        pos                                    += blockDim.x;
    }
    __syncthreads();

    pos                                         = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int limit                          = cSim.nfft1 * cSim.nfft2 * cSim.nfft3;

    while (pos < limit)
    {
        // Calculate buffer count
        int z                                   = pos / cSim.nfft1xnfft2;
        int y                                   = (pos - z * cSim.nfft1xnfft2) / cSim.nfft1;
        int x                                   = pos - z * cSim.nfft1xnfft2 - y * cSim.nfft1;
        int oddBufferCount                      = psOddZBufferOverlapFlag[z] + psOddYBufferOverlapFlag[y] + psOddXBufferOverlapFlag[x];
        int extraBuffers                        = cSim.extraChargeGridBuffers[oddBufferCount]; 
    
        // Clear first 8
        PMEFloat* pFloat                        = &cSim.pXYZ_q[pos];
        *pFloat                                 = (PMEFloat)0.0;
        pFloat                                 += cSim.XYZStride;
        *pFloat                                 = (PMEFloat)0.0;
        pFloat                                 += cSim.XYZStride;        
         *pFloat                                 = (PMEFloat)0.0;
        pFloat                                 += cSim.XYZStride;
        *pFloat                                 = (PMEFloat)0.0;
        pFloat                                 += cSim.XYZStride;
        *pFloat                                 = (PMEFloat)0.0;
        pFloat                                 += cSim.XYZStride;
        *pFloat                                 = (PMEFloat)0.0;
        pFloat                                 += cSim.XYZStride;
        *pFloat                                 = (PMEFloat)0.0;
        pFloat                                 += cSim.XYZStride;
        *pFloat                                 = (PMEFloat)0.0;
        pFloat                                 += cSim.XYZStride;        
        // Clear extra buffers
        while (extraBuffers > 3)
        
       
        {
            *pFloat                             = (PMEFloat)0.0;
            pFloat                             += cSim.XYZStride;
            *pFloat                             = (PMEFloat)0.0;
            pFloat                             += cSim.XYZStride;        
            *pFloat                             = (PMEFloat)0.0;
            pFloat                             += cSim.XYZStride;        
            *pFloat                             = (PMEFloat)0.0;
            pFloat                             += cSim.XYZStride;        
            extraBuffers                       -= 4;
        }       
        
        while (extraBuffers > 0)
        {
            *pFloat                             = (PMEFloat)0.0;
            pFloat                             += cSim.XYZStride;
            extraBuffers--;
        }
 
        pos                                    += blockDim.x * gridDim.x;
    }
}

extern "C" void kPMEClearChargeGridBuffer(gpuContext gpu)
{    
    if (gpu->bOddNLCells)
        kPMEClearChargeGridBuffer27_kernel<<<gpu->blocks, gpu->updateThreadsPerBlock, (gpu->sim.nfft1 + gpu->sim.nfft2 + gpu->sim.nfft3) * sizeof(int)>>>(); 
    else
        hipMemset(gpu->sim.pXYZ_q, 0, 8 * gpu->sim.XYZStride * sizeof(PMEFloat));
#ifdef MPI
    LAUNCHERROR_NONBLOCKING("kPMEClearChargeGridBuffer");
#else        
    LAUNCHERROR("kPMEClearChargeGridBuffer");
#endif
}

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_REDUCEFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_REDUCEFORCES_THREADS_PER_BLOCK, 1)
#endif
kPMEReduceChargeGridBuffer8_kernel()
{
    unsigned int pos                            = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int limit                          = cSim.nfft1 * cSim.nfft2 * cSim.nfft3;

    while (pos < limit)
    {
        PMEFloat* pFloat                        = &cSim.pXYZ_q[pos];
        PMEFloat value1                         = *pFloat;
        pFloat                                 += cSim.XYZStride;
        PMEFloat value2                         = *pFloat;
        pFloat                                 += cSim.XYZStride;
        PMEFloat value3                         = *pFloat;
        pFloat                                 += cSim.XYZStride;
        PMEFloat value4                         = *pFloat;
        pFloat                                 += cSim.XYZStride;
        PMEFloat value5                         = *pFloat;
        pFloat                                 += cSim.XYZStride;
        PMEFloat value6                         = *pFloat;
        pFloat                                 += cSim.XYZStride;
        PMEFloat value7                         = *pFloat;
        pFloat                                 += cSim.XYZStride;
        PMEFloat value8                         = *pFloat;
        cSim.pXYZ_q[pos]                        = value1 + value2 + value3 + value4 + value5 + value6 + value7 + value8;       
        pos                                    += blockDim.x * gridDim.x;
    }
}

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_REDUCEFORCES_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_REDUCEFORCES_THREADS_PER_BLOCK, 1)
#endif
kPMEReduceChargeGridBuffer27_kernel()
{
extern __shared__ int sOddBufferOverlapFlag[];
    int* psOddXBufferOverlapFlag                = &sOddBufferOverlapFlag[0];
    int* psOddYBufferOverlapFlag                = &sOddBufferOverlapFlag[cSim.nfft1];
    int* psOddZBufferOverlapFlag                = &sOddBufferOverlapFlag[cSim.nfft1 + cSim.nfft2];


    // Read axis dependent buffer flags
    unsigned int pos                            = threadIdx.x;
    unsigned int end                            = cSim.nfft1 + cSim.nfft2 + cSim.nfft3;
    while (pos < end)
    {
        sOddBufferOverlapFlag[pos]              = cSim.pNLOddBufferOverlapFlag[pos];
        pos                                    += blockDim.x;
    }
    __syncthreads();

    pos                                         = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int limit                          = cSim.nfft1 * cSim.nfft2 * cSim.nfft3;
    
    while (pos < limit)
    {
        // Calculate buffer count
        int z                                   = pos / cSim.nfft1xnfft2;
        int y                                   = (pos - z * cSim.nfft1xnfft2) / cSim.nfft1;
        int x                                   = pos - z * cSim.nfft1xnfft2 - y * cSim.nfft1;
        int oddBufferCount                      = psOddZBufferOverlapFlag[z] + psOddYBufferOverlapFlag[y] + psOddXBufferOverlapFlag[x];
        int extraBuffers                        = cSim.extraChargeGridBuffers[oddBufferCount]; 
    
        PMEFloat sum                            = (PMEFloat)0.0;
        PMEFloat* pFloat                        = &cSim.pXYZ_q[pos];
        PMEFloat value1                         = *pFloat;
        pFloat                                 += cSim.XYZStride;
        PMEFloat value2                         = *pFloat;
        pFloat                                 += cSim.XYZStride;
        PMEFloat value3                         = *pFloat;
        pFloat                                 += cSim.XYZStride;
        PMEFloat value4                         = *pFloat;
        pFloat                                 += cSim.XYZStride;
        PMEFloat value5                         = *pFloat;
        pFloat                                 += cSim.XYZStride;
        PMEFloat value6                         = *pFloat;
        pFloat                                 += cSim.XYZStride;
        PMEFloat value7                         = *pFloat;
        pFloat                                 += cSim.XYZStride;
        PMEFloat value8                         = *pFloat;
        pFloat                                 += cSim.XYZStride;
        
        while (extraBuffers > 3)
        {
            PMEFloat val1                       = *pFloat;
            pFloat                             += cSim.XYZStride;
            PMEFloat val2                       = *pFloat;
            pFloat                             += cSim.XYZStride;
            PMEFloat val3                       = *pFloat;
            pFloat                             += cSim.XYZStride;
            PMEFloat val4                       = *pFloat;
            pFloat                             += cSim.XYZStride;
            sum                                += val1 + val2 + val3 + val4; 
            extraBuffers                       -= 4;
        }
        
        while (extraBuffers > 0)
        {
            PMEFloat val                        = *pFloat;
            pFloat                             += cSim.XYZStride;
            sum                                += val;
            extraBuffers--;
        }        
        cSim.pXYZ_q[pos]                        = value1 + value2 + value3 + value4 + value5 + value6 + value7 + value8 + sum;
   
        pos                                    += blockDim.x * gridDim.x;
    }
}

extern "C" void kPMEReduceChargeGridBuffer(gpuContext gpu)
{



#if 0
    int* pOddXBufferOverlapFlag                 = &(gpu->pbNLOddBufferOverlapFlag->_pSysData[0]);
    int* pOddYBufferOverlapFlag                 = &(gpu->pbNLOddBufferOverlapFlag->_pSysData[gpu->sim.nfft1]);
    int* pOddZBufferOverlapFlag                 = &(gpu->pbNLOddBufferOverlapFlag->_pSysData[gpu->sim.nfft1 + gpu->sim.nfft2]);
#if 0   
    for (int i = 0; i < gpu->sim.nfft1; i++)
    {
        printf("%d %d %d %d\n", i, pOddXBufferOverlapFlag[i], pOddYBufferOverlapFlag[i], pOddZBufferOverlapFlag[i]);
    }
#endif    
    gpu->pbXYZ_q->Download();
    float* pXYZ_q = gpu->pbXYZ_q->_pSysData;
    for (int i = 0; i < gpu->sim.nfft1 * gpu->sim.nfft2 * gpu->sim.nfft3; i++)
    {
        int z                                   = i / gpu->sim.nfft1xnfft2;
        int y                                   = (i - z * gpu->sim.nfft1xnfft2) / gpu->sim.nfft1;
        int x                                   = i - z * gpu->sim.nfft1xnfft2 - y * gpu->sim.nfft1; 
        int oddBufferCount                      = pOddZBufferOverlapFlag[z] + pOddYBufferOverlapFlag[y] + pOddXBufferOverlapFlag[x];
        int buffers                             = gpu->sim.extraChargeGridBuffers[oddBufferCount] + 8; 
        
        
        int maxBuffer = 0;
        for (int j = 0; j < 27; j++)
            if (pXYZ_q[i + j * gpu->sim.XYZStride] > 0.0f)
                maxBuffer = j;
        if (maxBuffer > buffers)
            printf("%3d %3d %3d %3d %3d %3d %3d %3d\n", x, y, z, pOddXBufferOverlapFlag[x], pOddYBufferOverlapFlag[y], pOddZBufferOverlapFlag[z], maxBuffer, buffers);
        
    }
   // exit(-1);

#endif

    if (gpu->bOddNLCells)
        kPMEReduceChargeGridBuffer27_kernel<<<gpu->blocks, gpu->reduceForcesThreadsPerBlock, (gpu->sim.nfft1 + gpu->sim.nfft2 + gpu->sim.nfft3) * sizeof(int)>>>();
    else
        kPMEReduceChargeGridBuffer8_kernel<<<gpu->blocks, gpu->reduceForcesThreadsPerBlock>>>();   
    LAUNCHERROR("kPMEReduceChargeGridBuffer");


#if 0
    gpu->pbXYZ_qc->Download();
    hipfftComplex* xyz_qc = gpu->pbXYZ_qc->_pSysData;
  
    float sum = 0.0;
    for (int i = 0; i < gpu->sim.nfft1; i++)
    {
        for (int j = 0; j < gpu->sim.nfft2; j++)
        {
            for (int k = 0; k < gpu->sim.nfft3; k++)
            {
                printf("%3d %3d %3d %32.15f\n", i, j, k, xyz_qc[(k * gpu->sim.nfft2 + j) * gpu->sim.nfft1 + i].x); 
                sum += xyz_qc[(k * gpu->sim.nfft2 + j) * gpu->sim.nfft1 + i].x;
            }
        }
    }
    printf("%f\n", sum);
    printf("%d %6.1f\n", gpu->sim.atoms, sum / gpu->sim.atoms);
    exit(-1);
#endif

#if 0
    printf("%d %d %d %d\n", gpu->sim.xcells, gpu->sim.ycells, gpu->sim.zcells, gpu->sim.xycells);
    for (int pos = 0; pos < gpu->sim.cells; pos++)
    {
        int zCell                               = pos / gpu->sim.xycells;
        int yCell                               = (pos - zCell * gpu->sim.xycells) / gpu->sim.xcells;
        int xCell                               = pos - zCell * gpu->sim.xycells - yCell * gpu->sim.xcells; 
        printf("%d %d %d %d %d\n", pos, xCell, yCell, zCell, gpu->pbNLChargeGridBufferOffset->_pSysData[pos] / gpu->sim.XYZStride);
    }
    
    printf("%d\n", gpu->sim.XYZStride);
    printf("%d\n", gpu->sim.nfft1xnfft2);
    exit(-1);
#endif
}

struct FillChargeGridAtomData
{
    int ix;
    int iy;
    int iz;
    PMEFloat tx[4];
    PMEFloat ty[4];
    PMEFloat tz[4];
};
static const int LOADSIZE = 32;
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(64, 8)
#else
__launch_bounds__(64, 8)
#endif
kPMEFillChargeGridBuffer8_kernel()
{
__shared__ volatile FillChargeGridAtomData sAtom[LOADSIZE];

    // Determine grid offsets   
    const int tOffsetX                          = threadIdx.x & 0x03; 
    const int tOffsetY                          = (threadIdx.x & 0x0f) >> 2;
    const int tOffsetZ                          = threadIdx.x >> 4;
    const int iOffsetX                          = tOffsetX;
    const int iOffsetY                          = tOffsetY;
    const int iOffsetZ                          = tOffsetZ;
        
        
    // Load cell data
    if (threadIdx.x == 0)
    {
        uint2 cell                              = cSim.pNLNonbondCellStartEnd[blockIdx.x];
        sAtom[0].ix                             = cell.x;
        sAtom[0].iy                             = cell.y;
    }
    __syncthreads();
    uint2 cellStartEnd;
    cellStartEnd.x                              = sAtom[0].ix;
    cellStartEnd.y                              = sAtom[0].iy;
    __syncthreads();
        
    // Calculate buffer offset
    int zCell                                   = blockIdx.x / cSim.xycells;
    int yCell                                   = (blockIdx.x - zCell * cSim.xycells) / cSim.xcells;
    int xCell                                   = blockIdx.x - zCell * cSim.xycells - yCell * cSim.xcells; 
    int bufferOffset                            = (4 * (zCell & 0x1) + 2 * (yCell & 0x1) + (xCell & 0x1)) * cSim.XYZStride;
    PMEFloat* pXYZ_q                            = cSim.pXYZ_q + bufferOffset;

    // Iterate through cell
    unsigned int pos                            = cellStartEnd.x;
    while (pos  < cellStartEnd.y)
    {
        
        // Read Atom Data
        unsigned int maxatom                    = min(pos + LOADSIZE, cellStartEnd.y);
        unsigned int pos1                       = pos + threadIdx.x;
        if (pos1 < maxatom)
        {
            PMEFloat charge                     = cSim.pAtomChargeSP[pos1];
            int ix                              = cSim.pIFractX[pos1];
            int iy                              = cSim.pIFractY[pos1];
            int iz                              = cSim.pIFractZ[pos1];
            PMEFloat4 tx                        = cSim.pThetaX[pos1];
            PMEFloat4 ty                        = cSim.pThetaY[pos1];
            PMEFloat4 tz                        = cSim.pThetaZ[pos1];
            sAtom[threadIdx.x].ix               = ix;
            sAtom[threadIdx.x].iy               = iy;
            sAtom[threadIdx.x].iz               = iz;
            sAtom[threadIdx.x].tx[0]            = tx.x * charge;
            sAtom[threadIdx.x].tx[1]            = tx.y * charge;
            sAtom[threadIdx.x].tx[2]            = tx.z * charge;
            sAtom[threadIdx.x].tx[3]            = tx.w * charge;
            sAtom[threadIdx.x].ty[0]            = ty.x;
            sAtom[threadIdx.x].ty[1]            = ty.y;
            sAtom[threadIdx.x].ty[2]            = ty.z;
            sAtom[threadIdx.x].ty[3]            = ty.w;
            sAtom[threadIdx.x].tz[0]            = tz.x;
            sAtom[threadIdx.x].tz[1]            = tz.y;
            sAtom[threadIdx.x].tz[2]            = tz.z;
            sAtom[threadIdx.x].tz[3]            = tz.w;
        }
        __syncthreads();
      
        // Interpolate onto grid
        pos1                                    = 0;
        unsigned int lastAtom                   = min(LOADSIZE, cellStartEnd.y - pos);
        while (pos1 < lastAtom)
        {
            // Calculate values
            int ix                              = sAtom[pos1].ix + iOffsetX;
            int iy                              = sAtom[pos1].iy + iOffsetY;
            int iz                              = sAtom[pos1].iz + iOffsetZ;
            
            // Insure coordinates stay in bounds
            if (ix >= cSim.nfft1)
                ix                             -= cSim.nfft1;
            if (iy >= cSim.nfft2)
                iy                             -= cSim.nfft2;
            if (iz >= cSim.nfft3)
                iz                             -= cSim.nfft3;  
                
            // Calculate interpolation values and destinations    
            int gpos                            = (iz * cSim.nfft2 + iy) * cSim.nfft1 + ix; 
            PMEFloat qvalue                     = pXYZ_q[gpos];        
            PMEFloat value                      = sAtom[pos1].tx[tOffsetX] * sAtom[pos1].ty[tOffsetY] * sAtom[pos1].tz[tOffsetZ];
                    
            // Write memory and sync all threads          
            pXYZ_q[gpos]                        = qvalue + value;
            __threadfence_block();
            __syncthreads();
 
            pos1++;
         }
         pos                                   += LOADSIZE;
    }      
}

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(64, 8)
#else
__launch_bounds__(64, 8)
#endif
kPMEFillChargeGridBuffer27_kernel()
{
__shared__ volatile FillChargeGridAtomData sAtom[LOADSIZE];

    // Determine grid offsets   
    const int tOffsetX                          = threadIdx.x & 0x03; 
    const int tOffsetY                          = (threadIdx.x & 0x0f) >> 2;
    const int tOffsetZ                          = threadIdx.x >> 4;
    const int iOffsetX                          = tOffsetX;
    const int iOffsetY                          = tOffsetY;
    const int iOffsetZ                          = tOffsetZ;
        
        
    // Load cell data
    if (threadIdx.x == 0)
    {
        uint2 cell                              = cSim.pNLNonbondCellStartEnd[blockIdx.x];
        int bufferOffset                        = cSim.pNLChargeGridBufferOffset[blockIdx.x];
        sAtom[0].ix                             = cell.x;
        sAtom[0].iy                             = cell.y;
        sAtom[0].iz                             = bufferOffset;
    }
    __syncthreads();
    PMEFloat* pXYZ_q                            = cSim.pXYZ_q + sAtom[0].iz;
    uint2 cellStartEnd;
    cellStartEnd.x                              = sAtom[0].ix;
    cellStartEnd.y                              = sAtom[0].iy;        
    __syncthreads();
        
        
    // Iterate through cell
    unsigned int pos                            = cellStartEnd.x;
    while (pos  < cellStartEnd.y)
    {
        
        // Read Atom Data
        unsigned int maxatom                    = min(pos + LOADSIZE, cellStartEnd.y);
        unsigned int pos1                       = pos + threadIdx.x;
        if (pos1 < maxatom)
        {
            PMEFloat charge                     = cSim.pAtomChargeSP[pos1];
            int ix                              = cSim.pIFractX[pos1];
            int iy                              = cSim.pIFractY[pos1];
            int iz                              = cSim.pIFractZ[pos1];
            PMEFloat4 tx                        = cSim.pThetaX[pos1];
            PMEFloat4 ty                        = cSim.pThetaY[pos1];
            PMEFloat4 tz                        = cSim.pThetaZ[pos1];
            sAtom[threadIdx.x].ix               = ix;
            sAtom[threadIdx.x].iy               = iy;
            sAtom[threadIdx.x].iz               = iz;
            sAtom[threadIdx.x].tx[0]            = tx.x * charge;
            sAtom[threadIdx.x].tx[1]            = tx.y * charge;
            sAtom[threadIdx.x].tx[2]            = tx.z * charge;
            sAtom[threadIdx.x].tx[3]            = tx.w * charge;
            sAtom[threadIdx.x].ty[0]            = ty.x;
            sAtom[threadIdx.x].ty[1]            = ty.y;
            sAtom[threadIdx.x].ty[2]            = ty.z;
            sAtom[threadIdx.x].ty[3]            = ty.w;
            sAtom[threadIdx.x].tz[0]            = tz.x;
            sAtom[threadIdx.x].tz[1]            = tz.y;
            sAtom[threadIdx.x].tz[2]            = tz.z;
            sAtom[threadIdx.x].tz[3]            = tz.w;
        }
        __syncthreads();
      
        // Interpolate onto grid
        pos1                                    = 0;
        unsigned int lastAtom                   = min(LOADSIZE, cellStartEnd.y - pos);
        while (pos1 < lastAtom)
        {
            // Calculate values
            int ix                              = sAtom[pos1].ix + iOffsetX;
            int iy                              = sAtom[pos1].iy + iOffsetY;
            int iz                              = sAtom[pos1].iz + iOffsetZ;
            
            // Insure coordinates stay in bounds
            if (ix >= cSim.nfft1)
                ix                             -= cSim.nfft1;
            if (iy >= cSim.nfft2)
                iy                             -= cSim.nfft2;
            if (iz >= cSim.nfft3)
                iz                             -= cSim.nfft3;  
                
            // Calculate interpolation values and destinations    
            int gpos                            = (iz * cSim.nfft2 + iy) * cSim.nfft1 + ix; 
            PMEFloat qvalue                     = pXYZ_q[gpos];        
            PMEFloat value                      = sAtom[pos1].tx[tOffsetX] * sAtom[pos1].ty[tOffsetY] * sAtom[pos1].tz[tOffsetZ];
                    
            // Write memory and sync all threads          
            pXYZ_q[gpos]                        = qvalue + value;
            __threadfence_block();
            __syncthreads();
 
            pos1++;
         }
         pos                                   += LOADSIZE;
    }
}

extern "C" void PMEInitKernels(gpuContext gpu)
{
	if (gpu->sm_version >= SM_2X)
	{
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kPMEFillChargeGridBuffer27_kernel), hipFuncCachePreferL1);
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kPMEFillChargeGridBuffer8_kernel), hipFuncCachePreferL1);       
    }
}


extern "C" void kPMEFillChargeGridBuffer(gpuContext gpu)
{
	if (gpu->sm_version >= SM_2X)
	{ 
	    if (gpu->bOddNLCells)
    	    kPMEFillChargeGridBuffer27_kernel<<<gpu->sim.cells, 64>>>(); 
   	 	else
        	kPMEFillChargeGridBuffer8_kernel<<<gpu->sim.cells, 64>>>();
    }
	else {
 		if (gpu->bOddNLCells)
        	kPMEFillChargeGridBuffer27_kernel<<<gpu->sim.cells, 64>>>(); 
    	else
        	kPMEFillChargeGridBuffer8_kernel<<<gpu->sim.cells, 64>>>();  
	}
#ifdef MPI
    LAUNCHERROR_NONBLOCKING("kPMEFillChargeGridBuffer");
#else	
    LAUNCHERROR("kPMEFillChargeGridBuffer");
#endif
#if 0  
    gpu->pbNLNonbondCellStartEnd->Download();
    gpu->pbIFract->Download();
    uint2* pNLCellStartEnd = gpu->pbNLNonbondCellStartEnd->_pSysData;
    int* pIFractX = gpu->pbIFract->_pSysData;
    int* pIFractY = pIFractX + gpu->sim.stride;
    int* pIFractZ = pIFractX + gpu->sim.stride2;
    
    for (int i = 0; i < gpu->sim.cells; i++)
    {
        int start = pNLCellStartEnd[i].x;
        int end = pNLCellStartEnd[i].y;
        int zCell                               = i / gpu->sim.xycells;
        int yCell                               = (i - zCell * gpu->sim.xycells) / gpu->sim.xcells;
        int xCell                               = i - zCell * gpu->sim.xycells - yCell * gpu->sim.xcells; 
        //int bufferOffset                        = (4 * (zCell & 0x1) + 2 * (yCell & 0x1) + (xCell & 0x1));
        int xmin                                = xCell       * gpu->sim.nfft1 / gpu->sim.xcells - 4;
        if (xmin < 0)
            xmin += gpu->sim.nfft1;
        int xmax                                = (xCell + 1) * gpu->sim.nfft1 / gpu->sim.xcells;
        int ymin                                = yCell       * gpu->sim.nfft2 / gpu->sim.ycells - 4;
        if (ymin < 0)
            ymin += gpu->sim.nfft2;
        int ymax                                = (yCell + 1) * gpu->sim.nfft2 / gpu->sim.ycells;
        int zmin                                = zCell       * gpu->sim.nfft3 / gpu->sim.zcells - 4;
        if (zmin < 0)
            zmin += gpu->sim.nfft3;
        int zmax                                = (zCell + 1) * gpu->sim.nfft3 / gpu->sim.zcells;
        printf("C: %3d %3d %3d %3d %3d %3d %3d %3d %3d %3d\n", i, xCell, yCell, zCell, xmin, xmax, ymin, ymax, zmin, zmax);
        
        for (int j = start; j < end; j++)
        {
            int ix = pIFractX[j];
            int iy = pIFractY[j];
            int iz = pIFractZ[j];
            bool valid = true;
            if (xmin < xmax)
            {
                if ((ix < xmin) | (ix > xmax))
                    valid = false;
            }
            else
            {
                if ((ix > xmax) && (ix < xmin))
                    valid = false; 
            }
            
            if (ymin < ymax)
            {
                if ((iy < ymin) | (iy > ymax))
                    valid = false;
            }
            else
            {
                if ((iy > ymax) && (iy < ymin))
                    valid = false; 
            }
            
            if (zmin < zmax)
            {
                if ((iz < zmin) | (iz > zmax))
                    valid = false;
            }
            else
            {
                if ((iz > zmax) && (iz < zmin))
                    valid = false; 
            }
            if (!valid)
            {
                printf("F: %6d %3d %3d %3d\n", j , ix, iy, iz);
            }
           // printf("A: %6d %3d %3d %3d\n", j , ix, iy, iz);
        }
        
        
        
    }
    exit(-1);
#endif    
    
    
#if 0    
    for (int i = 0; i < gpu->sim.xcells; i++)
    {
        int ixstart = i * gpu->sim.nfft1 / gpu->sim.xcells - 4;
        int ixend = (i + 1) * gpu->sim.nfft1 / gpu->sim.xcells;
        printf("%d %d %d\n", i, ixstart, ixend);
    }
    exit(-1);
#endif
}
 
#define PME_ENERGY
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_UPDATE_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_UPDATE_THREADS_PER_BLOCK, 1)
#endif
kPMEScalarSumRCEnergy_kernel(PMEDouble ewaldcof, PMEDouble vol)
#include "kPSSE.h"
#undef PME_ENERGY

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_UPDATE_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_UPDATE_THREADS_PER_BLOCK, 1)
#endif
kPMEScalarSumRC_kernel(PMEDouble ewaldcof, PMEDouble vol)
#include "kPSSE.h"

#define PME_VIRIAL
#define PME_ENERGY
__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_UPDATE_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_UPDATE_THREADS_PER_BLOCK, 1)
#endif
kPMEScalarSumRCEnergyVirial_kernel(PMEDouble ewaldcof, PMEDouble vol)
#include "kPSSE.h"
#undef PME_ENERGY

__global__ void 
#if (__CUDA_ARCH__ >= 200)
__launch_bounds__(SM_2X_UPDATE_THREADS_PER_BLOCK, 1)
#else
__launch_bounds__(SM_13_UPDATE_THREADS_PER_BLOCK, 1)
#endif
kPMEScalarSumRCVirial_kernel(PMEDouble ewaldcof, PMEDouble vol)
#include "kPSSE.h"
#undef PME_VIRIAL

extern "C" void kPMEScalarSumRC(gpuContext gpu, PMEDouble ewaldcof, PMEDouble vol)
{
    if (gpu->sim.ntp > 0)
        kPMEScalarSumRCVirial_kernel<<<gpu->blocks, gpu->updateThreadsPerBlock>>>(ewaldcof, vol);     
    else
        kPMEScalarSumRC_kernel<<<gpu->blocks, gpu->updateThreadsPerBlock>>>(ewaldcof, vol);   
#ifdef MPI
    LAUNCHERROR_NONBLOCKING("kPMEScalarSumRC");
#else          
    LAUNCHERROR("kPMEScalarSumRC");
#endif
}

extern "C" void kPMEScalarSumRCEnergy(gpuContext gpu, PMEDouble ewaldcof, PMEDouble vol)
{
    if (gpu->sim.ntp > 0)
        kPMEScalarSumRCEnergyVirial_kernel<<<gpu->blocks, gpu->updateThreadsPerBlock>>>(ewaldcof, vol);
    else
        kPMEScalarSumRCEnergy_kernel<<<gpu->blocks, gpu->updateThreadsPerBlock>>>(ewaldcof, vol);
#ifdef MPI
    LAUNCHERROR_NONBLOCKING("kPMEScalarSumRCEnergy");
#else        
    LAUNCHERROR("kPMEScalarSumRCEnergy");
#endif
}

#ifdef use_DPDP
texture<int2, 1, hipReadModeElementType> texref;
#else
texture<PMEFloat, 1, hipReadModeElementType> texref;
#endif
static const int GRADSUMTHREADS = 64;

__global__ void 
#if (__CUDA_ARCH__ >= 200)
#ifdef use_DPDP
__launch_bounds__(GRADSUMTHREADS, 8)
#else
__launch_bounds__(GRADSUMTHREADS, 8)
#endif
#else
#ifdef use_DPDP
__launch_bounds__(GRADSUMTHREADS, 3)
#else
__launch_bounds__(GRADSUMTHREADS, 4)
#endif
#endif
kPMEGradSum64_kernel()
#include "kPGS.h"

#define PME_VIRIAL
__global__ void 
#if (__CUDA_ARCH__ >= 200)
#ifdef use_DPDP
__launch_bounds__(GRADSUMTHREADS, 8)
#else
__launch_bounds__(GRADSUMTHREADS, 8)
#endif
#else
#ifdef use_DPDP
__launch_bounds__(GRADSUMTHREADS, 3)
#else
__launch_bounds__(GRADSUMTHREADS, 4)
#endif
#endif
kPMEGradSum64Virial_kernel()
#include "kPGS.h"
#undef PME_VIRIAL


extern "C" void kPMEGradSum(gpuContext gpu)
{
    texref.normalized = 0;
    texref.filterMode = hipFilterModePoint;
    texref.addressMode[0] = hipAddressModeClamp;
    texref.channelDesc.x = 32;
#ifdef use_DPDP    
    texref.channelDesc.y = 32;
#else
    texref.channelDesc.y = 0;
#endif
    texref.channelDesc.z = 0;
    texref.channelDesc.w = 0;
#ifdef use_DPDP
    hipBindTexture(NULL, texref, (int2*)(gpu->sim.pXYZ_q), gpu->sim.nfft1 * gpu->sim.nfft2 * gpu->sim.nfft3 * sizeof(int2));
#else
    hipBindTexture(NULL, texref, (PMEFloat*)(gpu->sim.pXYZ_q), gpu->sim.nfft1 * gpu->sim.nfft2 * gpu->sim.nfft3 * sizeof(PMEFloat));
#endif

    int blocks;
    if (gpu->sm_version >= SM_2X)
        blocks = (gpu->sim.atoms + 32 - 1) / 32;
    else
        blocks = (gpu->sim.atoms + 16 - 1) / 16; 
    if (gpu->sim.ntp > 0)
        kPMEGradSum64Virial_kernel<<<blocks, GRADSUMTHREADS>>>();   
    else
        kPMEGradSum64_kernel<<<blocks, GRADSUMTHREADS>>>();   

    LAUNCHERROR("kPMEGradSum");
    hipUnbindTexture(texref);
}


